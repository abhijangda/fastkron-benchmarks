#include <hiptensor.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <cassert>
#include <cstdint>
#include <unordered_map>
#include <chrono>

bool CHECK_success(hipError_t status)
{
    return status == hipSuccess;
}

const char* CHECK_pretty(hipError_t status)
{
    return hipGetErrorName(status);
}

bool CHECK_success(hiptensorStatus_t status)
{
    return status == HIPTENSOR_STATUS_SUCCESS;
}

const char* CHECK_pretty(hiptensorStatus_t status)
{
    return hiptensorGetErrorString(status);
}

template<typename T>
T product(const std::vector<T> &values)
{
    T result = 1;
    for (auto& value : values)
    {
        result *= value;
    }
    return result;
}

template<typename T, typename U>
std::vector<T> multiply(const std::vector<T> &lhs, const std::vector<U> &rhs)
{
    std::vector<T> result;
    assert(lhs.size() == rhs.size() || lhs.empty() || rhs.empty());
    for (size_t i = 0; i < lhs.size(); i++)
    {
        result.push_back((lhs.empty() ? 1 : lhs[i]) * (rhs.empty() ? 1 : rhs[i]));
    }
    return result;
}

template<typename T, typename U>
std::vector<T> discretize(const std::vector<T> &in, const std::vector<U> &block)
{
    if (in.empty())
    {
        return in;
    }
    if (block.empty())
    {
        return in;
    }

    std::vector<T> result;
    assert(in.size() == block.size());
    for (size_t i = 0; i < in.size(); i++)
    {
        U b = block[i];
        result.push_back(b * ((in[i] + b - 1) / b));
    }
    return result;
}

#define CHECK(x) do { auto CHECK_err = (x); if (! CHECK_success(CHECK_err)) { \
  printf("Error (%s:%d): \"%s\" returned %s (%d)\n", __FILE__, __LINE__, \
    #x, CHECK_pretty(CHECK_err), CHECK_err); exit(-1);} } while(0)

template<typename K, typename V, typename K2>
std::vector<V> collect(const std::unordered_map<K, V> &map, const std::vector<K2> &index) {
    std::vector<V> result;
    for (auto& elem : index)
    {
        result.push_back(map.at(elem));
    }
    return result;
}

void printDeviceInfo(int deviceId)
{
    struct hipDeviceProp_t prop;
    int currentDeviceId = 0;
    CHECK(hipGetDevice(&currentDeviceId));
    CHECK(hipSetDevice(deviceId));
    CHECK(hipGetDeviceProperties(&prop, deviceId));
    printf( "device %d (%s): SMs %2d  Capabilities %d.%d, SmClock %.1f Mhz, MemSize (MB) %d, MemClock %.1f Mhz\n",
            deviceId,
            prop.name,
            prop.multiProcessorCount, prop.major, prop.minor,
            (float)prop.clockRate*1e-3,
            (int)(prop.totalGlobalMem/(1024*1024)),
            (float)prop.memoryClockRate*1e-3);
    CHECK(hipSetDevice(currentDeviceId));
}

int main(int argc, char** argv)
{
    printf("This sample uses the following GPUs:\n");
    if (argc < 7) {
        printf("Provide: m n p q gm gk but %d provided\n", argc);
     }

    uint m = atoi(argv[1]);
    uint n = atoi(argv[2]);
    uint p = atoi(argv[3]);
    uint q = atoi(argv[4]);
    uint gm = atoi(argv[5]);
    uint gk = atoi(argv[6]);
    
    uint p_power_n = 1;
    for (int i = 0; i < n; i++) p_power_n = p_power_n * p;

    std::vector<int32_t> devices;
    for (int i = 0; i < gm*gk; i++)
    {
        const int deviceId = i;
        printDeviceInfo(deviceId);
        devices.push_back(deviceId);
    }
    cutensorMgHandle_t handle;
    printf("Initializing cutensorMg handle ... ");
    CHECK(cutensorMgCreate(&handle, devices.size(), devices.data()));
    printf("done.\n");
    printf("using %d x %d gpus\n", gm, gk);
    std::unordered_map<int32_t, int64_t> extent;
    extent['m'] = m;
    extent['1'] = p_power_n/p;
    extent['2'] = p;
    extent['3'] = q;

    std::unordered_map<int32_t, int64_t> blocksize;
    blocksize['m'] = extent['m']/gm;
    blocksize['1'] = extent['1']/gk;
    blocksize['2'] = extent['2'];
    blocksize['3'] = extent['3'];

    std::unordered_map<int32_t, int32_t> deviceCount;
    deviceCount['m'] = gm;
    deviceCount['1'] = gk;
    deviceCount['2'] = 1;
    deviceCount['3'] = 1;

    std::vector<int32_t> modesA {'m', '1', '3'};
    std::vector<int32_t> modesB {'m', '1', '2'};
    std::vector<int32_t> modesC {'2', '3'};

    hipDataType kDataType = HIP_R_32F;
    const int64_t kElementSize = 4;

    printf("Creating distributed tensor descriptors ... ");

    auto fillUp = [](const std::vector<int32_t> &devices, const int32_t n)
    {
        std::vector<int32_t> ret; 
        int32_t numDevices = devices.size();
        for(int i=0; i < n; ++i)
        {
            ret.push_back(devices[i%numDevices]);
        }
        return ret;
    };

    cutensorMgTensorDescriptor_t descA;
    std::vector<int64_t> extentA = collect(extent, modesA);
    std::vector<int64_t> blocksizeA = collect(blocksize, modesA);
    std::vector<int32_t> deviceCountA = collect(deviceCount, modesA);
    std::vector<int32_t> devicesA = fillUp(devices, product(deviceCountA));
    assert(product(deviceCountA) == devicesA.size());
    CHECK(cutensorMgCreateTensorDescriptor(handle, &descA, modesA.size(),
        extentA.data(), NULL, blocksizeA.data(), NULL,
        deviceCountA.data(), devicesA.size(), devicesA.data(), kDataType));

    cutensorMgTensorDescriptor_t descB;
    std::vector<int64_t> extentB = collect(extent, modesB);
    std::vector<int64_t> blocksizeB = collect(blocksize, modesB);
    std::vector<int32_t> deviceCountB = collect(deviceCount, modesB);
    std::vector<int32_t> devicesB = fillUp(devices, product(deviceCountB));
    assert(product(deviceCountB) == devicesB.size());
    CHECK(cutensorMgCreateTensorDescriptor(handle, &descB, modesB.size(),
        extentB.data(), NULL, blocksizeB.data(), NULL,
        deviceCountB.data(), devicesB.size(), devicesB.data(), kDataType));

    cutensorMgTensorDescriptor_t descC;
    std::vector<int64_t> extentC = collect(extent, modesC);
    std::vector<int64_t> blocksizeC = collect(blocksize, modesC);
    std::vector<int32_t> deviceCountC = collect(deviceCount, modesC);
    std::vector<int32_t> devicesC = fillUp(devices, product(deviceCountC));
    assert(product(deviceCountC) == devicesC.size());
    CHECK(cutensorMgCreateTensorDescriptor(handle, &descC, modesC.size(),
        extentC.data(), NULL, blocksizeC.data(), NULL,
        deviceCountC.data(), devicesC.size(), devicesC.data(), kDataType));

    printf("done.\n");

    printf("Querying workspace size (per GPU) ... ");

    const hiptensorComputeDescriptor_t kComputeType = HIPTENSOR_COMPUTE_DESC_32F;
    const hiptensorWorksizePreference_t kWorksizePreference = 
        HIPTENSOR_WORKSPACE_DEFAULT;

    cutensorMgContractionDescriptor_t contractionDesc;
    CHECK(cutensorMgCreateContractionDescriptor(handle, &contractionDesc,
                descA, modesA.data(),
                descB, modesB.data(),
                descC, modesC.data(),
                descC, modesC.data(),
                kComputeType));

    cutensorMgContractionFind_t contractionFind;
    CHECK(cutensorMgCreateContractionFind(handle, &contractionFind,
                CUTENSORMG_ALGO_DEFAULT));

    std::vector<int64_t> workspaceSize(devices.size());
    int64_t workspaceHostSize;
    CHECK(cutensorMgContractionGetWorkspace(handle,
        contractionDesc, contractionFind, kWorksizePreference, workspaceSize.data(), &workspaceHostSize));

    printf("done.\n");

    printf("Initializing contraction plan ... \n");
 
    cutensorMgContractionPlan_t plan;
    CHECK(cutensorMgCreateContractionPlan(handle, &plan,
                contractionDesc, contractionFind, workspaceSize.data(), workspaceHostSize));

    printf("done.\n");

    printf("Allocating data ... ");

    int64_t elementsA = product(discretize(extentA, multiply(deviceCountA, blocksizeA))) / product(deviceCountA);
    std::vector<void*> memoryA;
    for (auto& device : devicesA)
    {
        void* memory;
        CHECK(hipSetDevice(device));
        CHECK(hipMalloc(&memory, elementsA * kElementSize));
        memoryA.push_back(memory);
    }

    int64_t elementsB = product(discretize(extentB, multiply(deviceCountB, blocksizeB))) / product(deviceCountB);
    std::vector<void*> memoryB;
    for (auto& device : devicesB)
    {
        void* memory;
        CHECK(hipSetDevice(device));
        CHECK(hipMalloc(&memory, elementsB * kElementSize));
        memoryB.push_back(memory);
    }

    int64_t elementsC = product(discretize(extentC, multiply(deviceCountC, blocksizeC))) / product(deviceCountC);
    std::vector<void*> memoryC;
    for (auto& device : devicesC)
    {
        void* memory;
        CHECK(hipSetDevice(device));
        CHECK(hipMalloc(&memory, elementsC * kElementSize));
        memoryC.push_back(memory);
    }

    std::vector<hipStream_t> streams;
    for (auto& device : devices)
    {
        hipStream_t stream;
        CHECK(hipSetDevice(device));
        CHECK(hipStreamCreate(&stream));
        streams.push_back(stream);
    }

    /*
     * Allocate workspace
     */
    // host
    void* workspaceHost = nullptr;
    CHECK(hipHostMalloc(&workspaceHost, workspaceHostSize));

    // devices
    std::vector<void*> workspace;
    for (int i = 0; i < devices.size(); i++)
    {
        void* memory;
        CHECK(hipSetDevice(devices[i]));
        CHECK(hipMalloc(&memory, workspaceSize[i]));
        workspace.push_back(memory);
    }

    printf("done.\n");

    printf("Performing distributed tensor contraction ...\n");

    float kAlpha = 1;
    float kBeta = 0;


    int currentDeviceId = -1;
    CHECK(hipGetDevice(&currentDeviceId));

    float minElapsed = 0;
    const int nRep = 20; // for stable timings
    for (int rep = 0; rep < nRep; rep++)
    {
        const auto start = std::chrono::steady_clock::now();
        CHECK(cutensorMgContraction(handle, plan, &kAlpha,
            const_cast<const void**>(memoryA.data()),
            const_cast<const void**>(memoryB.data()), &kBeta, 
            const_cast<const void**>(memoryC.data()), memoryC.data(),
            workspace.data(), workspaceHost, streams.data()));

        for (auto& deviceId : devices)
        {
            CHECK(hipSetDevice(deviceId));
            CHECK(hipDeviceSynchronize());
        }

        const auto end = std::chrono::steady_clock::now();
        std::chrono::duration<double, std::milli> dur = end - start;
        printf("%d = %f\n", rep, dur.count());
        if (minElapsed == 0 || minElapsed > dur.count()) {
            minElapsed = dur.count();
        }
    }
    CHECK(hipSetDevice(currentDeviceId));

    double gflops = (2.0 * extent['m'] * extent['1'] * extent['2']* extent['3']) /1e9;
    printf("cuTensor: %.2f GFLOPs/s , %.2f ms\n", gflops / (minElapsed/1e3), minElapsed);
    
    printf("Free resources ...\n");

    for (auto& stream : streams)
    {
        CHECK(hipStreamSynchronize(stream));
        CHECK(hipStreamDestroy(stream));
    }

    for (auto& memory : memoryA)
    {
        CHECK(hipFree(memory));
    }

    for (auto& memory : memoryB)
    {
        CHECK(hipFree(memory));
    }

    for (auto& memory : memoryC)
    {
        CHECK(hipFree(memory));
    }

    CHECK(hipHostFree(workspaceHost));

    CHECK(cutensorMgDestroyContractionDescriptor(contractionDesc));
    CHECK(cutensorMgDestroyContractionFind(contractionFind));
    CHECK(cutensorMgDestroyContractionPlan(plan));

    CHECK(cutensorMgDestroyTensorDescriptor(descA));
    CHECK(cutensorMgDestroyTensorDescriptor(descB));
    CHECK(cutensorMgDestroyTensorDescriptor(descC));

    CHECK(cutensorMgDestroy(handle));
    printf("Done: everything has completed successfully.\n");
}
