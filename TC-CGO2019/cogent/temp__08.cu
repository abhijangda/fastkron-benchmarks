// created by tc_code_include() in tc_code_include.py

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <sys/time.h>
#include <locale.h>
#include <algorithm>
using namespace std;

// created by tc_gen_definition_new()
#define SIZE_SLICE_1_F 16
#define SIZE_SLICE_1_A 16
#define SIZE_SLICE_1_E 4
#define SIZE_SLICE_1_C 1
#define SIZE_SLICE_1_D 1
#define SIZE_SLICE_1_B1 16
#define SIZE_SLICE_1_B2 4

#define SIZE_INT_UNIT_1 SIZE_SLICE_1_F

#define SIZE_TB_1_X 	SIZE_SLICE_1_A * SIZE_SLICE_1_C * SIZE_SLICE_1_D
#define SIZE_TB_1_Y 	SIZE_SLICE_1_B1
#define SIZE_REG_1_X 	SIZE_SLICE_1_E
#define SIZE_REG_1_Y 	SIZE_SLICE_1_B2

#define NUM_INDEX 		6
#define CEIL(a, b) 		(((a) + (b) - 1) / (b))

// created by tc_gen_code_Kernel()
__global__ void kernel__1_1(double* dev_t3, 
double* dev_t2, 
double* dev_v2, 
int size_a, int size_b1, int size_b2, int size_c, int size_d, int size_e, int size_f, 
int numBlk_a, int numBlk_b1, int numBlk_b2, int numBlk_c, int numBlk_d, int numBlk_e, 
int stride_int_t2, int stride_int_v2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ double sm_a[16][64];
	__shared__ double sm_b[16][64];


	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 3
	// # of indices mapped on TB_Y: 1
	int idx_d = threadIdx.x / (SIZE_SLICE_1_C * SIZE_SLICE_1_A);
	int remaining_idx = threadIdx.x % (SIZE_SLICE_1_C * SIZE_SLICE_1_A);
	int idx_c = remaining_idx / (SIZE_SLICE_1_A);
	remaining_idx = remaining_idx % (SIZE_SLICE_1_A);
	int idx_a = remaining_idx;
	int idx_b1 = threadIdx.y;

	int tmp_blkIdx;
	int blk_idx_e = blockIdx.x / (numBlk_d * numBlk_c * numBlk_b2 * numBlk_b1 * numBlk_a);
	tmp_blkIdx = blockIdx.x % (numBlk_d * numBlk_c * numBlk_b2 * numBlk_b1 * numBlk_a);

	int blk_idx_d = tmp_blkIdx / (numBlk_c * numBlk_b2 * numBlk_b1 * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_c * numBlk_b2 * numBlk_b1 * numBlk_a);

	int blk_idx_c = tmp_blkIdx / (numBlk_b2 * numBlk_b1 * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_b2 * numBlk_b1 * numBlk_a);

	int blk_idx_b2 = tmp_blkIdx / (numBlk_b1 * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_b1 * numBlk_a);

	int blk_idx_b1 = tmp_blkIdx / numBlk_a;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a);

	int  blk_idx_a = tmp_blkIdx;

	int t3_base_thread = blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_b1 * SIZE_SLICE_1_B1 + idx_b1 + (blk_idx_b2 * SIZE_SLICE_1_B2 + (blk_idx_c * SIZE_SLICE_1_C + idx_c + (blk_idx_d * SIZE_SLICE_1_D + idx_d + (blk_idx_e * SIZE_SLICE_1_E) * size_d) * size_c) * size_b2) * size_b1) * size_a;


	double temp_av;
	double temp_bv[4];
	double reg_tile[4][4];

	for (int i = 0; i < 4; i++)
	for (int j = 0; j < 4; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'x', 't2', ['e', 'f', 'c', 'a', 'd']], [16, 'STR_SD2_V2_H7', 'y', 'v2', ['b1', 'b2', 'f']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		// No Need to Put Boundary-Checks before For-Statement: : 
		for (int ll = 0; ll < 4; ll++)
		{
			// ['e', 'f', 'c', 'a', 'd']
			// Exception: Temp. version!: threadIdx.y + l
			// Exception: Temp. version!: 0 < rng_c
			sm_a[threadIdx.y][threadIdx.x + ll * 16] = dev_t2[blk_idx_e * SIZE_SLICE_1_E + ll + ((blk_idx_c * SIZE_SLICE_1_C + 0 + (blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_d * SIZE_SLICE_1_D + 0) * size_a) * size_c) * size_f) * size_e + (threadIdx.y + l) * stride_int_t2];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		// No Need to Put Boundary-Checks before For-Statement: : 
		for (int ll = 0; ll < 4; ll++)
		{
			// ['b1', 'b2', 'f']
			// Exception: Temp. version!: threadIdx.y + l
			// Exception: Temp. version!: idx_a < rng_b1
			sm_b[threadIdx.y][threadIdx.x + ll * 16] = dev_v2[blk_idx_b1 * SIZE_SLICE_1_B1 + idx_a + (blk_idx_b2 * SIZE_SLICE_1_B2 + ll) * size_b1 + (threadIdx.y + l) * stride_int_v2];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1; ll++)
		{
			temp_bv[0] = sm_b[ll][idx_b1 + 0];
			temp_bv[1] = sm_b[ll][idx_b1 + 16];
			temp_bv[2] = sm_b[ll][idx_b1 + 32];
			temp_bv[3] = sm_b[ll][idx_b1 + 48];

			for (int xx = 0; xx < 4; xx++) // (1)
			{
				temp_av = sm_a[ll][idx_c + (idx_a + (idx_d) * SIZE_SLICE_1_A) * SIZE_SLICE_1_C + (xx * 16)];

				reg_tile[0][xx] += temp_av * temp_bv[0];
				reg_tile[1][xx] += temp_av * temp_bv[1];
				reg_tile[2][xx] += temp_av * temp_bv[2];
				reg_tile[3][xx] += temp_av * temp_bv[3];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	#pragma unroll 4
	for (int i = 0; i < 4; i++)
	{
		for (int j = 0; j < 4; j++)
		{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
		}
	}
}

// created by tc_gen_code_Kernel()
__global__ void kernel__2_1(double* dev_t3, 
double* dev_t2, 
double* dev_v2, 
int size_a, int size_b1, int size_b2, int size_c, int size_d, int size_e, int size_f, 
int numBlk_a, int numBlk_b1, int numBlk_b2, int numBlk_c, int numBlk_d, int numBlk_e, 
int stride_int_t2, int stride_int_v2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ double sm_a[16][64];
	__shared__ double sm_b[16][64];


	int internal_upperbound   = 0;
	int internal_offset;

	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 3
	// # of indices mapped on TB_Y: 1
	int idx_d = threadIdx.x / (SIZE_SLICE_1_C * SIZE_SLICE_1_A);
	int remaining_idx = threadIdx.x % (SIZE_SLICE_1_C * SIZE_SLICE_1_A);
	int idx_c = remaining_idx / (SIZE_SLICE_1_A);
	remaining_idx = remaining_idx % (SIZE_SLICE_1_A);
	int idx_a = remaining_idx;
	int idx_b1 = threadIdx.y;

	int tmp_blkIdx;
	int blk_idx_e = blockIdx.x / (numBlk_d * numBlk_c * numBlk_b2 * numBlk_b1 * numBlk_a);
	tmp_blkIdx = blockIdx.x % (numBlk_d * numBlk_c * numBlk_b2 * numBlk_b1 * numBlk_a);

	int blk_idx_d = tmp_blkIdx / (numBlk_c * numBlk_b2 * numBlk_b1 * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_c * numBlk_b2 * numBlk_b1 * numBlk_a);

	int blk_idx_c = tmp_blkIdx / (numBlk_b2 * numBlk_b1 * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_b2 * numBlk_b1 * numBlk_a);

	int blk_idx_b2 = tmp_blkIdx / (numBlk_b1 * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_b1 * numBlk_a);

	int blk_idx_b1 = tmp_blkIdx / numBlk_a;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a);

	int  blk_idx_a = tmp_blkIdx;

	int t3_base_thread = blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_b1 * SIZE_SLICE_1_B1 + idx_b1 + (blk_idx_b2 * SIZE_SLICE_1_B2 + (blk_idx_c * SIZE_SLICE_1_C + idx_c + (blk_idx_d * SIZE_SLICE_1_D + idx_d + (blk_idx_e * SIZE_SLICE_1_E) * size_d) * size_c) * size_b2) * size_b1) * size_a;


	double temp_av;
	double temp_bv[4];
	double reg_tile[4][4];

	for (int i = 0; i < 4; i++)
	for (int j = 0; j < 4; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'x', 't2', ['e', 'f', 'c', 'a', 'd']], [16, 'STR_SD2_V2_H7', 'y', 'v2', ['b1', 'b2', 'f']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		// Part: Generalized Contraction Index (p7b)
		internal_offset = (l + SIZE_INT_UNIT_1) - size_internal;
		if (internal_offset > 0) internal_upperbound = internal_offset;

		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (threadIdx.y < SIZE_INT_UNIT_1 - internal_upperbound)
		for (int ll = 0; ll < 4; ll++)
		{
			// ['e', 'f', 'c', 'a', 'd']
			// Exception: Temp. version!: threadIdx.y + l
			// Exception: Temp. version!: 0 < rng_c
			sm_a[threadIdx.y][threadIdx.x + ll * 16] = dev_t2[blk_idx_e * SIZE_SLICE_1_E + ll + ((blk_idx_c * SIZE_SLICE_1_C + 0 + (blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_d * SIZE_SLICE_1_D + 0) * size_a) * size_c) * size_f) * size_e + (threadIdx.y + l) * stride_int_t2];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (threadIdx.y < SIZE_INT_UNIT_1 - internal_upperbound)
		for (int ll = 0; ll < 4; ll++)
		{
			// ['b1', 'b2', 'f']
			// Exception: Temp. version!: threadIdx.y + l
			// Exception: Temp. version!: idx_a < rng_b1
			sm_b[threadIdx.y][threadIdx.x + ll * 16] = dev_v2[blk_idx_b1 * SIZE_SLICE_1_B1 + idx_a + (blk_idx_b2 * SIZE_SLICE_1_B2 + ll) * size_b1 + (threadIdx.y + l) * stride_int_v2];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1 - internal_upperbound; ll++)
		{
			temp_bv[0] = sm_b[ll][idx_b1 + 0];
			temp_bv[1] = sm_b[ll][idx_b1 + 16];
			temp_bv[2] = sm_b[ll][idx_b1 + 32];
			temp_bv[3] = sm_b[ll][idx_b1 + 48];

			for (int xx = 0; xx < 4; xx++) // (1)
			{
				temp_av = sm_a[ll][idx_c + (idx_a + (idx_d) * SIZE_SLICE_1_A) * SIZE_SLICE_1_C + (xx * 16)];

				reg_tile[0][xx] += temp_av * temp_bv[0];
				reg_tile[1][xx] += temp_av * temp_bv[1];
				reg_tile[2][xx] += temp_av * temp_bv[2];
				reg_tile[3][xx] += temp_av * temp_bv[3];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	#pragma unroll 4
	for (int i = 0; i < 4; i++)
	{
		for (int j = 0; j < 4; j++)
		{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
		}
	}
}

// created by tc_gen_code_Kernel()
__global__ void kernel__3_1(double* dev_t3, 
double* dev_t2, 
double* dev_v2, 
int size_a, int size_b1, int size_b2, int size_c, int size_d, int size_e, int size_f, 
int numBlk_a, int numBlk_b1, int numBlk_b2, int numBlk_c, int numBlk_d, int numBlk_e, 
int stride_int_t2, int stride_int_v2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ double sm_a[16][64];
	__shared__ double sm_b[16][64];


	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 3
	// # of indices mapped on TB_Y: 1
	int idx_d = threadIdx.x / (SIZE_SLICE_1_C * SIZE_SLICE_1_A);
	int remaining_idx = threadIdx.x % (SIZE_SLICE_1_C * SIZE_SLICE_1_A);
	int idx_c = remaining_idx / (SIZE_SLICE_1_A);
	remaining_idx = remaining_idx % (SIZE_SLICE_1_A);
	int idx_a = remaining_idx;
	int idx_b1 = threadIdx.y;

	int tmp_blkIdx;
	int blk_idx_e = blockIdx.x / (numBlk_d * numBlk_c * numBlk_b2 * numBlk_b1 * numBlk_a);
	tmp_blkIdx = blockIdx.x % (numBlk_d * numBlk_c * numBlk_b2 * numBlk_b1 * numBlk_a);

	int blk_idx_d = tmp_blkIdx / (numBlk_c * numBlk_b2 * numBlk_b1 * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_c * numBlk_b2 * numBlk_b1 * numBlk_a);

	int blk_idx_c = tmp_blkIdx / (numBlk_b2 * numBlk_b1 * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_b2 * numBlk_b1 * numBlk_a);

	int blk_idx_b2 = tmp_blkIdx / (numBlk_b1 * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_b1 * numBlk_a);

	int blk_idx_b1 = tmp_blkIdx / numBlk_a;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a);

	int  blk_idx_a = tmp_blkIdx;

	int t3_base_thread = blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_b1 * SIZE_SLICE_1_B1 + idx_b1 + (blk_idx_b2 * SIZE_SLICE_1_B2 + (blk_idx_c * SIZE_SLICE_1_C + idx_c + (blk_idx_d * SIZE_SLICE_1_D + idx_d + (blk_idx_e * SIZE_SLICE_1_E) * size_d) * size_c) * size_b2) * size_b1) * size_a;

	// need to support partial tiles
	int rng_a, rng_b1, rng_b2, rng_c, rng_d, rng_e;
	if ((size_a - (blk_idx_a * SIZE_SLICE_1_A)) >= SIZE_SLICE_1_A)
	{
		rng_a = SIZE_SLICE_1_A;
	}
	else
	{
		rng_a = size_a % SIZE_SLICE_1_A;
	}
	if ((size_b1 - (blk_idx_b1 * SIZE_SLICE_1_B1)) >= SIZE_SLICE_1_B1)
	{
		rng_b1 = SIZE_SLICE_1_B1;
	}
	else
	{
		rng_b1 = size_b1 % SIZE_SLICE_1_B1;
	}
	if ((size_b2 - (blk_idx_b2 * SIZE_SLICE_1_B2)) >= SIZE_SLICE_1_B2)
	{
		rng_b2 = SIZE_SLICE_1_B2;
	}
	else
	{
		rng_b2 = size_b2 % SIZE_SLICE_1_B2;
	}
	if ((size_c - (blk_idx_c * SIZE_SLICE_1_C)) >= SIZE_SLICE_1_C)
	{
		rng_c = SIZE_SLICE_1_C;
	}
	else
	{
		rng_c = size_c % SIZE_SLICE_1_C;
	}
	if ((size_d - (blk_idx_d * SIZE_SLICE_1_D)) >= SIZE_SLICE_1_D)
	{
		rng_d = SIZE_SLICE_1_D;
	}
	else
	{
		rng_d = size_d % SIZE_SLICE_1_D;
	}
	if ((size_e - (blk_idx_e * SIZE_SLICE_1_E)) >= SIZE_SLICE_1_E)
	{
		rng_e = SIZE_SLICE_1_E;
	}
	else
	{
		rng_e = size_e % SIZE_SLICE_1_E;
	}

	double temp_av;
	double temp_bv[4];
	double reg_tile[4][4];

	for (int i = 0; i < 4; i++)
	for (int j = 0; j < 4; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'x', 't2', ['e', 'f', 'c', 'a', 'd']], [16, 'STR_SD2_V2_H7', 'y', 'v2', ['b1', 'b2', 'f']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (0 < rng_c && idx_a < rng_a && 0 < rng_d)
		for (int ll = 0; ll < rng_e; ll++)
		{
			// ['e', 'f', 'c', 'a', 'd']
			// Exception: Temp. version!: threadIdx.y + l
			// Exception: Temp. version!: 0 < rng_c
			sm_a[threadIdx.y][threadIdx.x + ll * 16] = dev_t2[blk_idx_e * SIZE_SLICE_1_E + ll + ((blk_idx_c * SIZE_SLICE_1_C + 0 + (blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_d * SIZE_SLICE_1_D + 0) * size_a) * size_c) * size_f) * size_e + (threadIdx.y + l) * stride_int_t2];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (idx_a < rng_b1)
		for (int ll = 0; ll < rng_b2; ll++)
		{
			// ['b1', 'b2', 'f']
			// Exception: Temp. version!: threadIdx.y + l
			// Exception: Temp. version!: idx_a < rng_b1
			sm_b[threadIdx.y][threadIdx.x + ll * 16] = dev_v2[blk_idx_b1 * SIZE_SLICE_1_B1 + idx_a + (blk_idx_b2 * SIZE_SLICE_1_B2 + ll) * size_b1 + (threadIdx.y + l) * stride_int_v2];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1; ll++)
		{
			temp_bv[0] = sm_b[ll][idx_b1 + 0];
			temp_bv[1] = sm_b[ll][idx_b1 + 16];
			temp_bv[2] = sm_b[ll][idx_b1 + 32];
			temp_bv[3] = sm_b[ll][idx_b1 + 48];

			for (int xx = 0; xx < 4; xx++) // (1)
			{
				temp_av = sm_a[ll][idx_c + (idx_a + (idx_d) * SIZE_SLICE_1_A) * SIZE_SLICE_1_C + (xx * 16)];

				reg_tile[0][xx] += temp_av * temp_bv[0];
				reg_tile[1][xx] += temp_av * temp_bv[1];
				reg_tile[2][xx] += temp_av * temp_bv[2];
				reg_tile[3][xx] += temp_av * temp_bv[3];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	if (idx_a < rng_a && idx_c < rng_c && idx_d < rng_d && idx_b1 < rng_b1)
	for (int i = 0; i < 4; i++)
	{
		for (int j = 0; j < 4; j++)
		{
			if(i < rng_b2 && j < rng_e)
			{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
			}
		}
	}
}

// created by tc_gen_code_Kernel()
__global__ void kernel__4_1(double* dev_t3, 
double* dev_t2, 
double* dev_v2, 
int size_a, int size_b1, int size_b2, int size_c, int size_d, int size_e, int size_f, 
int numBlk_a, int numBlk_b1, int numBlk_b2, int numBlk_c, int numBlk_d, int numBlk_e, 
int stride_int_t2, int stride_int_v2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ double sm_a[16][64];
	__shared__ double sm_b[16][64];


	int internal_upperbound   = 0;
	int internal_offset;

	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 3
	// # of indices mapped on TB_Y: 1
	int idx_d = threadIdx.x / (SIZE_SLICE_1_C * SIZE_SLICE_1_A);
	int remaining_idx = threadIdx.x % (SIZE_SLICE_1_C * SIZE_SLICE_1_A);
	int idx_c = remaining_idx / (SIZE_SLICE_1_A);
	remaining_idx = remaining_idx % (SIZE_SLICE_1_A);
	int idx_a = remaining_idx;
	int idx_b1 = threadIdx.y;

	int tmp_blkIdx;
	int blk_idx_e = blockIdx.x / (numBlk_d * numBlk_c * numBlk_b2 * numBlk_b1 * numBlk_a);
	tmp_blkIdx = blockIdx.x % (numBlk_d * numBlk_c * numBlk_b2 * numBlk_b1 * numBlk_a);

	int blk_idx_d = tmp_blkIdx / (numBlk_c * numBlk_b2 * numBlk_b1 * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_c * numBlk_b2 * numBlk_b1 * numBlk_a);

	int blk_idx_c = tmp_blkIdx / (numBlk_b2 * numBlk_b1 * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_b2 * numBlk_b1 * numBlk_a);

	int blk_idx_b2 = tmp_blkIdx / (numBlk_b1 * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_b1 * numBlk_a);

	int blk_idx_b1 = tmp_blkIdx / numBlk_a;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a);

	int  blk_idx_a = tmp_blkIdx;

	int t3_base_thread = blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_b1 * SIZE_SLICE_1_B1 + idx_b1 + (blk_idx_b2 * SIZE_SLICE_1_B2 + (blk_idx_c * SIZE_SLICE_1_C + idx_c + (blk_idx_d * SIZE_SLICE_1_D + idx_d + (blk_idx_e * SIZE_SLICE_1_E) * size_d) * size_c) * size_b2) * size_b1) * size_a;

	// need to support partial tiles
	int rng_a, rng_b1, rng_b2, rng_c, rng_d, rng_e;
	if ((size_a - (blk_idx_a * SIZE_SLICE_1_A)) >= SIZE_SLICE_1_A)
	{
		rng_a = SIZE_SLICE_1_A;
	}
	else
	{
		rng_a = size_a % SIZE_SLICE_1_A;
	}
	if ((size_b1 - (blk_idx_b1 * SIZE_SLICE_1_B1)) >= SIZE_SLICE_1_B1)
	{
		rng_b1 = SIZE_SLICE_1_B1;
	}
	else
	{
		rng_b1 = size_b1 % SIZE_SLICE_1_B1;
	}
	if ((size_b2 - (blk_idx_b2 * SIZE_SLICE_1_B2)) >= SIZE_SLICE_1_B2)
	{
		rng_b2 = SIZE_SLICE_1_B2;
	}
	else
	{
		rng_b2 = size_b2 % SIZE_SLICE_1_B2;
	}
	if ((size_c - (blk_idx_c * SIZE_SLICE_1_C)) >= SIZE_SLICE_1_C)
	{
		rng_c = SIZE_SLICE_1_C;
	}
	else
	{
		rng_c = size_c % SIZE_SLICE_1_C;
	}
	if ((size_d - (blk_idx_d * SIZE_SLICE_1_D)) >= SIZE_SLICE_1_D)
	{
		rng_d = SIZE_SLICE_1_D;
	}
	else
	{
		rng_d = size_d % SIZE_SLICE_1_D;
	}
	if ((size_e - (blk_idx_e * SIZE_SLICE_1_E)) >= SIZE_SLICE_1_E)
	{
		rng_e = SIZE_SLICE_1_E;
	}
	else
	{
		rng_e = size_e % SIZE_SLICE_1_E;
	}

	double temp_av;
	double temp_bv[4];
	double reg_tile[4][4];

	for (int i = 0; i < 4; i++)
	for (int j = 0; j < 4; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'x', 't2', ['e', 'f', 'c', 'a', 'd']], [16, 'STR_SD2_V2_H7', 'y', 'v2', ['b1', 'b2', 'f']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		// Part: Generalized Contraction Index (p7b)
		internal_offset = (l + SIZE_INT_UNIT_1) - size_internal;
		if (internal_offset > 0) internal_upperbound = internal_offset;

		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (0 < rng_c && idx_a < rng_a && 0 < rng_d && threadIdx.y < SIZE_INT_UNIT_1 - internal_upperbound)
		for (int ll = 0; ll < rng_e; ll++)
		{
			// ['e', 'f', 'c', 'a', 'd']
			// Exception: Temp. version!: threadIdx.y + l
			// Exception: Temp. version!: 0 < rng_c
			sm_a[threadIdx.y][threadIdx.x + ll * 16] = dev_t2[blk_idx_e * SIZE_SLICE_1_E + ll + ((blk_idx_c * SIZE_SLICE_1_C + 0 + (blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_d * SIZE_SLICE_1_D + 0) * size_a) * size_c) * size_f) * size_e + (threadIdx.y + l) * stride_int_t2];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (idx_a < rng_b1 && threadIdx.y < SIZE_INT_UNIT_1 - internal_upperbound)
		for (int ll = 0; ll < rng_b2; ll++)
		{
			// ['b1', 'b2', 'f']
			// Exception: Temp. version!: threadIdx.y + l
			// Exception: Temp. version!: idx_a < rng_b1
			sm_b[threadIdx.y][threadIdx.x + ll * 16] = dev_v2[blk_idx_b1 * SIZE_SLICE_1_B1 + idx_a + (blk_idx_b2 * SIZE_SLICE_1_B2 + ll) * size_b1 + (threadIdx.y + l) * stride_int_v2];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1 - internal_upperbound; ll++)
		{
			temp_bv[0] = sm_b[ll][idx_b1 + 0];
			temp_bv[1] = sm_b[ll][idx_b1 + 16];
			temp_bv[2] = sm_b[ll][idx_b1 + 32];
			temp_bv[3] = sm_b[ll][idx_b1 + 48];

			for (int xx = 0; xx < 4; xx++) // (1)
			{
				temp_av = sm_a[ll][idx_c + (idx_a + (idx_d) * SIZE_SLICE_1_A) * SIZE_SLICE_1_C + (xx * 16)];

				reg_tile[0][xx] += temp_av * temp_bv[0];
				reg_tile[1][xx] += temp_av * temp_bv[1];
				reg_tile[2][xx] += temp_av * temp_bv[2];
				reg_tile[3][xx] += temp_av * temp_bv[3];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	if (idx_a < rng_a && idx_c < rng_c && idx_d < rng_d && idx_b1 < rng_b1)
	for (int i = 0; i < 4; i++)
	{
		for (int j = 0; j < 4; j++)
		{
			if(i < rng_b2 && j < rng_e)
			{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
			}
		}
	}
}

// written by tc_interface.tc_gen_code_interface_Header()
extern "C"
void sd_t_d2_fusion(int size_a, int size_b1, int size_b2, int size_c, int size_d, int size_e, int size_f, double* t3, double* host_t2, double* host_v2, int cond_kernel_1, int opt_register_transpose)
{
	int num_thread_blocks_kernel_1;

	double* dev_t3;
	double* dev_t2;
	double* dev_v2;


	num_thread_blocks_kernel_1 = CEIL(size_a, SIZE_SLICE_1_A) * CEIL(size_b1, SIZE_SLICE_1_B1) * CEIL(size_b2, SIZE_SLICE_1_B2) * CEIL(size_c, SIZE_SLICE_1_C) * CEIL(size_d, SIZE_SLICE_1_D) * CEIL(size_e, SIZE_SLICE_1_E);
	// cudaMalloc()
	hipMalloc((void**) &dev_t3, sizeof(double) * size_a * size_b1 * size_b2 * size_c * size_d * size_e);
	hipMalloc((void**) &dev_t2, sizeof(double) * size_d * size_a * size_c * size_f * size_e);
	hipMalloc((void**) &dev_v2, sizeof(double) * size_f * size_b2 * size_b1);

	// cudaMemcpy()
	hipMemcpy(dev_t3, t3, sizeof(double) * size_a * size_b1 * size_b2 * size_c * size_d * size_e, hipMemcpyHostToDevice);
	hipMemcpy(dev_t2, host_t2, sizeof(double) * size_d * size_a * size_c * size_f * size_e, hipMemcpyHostToDevice);
	hipMemcpy(dev_v2, host_v2, sizeof(double) * size_f * size_b2 * size_b1, hipMemcpyHostToDevice);

	// Related to Kernels
	// There are 1 Basic Kernels
	long long int tmp_operations = 2 * (long long int)(size_a * size_b1 * size_b2 * size_c * size_d * size_e) * size_f;
	printf ("========================================= fusedKernels =============================================\n");
	printf ("		Grid Size  : %6d (1D)\n", num_thread_blocks_kernel_1);
	printf ("		Block-size : %2d, %2d (2D)\n", SIZE_TB_1_X, SIZE_TB_1_Y);
	printf ("		Reg.-size  : %2d, %2d (2D)\n", SIZE_REG_1_X, SIZE_REG_1_Y);
	printf ("		A thread deals with (%d x %d) elements (basically)\n", SIZE_TB_1_X * SIZE_REG_1_X, SIZE_TB_1_Y * SIZE_REG_1_Y);
	printf ("		# of Operations: %lld\n", tmp_operations);
	printf ("====================================================================================================\n");
	dim3 gridsize_1(num_thread_blocks_kernel_1);
	dim3 blocksize_1(SIZE_TB_1_X, SIZE_TB_1_Y);

	int stride_output_a = 1;
	int stride_output_b1 = stride_output_a * size_a;
	int stride_output_b2 = stride_output_b1 * size_b1;
	int stride_output_c = stride_output_b2 * size_b2;
	int stride_output_d = stride_output_c * size_c;
	int stride_output_e = stride_output_d * size_d;

	int stride_reg_x_1 = stride_output_e;
	int stride_reg_y_1 = stride_output_b2;

	int size_internal = size_f;

	int stride_int_t2 = size_e;
	int stride_int_v2 = size_b1 * size_b2;

	// Decision Tree for Kernel Types
	// No Chance to Utilize the Register Transpose
	if (size_a % SIZE_SLICE_1_A == 0 && size_b1 % SIZE_SLICE_1_B1 == 0 && size_b2 % SIZE_SLICE_1_B2 == 0 && size_c % SIZE_SLICE_1_C == 0 && size_d % SIZE_SLICE_1_D == 0 && size_e % SIZE_SLICE_1_E == 0)
	{
		// [2] Extenral Index: Full
		if (size_f % SIZE_SLICE_1_F == 0)
		{
			// [3] Internal Index: Full
			// >>> External: Full && Internal: Full
			printf ("External: Full, Internal: Full\n");
			kernel__1_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a, size_b1, size_b2, size_c, size_d, size_e, size_f, CEIL(size_a, SIZE_SLICE_1_A), CEIL(size_b1, SIZE_SLICE_1_B1), CEIL(size_b2, SIZE_SLICE_1_B2), CEIL(size_c, SIZE_SLICE_1_C), CEIL(size_d, SIZE_SLICE_1_D), CEIL(size_e, SIZE_SLICE_1_E), stride_int_t2, stride_int_v2, stride_reg_x_1, stride_reg_y_1, size_internal);
		}
		else
		{
			// [4] Internal Index: Partial
			// >>> External: Full && Internal: Partial
			printf ("External: Full, Internal: Partial\n");
			kernel__2_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a, size_b1, size_b2, size_c, size_d, size_e, size_f, CEIL(size_a, SIZE_SLICE_1_A), CEIL(size_b1, SIZE_SLICE_1_B1), CEIL(size_b2, SIZE_SLICE_1_B2), CEIL(size_c, SIZE_SLICE_1_C), CEIL(size_d, SIZE_SLICE_1_D), CEIL(size_e, SIZE_SLICE_1_E), stride_int_t2, stride_int_v2, stride_reg_x_1, stride_reg_y_1, size_internal);
		}
	}
	else
	{
		// [2] Extenral Index: Partial
		if (size_f % SIZE_SLICE_1_F == 0)
		{
			// [3] Internal Index: Full
			// >>> External: Partial && Internal: Full
			printf ("External: Partial, Internal: Full\n");
			kernel__3_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a, size_b1, size_b2, size_c, size_d, size_e, size_f, CEIL(size_a, SIZE_SLICE_1_A), CEIL(size_b1, SIZE_SLICE_1_B1), CEIL(size_b2, SIZE_SLICE_1_B2), CEIL(size_c, SIZE_SLICE_1_C), CEIL(size_d, SIZE_SLICE_1_D), CEIL(size_e, SIZE_SLICE_1_E), stride_int_t2, stride_int_v2, stride_reg_x_1, stride_reg_y_1, size_internal);
		}
		else
		{
			// [4] Internal Index: Partial
			// >>> External: Partial && Internal: Partial
			printf ("External: Partial, Internal: Partial\n");
			kernel__4_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a, size_b1, size_b2, size_c, size_d, size_e, size_f, CEIL(size_a, SIZE_SLICE_1_A), CEIL(size_b1, SIZE_SLICE_1_B1), CEIL(size_b2, SIZE_SLICE_1_B2), CEIL(size_c, SIZE_SLICE_1_C), CEIL(size_d, SIZE_SLICE_1_D), CEIL(size_e, SIZE_SLICE_1_E), stride_int_t2, stride_int_v2, stride_reg_x_1, stride_reg_y_1, size_internal);
		}
	}

	// Copy the Result from Device to Host
	hipMemcpy(t3, dev_t3, sizeof(double) * (size_a * size_b1 * size_b2 * size_c * size_d * size_e), hipMemcpyDeviceToHost);

	// cudaFree()
	hipFree(dev_t3);	hipFree(dev_t2);	hipFree(dev_v2);

	// Shoule be Fixed
	// HostFree

}

// This is written by tc_interface.tc_gen_code_interface()
// This Interface Should be Called to Run the Kernels
extern "C"
void sd_t_d2_fusion_(int size_a, int size_b, int size_c, int size_d, int size_e, int size_f, double* t3, double* t2, double* v2, int cond_kernel_1, int opt_register_transpose)
{
	// Pre-Processing for Split
	// Based on Tile-Sizes and Problem-Size
	// Currently, one index can be split into two indices
	int size_b1;
	int size_b2;

	if (size_b % 12 == 0)
	{
		//
		size_b1 = 12;
		size_b2 = size_b / 12;
	}
	else
	{
		//
		size_b1 = size_b;
		size_b2 = 1;
	}

	// Call An Application
	sd_t_d2_fusion(size_a, size_b1, size_b2, size_c, size_d, size_e, size_f, t3, t2, v2, cond_kernel_1, opt_register_transpose);
}
