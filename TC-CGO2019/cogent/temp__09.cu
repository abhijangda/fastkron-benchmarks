// created by tc_code_include() in tc_code_include.py

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <sys/time.h>
#include <locale.h>
#include <algorithm>
using namespace std;

// created by tc_gen_definition_new()
#define SIZE_SLICE_1_E 16
#define SIZE_SLICE_1_A1 16
#define SIZE_SLICE_1_A2 6
#define SIZE_SLICE_1_B 16
#define SIZE_SLICE_1_C 6
#define SIZE_SLICE_1_D 1

#define SIZE_INT_UNIT_1 SIZE_SLICE_1_E

#define SIZE_TB_1_X 	SIZE_SLICE_1_A1
#define SIZE_TB_1_Y 	SIZE_SLICE_1_B * SIZE_SLICE_1_D
#define SIZE_REG_1_X 	SIZE_SLICE_1_A2
#define SIZE_REG_1_Y 	SIZE_SLICE_1_C

#define NUM_INDEX 		5
#define CEIL(a, b) 		(((a) + (b) - 1) / (b))

// created by tc_gen_code_Kernel()
__global__ void kernel__1_1(double* dev_t3, 
double* dev_t2, 
double* dev_v2, 
int size_a1, int size_a2, int size_b, int size_c, int size_d, int size_e, 
int numBlk_a1, int numBlk_a2, int numBlk_b, int numBlk_c, int numBlk_d, 
int stride_int_t2, int stride_int_v2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ double sm_a[16][96];
	__shared__ double sm_b[16][96];


	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 1
	// # of indices mapped on TB_Y: 2
	int idx_a1 = threadIdx.x;
	int idx_b = threadIdx.y % SIZE_SLICE_1_B;
	int idx_d = threadIdx.y / SIZE_SLICE_1_B;

	int tmp_blkIdx;
	int blk_idx_d = blockIdx.x / (numBlk_c * numBlk_b * numBlk_a2 * numBlk_a1);
	tmp_blkIdx = blockIdx.x % (numBlk_c * numBlk_b * numBlk_a2 * numBlk_a1);

	int blk_idx_c = tmp_blkIdx / (numBlk_b * numBlk_a2 * numBlk_a1);
	tmp_blkIdx = tmp_blkIdx % (numBlk_b * numBlk_a2 * numBlk_a1);

	int blk_idx_b = tmp_blkIdx / (numBlk_a2 * numBlk_a1);
	tmp_blkIdx = tmp_blkIdx % (numBlk_a2 * numBlk_a1);

	int blk_idx_a2 = tmp_blkIdx / numBlk_a1;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a1);

	int  blk_idx_a1 = tmp_blkIdx;

	int t3_base_thread = blk_idx_a1 * SIZE_SLICE_1_A1 + idx_a1 + (blk_idx_a2 * SIZE_SLICE_1_A2 + (blk_idx_b * SIZE_SLICE_1_B + idx_b + (blk_idx_c * SIZE_SLICE_1_C + (blk_idx_d * SIZE_SLICE_1_D + idx_d) * size_c) * size_b) * size_a2) * size_a1;


	double temp_av;
	double temp_bv[6];
	double reg_tile[6][6];

	for (int i = 0; i < 6; i++)
	for (int j = 0; j < 6; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'x', 't2', ['e', 'a1', 'a2']], [16, 'STR_SD2_V2_H7', 'y', 'v2', ['e', 'b', 'c', 'd']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		// No Need to Put Boundary-Checks before For-Statement: : 
		for (int ll = 0; ll < 6; ll++)
		{
			// ['e', 'a1', 'a2']
			// Exception: Temp. version!: threadIdx.x + l
			// Exception: Temp. version!: idx_b < rng_a1
			sm_a[threadIdx.x][threadIdx.y + ll * 16] = dev_t2[(blk_idx_a1 * SIZE_SLICE_1_A1 + idx_b + (blk_idx_a2 * SIZE_SLICE_1_A2 + ll) * size_a1) * size_e + (threadIdx.x + l)];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		// No Need to Put Boundary-Checks before For-Statement: : 
		for (int ll = 0; ll < 6; ll++)
		{
			// ['e', 'b', 'c', 'd']
			// Exception: Temp. version!: threadIdx.x + l
			// Exception: Temp. version!: idx_b < rng_b
			sm_b[threadIdx.x][threadIdx.y + ll * 16] = dev_v2[(blk_idx_b * SIZE_SLICE_1_B + idx_b + (blk_idx_c * SIZE_SLICE_1_C + ll + (blk_idx_d * SIZE_SLICE_1_D + 0) * size_c) * size_b) * size_e + (threadIdx.x + l)];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1; ll++)
		{
			temp_bv[0] = sm_b[ll][idx_b + (idx_d) * SIZE_SLICE_1_B + 0];
			temp_bv[1] = sm_b[ll][idx_b + (idx_d) * SIZE_SLICE_1_B + 16];
			temp_bv[2] = sm_b[ll][idx_b + (idx_d) * SIZE_SLICE_1_B + 32];
			temp_bv[3] = sm_b[ll][idx_b + (idx_d) * SIZE_SLICE_1_B + 48];
			temp_bv[4] = sm_b[ll][idx_b + (idx_d) * SIZE_SLICE_1_B + 64];
			temp_bv[5] = sm_b[ll][idx_b + (idx_d) * SIZE_SLICE_1_B + 80];

			for (int xx = 0; xx < 6; xx++) // (1)
			{
				temp_av = sm_a[ll][idx_a1 + (xx * 16)];

				reg_tile[0][xx] += temp_av * temp_bv[0];
				reg_tile[1][xx] += temp_av * temp_bv[1];
				reg_tile[2][xx] += temp_av * temp_bv[2];
				reg_tile[3][xx] += temp_av * temp_bv[3];
				reg_tile[4][xx] += temp_av * temp_bv[4];
				reg_tile[5][xx] += temp_av * temp_bv[5];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	#pragma unroll 6
	for (int i = 0; i < 6; i++)
	{
		for (int j = 0; j < 6; j++)
		{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
		}
	}
}

// created by tc_gen_code_Kernel()
__global__ void kernel__2_1(double* dev_t3, 
double* dev_t2, 
double* dev_v2, 
int size_a1, int size_a2, int size_b, int size_c, int size_d, int size_e, 
int numBlk_a1, int numBlk_a2, int numBlk_b, int numBlk_c, int numBlk_d, 
int stride_int_t2, int stride_int_v2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ double sm_a[16][96];
	__shared__ double sm_b[16][96];


	int internal_upperbound   = 0;
	int internal_offset;

	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 1
	// # of indices mapped on TB_Y: 2
	int idx_a1 = threadIdx.x;
	int idx_b = threadIdx.y % SIZE_SLICE_1_B;
	int idx_d = threadIdx.y / SIZE_SLICE_1_B;

	int tmp_blkIdx;
	int blk_idx_d = blockIdx.x / (numBlk_c * numBlk_b * numBlk_a2 * numBlk_a1);
	tmp_blkIdx = blockIdx.x % (numBlk_c * numBlk_b * numBlk_a2 * numBlk_a1);

	int blk_idx_c = tmp_blkIdx / (numBlk_b * numBlk_a2 * numBlk_a1);
	tmp_blkIdx = tmp_blkIdx % (numBlk_b * numBlk_a2 * numBlk_a1);

	int blk_idx_b = tmp_blkIdx / (numBlk_a2 * numBlk_a1);
	tmp_blkIdx = tmp_blkIdx % (numBlk_a2 * numBlk_a1);

	int blk_idx_a2 = tmp_blkIdx / numBlk_a1;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a1);

	int  blk_idx_a1 = tmp_blkIdx;

	int t3_base_thread = blk_idx_a1 * SIZE_SLICE_1_A1 + idx_a1 + (blk_idx_a2 * SIZE_SLICE_1_A2 + (blk_idx_b * SIZE_SLICE_1_B + idx_b + (blk_idx_c * SIZE_SLICE_1_C + (blk_idx_d * SIZE_SLICE_1_D + idx_d) * size_c) * size_b) * size_a2) * size_a1;


	double temp_av;
	double temp_bv[6];
	double reg_tile[6][6];

	for (int i = 0; i < 6; i++)
	for (int j = 0; j < 6; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'x', 't2', ['e', 'a1', 'a2']], [16, 'STR_SD2_V2_H7', 'y', 'v2', ['e', 'b', 'c', 'd']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		// Part: Generalized Contraction Index (p7b)
		internal_offset = (l + SIZE_INT_UNIT_1) - size_internal;
		if (internal_offset > 0) internal_upperbound = internal_offset;

		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (threadIdx.x < SIZE_INT_UNIT_1 - internal_upperbound)
		for (int ll = 0; ll < 6; ll++)
		{
			// ['e', 'a1', 'a2']
			// Exception: Temp. version!: threadIdx.x + l
			// Exception: Temp. version!: idx_b < rng_a1
			sm_a[threadIdx.x][threadIdx.y + ll * 16] = dev_t2[(blk_idx_a1 * SIZE_SLICE_1_A1 + idx_b + (blk_idx_a2 * SIZE_SLICE_1_A2 + ll) * size_a1) * size_e + (threadIdx.x + l)];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (threadIdx.x < SIZE_INT_UNIT_1 - internal_upperbound)
		for (int ll = 0; ll < 6; ll++)
		{
			// ['e', 'b', 'c', 'd']
			// Exception: Temp. version!: threadIdx.x + l
			// Exception: Temp. version!: idx_b < rng_b
			sm_b[threadIdx.x][threadIdx.y + ll * 16] = dev_v2[(blk_idx_b * SIZE_SLICE_1_B + idx_b + (blk_idx_c * SIZE_SLICE_1_C + ll + (blk_idx_d * SIZE_SLICE_1_D + 0) * size_c) * size_b) * size_e + (threadIdx.x + l)];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1 - internal_upperbound; ll++)
		{
			temp_bv[0] = sm_b[ll][idx_b + (idx_d) * SIZE_SLICE_1_B + 0];
			temp_bv[1] = sm_b[ll][idx_b + (idx_d) * SIZE_SLICE_1_B + 16];
			temp_bv[2] = sm_b[ll][idx_b + (idx_d) * SIZE_SLICE_1_B + 32];
			temp_bv[3] = sm_b[ll][idx_b + (idx_d) * SIZE_SLICE_1_B + 48];
			temp_bv[4] = sm_b[ll][idx_b + (idx_d) * SIZE_SLICE_1_B + 64];
			temp_bv[5] = sm_b[ll][idx_b + (idx_d) * SIZE_SLICE_1_B + 80];

			for (int xx = 0; xx < 6; xx++) // (1)
			{
				temp_av = sm_a[ll][idx_a1 + (xx * 16)];

				reg_tile[0][xx] += temp_av * temp_bv[0];
				reg_tile[1][xx] += temp_av * temp_bv[1];
				reg_tile[2][xx] += temp_av * temp_bv[2];
				reg_tile[3][xx] += temp_av * temp_bv[3];
				reg_tile[4][xx] += temp_av * temp_bv[4];
				reg_tile[5][xx] += temp_av * temp_bv[5];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	#pragma unroll 6
	for (int i = 0; i < 6; i++)
	{
		for (int j = 0; j < 6; j++)
		{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
		}
	}
}

// created by tc_gen_code_Kernel()
__global__ void kernel__3_1(double* dev_t3, 
double* dev_t2, 
double* dev_v2, 
int size_a1, int size_a2, int size_b, int size_c, int size_d, int size_e, 
int numBlk_a1, int numBlk_a2, int numBlk_b, int numBlk_c, int numBlk_d, 
int stride_int_t2, int stride_int_v2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ double sm_a[16][96];
	__shared__ double sm_b[16][96];


	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 1
	// # of indices mapped on TB_Y: 2
	int idx_a1 = threadIdx.x;
	int idx_b = threadIdx.y % SIZE_SLICE_1_B;
	int idx_d = threadIdx.y / SIZE_SLICE_1_B;

	int tmp_blkIdx;
	int blk_idx_d = blockIdx.x / (numBlk_c * numBlk_b * numBlk_a2 * numBlk_a1);
	tmp_blkIdx = blockIdx.x % (numBlk_c * numBlk_b * numBlk_a2 * numBlk_a1);

	int blk_idx_c = tmp_blkIdx / (numBlk_b * numBlk_a2 * numBlk_a1);
	tmp_blkIdx = tmp_blkIdx % (numBlk_b * numBlk_a2 * numBlk_a1);

	int blk_idx_b = tmp_blkIdx / (numBlk_a2 * numBlk_a1);
	tmp_blkIdx = tmp_blkIdx % (numBlk_a2 * numBlk_a1);

	int blk_idx_a2 = tmp_blkIdx / numBlk_a1;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a1);

	int  blk_idx_a1 = tmp_blkIdx;

	int t3_base_thread = blk_idx_a1 * SIZE_SLICE_1_A1 + idx_a1 + (blk_idx_a2 * SIZE_SLICE_1_A2 + (blk_idx_b * SIZE_SLICE_1_B + idx_b + (blk_idx_c * SIZE_SLICE_1_C + (blk_idx_d * SIZE_SLICE_1_D + idx_d) * size_c) * size_b) * size_a2) * size_a1;

	// need to support partial tiles
	int rng_a1, rng_a2, rng_b, rng_c, rng_d;
	if ((size_a1 - (blk_idx_a1 * SIZE_SLICE_1_A1)) >= SIZE_SLICE_1_A1)
	{
		rng_a1 = SIZE_SLICE_1_A1;
	}
	else
	{
		rng_a1 = size_a1 % SIZE_SLICE_1_A1;
	}
	if ((size_a2 - (blk_idx_a2 * SIZE_SLICE_1_A2)) >= SIZE_SLICE_1_A2)
	{
		rng_a2 = SIZE_SLICE_1_A2;
	}
	else
	{
		rng_a2 = size_a2 % SIZE_SLICE_1_A2;
	}
	if ((size_b - (blk_idx_b * SIZE_SLICE_1_B)) >= SIZE_SLICE_1_B)
	{
		rng_b = SIZE_SLICE_1_B;
	}
	else
	{
		rng_b = size_b % SIZE_SLICE_1_B;
	}
	if ((size_c - (blk_idx_c * SIZE_SLICE_1_C)) >= SIZE_SLICE_1_C)
	{
		rng_c = SIZE_SLICE_1_C;
	}
	else
	{
		rng_c = size_c % SIZE_SLICE_1_C;
	}
	if ((size_d - (blk_idx_d * SIZE_SLICE_1_D)) >= SIZE_SLICE_1_D)
	{
		rng_d = SIZE_SLICE_1_D;
	}
	else
	{
		rng_d = size_d % SIZE_SLICE_1_D;
	}

	double temp_av;
	double temp_bv[6];
	double reg_tile[6][6];

	for (int i = 0; i < 6; i++)
	for (int j = 0; j < 6; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'x', 't2', ['e', 'a1', 'a2']], [16, 'STR_SD2_V2_H7', 'y', 'v2', ['e', 'b', 'c', 'd']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (idx_b < rng_a1)
		for (int ll = 0; ll < rng_a2; ll++)
		{
			// ['e', 'a1', 'a2']
			// Exception: Temp. version!: threadIdx.x + l
			// Exception: Temp. version!: idx_b < rng_a1
			sm_a[threadIdx.x][threadIdx.y + ll * 16] = dev_t2[(blk_idx_a1 * SIZE_SLICE_1_A1 + idx_b + (blk_idx_a2 * SIZE_SLICE_1_A2 + ll) * size_a1) * size_e + (threadIdx.x + l)];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (idx_b < rng_b && 0 < rng_d)
		for (int ll = 0; ll < rng_c; ll++)
		{
			// ['e', 'b', 'c', 'd']
			// Exception: Temp. version!: threadIdx.x + l
			// Exception: Temp. version!: idx_b < rng_b
			sm_b[threadIdx.x][threadIdx.y + ll * 16] = dev_v2[(blk_idx_b * SIZE_SLICE_1_B + idx_b + (blk_idx_c * SIZE_SLICE_1_C + ll + (blk_idx_d * SIZE_SLICE_1_D + 0) * size_c) * size_b) * size_e + (threadIdx.x + l)];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1; ll++)
		{
			temp_bv[0] = sm_b[ll][idx_b + (idx_d) * SIZE_SLICE_1_B + 0];
			temp_bv[1] = sm_b[ll][idx_b + (idx_d) * SIZE_SLICE_1_B + 16];
			temp_bv[2] = sm_b[ll][idx_b + (idx_d) * SIZE_SLICE_1_B + 32];
			temp_bv[3] = sm_b[ll][idx_b + (idx_d) * SIZE_SLICE_1_B + 48];
			temp_bv[4] = sm_b[ll][idx_b + (idx_d) * SIZE_SLICE_1_B + 64];
			temp_bv[5] = sm_b[ll][idx_b + (idx_d) * SIZE_SLICE_1_B + 80];

			for (int xx = 0; xx < 6; xx++) // (1)
			{
				temp_av = sm_a[ll][idx_a1 + (xx * 16)];

				reg_tile[0][xx] += temp_av * temp_bv[0];
				reg_tile[1][xx] += temp_av * temp_bv[1];
				reg_tile[2][xx] += temp_av * temp_bv[2];
				reg_tile[3][xx] += temp_av * temp_bv[3];
				reg_tile[4][xx] += temp_av * temp_bv[4];
				reg_tile[5][xx] += temp_av * temp_bv[5];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	if (idx_a1 < rng_a1 && idx_b < rng_b && idx_d < rng_d)
	for (int i = 0; i < 6; i++)
	{
		for (int j = 0; j < 6; j++)
		{
			if(i < rng_c && j < rng_a2)
			{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
			}
		}
	}
}

// created by tc_gen_code_Kernel()
__global__ void kernel__4_1(double* dev_t3, 
double* dev_t2, 
double* dev_v2, 
int size_a1, int size_a2, int size_b, int size_c, int size_d, int size_e, 
int numBlk_a1, int numBlk_a2, int numBlk_b, int numBlk_c, int numBlk_d, 
int stride_int_t2, int stride_int_v2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ double sm_a[16][96];
	__shared__ double sm_b[16][96];


	int internal_upperbound   = 0;
	int internal_offset;

	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 1
	// # of indices mapped on TB_Y: 2
	int idx_a1 = threadIdx.x;
	int idx_b = threadIdx.y % SIZE_SLICE_1_B;
	int idx_d = threadIdx.y / SIZE_SLICE_1_B;

	int tmp_blkIdx;
	int blk_idx_d = blockIdx.x / (numBlk_c * numBlk_b * numBlk_a2 * numBlk_a1);
	tmp_blkIdx = blockIdx.x % (numBlk_c * numBlk_b * numBlk_a2 * numBlk_a1);

	int blk_idx_c = tmp_blkIdx / (numBlk_b * numBlk_a2 * numBlk_a1);
	tmp_blkIdx = tmp_blkIdx % (numBlk_b * numBlk_a2 * numBlk_a1);

	int blk_idx_b = tmp_blkIdx / (numBlk_a2 * numBlk_a1);
	tmp_blkIdx = tmp_blkIdx % (numBlk_a2 * numBlk_a1);

	int blk_idx_a2 = tmp_blkIdx / numBlk_a1;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a1);

	int  blk_idx_a1 = tmp_blkIdx;

	int t3_base_thread = blk_idx_a1 * SIZE_SLICE_1_A1 + idx_a1 + (blk_idx_a2 * SIZE_SLICE_1_A2 + (blk_idx_b * SIZE_SLICE_1_B + idx_b + (blk_idx_c * SIZE_SLICE_1_C + (blk_idx_d * SIZE_SLICE_1_D + idx_d) * size_c) * size_b) * size_a2) * size_a1;

	// need to support partial tiles
	int rng_a1, rng_a2, rng_b, rng_c, rng_d;
	if ((size_a1 - (blk_idx_a1 * SIZE_SLICE_1_A1)) >= SIZE_SLICE_1_A1)
	{
		rng_a1 = SIZE_SLICE_1_A1;
	}
	else
	{
		rng_a1 = size_a1 % SIZE_SLICE_1_A1;
	}
	if ((size_a2 - (blk_idx_a2 * SIZE_SLICE_1_A2)) >= SIZE_SLICE_1_A2)
	{
		rng_a2 = SIZE_SLICE_1_A2;
	}
	else
	{
		rng_a2 = size_a2 % SIZE_SLICE_1_A2;
	}
	if ((size_b - (blk_idx_b * SIZE_SLICE_1_B)) >= SIZE_SLICE_1_B)
	{
		rng_b = SIZE_SLICE_1_B;
	}
	else
	{
		rng_b = size_b % SIZE_SLICE_1_B;
	}
	if ((size_c - (blk_idx_c * SIZE_SLICE_1_C)) >= SIZE_SLICE_1_C)
	{
		rng_c = SIZE_SLICE_1_C;
	}
	else
	{
		rng_c = size_c % SIZE_SLICE_1_C;
	}
	if ((size_d - (blk_idx_d * SIZE_SLICE_1_D)) >= SIZE_SLICE_1_D)
	{
		rng_d = SIZE_SLICE_1_D;
	}
	else
	{
		rng_d = size_d % SIZE_SLICE_1_D;
	}

	double temp_av;
	double temp_bv[6];
	double reg_tile[6][6];

	for (int i = 0; i < 6; i++)
	for (int j = 0; j < 6; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'x', 't2', ['e', 'a1', 'a2']], [16, 'STR_SD2_V2_H7', 'y', 'v2', ['e', 'b', 'c', 'd']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		// Part: Generalized Contraction Index (p7b)
		internal_offset = (l + SIZE_INT_UNIT_1) - size_internal;
		if (internal_offset > 0) internal_upperbound = internal_offset;

		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (idx_b < rng_a1 && threadIdx.x < SIZE_INT_UNIT_1 - internal_upperbound)
		for (int ll = 0; ll < rng_a2; ll++)
		{
			// ['e', 'a1', 'a2']
			// Exception: Temp. version!: threadIdx.x + l
			// Exception: Temp. version!: idx_b < rng_a1
			sm_a[threadIdx.x][threadIdx.y + ll * 16] = dev_t2[(blk_idx_a1 * SIZE_SLICE_1_A1 + idx_b + (blk_idx_a2 * SIZE_SLICE_1_A2 + ll) * size_a1) * size_e + (threadIdx.x + l)];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (idx_b < rng_b && 0 < rng_d && threadIdx.x < SIZE_INT_UNIT_1 - internal_upperbound)
		for (int ll = 0; ll < rng_c; ll++)
		{
			// ['e', 'b', 'c', 'd']
			// Exception: Temp. version!: threadIdx.x + l
			// Exception: Temp. version!: idx_b < rng_b
			sm_b[threadIdx.x][threadIdx.y + ll * 16] = dev_v2[(blk_idx_b * SIZE_SLICE_1_B + idx_b + (blk_idx_c * SIZE_SLICE_1_C + ll + (blk_idx_d * SIZE_SLICE_1_D + 0) * size_c) * size_b) * size_e + (threadIdx.x + l)];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1 - internal_upperbound; ll++)
		{
			temp_bv[0] = sm_b[ll][idx_b + (idx_d) * SIZE_SLICE_1_B + 0];
			temp_bv[1] = sm_b[ll][idx_b + (idx_d) * SIZE_SLICE_1_B + 16];
			temp_bv[2] = sm_b[ll][idx_b + (idx_d) * SIZE_SLICE_1_B + 32];
			temp_bv[3] = sm_b[ll][idx_b + (idx_d) * SIZE_SLICE_1_B + 48];
			temp_bv[4] = sm_b[ll][idx_b + (idx_d) * SIZE_SLICE_1_B + 64];
			temp_bv[5] = sm_b[ll][idx_b + (idx_d) * SIZE_SLICE_1_B + 80];

			for (int xx = 0; xx < 6; xx++) // (1)
			{
				temp_av = sm_a[ll][idx_a1 + (xx * 16)];

				reg_tile[0][xx] += temp_av * temp_bv[0];
				reg_tile[1][xx] += temp_av * temp_bv[1];
				reg_tile[2][xx] += temp_av * temp_bv[2];
				reg_tile[3][xx] += temp_av * temp_bv[3];
				reg_tile[4][xx] += temp_av * temp_bv[4];
				reg_tile[5][xx] += temp_av * temp_bv[5];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	if (idx_a1 < rng_a1 && idx_b < rng_b && idx_d < rng_d)
	for (int i = 0; i < 6; i++)
	{
		for (int j = 0; j < 6; j++)
		{
			if(i < rng_c && j < rng_a2)
			{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
			}
		}
	}
}

// written by tc_interface.tc_gen_code_interface_Header()
extern "C"
void sd_t_d2_fusion(int size_a1, int size_a2, int size_b, int size_c, int size_d, int size_e, double* t3, double* host_t2, double* host_v2, int cond_kernel_1, int opt_register_transpose)
{
	int num_thread_blocks_kernel_1;

	double* dev_t3;
	double* dev_t2;
	double* dev_v2;


	num_thread_blocks_kernel_1 = CEIL(size_a1, SIZE_SLICE_1_A1) * CEIL(size_a2, SIZE_SLICE_1_A2) * CEIL(size_b, SIZE_SLICE_1_B) * CEIL(size_c, SIZE_SLICE_1_C) * CEIL(size_d, SIZE_SLICE_1_D);
	// cudaMalloc()
	hipMalloc((void**) &dev_t3, sizeof(double) * size_a1 * size_a2 * size_b * size_c * size_d);
	hipMalloc((void**) &dev_t2, sizeof(double) * size_a2 * size_a1 * size_e);
	hipMalloc((void**) &dev_v2, sizeof(double) * size_d * size_c * size_b * size_e);

	// cudaMemcpy()
	hipMemcpy(dev_t3, t3, sizeof(double) * size_a1 * size_a2 * size_b * size_c * size_d, hipMemcpyHostToDevice);
	hipMemcpy(dev_t2, host_t2, sizeof(double) * size_a2 * size_a1 * size_e, hipMemcpyHostToDevice);
	hipMemcpy(dev_v2, host_v2, sizeof(double) * size_d * size_c * size_b * size_e, hipMemcpyHostToDevice);

	// Related to Kernels
	// There are 1 Basic Kernels
	long long int tmp_operations = 2 * (long long int)(size_a1 * size_a2 * size_b * size_c * size_d) * size_e;
	printf ("========================================= fusedKernels =============================================\n");
	printf ("		Grid Size  : %6d (1D)\n", num_thread_blocks_kernel_1);
	printf ("		Block-size : %2d, %2d (2D)\n", SIZE_TB_1_X, SIZE_TB_1_Y);
	printf ("		Reg.-size  : %2d, %2d (2D)\n", SIZE_REG_1_X, SIZE_REG_1_Y);
	printf ("		A thread deals with (%d x %d) elements (basically)\n", SIZE_TB_1_X * SIZE_REG_1_X, SIZE_TB_1_Y * SIZE_REG_1_Y);
	printf ("		# of Operations: %lld\n", tmp_operations);
	printf ("====================================================================================================\n");
	dim3 gridsize_1(num_thread_blocks_kernel_1);
	dim3 blocksize_1(SIZE_TB_1_X, SIZE_TB_1_Y);

	int stride_output_a1 = 1;
	int stride_output_a2 = stride_output_a1 * size_a1;
	int stride_output_b = stride_output_a2 * size_a2;
	int stride_output_c = stride_output_b * size_b;
	int stride_output_d = stride_output_c * size_c;

	int stride_reg_x_1 = stride_output_a2;
	int stride_reg_y_1 = stride_output_c;

	int size_internal = size_e;

	int stride_int_t2 = 1;
	int stride_int_v2 = 1;

	// Decision Tree for Kernel Types
	// No Chance to Utilize the Register Transpose
	if (size_a1 % SIZE_SLICE_1_A1 == 0 && size_a2 % SIZE_SLICE_1_A2 == 0 && size_b % SIZE_SLICE_1_B == 0 && size_c % SIZE_SLICE_1_C == 0 && size_d % SIZE_SLICE_1_D == 0)
	{
		// [2] Extenral Index: Full
		if (size_e % SIZE_SLICE_1_E == 0)
		{
			// [3] Internal Index: Full
			// >>> External: Full && Internal: Full
			printf ("External: Full, Internal: Full\n");
			kernel__1_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a1, size_a2, size_b, size_c, size_d, size_e, CEIL(size_a1, SIZE_SLICE_1_A1), CEIL(size_a2, SIZE_SLICE_1_A2), CEIL(size_b, SIZE_SLICE_1_B), CEIL(size_c, SIZE_SLICE_1_C), CEIL(size_d, SIZE_SLICE_1_D), stride_int_t2, stride_int_v2, stride_reg_x_1, stride_reg_y_1, size_internal);
		}
		else
		{
			// [4] Internal Index: Partial
			// >>> External: Full && Internal: Partial
			printf ("External: Full, Internal: Partial\n");
			kernel__2_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a1, size_a2, size_b, size_c, size_d, size_e, CEIL(size_a1, SIZE_SLICE_1_A1), CEIL(size_a2, SIZE_SLICE_1_A2), CEIL(size_b, SIZE_SLICE_1_B), CEIL(size_c, SIZE_SLICE_1_C), CEIL(size_d, SIZE_SLICE_1_D), stride_int_t2, stride_int_v2, stride_reg_x_1, stride_reg_y_1, size_internal);
		}
	}
	else
	{
		// [2] Extenral Index: Partial
		if (size_e % SIZE_SLICE_1_E == 0)
		{
			// [3] Internal Index: Full
			// >>> External: Partial && Internal: Full
			printf ("External: Partial, Internal: Full\n");
			kernel__3_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a1, size_a2, size_b, size_c, size_d, size_e, CEIL(size_a1, SIZE_SLICE_1_A1), CEIL(size_a2, SIZE_SLICE_1_A2), CEIL(size_b, SIZE_SLICE_1_B), CEIL(size_c, SIZE_SLICE_1_C), CEIL(size_d, SIZE_SLICE_1_D), stride_int_t2, stride_int_v2, stride_reg_x_1, stride_reg_y_1, size_internal);
		}
		else
		{
			// [4] Internal Index: Partial
			// >>> External: Partial && Internal: Partial
			printf ("External: Partial, Internal: Partial\n");
			kernel__4_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a1, size_a2, size_b, size_c, size_d, size_e, CEIL(size_a1, SIZE_SLICE_1_A1), CEIL(size_a2, SIZE_SLICE_1_A2), CEIL(size_b, SIZE_SLICE_1_B), CEIL(size_c, SIZE_SLICE_1_C), CEIL(size_d, SIZE_SLICE_1_D), stride_int_t2, stride_int_v2, stride_reg_x_1, stride_reg_y_1, size_internal);
		}
	}

	// Copy the Result from Device to Host
	hipMemcpy(t3, dev_t3, sizeof(double) * (size_a1 * size_a2 * size_b * size_c * size_d), hipMemcpyDeviceToHost);

	// cudaFree()
	hipFree(dev_t3);	hipFree(dev_t2);	hipFree(dev_v2);

	// Shoule be Fixed
	// HostFree

}

// This is written by tc_interface.tc_gen_code_interface()
// This Interface Should be Called to Run the Kernels
extern "C"
void sd_t_d2_fusion_(int size_a, int size_b, int size_c, int size_d, int size_e, double* t3, double* t2, double* v2, int cond_kernel_1, int opt_register_transpose)
{
	// Pre-Processing for Split
	// Based on Tile-Sizes and Problem-Size
	// Currently, one index can be split into two indices
	int size_a1;
	int size_a2;

	if (size_a % 12 == 0)
	{
		//
		size_a1 = 12;
		size_a2 = size_a / 12;
	}
	else
	{
		//
		size_a1 = size_a;
		size_a2 = 1;
	}

	// Call An Application
	sd_t_d2_fusion(size_a1, size_a2, size_b, size_c, size_d, size_e, t3, t2, v2, cond_kernel_1, opt_register_transpose);
}
