// created by tc_code_include() in tc_code_include.py

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <sys/time.h>
#include <locale.h>
#include <algorithm>
using namespace std;

// created by tc_gen_definition_new()
#define SIZE_SLICE_1_C 16
#define SIZE_SLICE_1_A1 16
#define SIZE_SLICE_1_A2 6
#define SIZE_SLICE_1_B1 16
#define SIZE_SLICE_1_B2 6

#define SIZE_INT_UNIT_1 SIZE_SLICE_1_C

#define SIZE_TB_1_X 	SIZE_SLICE_1_A1
#define SIZE_TB_1_Y 	SIZE_SLICE_1_B1
#define SIZE_REG_1_X 	SIZE_SLICE_1_A2
#define SIZE_REG_1_Y 	SIZE_SLICE_1_B2

#define NUM_INDEX 		4
#define CEIL(a, b) 		(((a) + (b) - 1) / (b))

// created by tc_gen_code_Kernel()
__global__ void kernel__1_1(double* dev_t3, 
double* dev_t2, 
double* dev_v2, 
int size_a1, int size_a2, int size_b1, int size_b2, int size_c, 
int numBlk_a1, int numBlk_a2, int numBlk_b1, int numBlk_b2, 
int stride_int_t2, int stride_int_v2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ double sm_a[16][96];
	__shared__ double sm_b[16][96];


	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 1
	// # of indices mapped on TB_Y: 1
	int idx_a1 = threadIdx.x;
	int idx_b1 = threadIdx.y;

	int tmp_blkIdx;
	int blk_idx_b2 = blockIdx.x / (numBlk_b1 * numBlk_a2 * numBlk_a1);
	tmp_blkIdx = blockIdx.x % (numBlk_b1 * numBlk_a2 * numBlk_a1);

	int blk_idx_b1 = tmp_blkIdx / (numBlk_a2 * numBlk_a1);
	tmp_blkIdx = tmp_blkIdx % (numBlk_a2 * numBlk_a1);

	int blk_idx_a2 = tmp_blkIdx / numBlk_a1;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a1);

	int  blk_idx_a1 = tmp_blkIdx;

	int t3_base_thread = blk_idx_a1 * SIZE_SLICE_1_A1 + idx_a1 + (blk_idx_a2 * SIZE_SLICE_1_A2 + (blk_idx_b1 * SIZE_SLICE_1_B1 + idx_b1 + (blk_idx_b2 * SIZE_SLICE_1_B2) * size_b1) * size_a2) * size_a1;


	double temp_av;
	double temp_bv[6];
	double reg_tile[6][6];

	for (int i = 0; i < 6; i++)
	for (int j = 0; j < 6; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'x', 't2', ['a1', 'a2', 'c']], [16, 'STR_SD2_V2_H7', 'y', 'v2', ['c', 'b1', 'b2']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		// No Need to Put Boundary-Checks before For-Statement: : 
		for (int ll = 0; ll < 6; ll++)
		{
			// ['a1', 'a2', 'c']
			// Exception: Temp. version!: threadIdx.y + l
			// Exception: Temp. version!: idx_a1 < rng_a1
			sm_a[threadIdx.y][threadIdx.x + ll * 16] = dev_t2[blk_idx_a1 * SIZE_SLICE_1_A1 + idx_a1 + (blk_idx_a2 * SIZE_SLICE_1_A2 + ll) * size_a1 + (threadIdx.y + l) * stride_int_t2];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		// No Need to Put Boundary-Checks before For-Statement: : 
		for (int ll = 0; ll < 6; ll++)
		{
			// ['c', 'b1', 'b2']
			// Exception: Temp. version!: threadIdx.x + l
			// Exception: Temp. version!: idx_b1 < rng_b1
			sm_b[threadIdx.x][threadIdx.y + ll * 16] = dev_v2[(blk_idx_b1 * SIZE_SLICE_1_B1 + idx_b1 + (blk_idx_b2 * SIZE_SLICE_1_B2 + ll) * size_b1) * size_c + (threadIdx.x + l)];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1; ll++)
		{
			temp_bv[0] = sm_b[ll][idx_b1 + 0];
			temp_bv[1] = sm_b[ll][idx_b1 + 16];
			temp_bv[2] = sm_b[ll][idx_b1 + 32];
			temp_bv[3] = sm_b[ll][idx_b1 + 48];
			temp_bv[4] = sm_b[ll][idx_b1 + 64];
			temp_bv[5] = sm_b[ll][idx_b1 + 80];

			for (int xx = 0; xx < 6; xx++) // (1)
			{
				temp_av = sm_a[ll][idx_a1 + (xx * 16)];

				reg_tile[0][xx] += temp_av * temp_bv[0];
				reg_tile[1][xx] += temp_av * temp_bv[1];
				reg_tile[2][xx] += temp_av * temp_bv[2];
				reg_tile[3][xx] += temp_av * temp_bv[3];
				reg_tile[4][xx] += temp_av * temp_bv[4];
				reg_tile[5][xx] += temp_av * temp_bv[5];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	#pragma unroll 6
	for (int i = 0; i < 6; i++)
	{
		for (int j = 0; j < 6; j++)
		{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
		}
	}
}

// created by tc_gen_code_Kernel()
__global__ void kernel__2_1(double* dev_t3, 
double* dev_t2, 
double* dev_v2, 
int size_a1, int size_a2, int size_b1, int size_b2, int size_c, 
int numBlk_a1, int numBlk_a2, int numBlk_b1, int numBlk_b2, 
int stride_int_t2, int stride_int_v2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ double sm_a[16][96];
	__shared__ double sm_b[16][96];


	int internal_upperbound   = 0;
	int internal_offset;

	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 1
	// # of indices mapped on TB_Y: 1
	int idx_a1 = threadIdx.x;
	int idx_b1 = threadIdx.y;

	int tmp_blkIdx;
	int blk_idx_b2 = blockIdx.x / (numBlk_b1 * numBlk_a2 * numBlk_a1);
	tmp_blkIdx = blockIdx.x % (numBlk_b1 * numBlk_a2 * numBlk_a1);

	int blk_idx_b1 = tmp_blkIdx / (numBlk_a2 * numBlk_a1);
	tmp_blkIdx = tmp_blkIdx % (numBlk_a2 * numBlk_a1);

	int blk_idx_a2 = tmp_blkIdx / numBlk_a1;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a1);

	int  blk_idx_a1 = tmp_blkIdx;

	int t3_base_thread = blk_idx_a1 * SIZE_SLICE_1_A1 + idx_a1 + (blk_idx_a2 * SIZE_SLICE_1_A2 + (blk_idx_b1 * SIZE_SLICE_1_B1 + idx_b1 + (blk_idx_b2 * SIZE_SLICE_1_B2) * size_b1) * size_a2) * size_a1;


	double temp_av;
	double temp_bv[6];
	double reg_tile[6][6];

	for (int i = 0; i < 6; i++)
	for (int j = 0; j < 6; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'x', 't2', ['a1', 'a2', 'c']], [16, 'STR_SD2_V2_H7', 'y', 'v2', ['c', 'b1', 'b2']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		// Part: Generalized Contraction Index (p7b)
		internal_offset = (l + SIZE_INT_UNIT_1) - size_internal;
		if (internal_offset > 0) internal_upperbound = internal_offset;

		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (threadIdx.y < SIZE_INT_UNIT_1 - internal_upperbound)
		for (int ll = 0; ll < 6; ll++)
		{
			// ['a1', 'a2', 'c']
			// Exception: Temp. version!: threadIdx.y + l
			// Exception: Temp. version!: idx_a1 < rng_a1
			sm_a[threadIdx.y][threadIdx.x + ll * 16] = dev_t2[blk_idx_a1 * SIZE_SLICE_1_A1 + idx_a1 + (blk_idx_a2 * SIZE_SLICE_1_A2 + ll) * size_a1 + (threadIdx.y + l) * stride_int_t2];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (threadIdx.x < SIZE_INT_UNIT_1 - internal_upperbound)
		for (int ll = 0; ll < 6; ll++)
		{
			// ['c', 'b1', 'b2']
			// Exception: Temp. version!: threadIdx.x + l
			// Exception: Temp. version!: idx_b1 < rng_b1
			sm_b[threadIdx.x][threadIdx.y + ll * 16] = dev_v2[(blk_idx_b1 * SIZE_SLICE_1_B1 + idx_b1 + (blk_idx_b2 * SIZE_SLICE_1_B2 + ll) * size_b1) * size_c + (threadIdx.x + l)];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1 - internal_upperbound; ll++)
		{
			temp_bv[0] = sm_b[ll][idx_b1 + 0];
			temp_bv[1] = sm_b[ll][idx_b1 + 16];
			temp_bv[2] = sm_b[ll][idx_b1 + 32];
			temp_bv[3] = sm_b[ll][idx_b1 + 48];
			temp_bv[4] = sm_b[ll][idx_b1 + 64];
			temp_bv[5] = sm_b[ll][idx_b1 + 80];

			for (int xx = 0; xx < 6; xx++) // (1)
			{
				temp_av = sm_a[ll][idx_a1 + (xx * 16)];

				reg_tile[0][xx] += temp_av * temp_bv[0];
				reg_tile[1][xx] += temp_av * temp_bv[1];
				reg_tile[2][xx] += temp_av * temp_bv[2];
				reg_tile[3][xx] += temp_av * temp_bv[3];
				reg_tile[4][xx] += temp_av * temp_bv[4];
				reg_tile[5][xx] += temp_av * temp_bv[5];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	#pragma unroll 6
	for (int i = 0; i < 6; i++)
	{
		for (int j = 0; j < 6; j++)
		{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
		}
	}
}

// created by tc_gen_code_Kernel()
__global__ void kernel__3_1(double* dev_t3, 
double* dev_t2, 
double* dev_v2, 
int size_a1, int size_a2, int size_b1, int size_b2, int size_c, 
int numBlk_a1, int numBlk_a2, int numBlk_b1, int numBlk_b2, 
int stride_int_t2, int stride_int_v2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ double sm_a[16][96];
	__shared__ double sm_b[16][96];


	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 1
	// # of indices mapped on TB_Y: 1
	int idx_a1 = threadIdx.x;
	int idx_b1 = threadIdx.y;

	int tmp_blkIdx;
	int blk_idx_b2 = blockIdx.x / (numBlk_b1 * numBlk_a2 * numBlk_a1);
	tmp_blkIdx = blockIdx.x % (numBlk_b1 * numBlk_a2 * numBlk_a1);

	int blk_idx_b1 = tmp_blkIdx / (numBlk_a2 * numBlk_a1);
	tmp_blkIdx = tmp_blkIdx % (numBlk_a2 * numBlk_a1);

	int blk_idx_a2 = tmp_blkIdx / numBlk_a1;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a1);

	int  blk_idx_a1 = tmp_blkIdx;

	int t3_base_thread = blk_idx_a1 * SIZE_SLICE_1_A1 + idx_a1 + (blk_idx_a2 * SIZE_SLICE_1_A2 + (blk_idx_b1 * SIZE_SLICE_1_B1 + idx_b1 + (blk_idx_b2 * SIZE_SLICE_1_B2) * size_b1) * size_a2) * size_a1;

	// need to support partial tiles
	int rng_a1, rng_a2, rng_b1, rng_b2;
	if ((size_a1 - (blk_idx_a1 * SIZE_SLICE_1_A1)) >= SIZE_SLICE_1_A1)
	{
		rng_a1 = SIZE_SLICE_1_A1;
	}
	else
	{
		rng_a1 = size_a1 % SIZE_SLICE_1_A1;
	}
	if ((size_a2 - (blk_idx_a2 * SIZE_SLICE_1_A2)) >= SIZE_SLICE_1_A2)
	{
		rng_a2 = SIZE_SLICE_1_A2;
	}
	else
	{
		rng_a2 = size_a2 % SIZE_SLICE_1_A2;
	}
	if ((size_b1 - (blk_idx_b1 * SIZE_SLICE_1_B1)) >= SIZE_SLICE_1_B1)
	{
		rng_b1 = SIZE_SLICE_1_B1;
	}
	else
	{
		rng_b1 = size_b1 % SIZE_SLICE_1_B1;
	}
	if ((size_b2 - (blk_idx_b2 * SIZE_SLICE_1_B2)) >= SIZE_SLICE_1_B2)
	{
		rng_b2 = SIZE_SLICE_1_B2;
	}
	else
	{
		rng_b2 = size_b2 % SIZE_SLICE_1_B2;
	}

	double temp_av;
	double temp_bv[6];
	double reg_tile[6][6];

	for (int i = 0; i < 6; i++)
	for (int j = 0; j < 6; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'x', 't2', ['a1', 'a2', 'c']], [16, 'STR_SD2_V2_H7', 'y', 'v2', ['c', 'b1', 'b2']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (idx_a1 < rng_a1)
		for (int ll = 0; ll < rng_a2; ll++)
		{
			// ['a1', 'a2', 'c']
			// Exception: Temp. version!: threadIdx.y + l
			// Exception: Temp. version!: idx_a1 < rng_a1
			sm_a[threadIdx.y][threadIdx.x + ll * 16] = dev_t2[blk_idx_a1 * SIZE_SLICE_1_A1 + idx_a1 + (blk_idx_a2 * SIZE_SLICE_1_A2 + ll) * size_a1 + (threadIdx.y + l) * stride_int_t2];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (idx_b1 < rng_b1)
		for (int ll = 0; ll < rng_b2; ll++)
		{
			// ['c', 'b1', 'b2']
			// Exception: Temp. version!: threadIdx.x + l
			// Exception: Temp. version!: idx_b1 < rng_b1
			sm_b[threadIdx.x][threadIdx.y + ll * 16] = dev_v2[(blk_idx_b1 * SIZE_SLICE_1_B1 + idx_b1 + (blk_idx_b2 * SIZE_SLICE_1_B2 + ll) * size_b1) * size_c + (threadIdx.x + l)];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1; ll++)
		{
			temp_bv[0] = sm_b[ll][idx_b1 + 0];
			temp_bv[1] = sm_b[ll][idx_b1 + 16];
			temp_bv[2] = sm_b[ll][idx_b1 + 32];
			temp_bv[3] = sm_b[ll][idx_b1 + 48];
			temp_bv[4] = sm_b[ll][idx_b1 + 64];
			temp_bv[5] = sm_b[ll][idx_b1 + 80];

			for (int xx = 0; xx < 6; xx++) // (1)
			{
				temp_av = sm_a[ll][idx_a1 + (xx * 16)];

				reg_tile[0][xx] += temp_av * temp_bv[0];
				reg_tile[1][xx] += temp_av * temp_bv[1];
				reg_tile[2][xx] += temp_av * temp_bv[2];
				reg_tile[3][xx] += temp_av * temp_bv[3];
				reg_tile[4][xx] += temp_av * temp_bv[4];
				reg_tile[5][xx] += temp_av * temp_bv[5];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	if (idx_a1 < rng_a1 && idx_b1 < rng_b1)
	for (int i = 0; i < 6; i++)
	{
		for (int j = 0; j < 6; j++)
		{
			if(i < rng_b2 && j < rng_a2)
			{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
			}
		}
	}
}

// created by tc_gen_code_Kernel()
__global__ void kernel__4_1(double* dev_t3, 
double* dev_t2, 
double* dev_v2, 
int size_a1, int size_a2, int size_b1, int size_b2, int size_c, 
int numBlk_a1, int numBlk_a2, int numBlk_b1, int numBlk_b2, 
int stride_int_t2, int stride_int_v2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ double sm_a[16][96];
	__shared__ double sm_b[16][96];


	int internal_upperbound   = 0;
	int internal_offset;

	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 1
	// # of indices mapped on TB_Y: 1
	int idx_a1 = threadIdx.x;
	int idx_b1 = threadIdx.y;

	int tmp_blkIdx;
	int blk_idx_b2 = blockIdx.x / (numBlk_b1 * numBlk_a2 * numBlk_a1);
	tmp_blkIdx = blockIdx.x % (numBlk_b1 * numBlk_a2 * numBlk_a1);

	int blk_idx_b1 = tmp_blkIdx / (numBlk_a2 * numBlk_a1);
	tmp_blkIdx = tmp_blkIdx % (numBlk_a2 * numBlk_a1);

	int blk_idx_a2 = tmp_blkIdx / numBlk_a1;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a1);

	int  blk_idx_a1 = tmp_blkIdx;

	int t3_base_thread = blk_idx_a1 * SIZE_SLICE_1_A1 + idx_a1 + (blk_idx_a2 * SIZE_SLICE_1_A2 + (blk_idx_b1 * SIZE_SLICE_1_B1 + idx_b1 + (blk_idx_b2 * SIZE_SLICE_1_B2) * size_b1) * size_a2) * size_a1;

	// need to support partial tiles
	int rng_a1, rng_a2, rng_b1, rng_b2;
	if ((size_a1 - (blk_idx_a1 * SIZE_SLICE_1_A1)) >= SIZE_SLICE_1_A1)
	{
		rng_a1 = SIZE_SLICE_1_A1;
	}
	else
	{
		rng_a1 = size_a1 % SIZE_SLICE_1_A1;
	}
	if ((size_a2 - (blk_idx_a2 * SIZE_SLICE_1_A2)) >= SIZE_SLICE_1_A2)
	{
		rng_a2 = SIZE_SLICE_1_A2;
	}
	else
	{
		rng_a2 = size_a2 % SIZE_SLICE_1_A2;
	}
	if ((size_b1 - (blk_idx_b1 * SIZE_SLICE_1_B1)) >= SIZE_SLICE_1_B1)
	{
		rng_b1 = SIZE_SLICE_1_B1;
	}
	else
	{
		rng_b1 = size_b1 % SIZE_SLICE_1_B1;
	}
	if ((size_b2 - (blk_idx_b2 * SIZE_SLICE_1_B2)) >= SIZE_SLICE_1_B2)
	{
		rng_b2 = SIZE_SLICE_1_B2;
	}
	else
	{
		rng_b2 = size_b2 % SIZE_SLICE_1_B2;
	}

	double temp_av;
	double temp_bv[6];
	double reg_tile[6][6];

	for (int i = 0; i < 6; i++)
	for (int j = 0; j < 6; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'x', 't2', ['a1', 'a2', 'c']], [16, 'STR_SD2_V2_H7', 'y', 'v2', ['c', 'b1', 'b2']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		// Part: Generalized Contraction Index (p7b)
		internal_offset = (l + SIZE_INT_UNIT_1) - size_internal;
		if (internal_offset > 0) internal_upperbound = internal_offset;

		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (idx_a1 < rng_a1 && threadIdx.y < SIZE_INT_UNIT_1 - internal_upperbound)
		for (int ll = 0; ll < rng_a2; ll++)
		{
			// ['a1', 'a2', 'c']
			// Exception: Temp. version!: threadIdx.y + l
			// Exception: Temp. version!: idx_a1 < rng_a1
			sm_a[threadIdx.y][threadIdx.x + ll * 16] = dev_t2[blk_idx_a1 * SIZE_SLICE_1_A1 + idx_a1 + (blk_idx_a2 * SIZE_SLICE_1_A2 + ll) * size_a1 + (threadIdx.y + l) * stride_int_t2];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (idx_b1 < rng_b1 && threadIdx.x < SIZE_INT_UNIT_1 - internal_upperbound)
		for (int ll = 0; ll < rng_b2; ll++)
		{
			// ['c', 'b1', 'b2']
			// Exception: Temp. version!: threadIdx.x + l
			// Exception: Temp. version!: idx_b1 < rng_b1
			sm_b[threadIdx.x][threadIdx.y + ll * 16] = dev_v2[(blk_idx_b1 * SIZE_SLICE_1_B1 + idx_b1 + (blk_idx_b2 * SIZE_SLICE_1_B2 + ll) * size_b1) * size_c + (threadIdx.x + l)];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1 - internal_upperbound; ll++)
		{
			temp_bv[0] = sm_b[ll][idx_b1 + 0];
			temp_bv[1] = sm_b[ll][idx_b1 + 16];
			temp_bv[2] = sm_b[ll][idx_b1 + 32];
			temp_bv[3] = sm_b[ll][idx_b1 + 48];
			temp_bv[4] = sm_b[ll][idx_b1 + 64];
			temp_bv[5] = sm_b[ll][idx_b1 + 80];

			for (int xx = 0; xx < 6; xx++) // (1)
			{
				temp_av = sm_a[ll][idx_a1 + (xx * 16)];

				reg_tile[0][xx] += temp_av * temp_bv[0];
				reg_tile[1][xx] += temp_av * temp_bv[1];
				reg_tile[2][xx] += temp_av * temp_bv[2];
				reg_tile[3][xx] += temp_av * temp_bv[3];
				reg_tile[4][xx] += temp_av * temp_bv[4];
				reg_tile[5][xx] += temp_av * temp_bv[5];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	if (idx_a1 < rng_a1 && idx_b1 < rng_b1)
	for (int i = 0; i < 6; i++)
	{
		for (int j = 0; j < 6; j++)
		{
			if(i < rng_b2 && j < rng_a2)
			{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
			}
		}
	}
}

// written by tc_interface.tc_gen_code_interface_Header()
extern "C"
void sd_t_d2_fusion(int size_a1, int size_a2, int size_b1, int size_b2, int size_c, double* t3, double* host_t2, double* host_v2, int cond_kernel_1, int opt_register_transpose)
{
	int num_thread_blocks_kernel_1;

	double* dev_t3;
	double* dev_t2;
	double* dev_v2;


	num_thread_blocks_kernel_1 = CEIL(size_a1, SIZE_SLICE_1_A1) * CEIL(size_a2, SIZE_SLICE_1_A2) * CEIL(size_b1, SIZE_SLICE_1_B1) * CEIL(size_b2, SIZE_SLICE_1_B2);
	// cudaMalloc()
	hipMalloc((void**) &dev_t3, sizeof(double) * size_a1 * size_a2 * size_b1 * size_b2);
	hipMalloc((void**) &dev_t2, sizeof(double) * size_c * size_a2 * size_a1);
	hipMalloc((void**) &dev_v2, sizeof(double) * size_b2 * size_b1 * size_c);

	// cudaMemcpy()
	hipMemcpy(dev_t3, t3, sizeof(double) * size_a1 * size_a2 * size_b1 * size_b2, hipMemcpyHostToDevice);
	hipMemcpy(dev_t2, host_t2, sizeof(double) * size_c * size_a2 * size_a1, hipMemcpyHostToDevice);
	hipMemcpy(dev_v2, host_v2, sizeof(double) * size_b2 * size_b1 * size_c, hipMemcpyHostToDevice);

	// Related to Kernels
	// There are 1 Basic Kernels
	long long int tmp_operations = 2 * (long long int)(size_a1 * size_a2 * size_b1 * size_b2) * size_c;
	printf ("========================================= fusedKernels =============================================\n");
	printf ("		Grid Size  : %6d (1D)\n", num_thread_blocks_kernel_1);
	printf ("		Block-size : %2d, %2d (2D)\n", SIZE_TB_1_X, SIZE_TB_1_Y);
	printf ("		Reg.-size  : %2d, %2d (2D)\n", SIZE_REG_1_X, SIZE_REG_1_Y);
	printf ("		A thread deals with (%d x %d) elements (basically)\n", SIZE_TB_1_X * SIZE_REG_1_X, SIZE_TB_1_Y * SIZE_REG_1_Y);
	printf ("		# of Operations: %lld\n", tmp_operations);
	printf ("====================================================================================================\n");
	dim3 gridsize_1(num_thread_blocks_kernel_1);
	dim3 blocksize_1(SIZE_TB_1_X, SIZE_TB_1_Y);

	int stride_output_a1 = 1;
	int stride_output_a2 = stride_output_a1 * size_a1;
	int stride_output_b1 = stride_output_a2 * size_a2;
	int stride_output_b2 = stride_output_b1 * size_b1;

	int stride_reg_x_1 = stride_output_a2;
	int stride_reg_y_1 = stride_output_b2;

	int size_internal = size_c;

	int stride_int_t2 = size_a1 * size_a2;
	int stride_int_v2 = 1;

	// Decision Tree for Kernel Types
	// No Chance to Utilize the Register Transpose
	if (size_a1 % SIZE_SLICE_1_A1 == 0 && size_a2 % SIZE_SLICE_1_A2 == 0 && size_b1 % SIZE_SLICE_1_B1 == 0 && size_b2 % SIZE_SLICE_1_B2 == 0)
	{
		// [2] Extenral Index: Full
		if (size_c % SIZE_SLICE_1_C == 0)
		{
			// [3] Internal Index: Full
			// >>> External: Full && Internal: Full
			printf ("External: Full, Internal: Full\n");
			kernel__1_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a1, size_a2, size_b1, size_b2, size_c, CEIL(size_a1, SIZE_SLICE_1_A1), CEIL(size_a2, SIZE_SLICE_1_A2), CEIL(size_b1, SIZE_SLICE_1_B1), CEIL(size_b2, SIZE_SLICE_1_B2), stride_int_t2, stride_int_v2, stride_reg_x_1, stride_reg_y_1, size_internal);
		}
		else
		{
			// [4] Internal Index: Partial
			// >>> External: Full && Internal: Partial
			printf ("External: Full, Internal: Partial\n");
			kernel__2_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a1, size_a2, size_b1, size_b2, size_c, CEIL(size_a1, SIZE_SLICE_1_A1), CEIL(size_a2, SIZE_SLICE_1_A2), CEIL(size_b1, SIZE_SLICE_1_B1), CEIL(size_b2, SIZE_SLICE_1_B2), stride_int_t2, stride_int_v2, stride_reg_x_1, stride_reg_y_1, size_internal);
		}
	}
	else
	{
		// [2] Extenral Index: Partial
		if (size_c % SIZE_SLICE_1_C == 0)
		{
			// [3] Internal Index: Full
			// >>> External: Partial && Internal: Full
			printf ("External: Partial, Internal: Full\n");
			kernel__3_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a1, size_a2, size_b1, size_b2, size_c, CEIL(size_a1, SIZE_SLICE_1_A1), CEIL(size_a2, SIZE_SLICE_1_A2), CEIL(size_b1, SIZE_SLICE_1_B1), CEIL(size_b2, SIZE_SLICE_1_B2), stride_int_t2, stride_int_v2, stride_reg_x_1, stride_reg_y_1, size_internal);
		}
		else
		{
			// [4] Internal Index: Partial
			// >>> External: Partial && Internal: Partial
			printf ("External: Partial, Internal: Partial\n");
			kernel__4_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a1, size_a2, size_b1, size_b2, size_c, CEIL(size_a1, SIZE_SLICE_1_A1), CEIL(size_a2, SIZE_SLICE_1_A2), CEIL(size_b1, SIZE_SLICE_1_B1), CEIL(size_b2, SIZE_SLICE_1_B2), stride_int_t2, stride_int_v2, stride_reg_x_1, stride_reg_y_1, size_internal);
		}
	}

	// Copy the Result from Device to Host
	hipMemcpy(t3, dev_t3, sizeof(double) * (size_a1 * size_a2 * size_b1 * size_b2), hipMemcpyDeviceToHost);

	// cudaFree()
	hipFree(dev_t3);	hipFree(dev_t2);	hipFree(dev_v2);

	// Shoule be Fixed
	// HostFree

}

// This is written by tc_interface.tc_gen_code_interface()
// This Interface Should be Called to Run the Kernels
extern "C"
void sd_t_d2_fusion_(int size_a, int size_b, int size_c, double* t3, double* t2, double* v2, int cond_kernel_1, int opt_register_transpose)
{
	// Pre-Processing for Split
	// Based on Tile-Sizes and Problem-Size
	// Currently, one index can be split into two indices
	int size_a1;
	int size_a2;

	if (size_a % 856 == 0)
	{
		//
		size_a1 = 856;
		size_a2 = size_a / 856;
	}
	else
	{
		//
		size_a1 = size_a;
		size_a2 = 1;
	}
	int size_b1;
	int size_b2;

	if (size_b % 32 == 0)
	{
		//
		size_b1 = 32;
		size_b2 = size_b / 32;
	}
	else
	{
		//
		size_b1 = size_b;
		size_b2 = 1;
	}

	// Call An Application
	sd_t_d2_fusion(size_a1, size_a2, size_b1, size_b2, size_c, t3, t2, v2, cond_kernel_1, opt_register_transpose);
}
