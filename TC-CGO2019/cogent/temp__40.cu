// created by tc_code_include() in tc_code_include.py

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <sys/time.h>
#include <locale.h>
#include <algorithm>
using namespace std;

// created by tc_gen_definition_new()
#define SIZE_SLICE_1_E 16
#define SIZE_SLICE_1_A 16
#define SIZE_SLICE_1_B 4
#define SIZE_SLICE_1_C 1
#define SIZE_SLICE_1_D1 8
#define SIZE_SLICE_1_D2 8

#define SIZE_INT_UNIT_1 SIZE_SLICE_1_E

#define SIZE_TB_1_X 	SIZE_SLICE_1_A * SIZE_SLICE_1_C
#define SIZE_TB_1_Y 	SIZE_SLICE_1_D1
#define SIZE_REG_1_X 	SIZE_SLICE_1_B
#define SIZE_REG_1_Y 	SIZE_SLICE_1_D2

#define NUM_INDEX 		5
#define CEIL(a, b) 		(((a) + (b) - 1) / (b))

// created by tc_gen_code_Kernel()
__global__ void kernel__1_1(float* dev_t3, 
float* dev_t2, 
float* dev_v2, 
int size_a, int size_d1, int size_d2, int size_b, int size_c, int size_e, 
int numBlk_a, int numBlk_d1, int numBlk_d2, int numBlk_b, int numBlk_c, 
int stride_int_t2, int stride_int_v2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ float sm_a[16][64];
	__shared__ float sm_b[16][64];


	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 2
	// # of indices mapped on TB_Y: 1
	int idx_a = threadIdx.x % SIZE_SLICE_1_A;
	int idx_c = threadIdx.x / SIZE_SLICE_1_A;
	int idx_d1 = threadIdx.y;

	int tmp_blkIdx;
	int blk_idx_c = blockIdx.x / (numBlk_b * numBlk_d2 * numBlk_d1 * numBlk_a);
	tmp_blkIdx = blockIdx.x % (numBlk_b * numBlk_d2 * numBlk_d1 * numBlk_a);

	int blk_idx_b = tmp_blkIdx / (numBlk_d2 * numBlk_d1 * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_d2 * numBlk_d1 * numBlk_a);

	int blk_idx_d2 = tmp_blkIdx / (numBlk_d1 * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_d1 * numBlk_a);

	int blk_idx_d1 = tmp_blkIdx / numBlk_a;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a);

	int  blk_idx_a = tmp_blkIdx;

	int t3_base_thread = blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_d1 * SIZE_SLICE_1_D1 + idx_d1 + (blk_idx_d2 * SIZE_SLICE_1_D2 + (blk_idx_b * SIZE_SLICE_1_B + (blk_idx_c * SIZE_SLICE_1_C + idx_c) * size_b) * size_d2) * size_d1) * size_a;


	float temp_av;
	float temp_bv[8];
	float reg_tile[8][4];

	for (int i = 0; i < 8; i++)
	for (int j = 0; j < 4; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'x', 't2', ['a', 'b', 'c', 'e']], [16, 'STR_SD2_V2_H7', 'y', 'v2', ['e', 'd1', 'd2']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		// No Need to Put Boundary-Checks before For-Statement: : 
		for (int ll = 0; ll < 4; ll++)
		{
			// ['a', 'b', 'c', 'e']
			// Exception: Temp. version!: threadIdx.y + l + 0
			// Exception: Temp. version!: idx_a < rng_a
			sm_a[threadIdx.y + 0][threadIdx.x + ll * 16] = dev_t2[blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_b * SIZE_SLICE_1_B + ll + (blk_idx_c * SIZE_SLICE_1_C + 0) * size_b) * size_a + (threadIdx.y + l + 0) * stride_int_t2];
			// Exception: Temp. version!: threadIdx.y + l + 8
			// Exception: Temp. version!: idx_a < rng_a
			// Exception: Full-Full
			sm_a[threadIdx.y + 8][threadIdx.x + ll * 16] = dev_t2[blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_b * SIZE_SLICE_1_B + ll + (blk_idx_c * SIZE_SLICE_1_C + 0) * size_b) * size_a + (threadIdx.y + l + 8) * stride_int_t2];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		// No Need to Put Boundary-Checks before For-Statement: : 
		for (int ll = 0; ll < 8; ll++)
		{
			// ['e', 'd1', 'd2']
			// Exception: Temp. version!: threadIdx.x + l
			// Exception: Temp. version!: idx_d1 < rng_d1
			sm_b[threadIdx.x][threadIdx.y + ll * 8] = dev_v2[(blk_idx_d1 * SIZE_SLICE_1_D1 + idx_d1 + (blk_idx_d2 * SIZE_SLICE_1_D2 + ll) * size_d1) * size_e + (threadIdx.x + l)];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1; ll++)
		{
			temp_bv[0] = sm_b[ll][idx_d1 + 0];
			temp_bv[1] = sm_b[ll][idx_d1 + 8];
			temp_bv[2] = sm_b[ll][idx_d1 + 16];
			temp_bv[3] = sm_b[ll][idx_d1 + 24];
			temp_bv[4] = sm_b[ll][idx_d1 + 32];
			temp_bv[5] = sm_b[ll][idx_d1 + 40];
			temp_bv[6] = sm_b[ll][idx_d1 + 48];
			temp_bv[7] = sm_b[ll][idx_d1 + 56];

			for (int xx = 0; xx < 4; xx++) // (1)
			{
				temp_av = sm_a[ll][idx_a + (idx_c) * SIZE_SLICE_1_A + (xx * 16)];

				reg_tile[0][xx] += temp_av * temp_bv[0];
				reg_tile[1][xx] += temp_av * temp_bv[1];
				reg_tile[2][xx] += temp_av * temp_bv[2];
				reg_tile[3][xx] += temp_av * temp_bv[3];
				reg_tile[4][xx] += temp_av * temp_bv[4];
				reg_tile[5][xx] += temp_av * temp_bv[5];
				reg_tile[6][xx] += temp_av * temp_bv[6];
				reg_tile[7][xx] += temp_av * temp_bv[7];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	#pragma unroll 8
	for (int i = 0; i < 8; i++)
	{
		for (int j = 0; j < 4; j++)
		{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
		}
	}
}

// created by tc_gen_code_Kernel()
__global__ void kernel__2_1(float* dev_t3, 
float* dev_t2, 
float* dev_v2, 
int size_a, int size_d1, int size_d2, int size_b, int size_c, int size_e, 
int numBlk_a, int numBlk_d1, int numBlk_d2, int numBlk_b, int numBlk_c, 
int stride_int_t2, int stride_int_v2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ float sm_a[16][64];
	__shared__ float sm_b[16][64];


	int internal_upperbound   = 0;
	int internal_offset;

	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 2
	// # of indices mapped on TB_Y: 1
	int idx_a = threadIdx.x % SIZE_SLICE_1_A;
	int idx_c = threadIdx.x / SIZE_SLICE_1_A;
	int idx_d1 = threadIdx.y;

	int tmp_blkIdx;
	int blk_idx_c = blockIdx.x / (numBlk_b * numBlk_d2 * numBlk_d1 * numBlk_a);
	tmp_blkIdx = blockIdx.x % (numBlk_b * numBlk_d2 * numBlk_d1 * numBlk_a);

	int blk_idx_b = tmp_blkIdx / (numBlk_d2 * numBlk_d1 * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_d2 * numBlk_d1 * numBlk_a);

	int blk_idx_d2 = tmp_blkIdx / (numBlk_d1 * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_d1 * numBlk_a);

	int blk_idx_d1 = tmp_blkIdx / numBlk_a;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a);

	int  blk_idx_a = tmp_blkIdx;

	int t3_base_thread = blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_d1 * SIZE_SLICE_1_D1 + idx_d1 + (blk_idx_d2 * SIZE_SLICE_1_D2 + (blk_idx_b * SIZE_SLICE_1_B + (blk_idx_c * SIZE_SLICE_1_C + idx_c) * size_b) * size_d2) * size_d1) * size_a;


	float temp_av;
	float temp_bv[8];
	float reg_tile[8][4];

	for (int i = 0; i < 8; i++)
	for (int j = 0; j < 4; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'x', 't2', ['a', 'b', 'c', 'e']], [16, 'STR_SD2_V2_H7', 'y', 'v2', ['e', 'd1', 'd2']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		// Part: Generalized Contraction Index (p7b)
		internal_offset = (l + SIZE_INT_UNIT_1) - size_internal;
		if (internal_offset > 0) internal_upperbound = internal_offset;

		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (threadIdx.y < SIZE_INT_UNIT_1 - internal_upperbound)
		for (int ll = 0; ll < 4; ll++)
		{
			// ['a', 'b', 'c', 'e']
			// Exception: Temp. version!: threadIdx.y + l + 0
			// Exception: Temp. version!: idx_a < rng_a
			sm_a[threadIdx.y + 0][threadIdx.x + ll * 16] = dev_t2[blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_b * SIZE_SLICE_1_B + ll + (blk_idx_c * SIZE_SLICE_1_C + 0) * size_b) * size_a + (threadIdx.y + l + 0) * stride_int_t2];
			// Exception: Temp. version!: threadIdx.y + l + 8
			// Exception: Temp. version!: idx_a < rng_a
			if (threadIdx.y + l + 8 < size_internal) 
			sm_a[threadIdx.y + 8][threadIdx.x + ll * 16] = dev_t2[blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_b * SIZE_SLICE_1_B + ll + (blk_idx_c * SIZE_SLICE_1_C + 0) * size_b) * size_a + (threadIdx.y + l + 8) * stride_int_t2];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (threadIdx.x < SIZE_INT_UNIT_1 - internal_upperbound)
		for (int ll = 0; ll < 8; ll++)
		{
			// ['e', 'd1', 'd2']
			// Exception: Temp. version!: threadIdx.x + l
			// Exception: Temp. version!: idx_d1 < rng_d1
			sm_b[threadIdx.x][threadIdx.y + ll * 8] = dev_v2[(blk_idx_d1 * SIZE_SLICE_1_D1 + idx_d1 + (blk_idx_d2 * SIZE_SLICE_1_D2 + ll) * size_d1) * size_e + (threadIdx.x + l)];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1 - internal_upperbound; ll++)
		{
			temp_bv[0] = sm_b[ll][idx_d1 + 0];
			temp_bv[1] = sm_b[ll][idx_d1 + 8];
			temp_bv[2] = sm_b[ll][idx_d1 + 16];
			temp_bv[3] = sm_b[ll][idx_d1 + 24];
			temp_bv[4] = sm_b[ll][idx_d1 + 32];
			temp_bv[5] = sm_b[ll][idx_d1 + 40];
			temp_bv[6] = sm_b[ll][idx_d1 + 48];
			temp_bv[7] = sm_b[ll][idx_d1 + 56];

			for (int xx = 0; xx < 4; xx++) // (1)
			{
				temp_av = sm_a[ll][idx_a + (idx_c) * SIZE_SLICE_1_A + (xx * 16)];

				reg_tile[0][xx] += temp_av * temp_bv[0];
				reg_tile[1][xx] += temp_av * temp_bv[1];
				reg_tile[2][xx] += temp_av * temp_bv[2];
				reg_tile[3][xx] += temp_av * temp_bv[3];
				reg_tile[4][xx] += temp_av * temp_bv[4];
				reg_tile[5][xx] += temp_av * temp_bv[5];
				reg_tile[6][xx] += temp_av * temp_bv[6];
				reg_tile[7][xx] += temp_av * temp_bv[7];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	#pragma unroll 8
	for (int i = 0; i < 8; i++)
	{
		for (int j = 0; j < 4; j++)
		{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
		}
	}
}

// created by tc_gen_code_Kernel()
__global__ void kernel__3_1(float* dev_t3, 
float* dev_t2, 
float* dev_v2, 
int size_a, int size_d1, int size_d2, int size_b, int size_c, int size_e, 
int numBlk_a, int numBlk_d1, int numBlk_d2, int numBlk_b, int numBlk_c, 
int stride_int_t2, int stride_int_v2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ float sm_a[16][64];
	__shared__ float sm_b[16][64];


	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 2
	// # of indices mapped on TB_Y: 1
	int idx_a = threadIdx.x % SIZE_SLICE_1_A;
	int idx_c = threadIdx.x / SIZE_SLICE_1_A;
	int idx_d1 = threadIdx.y;

	int tmp_blkIdx;
	int blk_idx_c = blockIdx.x / (numBlk_b * numBlk_d2 * numBlk_d1 * numBlk_a);
	tmp_blkIdx = blockIdx.x % (numBlk_b * numBlk_d2 * numBlk_d1 * numBlk_a);

	int blk_idx_b = tmp_blkIdx / (numBlk_d2 * numBlk_d1 * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_d2 * numBlk_d1 * numBlk_a);

	int blk_idx_d2 = tmp_blkIdx / (numBlk_d1 * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_d1 * numBlk_a);

	int blk_idx_d1 = tmp_blkIdx / numBlk_a;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a);

	int  blk_idx_a = tmp_blkIdx;

	int t3_base_thread = blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_d1 * SIZE_SLICE_1_D1 + idx_d1 + (blk_idx_d2 * SIZE_SLICE_1_D2 + (blk_idx_b * SIZE_SLICE_1_B + (blk_idx_c * SIZE_SLICE_1_C + idx_c) * size_b) * size_d2) * size_d1) * size_a;

	// need to support partial tiles
	int rng_a, rng_d1, rng_d2, rng_b, rng_c;
	if ((size_a - (blk_idx_a * SIZE_SLICE_1_A)) >= SIZE_SLICE_1_A)
	{
		rng_a = SIZE_SLICE_1_A;
	}
	else
	{
		rng_a = size_a % SIZE_SLICE_1_A;
	}
	if ((size_d1 - (blk_idx_d1 * SIZE_SLICE_1_D1)) >= SIZE_SLICE_1_D1)
	{
		rng_d1 = SIZE_SLICE_1_D1;
	}
	else
	{
		rng_d1 = size_d1 % SIZE_SLICE_1_D1;
	}
	if ((size_d2 - (blk_idx_d2 * SIZE_SLICE_1_D2)) >= SIZE_SLICE_1_D2)
	{
		rng_d2 = SIZE_SLICE_1_D2;
	}
	else
	{
		rng_d2 = size_d2 % SIZE_SLICE_1_D2;
	}
	if ((size_b - (blk_idx_b * SIZE_SLICE_1_B)) >= SIZE_SLICE_1_B)
	{
		rng_b = SIZE_SLICE_1_B;
	}
	else
	{
		rng_b = size_b % SIZE_SLICE_1_B;
	}
	if ((size_c - (blk_idx_c * SIZE_SLICE_1_C)) >= SIZE_SLICE_1_C)
	{
		rng_c = SIZE_SLICE_1_C;
	}
	else
	{
		rng_c = size_c % SIZE_SLICE_1_C;
	}

	float temp_av;
	float temp_bv[8];
	float reg_tile[8][4];

	for (int i = 0; i < 8; i++)
	for (int j = 0; j < 4; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'x', 't2', ['a', 'b', 'c', 'e']], [16, 'STR_SD2_V2_H7', 'y', 'v2', ['e', 'd1', 'd2']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (idx_a < rng_a && 0 < rng_c)
		for (int ll = 0; ll < rng_b; ll++)
		{
			// ['a', 'b', 'c', 'e']
			// Exception: Temp. version!: threadIdx.y + l + 0
			// Exception: Temp. version!: idx_a < rng_a
			sm_a[threadIdx.y + 0][threadIdx.x + ll * 16] = dev_t2[blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_b * SIZE_SLICE_1_B + ll + (blk_idx_c * SIZE_SLICE_1_C + 0) * size_b) * size_a + (threadIdx.y + l + 0) * stride_int_t2];
			// Exception: Temp. version!: threadIdx.y + l + 8
			// Exception: Temp. version!: idx_a < rng_a
			if (idx_a < rng_a) 
			sm_a[threadIdx.y + 8][threadIdx.x + ll * 16] = dev_t2[blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_b * SIZE_SLICE_1_B + ll + (blk_idx_c * SIZE_SLICE_1_C + 0) * size_b) * size_a + (threadIdx.y + l + 8) * stride_int_t2];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (idx_d1 < rng_d1)
		for (int ll = 0; ll < rng_d2; ll++)
		{
			// ['e', 'd1', 'd2']
			// Exception: Temp. version!: threadIdx.x + l
			// Exception: Temp. version!: idx_d1 < rng_d1
			sm_b[threadIdx.x][threadIdx.y + ll * 8] = dev_v2[(blk_idx_d1 * SIZE_SLICE_1_D1 + idx_d1 + (blk_idx_d2 * SIZE_SLICE_1_D2 + ll) * size_d1) * size_e + (threadIdx.x + l)];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1; ll++)
		{
			temp_bv[0] = sm_b[ll][idx_d1 + 0];
			temp_bv[1] = sm_b[ll][idx_d1 + 8];
			temp_bv[2] = sm_b[ll][idx_d1 + 16];
			temp_bv[3] = sm_b[ll][idx_d1 + 24];
			temp_bv[4] = sm_b[ll][idx_d1 + 32];
			temp_bv[5] = sm_b[ll][idx_d1 + 40];
			temp_bv[6] = sm_b[ll][idx_d1 + 48];
			temp_bv[7] = sm_b[ll][idx_d1 + 56];

			for (int xx = 0; xx < 4; xx++) // (1)
			{
				temp_av = sm_a[ll][idx_a + (idx_c) * SIZE_SLICE_1_A + (xx * 16)];

				reg_tile[0][xx] += temp_av * temp_bv[0];
				reg_tile[1][xx] += temp_av * temp_bv[1];
				reg_tile[2][xx] += temp_av * temp_bv[2];
				reg_tile[3][xx] += temp_av * temp_bv[3];
				reg_tile[4][xx] += temp_av * temp_bv[4];
				reg_tile[5][xx] += temp_av * temp_bv[5];
				reg_tile[6][xx] += temp_av * temp_bv[6];
				reg_tile[7][xx] += temp_av * temp_bv[7];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	if (idx_a < rng_a && idx_c < rng_c && idx_d1 < rng_d1)
	for (int i = 0; i < 8; i++)
	{
		for (int j = 0; j < 4; j++)
		{
			if(i < rng_d2 && j < rng_b)
			{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
			}
		}
	}
}

// created by tc_gen_code_Kernel()
__global__ void kernel__4_1(float* dev_t3, 
float* dev_t2, 
float* dev_v2, 
int size_a, int size_d1, int size_d2, int size_b, int size_c, int size_e, 
int numBlk_a, int numBlk_d1, int numBlk_d2, int numBlk_b, int numBlk_c, 
int stride_int_t2, int stride_int_v2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ float sm_a[16][64];
	__shared__ float sm_b[16][64];


	int internal_upperbound   = 0;
	int internal_offset;

	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 2
	// # of indices mapped on TB_Y: 1
	int idx_a = threadIdx.x % SIZE_SLICE_1_A;
	int idx_c = threadIdx.x / SIZE_SLICE_1_A;
	int idx_d1 = threadIdx.y;

	int tmp_blkIdx;
	int blk_idx_c = blockIdx.x / (numBlk_b * numBlk_d2 * numBlk_d1 * numBlk_a);
	tmp_blkIdx = blockIdx.x % (numBlk_b * numBlk_d2 * numBlk_d1 * numBlk_a);

	int blk_idx_b = tmp_blkIdx / (numBlk_d2 * numBlk_d1 * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_d2 * numBlk_d1 * numBlk_a);

	int blk_idx_d2 = tmp_blkIdx / (numBlk_d1 * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_d1 * numBlk_a);

	int blk_idx_d1 = tmp_blkIdx / numBlk_a;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a);

	int  blk_idx_a = tmp_blkIdx;

	int t3_base_thread = blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_d1 * SIZE_SLICE_1_D1 + idx_d1 + (blk_idx_d2 * SIZE_SLICE_1_D2 + (blk_idx_b * SIZE_SLICE_1_B + (blk_idx_c * SIZE_SLICE_1_C + idx_c) * size_b) * size_d2) * size_d1) * size_a;

	// need to support partial tiles
	int rng_a, rng_d1, rng_d2, rng_b, rng_c;
	if ((size_a - (blk_idx_a * SIZE_SLICE_1_A)) >= SIZE_SLICE_1_A)
	{
		rng_a = SIZE_SLICE_1_A;
	}
	else
	{
		rng_a = size_a % SIZE_SLICE_1_A;
	}
	if ((size_d1 - (blk_idx_d1 * SIZE_SLICE_1_D1)) >= SIZE_SLICE_1_D1)
	{
		rng_d1 = SIZE_SLICE_1_D1;
	}
	else
	{
		rng_d1 = size_d1 % SIZE_SLICE_1_D1;
	}
	if ((size_d2 - (blk_idx_d2 * SIZE_SLICE_1_D2)) >= SIZE_SLICE_1_D2)
	{
		rng_d2 = SIZE_SLICE_1_D2;
	}
	else
	{
		rng_d2 = size_d2 % SIZE_SLICE_1_D2;
	}
	if ((size_b - (blk_idx_b * SIZE_SLICE_1_B)) >= SIZE_SLICE_1_B)
	{
		rng_b = SIZE_SLICE_1_B;
	}
	else
	{
		rng_b = size_b % SIZE_SLICE_1_B;
	}
	if ((size_c - (blk_idx_c * SIZE_SLICE_1_C)) >= SIZE_SLICE_1_C)
	{
		rng_c = SIZE_SLICE_1_C;
	}
	else
	{
		rng_c = size_c % SIZE_SLICE_1_C;
	}

	float temp_av;
	float temp_bv[8];
	float reg_tile[8][4];

	for (int i = 0; i < 8; i++)
	for (int j = 0; j < 4; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'x', 't2', ['a', 'b', 'c', 'e']], [16, 'STR_SD2_V2_H7', 'y', 'v2', ['e', 'd1', 'd2']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		// Part: Generalized Contraction Index (p7b)
		internal_offset = (l + SIZE_INT_UNIT_1) - size_internal;
		if (internal_offset > 0) internal_upperbound = internal_offset;

		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (idx_a < rng_a && 0 < rng_c && threadIdx.y < SIZE_INT_UNIT_1 - internal_upperbound)
		for (int ll = 0; ll < rng_b; ll++)
		{
			// ['a', 'b', 'c', 'e']
			// Exception: Temp. version!: threadIdx.y + l + 0
			// Exception: Temp. version!: idx_a < rng_a
			sm_a[threadIdx.y + 0][threadIdx.x + ll * 16] = dev_t2[blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_b * SIZE_SLICE_1_B + ll + (blk_idx_c * SIZE_SLICE_1_C + 0) * size_b) * size_a + (threadIdx.y + l + 0) * stride_int_t2];
			// Exception: Temp. version!: threadIdx.y + l + 8
			// Exception: Temp. version!: idx_a < rng_a
			if (threadIdx.y + l + 8 < size_internal && idx_a < rng_a) 
			sm_a[threadIdx.y + 8][threadIdx.x + ll * 16] = dev_t2[blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_b * SIZE_SLICE_1_B + ll + (blk_idx_c * SIZE_SLICE_1_C + 0) * size_b) * size_a + (threadIdx.y + l + 8) * stride_int_t2];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (idx_d1 < rng_d1 && threadIdx.x < SIZE_INT_UNIT_1 - internal_upperbound)
		for (int ll = 0; ll < rng_d2; ll++)
		{
			// ['e', 'd1', 'd2']
			// Exception: Temp. version!: threadIdx.x + l
			// Exception: Temp. version!: idx_d1 < rng_d1
			sm_b[threadIdx.x][threadIdx.y + ll * 8] = dev_v2[(blk_idx_d1 * SIZE_SLICE_1_D1 + idx_d1 + (blk_idx_d2 * SIZE_SLICE_1_D2 + ll) * size_d1) * size_e + (threadIdx.x + l)];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1 - internal_upperbound; ll++)
		{
			temp_bv[0] = sm_b[ll][idx_d1 + 0];
			temp_bv[1] = sm_b[ll][idx_d1 + 8];
			temp_bv[2] = sm_b[ll][idx_d1 + 16];
			temp_bv[3] = sm_b[ll][idx_d1 + 24];
			temp_bv[4] = sm_b[ll][idx_d1 + 32];
			temp_bv[5] = sm_b[ll][idx_d1 + 40];
			temp_bv[6] = sm_b[ll][idx_d1 + 48];
			temp_bv[7] = sm_b[ll][idx_d1 + 56];

			for (int xx = 0; xx < 4; xx++) // (1)
			{
				temp_av = sm_a[ll][idx_a + (idx_c) * SIZE_SLICE_1_A + (xx * 16)];

				reg_tile[0][xx] += temp_av * temp_bv[0];
				reg_tile[1][xx] += temp_av * temp_bv[1];
				reg_tile[2][xx] += temp_av * temp_bv[2];
				reg_tile[3][xx] += temp_av * temp_bv[3];
				reg_tile[4][xx] += temp_av * temp_bv[4];
				reg_tile[5][xx] += temp_av * temp_bv[5];
				reg_tile[6][xx] += temp_av * temp_bv[6];
				reg_tile[7][xx] += temp_av * temp_bv[7];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	if (idx_a < rng_a && idx_c < rng_c && idx_d1 < rng_d1)
	for (int i = 0; i < 8; i++)
	{
		for (int j = 0; j < 4; j++)
		{
			if(i < rng_d2 && j < rng_b)
			{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
			}
		}
	}
}

// written by tc_interface.tc_gen_code_interface_Header()
extern "C"
void sd_t_d2_fusion(int size_a, int size_d1, int size_d2, int size_b, int size_c, int size_e, float* t3, float* host_t2, float* host_v2, int cond_kernel_1, int opt_register_transpose)
{
	int num_thread_blocks_kernel_1;

	float* dev_t3;
	float* dev_t2;
	float* dev_v2;


	num_thread_blocks_kernel_1 = CEIL(size_a, SIZE_SLICE_1_A) * CEIL(size_d1, SIZE_SLICE_1_D1) * CEIL(size_d2, SIZE_SLICE_1_D2) * CEIL(size_b, SIZE_SLICE_1_B) * CEIL(size_c, SIZE_SLICE_1_C);
	// cudaMalloc()
	hipMalloc((void**) &dev_t3, sizeof(float) * size_a * size_d1 * size_d2 * size_b * size_c);
	hipMalloc((void**) &dev_t2, sizeof(float) * size_e * size_c * size_b * size_a);
	hipMalloc((void**) &dev_v2, sizeof(float) * size_d2 * size_d1 * size_e);

	// cudaMemcpy()
	hipMemcpy(dev_t3, t3, sizeof(float) * size_a * size_d1 * size_d2 * size_b * size_c, hipMemcpyHostToDevice);
	hipMemcpy(dev_t2, host_t2, sizeof(float) * size_e * size_c * size_b * size_a, hipMemcpyHostToDevice);
	hipMemcpy(dev_v2, host_v2, sizeof(float) * size_d2 * size_d1 * size_e, hipMemcpyHostToDevice);

	// Related to Kernels
	// There are 1 Basic Kernels
	long long int tmp_operations = (long long int)(size_a * size_d1 * size_d2 * size_b * size_c) * size_e;
	printf ("========================================= fusedKernels =============================================\n");
	printf ("		Grid Size  : %6d (1D)\n", num_thread_blocks_kernel_1);
	printf ("		Block-size : %2d, %2d (2D)\n", SIZE_TB_1_X, SIZE_TB_1_Y);
	printf ("		Reg.-size  : %2d, %2d (2D)\n", SIZE_REG_1_X, SIZE_REG_1_Y);
	printf ("		A thread deals with (%d x %d) elements (basically)\n", SIZE_TB_1_X * SIZE_REG_1_X, SIZE_TB_1_Y * SIZE_REG_1_Y);
	printf ("		# of Operations: %lld\n", tmp_operations);
	printf ("====================================================================================================\n");
	dim3 gridsize_1(num_thread_blocks_kernel_1);
	dim3 blocksize_1(SIZE_TB_1_X, SIZE_TB_1_Y);

	int stride_output_a = 1;
	int stride_output_d1 = stride_output_a * size_a;
	int stride_output_d2 = stride_output_d1 * size_d1;
	int stride_output_b = stride_output_d2 * size_d2;
	int stride_output_c = stride_output_b * size_b;

	int stride_reg_x_1 = stride_output_b;
	int stride_reg_y_1 = stride_output_d2;

	int size_internal = size_e;

	int stride_int_t2 = size_a * size_b * size_c;
	int stride_int_v2 = 1;

	for (int i = 0; i < 100; i++) {
	// Decision Tree for Kernel Types
	// No Chance to Utilize the Register Transpose
	if (size_a % SIZE_SLICE_1_A == 0 && size_d1 % SIZE_SLICE_1_D1 == 0 && size_d2 % SIZE_SLICE_1_D2 == 0 && size_b % SIZE_SLICE_1_B == 0 && size_c % SIZE_SLICE_1_C == 0)
	{
		// [2] Extenral Index: Full
		if (size_e % SIZE_SLICE_1_E == 0)
		{
			// [3] Internal Index: Full
			// >>> External: Full && Internal: Full
			printf ("External: Full, Internal: Full\n");
			kernel__1_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a, size_d1, size_d2, size_b, size_c, size_e, CEIL(size_a, SIZE_SLICE_1_A), CEIL(size_d1, SIZE_SLICE_1_D1), CEIL(size_d2, SIZE_SLICE_1_D2), CEIL(size_b, SIZE_SLICE_1_B), CEIL(size_c, SIZE_SLICE_1_C), stride_int_t2, stride_int_v2, stride_reg_x_1, stride_reg_y_1, size_internal);
		}
		else
		{
			// [4] Internal Index: Partial
			// >>> External: Full && Internal: Partial
			printf ("External: Full, Internal: Partial\n");
			kernel__2_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a, size_d1, size_d2, size_b, size_c, size_e, CEIL(size_a, SIZE_SLICE_1_A), CEIL(size_d1, SIZE_SLICE_1_D1), CEIL(size_d2, SIZE_SLICE_1_D2), CEIL(size_b, SIZE_SLICE_1_B), CEIL(size_c, SIZE_SLICE_1_C), stride_int_t2, stride_int_v2, stride_reg_x_1, stride_reg_y_1, size_internal);
		}
	}
	else
	{
		// [2] Extenral Index: Partial
		if (size_e % SIZE_SLICE_1_E == 0)
		{
			// [3] Internal Index: Full
			// >>> External: Partial && Internal: Full
			printf ("External: Partial, Internal: Full\n");
			kernel__3_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a, size_d1, size_d2, size_b, size_c, size_e, CEIL(size_a, SIZE_SLICE_1_A), CEIL(size_d1, SIZE_SLICE_1_D1), CEIL(size_d2, SIZE_SLICE_1_D2), CEIL(size_b, SIZE_SLICE_1_B), CEIL(size_c, SIZE_SLICE_1_C), stride_int_t2, stride_int_v2, stride_reg_x_1, stride_reg_y_1, size_internal);
		}
		else
		{
			// [4] Internal Index: Partial
			// >>> External: Partial && Internal: Partial
			printf ("External: Partial, Internal: Partial\n");
			kernel__4_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a, size_d1, size_d2, size_b, size_c, size_e, CEIL(size_a, SIZE_SLICE_1_A), CEIL(size_d1, SIZE_SLICE_1_D1), CEIL(size_d2, SIZE_SLICE_1_D2), CEIL(size_b, SIZE_SLICE_1_B), CEIL(size_c, SIZE_SLICE_1_C), stride_int_t2, stride_int_v2, stride_reg_x_1, stride_reg_y_1, size_internal);
		}
	}

	}
	// Copy the Result from Device to Host
	hipMemcpy(t3, dev_t3, sizeof(float) * (size_a * size_d1 * size_d2 * size_b * size_c), hipMemcpyDeviceToHost);

	// cudaFree()
	hipFree(dev_t3);	hipFree(dev_t2);	hipFree(dev_v2);

	// Shoule be Fixed
	// HostFree

}

// This is written by tc_interface.tc_gen_code_interface()
// This Interface Should be Called to Run the Kernels
extern "C"
void sd_t_d2_fusion_(int size_a, int size_d, int size_b, int size_c, int size_e, float* t3, float* t2, float* v2, int cond_kernel_1, int opt_register_transpose)
{
	// Pre-Processing for Split
	// Based on Tile-Sizes and Problem-Size
	// Currently, one index can be split into two indices
	int size_d1;
	int size_d2;

	if (size_d % 8 == 0)
	{
		//
		size_d1 = 8;
		size_d2 = size_d / 8;
	}
	else
	{
		//
		size_d1 = size_d;
		size_d2 = 1;
	}

	// Call An Application
	sd_t_d2_fusion(size_a, size_d1, size_d2, size_b, size_c, size_e, t3, t2, v2, cond_kernel_1, opt_register_transpose);
}
