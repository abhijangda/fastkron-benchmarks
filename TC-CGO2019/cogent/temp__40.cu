// created by tc_code_include() in tc_code_include.py

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <sys/time.h>
#include <locale.h>
#include <algorithm>
using namespace std;

// created by tc_gen_definition_new()
#define SIZE_SLICE_1_G 8
#define SIZE_SLICE_1_A 16
#define SIZE_SLICE_1_B 4
#define SIZE_SLICE_1_C 1
#define SIZE_SLICE_1_D 1
#define SIZE_SLICE_1_E 1
#define SIZE_SLICE_1_F1 8
#define SIZE_SLICE_1_F2 8

#define SIZE_INT_UNIT_1 SIZE_SLICE_1_G

#define SIZE_TB_1_X 	SIZE_SLICE_1_A * SIZE_SLICE_1_C * SIZE_SLICE_1_D * SIZE_SLICE_1_E
#define SIZE_TB_1_Y 	SIZE_SLICE_1_F1
#define SIZE_REG_1_X 	SIZE_SLICE_1_B
#define SIZE_REG_1_Y 	SIZE_SLICE_1_F2

#define NUM_INDEX 		7
#define CEIL(a, b) 		(((a) + (b) - 1) / (b))

// created by tc_gen_code_Kernel()
__global__ void kernel__1_1(float* dev_t3, 
float* dev_t2, 
float* dev_v2, 
int size_a, int size_f1, int size_f2, int size_b, int size_c, int size_d, int size_e, int size_g, 
int numBlk_a, int numBlk_f1, int numBlk_f2, int numBlk_b, int numBlk_c, int numBlk_d, int numBlk_e, 
int stride_int_t2, int stride_int_v2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ float sm_a[8][64];
	__shared__ float sm_b[8][64];


	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 4
	// # of indices mapped on TB_Y: 1
	int idx_e = threadIdx.x / (SIZE_SLICE_1_D * SIZE_SLICE_1_A);
	int remaining_idx = threadIdx.x % (SIZE_SLICE_1_D * SIZE_SLICE_1_A);
	int idx_d = remaining_idx / (SIZE_SLICE_1_C * SIZE_SLICE_1_A);
	remaining_idx = remaining_idx % (SIZE_SLICE_1_C * SIZE_SLICE_1_A);
	int idx_c = remaining_idx / (SIZE_SLICE_1_A);
	remaining_idx = remaining_idx % (SIZE_SLICE_1_A);
	int idx_a = remaining_idx;
	int idx_f1 = threadIdx.y;

	int tmp_blkIdx;
	int blk_idx_e = blockIdx.x / (numBlk_d * numBlk_c * numBlk_b * numBlk_f2 * numBlk_f1 * numBlk_a);
	tmp_blkIdx = blockIdx.x % (numBlk_d * numBlk_c * numBlk_b * numBlk_f2 * numBlk_f1 * numBlk_a);

	int blk_idx_d = tmp_blkIdx / (numBlk_c * numBlk_b * numBlk_f2 * numBlk_f1 * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_c * numBlk_b * numBlk_f2 * numBlk_f1 * numBlk_a);

	int blk_idx_c = tmp_blkIdx / (numBlk_b * numBlk_f2 * numBlk_f1 * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_b * numBlk_f2 * numBlk_f1 * numBlk_a);

	int blk_idx_b = tmp_blkIdx / (numBlk_f2 * numBlk_f1 * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_f2 * numBlk_f1 * numBlk_a);

	int blk_idx_f2 = tmp_blkIdx / (numBlk_f1 * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_f1 * numBlk_a);

	int blk_idx_f1 = tmp_blkIdx / numBlk_a;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a);

	int  blk_idx_a = tmp_blkIdx;

	int t3_base_thread = blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_f1 * SIZE_SLICE_1_F1 + idx_f1 + (blk_idx_f2 * SIZE_SLICE_1_F2 + (blk_idx_b * SIZE_SLICE_1_B + (blk_idx_c * SIZE_SLICE_1_C + idx_c + (blk_idx_d * SIZE_SLICE_1_D + idx_d + (blk_idx_e * SIZE_SLICE_1_E + idx_e) * size_d) * size_c) * size_b) * size_f2) * size_f1) * size_a;


	float temp_av;
	float temp_bv[8];
	float reg_tile[8][4];

	for (int i = 0; i < 8; i++)
	for (int j = 0; j < 4; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'x', 't2', ['a', 'b', 'c', 'd', 'e', 'g']], [16, 'STR_SD2_V2_H7', 'y', 'v2', ['g', 'f1', 'f2']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		// No Need to Put Boundary-Checks before For-Statement: : 
		for (int ll = 0; ll < 4; ll++)
		{
			// ['a', 'b', 'c', 'd', 'e', 'g']
			// Exception: Temp. version!: threadIdx.y + l
			// Exception: Temp. version!: idx_a < rng_a
			sm_a[threadIdx.y][threadIdx.x + ll * 16] = dev_t2[blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_b * SIZE_SLICE_1_B + ll + (blk_idx_c * SIZE_SLICE_1_C + 0 + (blk_idx_d * SIZE_SLICE_1_D + 0 + (blk_idx_e * SIZE_SLICE_1_E + 0) * size_d) * size_c) * size_b) * size_a + (threadIdx.y + l) * stride_int_t2];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (threadIdx.x < 8 && threadIdx.x < 8)
		for (int ll = 0; ll < 8; ll++)
		{
			// ['g', 'f1', 'f2']
			// Exception: Temp. version!: threadIdx.x + l
			// Exception: Temp. version!: idx_f1 < rng_f1
			sm_b[threadIdx.x][threadIdx.y + ll * 8] = dev_v2[(blk_idx_f1 * SIZE_SLICE_1_F1 + idx_f1 + (blk_idx_f2 * SIZE_SLICE_1_F2 + ll) * size_f1) * size_g + (threadIdx.x + l)];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1; ll++)
		{
			temp_bv[0] = sm_b[ll][idx_f1 + 0];
			temp_bv[1] = sm_b[ll][idx_f1 + 8];
			temp_bv[2] = sm_b[ll][idx_f1 + 16];
			temp_bv[3] = sm_b[ll][idx_f1 + 24];
			temp_bv[4] = sm_b[ll][idx_f1 + 32];
			temp_bv[5] = sm_b[ll][idx_f1 + 40];
			temp_bv[6] = sm_b[ll][idx_f1 + 48];
			temp_bv[7] = sm_b[ll][idx_f1 + 56];

			for (int xx = 0; xx < 4; xx++) // (1)
			{
				temp_av = sm_a[ll][idx_a + (idx_c + (idx_d + (idx_e) * SIZE_SLICE_1_D) * SIZE_SLICE_1_C) * SIZE_SLICE_1_A + (xx * 16)];

				reg_tile[0][xx] += temp_av * temp_bv[0];
				reg_tile[1][xx] += temp_av * temp_bv[1];
				reg_tile[2][xx] += temp_av * temp_bv[2];
				reg_tile[3][xx] += temp_av * temp_bv[3];
				reg_tile[4][xx] += temp_av * temp_bv[4];
				reg_tile[5][xx] += temp_av * temp_bv[5];
				reg_tile[6][xx] += temp_av * temp_bv[6];
				reg_tile[7][xx] += temp_av * temp_bv[7];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	#pragma unroll 8
	for (int i = 0; i < 8; i++)
	{
		for (int j = 0; j < 4; j++)
		{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
		}
	}
}

// created by tc_gen_code_Kernel()
__global__ void kernel__2_1(float* dev_t3, 
float* dev_t2, 
float* dev_v2, 
int size_a, int size_f1, int size_f2, int size_b, int size_c, int size_d, int size_e, int size_g, 
int numBlk_a, int numBlk_f1, int numBlk_f2, int numBlk_b, int numBlk_c, int numBlk_d, int numBlk_e, 
int stride_int_t2, int stride_int_v2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ float sm_a[8][64];
	__shared__ float sm_b[8][64];


	int internal_upperbound   = 0;
	int internal_offset;

	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 4
	// # of indices mapped on TB_Y: 1
	int idx_e = threadIdx.x / (SIZE_SLICE_1_D * SIZE_SLICE_1_A);
	int remaining_idx = threadIdx.x % (SIZE_SLICE_1_D * SIZE_SLICE_1_A);
	int idx_d = remaining_idx / (SIZE_SLICE_1_C * SIZE_SLICE_1_A);
	remaining_idx = remaining_idx % (SIZE_SLICE_1_C * SIZE_SLICE_1_A);
	int idx_c = remaining_idx / (SIZE_SLICE_1_A);
	remaining_idx = remaining_idx % (SIZE_SLICE_1_A);
	int idx_a = remaining_idx;
	int idx_f1 = threadIdx.y;

	int tmp_blkIdx;
	int blk_idx_e = blockIdx.x / (numBlk_d * numBlk_c * numBlk_b * numBlk_f2 * numBlk_f1 * numBlk_a);
	tmp_blkIdx = blockIdx.x % (numBlk_d * numBlk_c * numBlk_b * numBlk_f2 * numBlk_f1 * numBlk_a);

	int blk_idx_d = tmp_blkIdx / (numBlk_c * numBlk_b * numBlk_f2 * numBlk_f1 * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_c * numBlk_b * numBlk_f2 * numBlk_f1 * numBlk_a);

	int blk_idx_c = tmp_blkIdx / (numBlk_b * numBlk_f2 * numBlk_f1 * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_b * numBlk_f2 * numBlk_f1 * numBlk_a);

	int blk_idx_b = tmp_blkIdx / (numBlk_f2 * numBlk_f1 * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_f2 * numBlk_f1 * numBlk_a);

	int blk_idx_f2 = tmp_blkIdx / (numBlk_f1 * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_f1 * numBlk_a);

	int blk_idx_f1 = tmp_blkIdx / numBlk_a;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a);

	int  blk_idx_a = tmp_blkIdx;

	int t3_base_thread = blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_f1 * SIZE_SLICE_1_F1 + idx_f1 + (blk_idx_f2 * SIZE_SLICE_1_F2 + (blk_idx_b * SIZE_SLICE_1_B + (blk_idx_c * SIZE_SLICE_1_C + idx_c + (blk_idx_d * SIZE_SLICE_1_D + idx_d + (blk_idx_e * SIZE_SLICE_1_E + idx_e) * size_d) * size_c) * size_b) * size_f2) * size_f1) * size_a;


	float temp_av;
	float temp_bv[8];
	float reg_tile[8][4];

	for (int i = 0; i < 8; i++)
	for (int j = 0; j < 4; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'x', 't2', ['a', 'b', 'c', 'd', 'e', 'g']], [16, 'STR_SD2_V2_H7', 'y', 'v2', ['g', 'f1', 'f2']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		// Part: Generalized Contraction Index (p7b)
		internal_offset = (l + SIZE_INT_UNIT_1) - size_internal;
		if (internal_offset > 0) internal_upperbound = internal_offset;

		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (threadIdx.y < SIZE_INT_UNIT_1 - internal_upperbound)
		for (int ll = 0; ll < 4; ll++)
		{
			// ['a', 'b', 'c', 'd', 'e', 'g']
			// Exception: Temp. version!: threadIdx.y + l
			// Exception: Temp. version!: idx_a < rng_a
			sm_a[threadIdx.y][threadIdx.x + ll * 16] = dev_t2[blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_b * SIZE_SLICE_1_B + ll + (blk_idx_c * SIZE_SLICE_1_C + 0 + (blk_idx_d * SIZE_SLICE_1_D + 0 + (blk_idx_e * SIZE_SLICE_1_E + 0) * size_d) * size_c) * size_b) * size_a + (threadIdx.y + l) * stride_int_t2];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (threadIdx.x < SIZE_INT_UNIT_1 - internal_upperbound && threadIdx.x < 8 && threadIdx.x < 8)
		for (int ll = 0; ll < 8; ll++)
		{
			// ['g', 'f1', 'f2']
			// Exception: Temp. version!: threadIdx.x + l
			// Exception: Temp. version!: idx_f1 < rng_f1
			sm_b[threadIdx.x][threadIdx.y + ll * 8] = dev_v2[(blk_idx_f1 * SIZE_SLICE_1_F1 + idx_f1 + (blk_idx_f2 * SIZE_SLICE_1_F2 + ll) * size_f1) * size_g + (threadIdx.x + l)];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1 - internal_upperbound; ll++)
		{
			temp_bv[0] = sm_b[ll][idx_f1 + 0];
			temp_bv[1] = sm_b[ll][idx_f1 + 8];
			temp_bv[2] = sm_b[ll][idx_f1 + 16];
			temp_bv[3] = sm_b[ll][idx_f1 + 24];
			temp_bv[4] = sm_b[ll][idx_f1 + 32];
			temp_bv[5] = sm_b[ll][idx_f1 + 40];
			temp_bv[6] = sm_b[ll][idx_f1 + 48];
			temp_bv[7] = sm_b[ll][idx_f1 + 56];

			for (int xx = 0; xx < 4; xx++) // (1)
			{
				temp_av = sm_a[ll][idx_a + (idx_c + (idx_d + (idx_e) * SIZE_SLICE_1_D) * SIZE_SLICE_1_C) * SIZE_SLICE_1_A + (xx * 16)];

				reg_tile[0][xx] += temp_av * temp_bv[0];
				reg_tile[1][xx] += temp_av * temp_bv[1];
				reg_tile[2][xx] += temp_av * temp_bv[2];
				reg_tile[3][xx] += temp_av * temp_bv[3];
				reg_tile[4][xx] += temp_av * temp_bv[4];
				reg_tile[5][xx] += temp_av * temp_bv[5];
				reg_tile[6][xx] += temp_av * temp_bv[6];
				reg_tile[7][xx] += temp_av * temp_bv[7];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	#pragma unroll 8
	for (int i = 0; i < 8; i++)
	{
		for (int j = 0; j < 4; j++)
		{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
		}
	}
}

// created by tc_gen_code_Kernel()
__global__ void kernel__3_1(float* dev_t3, 
float* dev_t2, 
float* dev_v2, 
int size_a, int size_f1, int size_f2, int size_b, int size_c, int size_d, int size_e, int size_g, 
int numBlk_a, int numBlk_f1, int numBlk_f2, int numBlk_b, int numBlk_c, int numBlk_d, int numBlk_e, 
int stride_int_t2, int stride_int_v2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ float sm_a[8][64];
	__shared__ float sm_b[8][64];


	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 4
	// # of indices mapped on TB_Y: 1
	int idx_e = threadIdx.x / (SIZE_SLICE_1_D * SIZE_SLICE_1_A);
	int remaining_idx = threadIdx.x % (SIZE_SLICE_1_D * SIZE_SLICE_1_A);
	int idx_d = remaining_idx / (SIZE_SLICE_1_C * SIZE_SLICE_1_A);
	remaining_idx = remaining_idx % (SIZE_SLICE_1_C * SIZE_SLICE_1_A);
	int idx_c = remaining_idx / (SIZE_SLICE_1_A);
	remaining_idx = remaining_idx % (SIZE_SLICE_1_A);
	int idx_a = remaining_idx;
	int idx_f1 = threadIdx.y;

	int tmp_blkIdx;
	int blk_idx_e = blockIdx.x / (numBlk_d * numBlk_c * numBlk_b * numBlk_f2 * numBlk_f1 * numBlk_a);
	tmp_blkIdx = blockIdx.x % (numBlk_d * numBlk_c * numBlk_b * numBlk_f2 * numBlk_f1 * numBlk_a);

	int blk_idx_d = tmp_blkIdx / (numBlk_c * numBlk_b * numBlk_f2 * numBlk_f1 * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_c * numBlk_b * numBlk_f2 * numBlk_f1 * numBlk_a);

	int blk_idx_c = tmp_blkIdx / (numBlk_b * numBlk_f2 * numBlk_f1 * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_b * numBlk_f2 * numBlk_f1 * numBlk_a);

	int blk_idx_b = tmp_blkIdx / (numBlk_f2 * numBlk_f1 * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_f2 * numBlk_f1 * numBlk_a);

	int blk_idx_f2 = tmp_blkIdx / (numBlk_f1 * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_f1 * numBlk_a);

	int blk_idx_f1 = tmp_blkIdx / numBlk_a;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a);

	int  blk_idx_a = tmp_blkIdx;

	int t3_base_thread = blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_f1 * SIZE_SLICE_1_F1 + idx_f1 + (blk_idx_f2 * SIZE_SLICE_1_F2 + (blk_idx_b * SIZE_SLICE_1_B + (blk_idx_c * SIZE_SLICE_1_C + idx_c + (blk_idx_d * SIZE_SLICE_1_D + idx_d + (blk_idx_e * SIZE_SLICE_1_E + idx_e) * size_d) * size_c) * size_b) * size_f2) * size_f1) * size_a;

	// need to support partial tiles
	int rng_a, rng_f1, rng_f2, rng_b, rng_c, rng_d, rng_e;
	if ((size_a - (blk_idx_a * SIZE_SLICE_1_A)) >= SIZE_SLICE_1_A)
	{
		rng_a = SIZE_SLICE_1_A;
	}
	else
	{
		rng_a = size_a % SIZE_SLICE_1_A;
	}
	if ((size_f1 - (blk_idx_f1 * SIZE_SLICE_1_F1)) >= SIZE_SLICE_1_F1)
	{
		rng_f1 = SIZE_SLICE_1_F1;
	}
	else
	{
		rng_f1 = size_f1 % SIZE_SLICE_1_F1;
	}
	if ((size_f2 - (blk_idx_f2 * SIZE_SLICE_1_F2)) >= SIZE_SLICE_1_F2)
	{
		rng_f2 = SIZE_SLICE_1_F2;
	}
	else
	{
		rng_f2 = size_f2 % SIZE_SLICE_1_F2;
	}
	if ((size_b - (blk_idx_b * SIZE_SLICE_1_B)) >= SIZE_SLICE_1_B)
	{
		rng_b = SIZE_SLICE_1_B;
	}
	else
	{
		rng_b = size_b % SIZE_SLICE_1_B;
	}
	if ((size_c - (blk_idx_c * SIZE_SLICE_1_C)) >= SIZE_SLICE_1_C)
	{
		rng_c = SIZE_SLICE_1_C;
	}
	else
	{
		rng_c = size_c % SIZE_SLICE_1_C;
	}
	if ((size_d - (blk_idx_d * SIZE_SLICE_1_D)) >= SIZE_SLICE_1_D)
	{
		rng_d = SIZE_SLICE_1_D;
	}
	else
	{
		rng_d = size_d % SIZE_SLICE_1_D;
	}
	if ((size_e - (blk_idx_e * SIZE_SLICE_1_E)) >= SIZE_SLICE_1_E)
	{
		rng_e = SIZE_SLICE_1_E;
	}
	else
	{
		rng_e = size_e % SIZE_SLICE_1_E;
	}

	float temp_av;
	float temp_bv[8];
	float reg_tile[8][4];

	for (int i = 0; i < 8; i++)
	for (int j = 0; j < 4; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'x', 't2', ['a', 'b', 'c', 'd', 'e', 'g']], [16, 'STR_SD2_V2_H7', 'y', 'v2', ['g', 'f1', 'f2']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (idx_a < rng_a && 0 < rng_c && 0 < rng_d && 0 < rng_e)
		for (int ll = 0; ll < rng_b; ll++)
		{
			// ['a', 'b', 'c', 'd', 'e', 'g']
			// Exception: Temp. version!: threadIdx.y + l
			// Exception: Temp. version!: idx_a < rng_a
			sm_a[threadIdx.y][threadIdx.x + ll * 16] = dev_t2[blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_b * SIZE_SLICE_1_B + ll + (blk_idx_c * SIZE_SLICE_1_C + 0 + (blk_idx_d * SIZE_SLICE_1_D + 0 + (blk_idx_e * SIZE_SLICE_1_E + 0) * size_d) * size_c) * size_b) * size_a + (threadIdx.y + l) * stride_int_t2];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (idx_f1 < rng_f1 && threadIdx.x < 8 && threadIdx.x < 8)
		for (int ll = 0; ll < rng_f2; ll++)
		{
			// ['g', 'f1', 'f2']
			// Exception: Temp. version!: threadIdx.x + l
			// Exception: Temp. version!: idx_f1 < rng_f1
			sm_b[threadIdx.x][threadIdx.y + ll * 8] = dev_v2[(blk_idx_f1 * SIZE_SLICE_1_F1 + idx_f1 + (blk_idx_f2 * SIZE_SLICE_1_F2 + ll) * size_f1) * size_g + (threadIdx.x + l)];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1; ll++)
		{
			temp_bv[0] = sm_b[ll][idx_f1 + 0];
			temp_bv[1] = sm_b[ll][idx_f1 + 8];
			temp_bv[2] = sm_b[ll][idx_f1 + 16];
			temp_bv[3] = sm_b[ll][idx_f1 + 24];
			temp_bv[4] = sm_b[ll][idx_f1 + 32];
			temp_bv[5] = sm_b[ll][idx_f1 + 40];
			temp_bv[6] = sm_b[ll][idx_f1 + 48];
			temp_bv[7] = sm_b[ll][idx_f1 + 56];

			for (int xx = 0; xx < 4; xx++) // (1)
			{
				temp_av = sm_a[ll][idx_a + (idx_c + (idx_d + (idx_e) * SIZE_SLICE_1_D) * SIZE_SLICE_1_C) * SIZE_SLICE_1_A + (xx * 16)];

				reg_tile[0][xx] += temp_av * temp_bv[0];
				reg_tile[1][xx] += temp_av * temp_bv[1];
				reg_tile[2][xx] += temp_av * temp_bv[2];
				reg_tile[3][xx] += temp_av * temp_bv[3];
				reg_tile[4][xx] += temp_av * temp_bv[4];
				reg_tile[5][xx] += temp_av * temp_bv[5];
				reg_tile[6][xx] += temp_av * temp_bv[6];
				reg_tile[7][xx] += temp_av * temp_bv[7];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	if (idx_a < rng_a && idx_c < rng_c && idx_d < rng_d && idx_e < rng_e && idx_f1 < rng_f1)
	for (int i = 0; i < 8; i++)
	{
		for (int j = 0; j < 4; j++)
		{
			if(i < rng_f2 && j < rng_b)
			{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
			}
		}
	}
}

// created by tc_gen_code_Kernel()
__global__ void kernel__4_1(float* dev_t3, 
float* dev_t2, 
float* dev_v2, 
int size_a, int size_f1, int size_f2, int size_b, int size_c, int size_d, int size_e, int size_g, 
int numBlk_a, int numBlk_f1, int numBlk_f2, int numBlk_b, int numBlk_c, int numBlk_d, int numBlk_e, 
int stride_int_t2, int stride_int_v2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ float sm_a[8][64];
	__shared__ float sm_b[8][64];


	int internal_upperbound   = 0;
	int internal_offset;

	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 4
	// # of indices mapped on TB_Y: 1
	int idx_e = threadIdx.x / (SIZE_SLICE_1_D * SIZE_SLICE_1_A);
	int remaining_idx = threadIdx.x % (SIZE_SLICE_1_D * SIZE_SLICE_1_A);
	int idx_d = remaining_idx / (SIZE_SLICE_1_C * SIZE_SLICE_1_A);
	remaining_idx = remaining_idx % (SIZE_SLICE_1_C * SIZE_SLICE_1_A);
	int idx_c = remaining_idx / (SIZE_SLICE_1_A);
	remaining_idx = remaining_idx % (SIZE_SLICE_1_A);
	int idx_a = remaining_idx;
	int idx_f1 = threadIdx.y;

	int tmp_blkIdx;
	int blk_idx_e = blockIdx.x / (numBlk_d * numBlk_c * numBlk_b * numBlk_f2 * numBlk_f1 * numBlk_a);
	tmp_blkIdx = blockIdx.x % (numBlk_d * numBlk_c * numBlk_b * numBlk_f2 * numBlk_f1 * numBlk_a);

	int blk_idx_d = tmp_blkIdx / (numBlk_c * numBlk_b * numBlk_f2 * numBlk_f1 * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_c * numBlk_b * numBlk_f2 * numBlk_f1 * numBlk_a);

	int blk_idx_c = tmp_blkIdx / (numBlk_b * numBlk_f2 * numBlk_f1 * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_b * numBlk_f2 * numBlk_f1 * numBlk_a);

	int blk_idx_b = tmp_blkIdx / (numBlk_f2 * numBlk_f1 * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_f2 * numBlk_f1 * numBlk_a);

	int blk_idx_f2 = tmp_blkIdx / (numBlk_f1 * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_f1 * numBlk_a);

	int blk_idx_f1 = tmp_blkIdx / numBlk_a;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a);

	int  blk_idx_a = tmp_blkIdx;

	int t3_base_thread = blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_f1 * SIZE_SLICE_1_F1 + idx_f1 + (blk_idx_f2 * SIZE_SLICE_1_F2 + (blk_idx_b * SIZE_SLICE_1_B + (blk_idx_c * SIZE_SLICE_1_C + idx_c + (blk_idx_d * SIZE_SLICE_1_D + idx_d + (blk_idx_e * SIZE_SLICE_1_E + idx_e) * size_d) * size_c) * size_b) * size_f2) * size_f1) * size_a;

	// need to support partial tiles
	int rng_a, rng_f1, rng_f2, rng_b, rng_c, rng_d, rng_e;
	if ((size_a - (blk_idx_a * SIZE_SLICE_1_A)) >= SIZE_SLICE_1_A)
	{
		rng_a = SIZE_SLICE_1_A;
	}
	else
	{
		rng_a = size_a % SIZE_SLICE_1_A;
	}
	if ((size_f1 - (blk_idx_f1 * SIZE_SLICE_1_F1)) >= SIZE_SLICE_1_F1)
	{
		rng_f1 = SIZE_SLICE_1_F1;
	}
	else
	{
		rng_f1 = size_f1 % SIZE_SLICE_1_F1;
	}
	if ((size_f2 - (blk_idx_f2 * SIZE_SLICE_1_F2)) >= SIZE_SLICE_1_F2)
	{
		rng_f2 = SIZE_SLICE_1_F2;
	}
	else
	{
		rng_f2 = size_f2 % SIZE_SLICE_1_F2;
	}
	if ((size_b - (blk_idx_b * SIZE_SLICE_1_B)) >= SIZE_SLICE_1_B)
	{
		rng_b = SIZE_SLICE_1_B;
	}
	else
	{
		rng_b = size_b % SIZE_SLICE_1_B;
	}
	if ((size_c - (blk_idx_c * SIZE_SLICE_1_C)) >= SIZE_SLICE_1_C)
	{
		rng_c = SIZE_SLICE_1_C;
	}
	else
	{
		rng_c = size_c % SIZE_SLICE_1_C;
	}
	if ((size_d - (blk_idx_d * SIZE_SLICE_1_D)) >= SIZE_SLICE_1_D)
	{
		rng_d = SIZE_SLICE_1_D;
	}
	else
	{
		rng_d = size_d % SIZE_SLICE_1_D;
	}
	if ((size_e - (blk_idx_e * SIZE_SLICE_1_E)) >= SIZE_SLICE_1_E)
	{
		rng_e = SIZE_SLICE_1_E;
	}
	else
	{
		rng_e = size_e % SIZE_SLICE_1_E;
	}

	float temp_av;
	float temp_bv[8];
	float reg_tile[8][4];

	for (int i = 0; i < 8; i++)
	for (int j = 0; j < 4; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'x', 't2', ['a', 'b', 'c', 'd', 'e', 'g']], [16, 'STR_SD2_V2_H7', 'y', 'v2', ['g', 'f1', 'f2']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		// Part: Generalized Contraction Index (p7b)
		internal_offset = (l + SIZE_INT_UNIT_1) - size_internal;
		if (internal_offset > 0) internal_upperbound = internal_offset;

		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (idx_a < rng_a && 0 < rng_c && 0 < rng_d && 0 < rng_e && threadIdx.y < SIZE_INT_UNIT_1 - internal_upperbound)
		for (int ll = 0; ll < rng_b; ll++)
		{
			// ['a', 'b', 'c', 'd', 'e', 'g']
			// Exception: Temp. version!: threadIdx.y + l
			// Exception: Temp. version!: idx_a < rng_a
			sm_a[threadIdx.y][threadIdx.x + ll * 16] = dev_t2[blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_b * SIZE_SLICE_1_B + ll + (blk_idx_c * SIZE_SLICE_1_C + 0 + (blk_idx_d * SIZE_SLICE_1_D + 0 + (blk_idx_e * SIZE_SLICE_1_E + 0) * size_d) * size_c) * size_b) * size_a + (threadIdx.y + l) * stride_int_t2];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (idx_f1 < rng_f1 && threadIdx.x < SIZE_INT_UNIT_1 - internal_upperbound && threadIdx.x < 8 && threadIdx.x < 8)
		for (int ll = 0; ll < rng_f2; ll++)
		{
			// ['g', 'f1', 'f2']
			// Exception: Temp. version!: threadIdx.x + l
			// Exception: Temp. version!: idx_f1 < rng_f1
			sm_b[threadIdx.x][threadIdx.y + ll * 8] = dev_v2[(blk_idx_f1 * SIZE_SLICE_1_F1 + idx_f1 + (blk_idx_f2 * SIZE_SLICE_1_F2 + ll) * size_f1) * size_g + (threadIdx.x + l)];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1 - internal_upperbound; ll++)
		{
			temp_bv[0] = sm_b[ll][idx_f1 + 0];
			temp_bv[1] = sm_b[ll][idx_f1 + 8];
			temp_bv[2] = sm_b[ll][idx_f1 + 16];
			temp_bv[3] = sm_b[ll][idx_f1 + 24];
			temp_bv[4] = sm_b[ll][idx_f1 + 32];
			temp_bv[5] = sm_b[ll][idx_f1 + 40];
			temp_bv[6] = sm_b[ll][idx_f1 + 48];
			temp_bv[7] = sm_b[ll][idx_f1 + 56];

			for (int xx = 0; xx < 4; xx++) // (1)
			{
				temp_av = sm_a[ll][idx_a + (idx_c + (idx_d + (idx_e) * SIZE_SLICE_1_D) * SIZE_SLICE_1_C) * SIZE_SLICE_1_A + (xx * 16)];

				reg_tile[0][xx] += temp_av * temp_bv[0];
				reg_tile[1][xx] += temp_av * temp_bv[1];
				reg_tile[2][xx] += temp_av * temp_bv[2];
				reg_tile[3][xx] += temp_av * temp_bv[3];
				reg_tile[4][xx] += temp_av * temp_bv[4];
				reg_tile[5][xx] += temp_av * temp_bv[5];
				reg_tile[6][xx] += temp_av * temp_bv[6];
				reg_tile[7][xx] += temp_av * temp_bv[7];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	if (idx_a < rng_a && idx_c < rng_c && idx_d < rng_d && idx_e < rng_e && idx_f1 < rng_f1)
	for (int i = 0; i < 8; i++)
	{
		for (int j = 0; j < 4; j++)
		{
			if(i < rng_f2 && j < rng_b)
			{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
			}
		}
	}
}

// written by tc_interface.tc_gen_code_interface_Header()
extern "C"
void sd_t_d2_fusion(int size_a, int size_f1, int size_f2, int size_b, int size_c, int size_d, int size_e, int size_g, float* t3, float* host_t2, float* host_v2, int cond_kernel_1, int opt_register_transpose)
{
	int num_thread_blocks_kernel_1;

	float* dev_t3;
	float* dev_t2;
	float* dev_v2;


	num_thread_blocks_kernel_1 = CEIL(size_a, SIZE_SLICE_1_A) * CEIL(size_f1, SIZE_SLICE_1_F1) * CEIL(size_f2, SIZE_SLICE_1_F2) * CEIL(size_b, SIZE_SLICE_1_B) * CEIL(size_c, SIZE_SLICE_1_C) * CEIL(size_d, SIZE_SLICE_1_D) * CEIL(size_e, SIZE_SLICE_1_E);
	// cudaMalloc()
	hipMalloc((void**) &dev_t3, sizeof(float) * size_a * size_f1 * size_f2 * size_b * size_c * size_d * size_e);
	hipMalloc((void**) &dev_t2, sizeof(float) * size_g * size_e * size_d * size_c * size_b * size_a);
	hipMalloc((void**) &dev_v2, sizeof(float) * size_f2 * size_f1 * size_g);

	// cudaMemcpy()
	hipMemcpy(dev_t3, t3, sizeof(float) * size_a * size_f1 * size_f2 * size_b * size_c * size_d * size_e, hipMemcpyHostToDevice);
	hipMemcpy(dev_t2, host_t2, sizeof(float) * size_g * size_e * size_d * size_c * size_b * size_a, hipMemcpyHostToDevice);
	hipMemcpy(dev_v2, host_v2, sizeof(float) * size_f2 * size_f1 * size_g, hipMemcpyHostToDevice);

	// Related to Kernels
	// There are 1 Basic Kernels
	long long int tmp_operations = (long long int)(size_a * size_f1 * size_f2 * size_b * size_c * size_d * size_e) * size_g;
	printf ("========================================= fusedKernels =============================================\n");
	printf ("		Grid Size  : %6d (1D)\n", num_thread_blocks_kernel_1);
	printf ("		Block-size : %2d, %2d (2D)\n", SIZE_TB_1_X, SIZE_TB_1_Y);
	printf ("		Reg.-size  : %2d, %2d (2D)\n", SIZE_REG_1_X, SIZE_REG_1_Y);
	printf ("		A thread deals with (%d x %d) elements (basically)\n", SIZE_TB_1_X * SIZE_REG_1_X, SIZE_TB_1_Y * SIZE_REG_1_Y);
	printf ("		# of Operations: %lld\n", tmp_operations);
	printf ("====================================================================================================\n");
	dim3 gridsize_1(num_thread_blocks_kernel_1);
	dim3 blocksize_1(SIZE_TB_1_X, SIZE_TB_1_Y);

	int stride_output_a = 1;
	int stride_output_f1 = stride_output_a * size_a;
	int stride_output_f2 = stride_output_f1 * size_f1;
	int stride_output_b = stride_output_f2 * size_f2;
	int stride_output_c = stride_output_b * size_b;
	int stride_output_d = stride_output_c * size_c;
	int stride_output_e = stride_output_d * size_d;

	int stride_reg_x_1 = stride_output_b;
	int stride_reg_y_1 = stride_output_f2;

	int size_internal = size_g;

	int stride_int_t2 = size_a * size_b * size_c * size_d * size_e;
	int stride_int_v2 = 1;

	 hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);	for (int i = 0; i < 110; i++) {
	if (i==9)hipEventRecord(start);
	// Decision Tree for Kernel Types
	// No Chance to Utilize the Register Transpose
	if (size_a % SIZE_SLICE_1_A == 0 && size_f1 % SIZE_SLICE_1_F1 == 0 && size_f2 % SIZE_SLICE_1_F2 == 0 && size_b % SIZE_SLICE_1_B == 0 && size_c % SIZE_SLICE_1_C == 0 && size_d % SIZE_SLICE_1_D == 0 && size_e % SIZE_SLICE_1_E == 0)
	{
		// [2] Extenral Index: Full
		if (size_g % SIZE_SLICE_1_G == 0)
		{
			// [3] Internal Index: Full
			// >>> External: Full && Internal: Full
			printf ("External: Full, Internal: Full\n");
			kernel__1_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a, size_f1, size_f2, size_b, size_c, size_d, size_e, size_g, CEIL(size_a, SIZE_SLICE_1_A), CEIL(size_f1, SIZE_SLICE_1_F1), CEIL(size_f2, SIZE_SLICE_1_F2), CEIL(size_b, SIZE_SLICE_1_B), CEIL(size_c, SIZE_SLICE_1_C), CEIL(size_d, SIZE_SLICE_1_D), CEIL(size_e, SIZE_SLICE_1_E), stride_int_t2, stride_int_v2, stride_reg_x_1, stride_reg_y_1, size_internal);
		}
		else
		{
			// [4] Internal Index: Partial
			// >>> External: Full && Internal: Partial
			printf ("External: Full, Internal: Partial\n");
			kernel__2_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a, size_f1, size_f2, size_b, size_c, size_d, size_e, size_g, CEIL(size_a, SIZE_SLICE_1_A), CEIL(size_f1, SIZE_SLICE_1_F1), CEIL(size_f2, SIZE_SLICE_1_F2), CEIL(size_b, SIZE_SLICE_1_B), CEIL(size_c, SIZE_SLICE_1_C), CEIL(size_d, SIZE_SLICE_1_D), CEIL(size_e, SIZE_SLICE_1_E), stride_int_t2, stride_int_v2, stride_reg_x_1, stride_reg_y_1, size_internal);
		}
	}
	else
	{
		// [2] Extenral Index: Partial
		if (size_g % SIZE_SLICE_1_G == 0)
		{
			// [3] Internal Index: Full
			// >>> External: Partial && Internal: Full
			printf ("External: Partial, Internal: Full\n");
			kernel__3_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a, size_f1, size_f2, size_b, size_c, size_d, size_e, size_g, CEIL(size_a, SIZE_SLICE_1_A), CEIL(size_f1, SIZE_SLICE_1_F1), CEIL(size_f2, SIZE_SLICE_1_F2), CEIL(size_b, SIZE_SLICE_1_B), CEIL(size_c, SIZE_SLICE_1_C), CEIL(size_d, SIZE_SLICE_1_D), CEIL(size_e, SIZE_SLICE_1_E), stride_int_t2, stride_int_v2, stride_reg_x_1, stride_reg_y_1, size_internal);
		}
		else
		{
			// [4] Internal Index: Partial
			// >>> External: Partial && Internal: Partial
			printf ("External: Partial, Internal: Partial\n");
			kernel__4_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a, size_f1, size_f2, size_b, size_c, size_d, size_e, size_g, CEIL(size_a, SIZE_SLICE_1_A), CEIL(size_f1, SIZE_SLICE_1_F1), CEIL(size_f2, SIZE_SLICE_1_F2), CEIL(size_b, SIZE_SLICE_1_B), CEIL(size_c, SIZE_SLICE_1_C), CEIL(size_d, SIZE_SLICE_1_D), CEIL(size_e, SIZE_SLICE_1_E), stride_int_t2, stride_int_v2, stride_reg_x_1, stride_reg_y_1, size_internal);
		}
	}

	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);

	float milliseconds = 0;

	hipEventElapsedTime(&milliseconds, start, stop);

	printf("Time of one iteration: %f milliseconds\n", milliseconds/100.);
	// Copy the Result from Device to Host
	hipMemcpy(t3, dev_t3, sizeof(float) * (size_a * size_f1 * size_f2 * size_b * size_c * size_d * size_e), hipMemcpyDeviceToHost);

	// cudaFree()
	hipFree(dev_t3);	hipFree(dev_t2);	hipFree(dev_v2);

	// Shoule be Fixed
	// HostFree

}

// This is written by tc_interface.tc_gen_code_interface()
// This Interface Should be Called to Run the Kernels
extern "C"
void sd_t_d2_fusion_(int size_a, int size_f, int size_b, int size_c, int size_d, int size_e, int size_g, float* t3, float* t2, float* v2, int cond_kernel_1, int opt_register_transpose)
{
	// Pre-Processing for Split
	// Based on Tile-Sizes and Problem-Size
	// Currently, one index can be split into two indices
	int size_f1;
	int size_f2;

	if (size_f % 8 == 0)
	{
		//
		size_f1 = 8;
		size_f2 = size_f / 8;
	}
	else
	{
		//
		size_f1 = size_f;
		size_f2 = 1;
	}

	// Call An Application
	sd_t_d2_fusion(size_a, size_f1, size_f2, size_b, size_c, size_d, size_e, size_g, t3, t2, v2, cond_kernel_1, opt_register_transpose);
}
