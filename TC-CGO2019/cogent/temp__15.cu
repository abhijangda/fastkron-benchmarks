// created by tc_code_include() in tc_code_include.py

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <sys/time.h>
#include <locale.h>
#include <algorithm>
using namespace std;

// created by tc_gen_definition_new()
#define SIZE_SLICE_1_D 16
#define SIZE_SLICE_1_A 16
#define SIZE_SLICE_1_C 4
#define SIZE_SLICE_1_B1 8
#define SIZE_SLICE_1_B2 8

#define SIZE_INT_UNIT_1 SIZE_SLICE_1_D

#define SIZE_TB_1_X 	SIZE_SLICE_1_A
#define SIZE_TB_1_Y 	SIZE_SLICE_1_B1
#define SIZE_REG_1_X 	SIZE_SLICE_1_C
#define SIZE_REG_1_Y 	SIZE_SLICE_1_B2

#define NUM_INDEX 		4
#define CEIL(a, b) 		(((a) + (b) - 1) / (b))

// created by tc_gen_code_Kernel()
__global__ void kernel__1_1(double* dev_t3, 
double* dev_t2, 
double* dev_v2, 
int size_a, int size_b1, int size_b2, int size_c, int size_d, 
int numBlk_a, int numBlk_b1, int numBlk_b2, int numBlk_c, 
int stride_int_t2, int stride_int_v2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ double sm_a[16][64];
	__shared__ double sm_b[16][64];


	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 1
	// # of indices mapped on TB_Y: 1
	int idx_a = threadIdx.x;
	int idx_b1 = threadIdx.y;

	int tmp_blkIdx;
	int blk_idx_c = blockIdx.x / (numBlk_b2 * numBlk_b1 * numBlk_a);
	tmp_blkIdx = blockIdx.x % (numBlk_b2 * numBlk_b1 * numBlk_a);

	int blk_idx_b2 = tmp_blkIdx / (numBlk_b1 * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_b1 * numBlk_a);

	int blk_idx_b1 = tmp_blkIdx / numBlk_a;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a);

	int  blk_idx_a = tmp_blkIdx;

	int t3_base_thread = blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_b1 * SIZE_SLICE_1_B1 + idx_b1 + (blk_idx_b2 * SIZE_SLICE_1_B2 + (blk_idx_c * SIZE_SLICE_1_C) * size_b2) * size_b1) * size_a;


	double temp_av;
	double temp_bv[8];
	double reg_tile[8][4];

	for (int i = 0; i < 8; i++)
	for (int j = 0; j < 4; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'x', 't2', ['a', 'c', 'd']], [16, 'STR_SD2_V2_H7', 'y', 'v2', ['d', 'b1', 'b2']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		// No Need to Put Boundary-Checks before For-Statement: : 
		for (int ll = 0; ll < 4; ll++)
		{
			// ['a', 'c', 'd']
			// Exception: Temp. version!: threadIdx.y + l + 0
			// Exception: Temp. version!: idx_a < rng_a
			sm_a[threadIdx.y + 0][threadIdx.x + ll * 16] = dev_t2[blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_c * SIZE_SLICE_1_C + ll) * size_a + (threadIdx.y + l + 0) * stride_int_t2];
			// Exception: Temp. version!: threadIdx.y + l + 8
			// Exception: Temp. version!: idx_a < rng_a
			// Exception: Full-Full
			sm_a[threadIdx.y + 8][threadIdx.x + ll * 16] = dev_t2[blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_c * SIZE_SLICE_1_C + ll) * size_a + (threadIdx.y + l + 8) * stride_int_t2];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		// No Need to Put Boundary-Checks before For-Statement: : 
		for (int ll = 0; ll < 8; ll++)
		{
			// ['d', 'b1', 'b2']
			// Exception: Temp. version!: threadIdx.x + l
			// Exception: Temp. version!: idx_b1 < rng_b1
			sm_b[threadIdx.x][threadIdx.y + ll * 8] = dev_v2[(blk_idx_b1 * SIZE_SLICE_1_B1 + idx_b1 + (blk_idx_b2 * SIZE_SLICE_1_B2 + ll) * size_b1) * size_d + (threadIdx.x + l)];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1; ll++)
		{
			temp_bv[0] = sm_b[ll][idx_b1 + 0];
			temp_bv[1] = sm_b[ll][idx_b1 + 8];
			temp_bv[2] = sm_b[ll][idx_b1 + 16];
			temp_bv[3] = sm_b[ll][idx_b1 + 24];
			temp_bv[4] = sm_b[ll][idx_b1 + 32];
			temp_bv[5] = sm_b[ll][idx_b1 + 40];
			temp_bv[6] = sm_b[ll][idx_b1 + 48];
			temp_bv[7] = sm_b[ll][idx_b1 + 56];

			for (int xx = 0; xx < 4; xx++) // (1)
			{
				temp_av = sm_a[ll][idx_a + (xx * 16)];

				reg_tile[0][xx] += temp_av * temp_bv[0];
				reg_tile[1][xx] += temp_av * temp_bv[1];
				reg_tile[2][xx] += temp_av * temp_bv[2];
				reg_tile[3][xx] += temp_av * temp_bv[3];
				reg_tile[4][xx] += temp_av * temp_bv[4];
				reg_tile[5][xx] += temp_av * temp_bv[5];
				reg_tile[6][xx] += temp_av * temp_bv[6];
				reg_tile[7][xx] += temp_av * temp_bv[7];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	#pragma unroll 8
	for (int i = 0; i < 8; i++)
	{
		for (int j = 0; j < 4; j++)
		{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
		}
	}
}

// created by tc_gen_code_Kernel()
__global__ void kernel__2_1(double* dev_t3, 
double* dev_t2, 
double* dev_v2, 
int size_a, int size_b1, int size_b2, int size_c, int size_d, 
int numBlk_a, int numBlk_b1, int numBlk_b2, int numBlk_c, 
int stride_int_t2, int stride_int_v2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ double sm_a[16][64];
	__shared__ double sm_b[16][64];


	int internal_upperbound   = 0;
	int internal_offset;

	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 1
	// # of indices mapped on TB_Y: 1
	int idx_a = threadIdx.x;
	int idx_b1 = threadIdx.y;

	int tmp_blkIdx;
	int blk_idx_c = blockIdx.x / (numBlk_b2 * numBlk_b1 * numBlk_a);
	tmp_blkIdx = blockIdx.x % (numBlk_b2 * numBlk_b1 * numBlk_a);

	int blk_idx_b2 = tmp_blkIdx / (numBlk_b1 * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_b1 * numBlk_a);

	int blk_idx_b1 = tmp_blkIdx / numBlk_a;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a);

	int  blk_idx_a = tmp_blkIdx;

	int t3_base_thread = blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_b1 * SIZE_SLICE_1_B1 + idx_b1 + (blk_idx_b2 * SIZE_SLICE_1_B2 + (blk_idx_c * SIZE_SLICE_1_C) * size_b2) * size_b1) * size_a;


	double temp_av;
	double temp_bv[8];
	double reg_tile[8][4];

	for (int i = 0; i < 8; i++)
	for (int j = 0; j < 4; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'x', 't2', ['a', 'c', 'd']], [16, 'STR_SD2_V2_H7', 'y', 'v2', ['d', 'b1', 'b2']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		// Part: Generalized Contraction Index (p7b)
		internal_offset = (l + SIZE_INT_UNIT_1) - size_internal;
		if (internal_offset > 0) internal_upperbound = internal_offset;

		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (threadIdx.y < SIZE_INT_UNIT_1 - internal_upperbound)
		for (int ll = 0; ll < 4; ll++)
		{
			// ['a', 'c', 'd']
			// Exception: Temp. version!: threadIdx.y + l + 0
			// Exception: Temp. version!: idx_a < rng_a
			sm_a[threadIdx.y + 0][threadIdx.x + ll * 16] = dev_t2[blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_c * SIZE_SLICE_1_C + ll) * size_a + (threadIdx.y + l + 0) * stride_int_t2];
			// Exception: Temp. version!: threadIdx.y + l + 8
			// Exception: Temp. version!: idx_a < rng_a
			if (threadIdx.y + l + 8 < size_internal) 
			sm_a[threadIdx.y + 8][threadIdx.x + ll * 16] = dev_t2[blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_c * SIZE_SLICE_1_C + ll) * size_a + (threadIdx.y + l + 8) * stride_int_t2];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (threadIdx.x < SIZE_INT_UNIT_1 - internal_upperbound)
		for (int ll = 0; ll < 8; ll++)
		{
			// ['d', 'b1', 'b2']
			// Exception: Temp. version!: threadIdx.x + l
			// Exception: Temp. version!: idx_b1 < rng_b1
			sm_b[threadIdx.x][threadIdx.y + ll * 8] = dev_v2[(blk_idx_b1 * SIZE_SLICE_1_B1 + idx_b1 + (blk_idx_b2 * SIZE_SLICE_1_B2 + ll) * size_b1) * size_d + (threadIdx.x + l)];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1 - internal_upperbound; ll++)
		{
			temp_bv[0] = sm_b[ll][idx_b1 + 0];
			temp_bv[1] = sm_b[ll][idx_b1 + 8];
			temp_bv[2] = sm_b[ll][idx_b1 + 16];
			temp_bv[3] = sm_b[ll][idx_b1 + 24];
			temp_bv[4] = sm_b[ll][idx_b1 + 32];
			temp_bv[5] = sm_b[ll][idx_b1 + 40];
			temp_bv[6] = sm_b[ll][idx_b1 + 48];
			temp_bv[7] = sm_b[ll][idx_b1 + 56];

			for (int xx = 0; xx < 4; xx++) // (1)
			{
				temp_av = sm_a[ll][idx_a + (xx * 16)];

				reg_tile[0][xx] += temp_av * temp_bv[0];
				reg_tile[1][xx] += temp_av * temp_bv[1];
				reg_tile[2][xx] += temp_av * temp_bv[2];
				reg_tile[3][xx] += temp_av * temp_bv[3];
				reg_tile[4][xx] += temp_av * temp_bv[4];
				reg_tile[5][xx] += temp_av * temp_bv[5];
				reg_tile[6][xx] += temp_av * temp_bv[6];
				reg_tile[7][xx] += temp_av * temp_bv[7];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	#pragma unroll 8
	for (int i = 0; i < 8; i++)
	{
		for (int j = 0; j < 4; j++)
		{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
		}
	}
}

// created by tc_gen_code_Kernel()
__global__ void kernel__3_1(double* dev_t3, 
double* dev_t2, 
double* dev_v2, 
int size_a, int size_b1, int size_b2, int size_c, int size_d, 
int numBlk_a, int numBlk_b1, int numBlk_b2, int numBlk_c, 
int stride_int_t2, int stride_int_v2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ double sm_a[16][64];
	__shared__ double sm_b[16][64];


	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 1
	// # of indices mapped on TB_Y: 1
	int idx_a = threadIdx.x;
	int idx_b1 = threadIdx.y;

	int tmp_blkIdx;
	int blk_idx_c = blockIdx.x / (numBlk_b2 * numBlk_b1 * numBlk_a);
	tmp_blkIdx = blockIdx.x % (numBlk_b2 * numBlk_b1 * numBlk_a);

	int blk_idx_b2 = tmp_blkIdx / (numBlk_b1 * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_b1 * numBlk_a);

	int blk_idx_b1 = tmp_blkIdx / numBlk_a;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a);

	int  blk_idx_a = tmp_blkIdx;

	int t3_base_thread = blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_b1 * SIZE_SLICE_1_B1 + idx_b1 + (blk_idx_b2 * SIZE_SLICE_1_B2 + (blk_idx_c * SIZE_SLICE_1_C) * size_b2) * size_b1) * size_a;

	// need to support partial tiles
	int rng_a, rng_b1, rng_b2, rng_c;
	if ((size_a - (blk_idx_a * SIZE_SLICE_1_A)) >= SIZE_SLICE_1_A)
	{
		rng_a = SIZE_SLICE_1_A;
	}
	else
	{
		rng_a = size_a % SIZE_SLICE_1_A;
	}
	if ((size_b1 - (blk_idx_b1 * SIZE_SLICE_1_B1)) >= SIZE_SLICE_1_B1)
	{
		rng_b1 = SIZE_SLICE_1_B1;
	}
	else
	{
		rng_b1 = size_b1 % SIZE_SLICE_1_B1;
	}
	if ((size_b2 - (blk_idx_b2 * SIZE_SLICE_1_B2)) >= SIZE_SLICE_1_B2)
	{
		rng_b2 = SIZE_SLICE_1_B2;
	}
	else
	{
		rng_b2 = size_b2 % SIZE_SLICE_1_B2;
	}
	if ((size_c - (blk_idx_c * SIZE_SLICE_1_C)) >= SIZE_SLICE_1_C)
	{
		rng_c = SIZE_SLICE_1_C;
	}
	else
	{
		rng_c = size_c % SIZE_SLICE_1_C;
	}

	double temp_av;
	double temp_bv[8];
	double reg_tile[8][4];

	for (int i = 0; i < 8; i++)
	for (int j = 0; j < 4; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'x', 't2', ['a', 'c', 'd']], [16, 'STR_SD2_V2_H7', 'y', 'v2', ['d', 'b1', 'b2']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (idx_a < rng_a)
		for (int ll = 0; ll < rng_c; ll++)
		{
			// ['a', 'c', 'd']
			// Exception: Temp. version!: threadIdx.y + l + 0
			// Exception: Temp. version!: idx_a < rng_a
			sm_a[threadIdx.y + 0][threadIdx.x + ll * 16] = dev_t2[blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_c * SIZE_SLICE_1_C + ll) * size_a + (threadIdx.y + l + 0) * stride_int_t2];
			// Exception: Temp. version!: threadIdx.y + l + 8
			// Exception: Temp. version!: idx_a < rng_a
			if (idx_a < rng_a) 
			sm_a[threadIdx.y + 8][threadIdx.x + ll * 16] = dev_t2[blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_c * SIZE_SLICE_1_C + ll) * size_a + (threadIdx.y + l + 8) * stride_int_t2];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (idx_b1 < rng_b1)
		for (int ll = 0; ll < rng_b2; ll++)
		{
			// ['d', 'b1', 'b2']
			// Exception: Temp. version!: threadIdx.x + l
			// Exception: Temp. version!: idx_b1 < rng_b1
			sm_b[threadIdx.x][threadIdx.y + ll * 8] = dev_v2[(blk_idx_b1 * SIZE_SLICE_1_B1 + idx_b1 + (blk_idx_b2 * SIZE_SLICE_1_B2 + ll) * size_b1) * size_d + (threadIdx.x + l)];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1; ll++)
		{
			temp_bv[0] = sm_b[ll][idx_b1 + 0];
			temp_bv[1] = sm_b[ll][idx_b1 + 8];
			temp_bv[2] = sm_b[ll][idx_b1 + 16];
			temp_bv[3] = sm_b[ll][idx_b1 + 24];
			temp_bv[4] = sm_b[ll][idx_b1 + 32];
			temp_bv[5] = sm_b[ll][idx_b1 + 40];
			temp_bv[6] = sm_b[ll][idx_b1 + 48];
			temp_bv[7] = sm_b[ll][idx_b1 + 56];

			for (int xx = 0; xx < 4; xx++) // (1)
			{
				temp_av = sm_a[ll][idx_a + (xx * 16)];

				reg_tile[0][xx] += temp_av * temp_bv[0];
				reg_tile[1][xx] += temp_av * temp_bv[1];
				reg_tile[2][xx] += temp_av * temp_bv[2];
				reg_tile[3][xx] += temp_av * temp_bv[3];
				reg_tile[4][xx] += temp_av * temp_bv[4];
				reg_tile[5][xx] += temp_av * temp_bv[5];
				reg_tile[6][xx] += temp_av * temp_bv[6];
				reg_tile[7][xx] += temp_av * temp_bv[7];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	if (idx_a < rng_a && idx_b1 < rng_b1)
	for (int i = 0; i < 8; i++)
	{
		for (int j = 0; j < 4; j++)
		{
			if(i < rng_b2 && j < rng_c)
			{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
			}
		}
	}
}

// created by tc_gen_code_Kernel()
__global__ void kernel__4_1(double* dev_t3, 
double* dev_t2, 
double* dev_v2, 
int size_a, int size_b1, int size_b2, int size_c, int size_d, 
int numBlk_a, int numBlk_b1, int numBlk_b2, int numBlk_c, 
int stride_int_t2, int stride_int_v2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ double sm_a[16][64];
	__shared__ double sm_b[16][64];


	int internal_upperbound   = 0;
	int internal_offset;

	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 1
	// # of indices mapped on TB_Y: 1
	int idx_a = threadIdx.x;
	int idx_b1 = threadIdx.y;

	int tmp_blkIdx;
	int blk_idx_c = blockIdx.x / (numBlk_b2 * numBlk_b1 * numBlk_a);
	tmp_blkIdx = blockIdx.x % (numBlk_b2 * numBlk_b1 * numBlk_a);

	int blk_idx_b2 = tmp_blkIdx / (numBlk_b1 * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_b1 * numBlk_a);

	int blk_idx_b1 = tmp_blkIdx / numBlk_a;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a);

	int  blk_idx_a = tmp_blkIdx;

	int t3_base_thread = blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_b1 * SIZE_SLICE_1_B1 + idx_b1 + (blk_idx_b2 * SIZE_SLICE_1_B2 + (blk_idx_c * SIZE_SLICE_1_C) * size_b2) * size_b1) * size_a;

	// need to support partial tiles
	int rng_a, rng_b1, rng_b2, rng_c;
	if ((size_a - (blk_idx_a * SIZE_SLICE_1_A)) >= SIZE_SLICE_1_A)
	{
		rng_a = SIZE_SLICE_1_A;
	}
	else
	{
		rng_a = size_a % SIZE_SLICE_1_A;
	}
	if ((size_b1 - (blk_idx_b1 * SIZE_SLICE_1_B1)) >= SIZE_SLICE_1_B1)
	{
		rng_b1 = SIZE_SLICE_1_B1;
	}
	else
	{
		rng_b1 = size_b1 % SIZE_SLICE_1_B1;
	}
	if ((size_b2 - (blk_idx_b2 * SIZE_SLICE_1_B2)) >= SIZE_SLICE_1_B2)
	{
		rng_b2 = SIZE_SLICE_1_B2;
	}
	else
	{
		rng_b2 = size_b2 % SIZE_SLICE_1_B2;
	}
	if ((size_c - (blk_idx_c * SIZE_SLICE_1_C)) >= SIZE_SLICE_1_C)
	{
		rng_c = SIZE_SLICE_1_C;
	}
	else
	{
		rng_c = size_c % SIZE_SLICE_1_C;
	}

	double temp_av;
	double temp_bv[8];
	double reg_tile[8][4];

	for (int i = 0; i < 8; i++)
	for (int j = 0; j < 4; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'x', 't2', ['a', 'c', 'd']], [16, 'STR_SD2_V2_H7', 'y', 'v2', ['d', 'b1', 'b2']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		// Part: Generalized Contraction Index (p7b)
		internal_offset = (l + SIZE_INT_UNIT_1) - size_internal;
		if (internal_offset > 0) internal_upperbound = internal_offset;

		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (idx_a < rng_a && threadIdx.y < SIZE_INT_UNIT_1 - internal_upperbound)
		for (int ll = 0; ll < rng_c; ll++)
		{
			// ['a', 'c', 'd']
			// Exception: Temp. version!: threadIdx.y + l + 0
			// Exception: Temp. version!: idx_a < rng_a
			sm_a[threadIdx.y + 0][threadIdx.x + ll * 16] = dev_t2[blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_c * SIZE_SLICE_1_C + ll) * size_a + (threadIdx.y + l + 0) * stride_int_t2];
			// Exception: Temp. version!: threadIdx.y + l + 8
			// Exception: Temp. version!: idx_a < rng_a
			if (threadIdx.y + l + 8 < size_internal && idx_a < rng_a) 
			sm_a[threadIdx.y + 8][threadIdx.x + ll * 16] = dev_t2[blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_c * SIZE_SLICE_1_C + ll) * size_a + (threadIdx.y + l + 8) * stride_int_t2];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (idx_b1 < rng_b1 && threadIdx.x < SIZE_INT_UNIT_1 - internal_upperbound)
		for (int ll = 0; ll < rng_b2; ll++)
		{
			// ['d', 'b1', 'b2']
			// Exception: Temp. version!: threadIdx.x + l
			// Exception: Temp. version!: idx_b1 < rng_b1
			sm_b[threadIdx.x][threadIdx.y + ll * 8] = dev_v2[(blk_idx_b1 * SIZE_SLICE_1_B1 + idx_b1 + (blk_idx_b2 * SIZE_SLICE_1_B2 + ll) * size_b1) * size_d + (threadIdx.x + l)];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1 - internal_upperbound; ll++)
		{
			temp_bv[0] = sm_b[ll][idx_b1 + 0];
			temp_bv[1] = sm_b[ll][idx_b1 + 8];
			temp_bv[2] = sm_b[ll][idx_b1 + 16];
			temp_bv[3] = sm_b[ll][idx_b1 + 24];
			temp_bv[4] = sm_b[ll][idx_b1 + 32];
			temp_bv[5] = sm_b[ll][idx_b1 + 40];
			temp_bv[6] = sm_b[ll][idx_b1 + 48];
			temp_bv[7] = sm_b[ll][idx_b1 + 56];

			for (int xx = 0; xx < 4; xx++) // (1)
			{
				temp_av = sm_a[ll][idx_a + (xx * 16)];

				reg_tile[0][xx] += temp_av * temp_bv[0];
				reg_tile[1][xx] += temp_av * temp_bv[1];
				reg_tile[2][xx] += temp_av * temp_bv[2];
				reg_tile[3][xx] += temp_av * temp_bv[3];
				reg_tile[4][xx] += temp_av * temp_bv[4];
				reg_tile[5][xx] += temp_av * temp_bv[5];
				reg_tile[6][xx] += temp_av * temp_bv[6];
				reg_tile[7][xx] += temp_av * temp_bv[7];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	if (idx_a < rng_a && idx_b1 < rng_b1)
	for (int i = 0; i < 8; i++)
	{
		for (int j = 0; j < 4; j++)
		{
			if(i < rng_b2 && j < rng_c)
			{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
			}
		}
	}
}

// written by tc_interface.tc_gen_code_interface_Header()
extern "C"
void sd_t_d2_fusion(int size_a, int size_b1, int size_b2, int size_c, int size_d, double* t3, double* host_t2, double* host_v2, int cond_kernel_1, int opt_register_transpose)
{
	int num_thread_blocks_kernel_1;

	double* dev_t3;
	double* dev_t2;
	double* dev_v2;


	num_thread_blocks_kernel_1 = CEIL(size_a, SIZE_SLICE_1_A) * CEIL(size_b1, SIZE_SLICE_1_B1) * CEIL(size_b2, SIZE_SLICE_1_B2) * CEIL(size_c, SIZE_SLICE_1_C);
	// cudaMalloc()
	hipMalloc((void**) &dev_t3, sizeof(double) * size_a * size_b1 * size_b2 * size_c);
	hipMalloc((void**) &dev_t2, sizeof(double) * size_d * size_c * size_a);
	hipMalloc((void**) &dev_v2, sizeof(double) * size_b2 * size_b1 * size_d);

	// cudaMemcpy()
	hipMemcpy(dev_t3, t3, sizeof(double) * size_a * size_b1 * size_b2 * size_c, hipMemcpyHostToDevice);
	hipMemcpy(dev_t2, host_t2, sizeof(double) * size_d * size_c * size_a, hipMemcpyHostToDevice);
	hipMemcpy(dev_v2, host_v2, sizeof(double) * size_b2 * size_b1 * size_d, hipMemcpyHostToDevice);

	// Related to Kernels
	// There are 1 Basic Kernels
	long long int tmp_operations = 2 * (long long int)(size_a * size_b1 * size_b2 * size_c) * size_d;
	printf ("========================================= fusedKernels =============================================\n");
	printf ("		Grid Size  : %6d (1D)\n", num_thread_blocks_kernel_1);
	printf ("		Block-size : %2d, %2d (2D)\n", SIZE_TB_1_X, SIZE_TB_1_Y);
	printf ("		Reg.-size  : %2d, %2d (2D)\n", SIZE_REG_1_X, SIZE_REG_1_Y);
	printf ("		A thread deals with (%d x %d) elements (basically)\n", SIZE_TB_1_X * SIZE_REG_1_X, SIZE_TB_1_Y * SIZE_REG_1_Y);
	printf ("		# of Operations: %lld\n", tmp_operations);
	printf ("====================================================================================================\n");
	dim3 gridsize_1(num_thread_blocks_kernel_1);
	dim3 blocksize_1(SIZE_TB_1_X, SIZE_TB_1_Y);

	int stride_output_a = 1;
	int stride_output_b1 = stride_output_a * size_a;
	int stride_output_b2 = stride_output_b1 * size_b1;
	int stride_output_c = stride_output_b2 * size_b2;

	int stride_reg_x_1 = stride_output_c;
	int stride_reg_y_1 = stride_output_b2;

	int size_internal = size_d;

	int stride_int_t2 = size_a * size_c;
	int stride_int_v2 = 1;

	// Decision Tree for Kernel Types
	// No Chance to Utilize the Register Transpose
	if (size_a % SIZE_SLICE_1_A == 0 && size_b1 % SIZE_SLICE_1_B1 == 0 && size_b2 % SIZE_SLICE_1_B2 == 0 && size_c % SIZE_SLICE_1_C == 0)
	{
		// [2] Extenral Index: Full
		if (size_d % SIZE_SLICE_1_D == 0)
		{
			// [3] Internal Index: Full
			// >>> External: Full && Internal: Full
			printf ("External: Full, Internal: Full\n");
			kernel__1_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a, size_b1, size_b2, size_c, size_d, CEIL(size_a, SIZE_SLICE_1_A), CEIL(size_b1, SIZE_SLICE_1_B1), CEIL(size_b2, SIZE_SLICE_1_B2), CEIL(size_c, SIZE_SLICE_1_C), stride_int_t2, stride_int_v2, stride_reg_x_1, stride_reg_y_1, size_internal);
		}
		else
		{
			// [4] Internal Index: Partial
			// >>> External: Full && Internal: Partial
			printf ("External: Full, Internal: Partial\n");
			kernel__2_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a, size_b1, size_b2, size_c, size_d, CEIL(size_a, SIZE_SLICE_1_A), CEIL(size_b1, SIZE_SLICE_1_B1), CEIL(size_b2, SIZE_SLICE_1_B2), CEIL(size_c, SIZE_SLICE_1_C), stride_int_t2, stride_int_v2, stride_reg_x_1, stride_reg_y_1, size_internal);
		}
	}
	else
	{
		// [2] Extenral Index: Partial
		if (size_d % SIZE_SLICE_1_D == 0)
		{
			// [3] Internal Index: Full
			// >>> External: Partial && Internal: Full
			printf ("External: Partial, Internal: Full\n");
			kernel__3_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a, size_b1, size_b2, size_c, size_d, CEIL(size_a, SIZE_SLICE_1_A), CEIL(size_b1, SIZE_SLICE_1_B1), CEIL(size_b2, SIZE_SLICE_1_B2), CEIL(size_c, SIZE_SLICE_1_C), stride_int_t2, stride_int_v2, stride_reg_x_1, stride_reg_y_1, size_internal);
		}
		else
		{
			// [4] Internal Index: Partial
			// >>> External: Partial && Internal: Partial
			printf ("External: Partial, Internal: Partial\n");
			kernel__4_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a, size_b1, size_b2, size_c, size_d, CEIL(size_a, SIZE_SLICE_1_A), CEIL(size_b1, SIZE_SLICE_1_B1), CEIL(size_b2, SIZE_SLICE_1_B2), CEIL(size_c, SIZE_SLICE_1_C), stride_int_t2, stride_int_v2, stride_reg_x_1, stride_reg_y_1, size_internal);
		}
	}

	// Copy the Result from Device to Host
	hipMemcpy(t3, dev_t3, sizeof(double) * (size_a * size_b1 * size_b2 * size_c), hipMemcpyDeviceToHost);

	// cudaFree()
	hipFree(dev_t3);	hipFree(dev_t2);	hipFree(dev_v2);

	// Shoule be Fixed
	// HostFree

}

// This is written by tc_interface.tc_gen_code_interface()
// This Interface Should be Called to Run the Kernels
extern "C"
void sd_t_d2_fusion_(int size_a, int size_b, int size_c, int size_d, double* t3, double* t2, double* v2, int cond_kernel_1, int opt_register_transpose)
{
	// Pre-Processing for Split
	// Based on Tile-Sizes and Problem-Size
	// Currently, one index can be split into two indices
	int size_b1;
	int size_b2;

	if (size_b % 37 == 0)
	{
		//
		size_b1 = 37;
		size_b2 = size_b / 37;
	}
	else
	{
		//
		size_b1 = size_b;
		size_b2 = 1;
	}

	// Call An Application
	sd_t_d2_fusion(size_a, size_b1, size_b2, size_c, size_d, t3, t2, v2, cond_kernel_1, opt_register_transpose);
}
