// created by tc_code_include() in tc_code_include.py

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <sys/time.h>
#include <locale.h>
#include <algorithm>
using namespace std;

// created by tc_gen_definition_new()
#define SIZE_SLICE_1_E 8
#define SIZE_SLICE_1_A 8
#define SIZE_SLICE_1_D 8
#define SIZE_SLICE_1_B 1
#define SIZE_SLICE_1_C1 16
#define SIZE_SLICE_1_C2 4

#define SIZE_INT_UNIT_1 SIZE_SLICE_1_E

#define SIZE_TB_1_X 	SIZE_SLICE_1_A * SIZE_SLICE_1_B
#define SIZE_TB_1_Y 	SIZE_SLICE_1_C1
#define SIZE_REG_1_X 	SIZE_SLICE_1_D
#define SIZE_REG_1_Y 	SIZE_SLICE_1_C2

#define NUM_INDEX 		5
#define CEIL(a, b) 		(((a) + (b) - 1) / (b))

// created by tc_gen_code_Kernel()
__global__ void kernel__1_1(double* dev_t3, 
double* dev_t2, 
double* dev_v2, 
int size_a, int size_b, int size_c1, int size_c2, int size_d, int size_e, 
int numBlk_a, int numBlk_b, int numBlk_c1, int numBlk_c2, int numBlk_d, 
int stride_int_t2, int stride_int_v2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ double sm_a[8][64];
	__shared__ double sm_b[8][64];


	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 2
	// # of indices mapped on TB_Y: 1
	int idx_a = threadIdx.x % SIZE_SLICE_1_A;
	int idx_b = threadIdx.x / SIZE_SLICE_1_A;
	int idx_c1 = threadIdx.y;

	int tmp_blkIdx;
	int blk_idx_d = blockIdx.x / (numBlk_c2 * numBlk_c1 * numBlk_b * numBlk_a);
	tmp_blkIdx = blockIdx.x % (numBlk_c2 * numBlk_c1 * numBlk_b * numBlk_a);

	int blk_idx_c2 = tmp_blkIdx / (numBlk_c1 * numBlk_b * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_c1 * numBlk_b * numBlk_a);

	int blk_idx_c1 = tmp_blkIdx / (numBlk_b * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_b * numBlk_a);

	int blk_idx_b = tmp_blkIdx / numBlk_a;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a);

	int  blk_idx_a = tmp_blkIdx;

	int t3_base_thread = blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_b * SIZE_SLICE_1_B + idx_b + (blk_idx_c1 * SIZE_SLICE_1_C1 + idx_c1 + (blk_idx_c2 * SIZE_SLICE_1_C2 + (blk_idx_d * SIZE_SLICE_1_D) * size_c2) * size_c1) * size_b) * size_a;


	double temp_av;
	double temp_bv[8];
	double reg_tile[4][8];

	for (int i = 0; i < 4; i++)
	for (int j = 0; j < 8; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'x', 't2', ['d', 'b', 'e', 'a']], [16, 'STR_SD2_V2_H7', 'y', 'v2', ['e', 'c1', 'c2']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (threadIdx.y < 8 && threadIdx.y < 8)
		for (int ll = 0; ll < 8; ll++)
		{
			// ['d', 'b', 'e', 'a']
			// Exception: Temp. version!: threadIdx.y + l
			// Exception: Temp. version!: 0 < rng_b
			sm_a[threadIdx.y][threadIdx.x + ll * 8] = dev_t2[blk_idx_d * SIZE_SLICE_1_D + ll + (blk_idx_b * SIZE_SLICE_1_B + 0 + ((blk_idx_a * SIZE_SLICE_1_A + idx_a) * size_e) * size_b) * size_d + (threadIdx.y + l) * stride_int_t2];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		// No Need to Put Boundary-Checks before For-Statement: : 
		for (int ll = 0; ll < 4; ll++)
		{
			// ['e', 'c1', 'c2']
			// Exception: Temp. version!: threadIdx.x + l
			// Exception: Temp. version!: idx_c1 < rng_c1
			sm_b[threadIdx.x][threadIdx.y + ll * 16] = dev_v2[(blk_idx_c1 * SIZE_SLICE_1_C1 + idx_c1 + (blk_idx_c2 * SIZE_SLICE_1_C2 + ll) * size_c1) * size_e + (threadIdx.x + l)];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1; ll++)
		{
			temp_bv[0] = sm_a[ll][idx_a + (idx_b) * SIZE_SLICE_1_A + 0];
			temp_bv[1] = sm_a[ll][idx_a + (idx_b) * SIZE_SLICE_1_A + 8];
			temp_bv[2] = sm_a[ll][idx_a + (idx_b) * SIZE_SLICE_1_A + 16];
			temp_bv[3] = sm_a[ll][idx_a + (idx_b) * SIZE_SLICE_1_A + 24];
			temp_bv[4] = sm_a[ll][idx_a + (idx_b) * SIZE_SLICE_1_A + 32];
			temp_bv[5] = sm_a[ll][idx_a + (idx_b) * SIZE_SLICE_1_A + 40];
			temp_bv[6] = sm_a[ll][idx_a + (idx_b) * SIZE_SLICE_1_A + 48];
			temp_bv[7] = sm_a[ll][idx_a + (idx_b) * SIZE_SLICE_1_A + 56];

			for (int yy = 0; yy < 4; yy++) // (2)
			{
				temp_av = sm_b[ll][idx_c1 + (yy * 16)];

				reg_tile[yy][0] += temp_av * temp_bv[0];
				reg_tile[yy][1] += temp_av * temp_bv[1];
				reg_tile[yy][2] += temp_av * temp_bv[2];
				reg_tile[yy][3] += temp_av * temp_bv[3];
				reg_tile[yy][4] += temp_av * temp_bv[4];
				reg_tile[yy][5] += temp_av * temp_bv[5];
				reg_tile[yy][6] += temp_av * temp_bv[6];
				reg_tile[yy][7] += temp_av * temp_bv[7];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	#pragma unroll 4
	for (int i = 0; i < 4; i++)
	{
		for (int j = 0; j < 8; j++)
		{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
		}
	}
}

// created by tc_gen_code_Kernel()
__global__ void kernel__2_1(double* dev_t3, 
double* dev_t2, 
double* dev_v2, 
int size_a, int size_b, int size_c1, int size_c2, int size_d, int size_e, 
int numBlk_a, int numBlk_b, int numBlk_c1, int numBlk_c2, int numBlk_d, 
int stride_int_t2, int stride_int_v2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ double sm_a[8][64];
	__shared__ double sm_b[8][64];


	int internal_upperbound   = 0;
	int internal_offset;

	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 2
	// # of indices mapped on TB_Y: 1
	int idx_a = threadIdx.x % SIZE_SLICE_1_A;
	int idx_b = threadIdx.x / SIZE_SLICE_1_A;
	int idx_c1 = threadIdx.y;

	int tmp_blkIdx;
	int blk_idx_d = blockIdx.x / (numBlk_c2 * numBlk_c1 * numBlk_b * numBlk_a);
	tmp_blkIdx = blockIdx.x % (numBlk_c2 * numBlk_c1 * numBlk_b * numBlk_a);

	int blk_idx_c2 = tmp_blkIdx / (numBlk_c1 * numBlk_b * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_c1 * numBlk_b * numBlk_a);

	int blk_idx_c1 = tmp_blkIdx / (numBlk_b * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_b * numBlk_a);

	int blk_idx_b = tmp_blkIdx / numBlk_a;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a);

	int  blk_idx_a = tmp_blkIdx;

	int t3_base_thread = blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_b * SIZE_SLICE_1_B + idx_b + (blk_idx_c1 * SIZE_SLICE_1_C1 + idx_c1 + (blk_idx_c2 * SIZE_SLICE_1_C2 + (blk_idx_d * SIZE_SLICE_1_D) * size_c2) * size_c1) * size_b) * size_a;


	double temp_av;
	double temp_bv[8];
	double reg_tile[4][8];

	for (int i = 0; i < 4; i++)
	for (int j = 0; j < 8; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'x', 't2', ['d', 'b', 'e', 'a']], [16, 'STR_SD2_V2_H7', 'y', 'v2', ['e', 'c1', 'c2']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		// Part: Generalized Contraction Index (p7b)
		internal_offset = (l + SIZE_INT_UNIT_1) - size_internal;
		if (internal_offset > 0) internal_upperbound = internal_offset;

		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (threadIdx.y < SIZE_INT_UNIT_1 - internal_upperbound && threadIdx.y < 8 && threadIdx.y < 8)
		for (int ll = 0; ll < 8; ll++)
		{
			// ['d', 'b', 'e', 'a']
			// Exception: Temp. version!: threadIdx.y + l
			// Exception: Temp. version!: 0 < rng_b
			sm_a[threadIdx.y][threadIdx.x + ll * 8] = dev_t2[blk_idx_d * SIZE_SLICE_1_D + ll + (blk_idx_b * SIZE_SLICE_1_B + 0 + ((blk_idx_a * SIZE_SLICE_1_A + idx_a) * size_e) * size_b) * size_d + (threadIdx.y + l) * stride_int_t2];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (threadIdx.x < SIZE_INT_UNIT_1 - internal_upperbound)
		for (int ll = 0; ll < 4; ll++)
		{
			// ['e', 'c1', 'c2']
			// Exception: Temp. version!: threadIdx.x + l
			// Exception: Temp. version!: idx_c1 < rng_c1
			sm_b[threadIdx.x][threadIdx.y + ll * 16] = dev_v2[(blk_idx_c1 * SIZE_SLICE_1_C1 + idx_c1 + (blk_idx_c2 * SIZE_SLICE_1_C2 + ll) * size_c1) * size_e + (threadIdx.x + l)];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1 - internal_upperbound; ll++)
		{
			temp_bv[0] = sm_a[ll][idx_a + (idx_b) * SIZE_SLICE_1_A + 0];
			temp_bv[1] = sm_a[ll][idx_a + (idx_b) * SIZE_SLICE_1_A + 8];
			temp_bv[2] = sm_a[ll][idx_a + (idx_b) * SIZE_SLICE_1_A + 16];
			temp_bv[3] = sm_a[ll][idx_a + (idx_b) * SIZE_SLICE_1_A + 24];
			temp_bv[4] = sm_a[ll][idx_a + (idx_b) * SIZE_SLICE_1_A + 32];
			temp_bv[5] = sm_a[ll][idx_a + (idx_b) * SIZE_SLICE_1_A + 40];
			temp_bv[6] = sm_a[ll][idx_a + (idx_b) * SIZE_SLICE_1_A + 48];
			temp_bv[7] = sm_a[ll][idx_a + (idx_b) * SIZE_SLICE_1_A + 56];

			for (int yy = 0; yy < 4; yy++) // (2)
			{
				temp_av = sm_b[ll][idx_c1 + (yy * 16)];

				reg_tile[yy][0] += temp_av * temp_bv[0];
				reg_tile[yy][1] += temp_av * temp_bv[1];
				reg_tile[yy][2] += temp_av * temp_bv[2];
				reg_tile[yy][3] += temp_av * temp_bv[3];
				reg_tile[yy][4] += temp_av * temp_bv[4];
				reg_tile[yy][5] += temp_av * temp_bv[5];
				reg_tile[yy][6] += temp_av * temp_bv[6];
				reg_tile[yy][7] += temp_av * temp_bv[7];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	#pragma unroll 4
	for (int i = 0; i < 4; i++)
	{
		for (int j = 0; j < 8; j++)
		{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
		}
	}
}

// created by tc_gen_code_Kernel()
__global__ void kernel__3_1(double* dev_t3, 
double* dev_t2, 
double* dev_v2, 
int size_a, int size_b, int size_c1, int size_c2, int size_d, int size_e, 
int numBlk_a, int numBlk_b, int numBlk_c1, int numBlk_c2, int numBlk_d, 
int stride_int_t2, int stride_int_v2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ double sm_a[8][64];
	__shared__ double sm_b[8][64];


	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 2
	// # of indices mapped on TB_Y: 1
	int idx_a = threadIdx.x % SIZE_SLICE_1_A;
	int idx_b = threadIdx.x / SIZE_SLICE_1_A;
	int idx_c1 = threadIdx.y;

	int tmp_blkIdx;
	int blk_idx_d = blockIdx.x / (numBlk_c2 * numBlk_c1 * numBlk_b * numBlk_a);
	tmp_blkIdx = blockIdx.x % (numBlk_c2 * numBlk_c1 * numBlk_b * numBlk_a);

	int blk_idx_c2 = tmp_blkIdx / (numBlk_c1 * numBlk_b * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_c1 * numBlk_b * numBlk_a);

	int blk_idx_c1 = tmp_blkIdx / (numBlk_b * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_b * numBlk_a);

	int blk_idx_b = tmp_blkIdx / numBlk_a;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a);

	int  blk_idx_a = tmp_blkIdx;

	int t3_base_thread = blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_b * SIZE_SLICE_1_B + idx_b + (blk_idx_c1 * SIZE_SLICE_1_C1 + idx_c1 + (blk_idx_c2 * SIZE_SLICE_1_C2 + (blk_idx_d * SIZE_SLICE_1_D) * size_c2) * size_c1) * size_b) * size_a;

	// need to support partial tiles
	int rng_a, rng_b, rng_c1, rng_c2, rng_d;
	if ((size_a - (blk_idx_a * SIZE_SLICE_1_A)) >= SIZE_SLICE_1_A)
	{
		rng_a = SIZE_SLICE_1_A;
	}
	else
	{
		rng_a = size_a % SIZE_SLICE_1_A;
	}
	if ((size_b - (blk_idx_b * SIZE_SLICE_1_B)) >= SIZE_SLICE_1_B)
	{
		rng_b = SIZE_SLICE_1_B;
	}
	else
	{
		rng_b = size_b % SIZE_SLICE_1_B;
	}
	if ((size_c1 - (blk_idx_c1 * SIZE_SLICE_1_C1)) >= SIZE_SLICE_1_C1)
	{
		rng_c1 = SIZE_SLICE_1_C1;
	}
	else
	{
		rng_c1 = size_c1 % SIZE_SLICE_1_C1;
	}
	if ((size_c2 - (blk_idx_c2 * SIZE_SLICE_1_C2)) >= SIZE_SLICE_1_C2)
	{
		rng_c2 = SIZE_SLICE_1_C2;
	}
	else
	{
		rng_c2 = size_c2 % SIZE_SLICE_1_C2;
	}
	if ((size_d - (blk_idx_d * SIZE_SLICE_1_D)) >= SIZE_SLICE_1_D)
	{
		rng_d = SIZE_SLICE_1_D;
	}
	else
	{
		rng_d = size_d % SIZE_SLICE_1_D;
	}

	double temp_av;
	double temp_bv[8];
	double reg_tile[4][8];

	for (int i = 0; i < 4; i++)
	for (int j = 0; j < 8; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'x', 't2', ['d', 'b', 'e', 'a']], [16, 'STR_SD2_V2_H7', 'y', 'v2', ['e', 'c1', 'c2']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (0 < rng_b && idx_a < rng_a && threadIdx.y < 8 && threadIdx.y < 8)
		for (int ll = 0; ll < rng_d; ll++)
		{
			// ['d', 'b', 'e', 'a']
			// Exception: Temp. version!: threadIdx.y + l
			// Exception: Temp. version!: 0 < rng_b
			sm_a[threadIdx.y][threadIdx.x + ll * 8] = dev_t2[blk_idx_d * SIZE_SLICE_1_D + ll + (blk_idx_b * SIZE_SLICE_1_B + 0 + ((blk_idx_a * SIZE_SLICE_1_A + idx_a) * size_e) * size_b) * size_d + (threadIdx.y + l) * stride_int_t2];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (idx_c1 < rng_c1)
		for (int ll = 0; ll < rng_c2; ll++)
		{
			// ['e', 'c1', 'c2']
			// Exception: Temp. version!: threadIdx.x + l
			// Exception: Temp. version!: idx_c1 < rng_c1
			sm_b[threadIdx.x][threadIdx.y + ll * 16] = dev_v2[(blk_idx_c1 * SIZE_SLICE_1_C1 + idx_c1 + (blk_idx_c2 * SIZE_SLICE_1_C2 + ll) * size_c1) * size_e + (threadIdx.x + l)];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1; ll++)
		{
			temp_bv[0] = sm_a[ll][idx_a + (idx_b) * SIZE_SLICE_1_A + 0];
			temp_bv[1] = sm_a[ll][idx_a + (idx_b) * SIZE_SLICE_1_A + 8];
			temp_bv[2] = sm_a[ll][idx_a + (idx_b) * SIZE_SLICE_1_A + 16];
			temp_bv[3] = sm_a[ll][idx_a + (idx_b) * SIZE_SLICE_1_A + 24];
			temp_bv[4] = sm_a[ll][idx_a + (idx_b) * SIZE_SLICE_1_A + 32];
			temp_bv[5] = sm_a[ll][idx_a + (idx_b) * SIZE_SLICE_1_A + 40];
			temp_bv[6] = sm_a[ll][idx_a + (idx_b) * SIZE_SLICE_1_A + 48];
			temp_bv[7] = sm_a[ll][idx_a + (idx_b) * SIZE_SLICE_1_A + 56];

			for (int yy = 0; yy < 4; yy++) // (2)
			{
				temp_av = sm_b[ll][idx_c1 + (yy * 16)];

				reg_tile[yy][0] += temp_av * temp_bv[0];
				reg_tile[yy][1] += temp_av * temp_bv[1];
				reg_tile[yy][2] += temp_av * temp_bv[2];
				reg_tile[yy][3] += temp_av * temp_bv[3];
				reg_tile[yy][4] += temp_av * temp_bv[4];
				reg_tile[yy][5] += temp_av * temp_bv[5];
				reg_tile[yy][6] += temp_av * temp_bv[6];
				reg_tile[yy][7] += temp_av * temp_bv[7];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	if (idx_a < rng_a && idx_b < rng_b && idx_c1 < rng_c1)
	for (int i = 0; i < 4; i++)
	{
		for (int j = 0; j < 8; j++)
		{
			if(i < rng_c2 && j < rng_d)
			{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
			}
		}
	}
}

// created by tc_gen_code_Kernel()
__global__ void kernel__4_1(double* dev_t3, 
double* dev_t2, 
double* dev_v2, 
int size_a, int size_b, int size_c1, int size_c2, int size_d, int size_e, 
int numBlk_a, int numBlk_b, int numBlk_c1, int numBlk_c2, int numBlk_d, 
int stride_int_t2, int stride_int_v2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ double sm_a[8][64];
	__shared__ double sm_b[8][64];


	int internal_upperbound   = 0;
	int internal_offset;

	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 2
	// # of indices mapped on TB_Y: 1
	int idx_a = threadIdx.x % SIZE_SLICE_1_A;
	int idx_b = threadIdx.x / SIZE_SLICE_1_A;
	int idx_c1 = threadIdx.y;

	int tmp_blkIdx;
	int blk_idx_d = blockIdx.x / (numBlk_c2 * numBlk_c1 * numBlk_b * numBlk_a);
	tmp_blkIdx = blockIdx.x % (numBlk_c2 * numBlk_c1 * numBlk_b * numBlk_a);

	int blk_idx_c2 = tmp_blkIdx / (numBlk_c1 * numBlk_b * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_c1 * numBlk_b * numBlk_a);

	int blk_idx_c1 = tmp_blkIdx / (numBlk_b * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_b * numBlk_a);

	int blk_idx_b = tmp_blkIdx / numBlk_a;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a);

	int  blk_idx_a = tmp_blkIdx;

	int t3_base_thread = blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_b * SIZE_SLICE_1_B + idx_b + (blk_idx_c1 * SIZE_SLICE_1_C1 + idx_c1 + (blk_idx_c2 * SIZE_SLICE_1_C2 + (blk_idx_d * SIZE_SLICE_1_D) * size_c2) * size_c1) * size_b) * size_a;

	// need to support partial tiles
	int rng_a, rng_b, rng_c1, rng_c2, rng_d;
	if ((size_a - (blk_idx_a * SIZE_SLICE_1_A)) >= SIZE_SLICE_1_A)
	{
		rng_a = SIZE_SLICE_1_A;
	}
	else
	{
		rng_a = size_a % SIZE_SLICE_1_A;
	}
	if ((size_b - (blk_idx_b * SIZE_SLICE_1_B)) >= SIZE_SLICE_1_B)
	{
		rng_b = SIZE_SLICE_1_B;
	}
	else
	{
		rng_b = size_b % SIZE_SLICE_1_B;
	}
	if ((size_c1 - (blk_idx_c1 * SIZE_SLICE_1_C1)) >= SIZE_SLICE_1_C1)
	{
		rng_c1 = SIZE_SLICE_1_C1;
	}
	else
	{
		rng_c1 = size_c1 % SIZE_SLICE_1_C1;
	}
	if ((size_c2 - (blk_idx_c2 * SIZE_SLICE_1_C2)) >= SIZE_SLICE_1_C2)
	{
		rng_c2 = SIZE_SLICE_1_C2;
	}
	else
	{
		rng_c2 = size_c2 % SIZE_SLICE_1_C2;
	}
	if ((size_d - (blk_idx_d * SIZE_SLICE_1_D)) >= SIZE_SLICE_1_D)
	{
		rng_d = SIZE_SLICE_1_D;
	}
	else
	{
		rng_d = size_d % SIZE_SLICE_1_D;
	}

	double temp_av;
	double temp_bv[8];
	double reg_tile[4][8];

	for (int i = 0; i < 4; i++)
	for (int j = 0; j < 8; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'x', 't2', ['d', 'b', 'e', 'a']], [16, 'STR_SD2_V2_H7', 'y', 'v2', ['e', 'c1', 'c2']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		// Part: Generalized Contraction Index (p7b)
		internal_offset = (l + SIZE_INT_UNIT_1) - size_internal;
		if (internal_offset > 0) internal_upperbound = internal_offset;

		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (0 < rng_b && idx_a < rng_a && threadIdx.y < SIZE_INT_UNIT_1 - internal_upperbound && threadIdx.y < 8 && threadIdx.y < 8)
		for (int ll = 0; ll < rng_d; ll++)
		{
			// ['d', 'b', 'e', 'a']
			// Exception: Temp. version!: threadIdx.y + l
			// Exception: Temp. version!: 0 < rng_b
			sm_a[threadIdx.y][threadIdx.x + ll * 8] = dev_t2[blk_idx_d * SIZE_SLICE_1_D + ll + (blk_idx_b * SIZE_SLICE_1_B + 0 + ((blk_idx_a * SIZE_SLICE_1_A + idx_a) * size_e) * size_b) * size_d + (threadIdx.y + l) * stride_int_t2];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (idx_c1 < rng_c1 && threadIdx.x < SIZE_INT_UNIT_1 - internal_upperbound)
		for (int ll = 0; ll < rng_c2; ll++)
		{
			// ['e', 'c1', 'c2']
			// Exception: Temp. version!: threadIdx.x + l
			// Exception: Temp. version!: idx_c1 < rng_c1
			sm_b[threadIdx.x][threadIdx.y + ll * 16] = dev_v2[(blk_idx_c1 * SIZE_SLICE_1_C1 + idx_c1 + (blk_idx_c2 * SIZE_SLICE_1_C2 + ll) * size_c1) * size_e + (threadIdx.x + l)];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1 - internal_upperbound; ll++)
		{
			temp_bv[0] = sm_a[ll][idx_a + (idx_b) * SIZE_SLICE_1_A + 0];
			temp_bv[1] = sm_a[ll][idx_a + (idx_b) * SIZE_SLICE_1_A + 8];
			temp_bv[2] = sm_a[ll][idx_a + (idx_b) * SIZE_SLICE_1_A + 16];
			temp_bv[3] = sm_a[ll][idx_a + (idx_b) * SIZE_SLICE_1_A + 24];
			temp_bv[4] = sm_a[ll][idx_a + (idx_b) * SIZE_SLICE_1_A + 32];
			temp_bv[5] = sm_a[ll][idx_a + (idx_b) * SIZE_SLICE_1_A + 40];
			temp_bv[6] = sm_a[ll][idx_a + (idx_b) * SIZE_SLICE_1_A + 48];
			temp_bv[7] = sm_a[ll][idx_a + (idx_b) * SIZE_SLICE_1_A + 56];

			for (int yy = 0; yy < 4; yy++) // (2)
			{
				temp_av = sm_b[ll][idx_c1 + (yy * 16)];

				reg_tile[yy][0] += temp_av * temp_bv[0];
				reg_tile[yy][1] += temp_av * temp_bv[1];
				reg_tile[yy][2] += temp_av * temp_bv[2];
				reg_tile[yy][3] += temp_av * temp_bv[3];
				reg_tile[yy][4] += temp_av * temp_bv[4];
				reg_tile[yy][5] += temp_av * temp_bv[5];
				reg_tile[yy][6] += temp_av * temp_bv[6];
				reg_tile[yy][7] += temp_av * temp_bv[7];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	if (idx_a < rng_a && idx_b < rng_b && idx_c1 < rng_c1)
	for (int i = 0; i < 4; i++)
	{
		for (int j = 0; j < 8; j++)
		{
			if(i < rng_c2 && j < rng_d)
			{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
			}
		}
	}
}

// written by tc_interface.tc_gen_code_interface_Header()
extern "C"
void sd_t_d2_fusion(int size_a, int size_b, int size_c1, int size_c2, int size_d, int size_e, double* t3, double* host_t2, double* host_v2, int cond_kernel_1, int opt_register_transpose)
{
	int num_thread_blocks_kernel_1;

	double* dev_t3;
	double* dev_t2;
	double* dev_v2;


	num_thread_blocks_kernel_1 = CEIL(size_a, SIZE_SLICE_1_A) * CEIL(size_b, SIZE_SLICE_1_B) * CEIL(size_c1, SIZE_SLICE_1_C1) * CEIL(size_c2, SIZE_SLICE_1_C2) * CEIL(size_d, SIZE_SLICE_1_D);
	// cudaMalloc()
	hipMalloc((void**) &dev_t3, sizeof(double) * size_a * size_b * size_c1 * size_c2 * size_d);
	hipMalloc((void**) &dev_t2, sizeof(double) * size_a * size_e * size_b * size_d);
	hipMalloc((void**) &dev_v2, sizeof(double) * size_c2 * size_c1 * size_e);

	// cudaMemcpy()
	hipMemcpy(dev_t3, t3, sizeof(double) * size_a * size_b * size_c1 * size_c2 * size_d, hipMemcpyHostToDevice);
	hipMemcpy(dev_t2, host_t2, sizeof(double) * size_a * size_e * size_b * size_d, hipMemcpyHostToDevice);
	hipMemcpy(dev_v2, host_v2, sizeof(double) * size_c2 * size_c1 * size_e, hipMemcpyHostToDevice);

	// Related to Kernels
	// There are 1 Basic Kernels
	long long int tmp_operations = 2 * (long long int)(size_a * size_b * size_c1 * size_c2 * size_d) * size_e;
	printf ("========================================= fusedKernels =============================================\n");
	printf ("		Grid Size  : %6d (1D)\n", num_thread_blocks_kernel_1);
	printf ("		Block-size : %2d, %2d (2D)\n", SIZE_TB_1_X, SIZE_TB_1_Y);
	printf ("		Reg.-size  : %2d, %2d (2D)\n", SIZE_REG_1_X, SIZE_REG_1_Y);
	printf ("		A thread deals with (%d x %d) elements (basically)\n", SIZE_TB_1_X * SIZE_REG_1_X, SIZE_TB_1_Y * SIZE_REG_1_Y);
	printf ("		# of Operations: %lld\n", tmp_operations);
	printf ("====================================================================================================\n");
	dim3 gridsize_1(num_thread_blocks_kernel_1);
	dim3 blocksize_1(SIZE_TB_1_X, SIZE_TB_1_Y);

	int stride_output_a = 1;
	int stride_output_b = stride_output_a * size_a;
	int stride_output_c1 = stride_output_b * size_b;
	int stride_output_c2 = stride_output_c1 * size_c1;
	int stride_output_d = stride_output_c2 * size_c2;

	int stride_reg_x_1 = stride_output_d;
	int stride_reg_y_1 = stride_output_c2;

	int size_internal = size_e;

	int stride_int_t2 = size_d * size_b;
	int stride_int_v2 = 1;

	// Decision Tree for Kernel Types
	// No Chance to Utilize the Register Transpose
	if (size_a % SIZE_SLICE_1_A == 0 && size_b % SIZE_SLICE_1_B == 0 && size_c1 % SIZE_SLICE_1_C1 == 0 && size_c2 % SIZE_SLICE_1_C2 == 0 && size_d % SIZE_SLICE_1_D == 0)
	{
		// [2] Extenral Index: Full
		if (size_e % SIZE_SLICE_1_E == 0)
		{
			// [3] Internal Index: Full
			// >>> External: Full && Internal: Full
			printf ("External: Full, Internal: Full\n");
			kernel__1_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a, size_b, size_c1, size_c2, size_d, size_e, CEIL(size_a, SIZE_SLICE_1_A), CEIL(size_b, SIZE_SLICE_1_B), CEIL(size_c1, SIZE_SLICE_1_C1), CEIL(size_c2, SIZE_SLICE_1_C2), CEIL(size_d, SIZE_SLICE_1_D), stride_int_t2, stride_int_v2, stride_reg_x_1, stride_reg_y_1, size_internal);
		}
		else
		{
			// [4] Internal Index: Partial
			// >>> External: Full && Internal: Partial
			printf ("External: Full, Internal: Partial\n");
			kernel__2_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a, size_b, size_c1, size_c2, size_d, size_e, CEIL(size_a, SIZE_SLICE_1_A), CEIL(size_b, SIZE_SLICE_1_B), CEIL(size_c1, SIZE_SLICE_1_C1), CEIL(size_c2, SIZE_SLICE_1_C2), CEIL(size_d, SIZE_SLICE_1_D), stride_int_t2, stride_int_v2, stride_reg_x_1, stride_reg_y_1, size_internal);
		}
	}
	else
	{
		// [2] Extenral Index: Partial
		if (size_e % SIZE_SLICE_1_E == 0)
		{
			// [3] Internal Index: Full
			// >>> External: Partial && Internal: Full
			printf ("External: Partial, Internal: Full\n");
			kernel__3_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a, size_b, size_c1, size_c2, size_d, size_e, CEIL(size_a, SIZE_SLICE_1_A), CEIL(size_b, SIZE_SLICE_1_B), CEIL(size_c1, SIZE_SLICE_1_C1), CEIL(size_c2, SIZE_SLICE_1_C2), CEIL(size_d, SIZE_SLICE_1_D), stride_int_t2, stride_int_v2, stride_reg_x_1, stride_reg_y_1, size_internal);
		}
		else
		{
			// [4] Internal Index: Partial
			// >>> External: Partial && Internal: Partial
			printf ("External: Partial, Internal: Partial\n");
			kernel__4_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a, size_b, size_c1, size_c2, size_d, size_e, CEIL(size_a, SIZE_SLICE_1_A), CEIL(size_b, SIZE_SLICE_1_B), CEIL(size_c1, SIZE_SLICE_1_C1), CEIL(size_c2, SIZE_SLICE_1_C2), CEIL(size_d, SIZE_SLICE_1_D), stride_int_t2, stride_int_v2, stride_reg_x_1, stride_reg_y_1, size_internal);
		}
	}

	// Copy the Result from Device to Host
	hipMemcpy(t3, dev_t3, sizeof(double) * (size_a * size_b * size_c1 * size_c2 * size_d), hipMemcpyDeviceToHost);

	// cudaFree()
	hipFree(dev_t3);	hipFree(dev_t2);	hipFree(dev_v2);

	// Shoule be Fixed
	// HostFree

}

// This is written by tc_interface.tc_gen_code_interface()
// This Interface Should be Called to Run the Kernels
extern "C"
void sd_t_d2_fusion_(int size_a, int size_b, int size_c, int size_d, int size_e, double* t3, double* t2, double* v2, int cond_kernel_1, int opt_register_transpose)
{
	// Pre-Processing for Split
	// Based on Tile-Sizes and Problem-Size
	// Currently, one index can be split into two indices
	int size_c1;
	int size_c2;

	if (size_c % 12 == 0)
	{
		//
		size_c1 = 12;
		size_c2 = size_c / 12;
	}
	else
	{
		//
		size_c1 = size_c;
		size_c2 = 1;
	}

	// Call An Application
	sd_t_d2_fusion(size_a, size_b, size_c1, size_c2, size_d, size_e, t3, t2, v2, cond_kernel_1, opt_register_transpose);
}
