// created by tc_code_include() in tc_code_include.py

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <sys/time.h>
#include <locale.h>
#include <algorithm>
using namespace std;

// created by tc_gen_definition_new()
#define SIZE_SLICE_1_D 16
#define SIZE_SLICE_1_A1 16
#define SIZE_SLICE_1_A2 6
#define SIZE_SLICE_1_B 16
#define SIZE_SLICE_1_C 6

#define SIZE_INT_UNIT_1 SIZE_SLICE_1_D

#define SIZE_TB_1_X 	SIZE_SLICE_1_A1
#define SIZE_TB_1_Y 	SIZE_SLICE_1_B
#define SIZE_REG_1_X 	SIZE_SLICE_1_A2
#define SIZE_REG_1_Y 	SIZE_SLICE_1_C

#define NUM_INDEX 		4
#define CEIL(a, b) 		(((a) + (b) - 1) / (b))

// created by tc_gen_code_Kernel()
__global__ void kernel__1_1(double* dev_t3, 
double* dev_t2, 
double* dev_v2, 
int size_a1, int size_a2, int size_b, int size_c, int size_d, 
int numBlk_a1, int numBlk_a2, int numBlk_b, int numBlk_c, 
int stride_int_t2, int stride_int_v2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ double sm_a[16][96];
	__shared__ double sm_b[16][96];


	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 1
	// # of indices mapped on TB_Y: 1
	int idx_a1 = threadIdx.x;
	int idx_b = threadIdx.y;

	int tmp_blkIdx;
	int blk_idx_c = blockIdx.x / (numBlk_b * numBlk_a2 * numBlk_a1);
	tmp_blkIdx = blockIdx.x % (numBlk_b * numBlk_a2 * numBlk_a1);

	int blk_idx_b = tmp_blkIdx / (numBlk_a2 * numBlk_a1);
	tmp_blkIdx = tmp_blkIdx % (numBlk_a2 * numBlk_a1);

	int blk_idx_a2 = tmp_blkIdx / numBlk_a1;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a1);

	int  blk_idx_a1 = tmp_blkIdx;

	int t3_base_thread = blk_idx_a1 * SIZE_SLICE_1_A1 + idx_a1 + (blk_idx_a2 * SIZE_SLICE_1_A2 + (blk_idx_b * SIZE_SLICE_1_B + idx_b + (blk_idx_c * SIZE_SLICE_1_C) * size_b) * size_a2) * size_a1;


	double temp_av;
	double temp_bv[6];
	double reg_tile[6][6];

	for (int i = 0; i < 6; i++)
	for (int j = 0; j < 6; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'x', 't2', ['a1', 'a2', 'd']], [16, 'STR_SD2_V2_H7', 'y', 'v2', ['b', 'd', 'c']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		// No Need to Put Boundary-Checks before For-Statement: : 
		for (int ll = 0; ll < 6; ll++)
		{
			// ['a1', 'a2', 'd']
			// Exception: Temp. version!: threadIdx.y + l
			// Exception: Temp. version!: idx_a1 < rng_a1
			sm_a[threadIdx.y][threadIdx.x + ll * 16] = dev_t2[blk_idx_a1 * SIZE_SLICE_1_A1 + idx_a1 + (blk_idx_a2 * SIZE_SLICE_1_A2 + ll) * size_a1 + (threadIdx.y + l) * stride_int_t2];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		// No Need to Put Boundary-Checks before For-Statement: : 
		for (int ll = 0; ll < 6; ll++)
		{
			// ['b', 'd', 'c']
			// Exception: Temp. version!: threadIdx.y + l
			// Exception: Temp. version!: idx_a1 < rng_b
			sm_b[threadIdx.y][threadIdx.x + ll * 16] = dev_v2[blk_idx_b * SIZE_SLICE_1_B + idx_a1 + ((blk_idx_c * SIZE_SLICE_1_C + ll) * size_d) * size_b + (threadIdx.y + l) * stride_int_v2];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1; ll++)
		{
			temp_bv[0] = sm_b[ll][idx_b + 0];
			temp_bv[1] = sm_b[ll][idx_b + 16];
			temp_bv[2] = sm_b[ll][idx_b + 32];
			temp_bv[3] = sm_b[ll][idx_b + 48];
			temp_bv[4] = sm_b[ll][idx_b + 64];
			temp_bv[5] = sm_b[ll][idx_b + 80];

			for (int xx = 0; xx < 6; xx++) // (1)
			{
				temp_av = sm_a[ll][idx_a1 + (xx * 16)];

				reg_tile[0][xx] += temp_av * temp_bv[0];
				reg_tile[1][xx] += temp_av * temp_bv[1];
				reg_tile[2][xx] += temp_av * temp_bv[2];
				reg_tile[3][xx] += temp_av * temp_bv[3];
				reg_tile[4][xx] += temp_av * temp_bv[4];
				reg_tile[5][xx] += temp_av * temp_bv[5];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	#pragma unroll 6
	for (int i = 0; i < 6; i++)
	{
		for (int j = 0; j < 6; j++)
		{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
		}
	}
}

// created by tc_gen_code_Kernel()
__global__ void kernel__2_1(double* dev_t3, 
double* dev_t2, 
double* dev_v2, 
int size_a1, int size_a2, int size_b, int size_c, int size_d, 
int numBlk_a1, int numBlk_a2, int numBlk_b, int numBlk_c, 
int stride_int_t2, int stride_int_v2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ double sm_a[16][96];
	__shared__ double sm_b[16][96];


	int internal_upperbound   = 0;
	int internal_offset;

	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 1
	// # of indices mapped on TB_Y: 1
	int idx_a1 = threadIdx.x;
	int idx_b = threadIdx.y;

	int tmp_blkIdx;
	int blk_idx_c = blockIdx.x / (numBlk_b * numBlk_a2 * numBlk_a1);
	tmp_blkIdx = blockIdx.x % (numBlk_b * numBlk_a2 * numBlk_a1);

	int blk_idx_b = tmp_blkIdx / (numBlk_a2 * numBlk_a1);
	tmp_blkIdx = tmp_blkIdx % (numBlk_a2 * numBlk_a1);

	int blk_idx_a2 = tmp_blkIdx / numBlk_a1;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a1);

	int  blk_idx_a1 = tmp_blkIdx;

	int t3_base_thread = blk_idx_a1 * SIZE_SLICE_1_A1 + idx_a1 + (blk_idx_a2 * SIZE_SLICE_1_A2 + (blk_idx_b * SIZE_SLICE_1_B + idx_b + (blk_idx_c * SIZE_SLICE_1_C) * size_b) * size_a2) * size_a1;


	double temp_av;
	double temp_bv[6];
	double reg_tile[6][6];

	for (int i = 0; i < 6; i++)
	for (int j = 0; j < 6; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'x', 't2', ['a1', 'a2', 'd']], [16, 'STR_SD2_V2_H7', 'y', 'v2', ['b', 'd', 'c']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		// Part: Generalized Contraction Index (p7b)
		internal_offset = (l + SIZE_INT_UNIT_1) - size_internal;
		if (internal_offset > 0) internal_upperbound = internal_offset;

		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (threadIdx.y < SIZE_INT_UNIT_1 - internal_upperbound)
		for (int ll = 0; ll < 6; ll++)
		{
			// ['a1', 'a2', 'd']
			// Exception: Temp. version!: threadIdx.y + l
			// Exception: Temp. version!: idx_a1 < rng_a1
			sm_a[threadIdx.y][threadIdx.x + ll * 16] = dev_t2[blk_idx_a1 * SIZE_SLICE_1_A1 + idx_a1 + (blk_idx_a2 * SIZE_SLICE_1_A2 + ll) * size_a1 + (threadIdx.y + l) * stride_int_t2];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (threadIdx.y < SIZE_INT_UNIT_1 - internal_upperbound)
		for (int ll = 0; ll < 6; ll++)
		{
			// ['b', 'd', 'c']
			// Exception: Temp. version!: threadIdx.y + l
			// Exception: Temp. version!: idx_a1 < rng_b
			sm_b[threadIdx.y][threadIdx.x + ll * 16] = dev_v2[blk_idx_b * SIZE_SLICE_1_B + idx_a1 + ((blk_idx_c * SIZE_SLICE_1_C + ll) * size_d) * size_b + (threadIdx.y + l) * stride_int_v2];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1 - internal_upperbound; ll++)
		{
			temp_bv[0] = sm_b[ll][idx_b + 0];
			temp_bv[1] = sm_b[ll][idx_b + 16];
			temp_bv[2] = sm_b[ll][idx_b + 32];
			temp_bv[3] = sm_b[ll][idx_b + 48];
			temp_bv[4] = sm_b[ll][idx_b + 64];
			temp_bv[5] = sm_b[ll][idx_b + 80];

			for (int xx = 0; xx < 6; xx++) // (1)
			{
				temp_av = sm_a[ll][idx_a1 + (xx * 16)];

				reg_tile[0][xx] += temp_av * temp_bv[0];
				reg_tile[1][xx] += temp_av * temp_bv[1];
				reg_tile[2][xx] += temp_av * temp_bv[2];
				reg_tile[3][xx] += temp_av * temp_bv[3];
				reg_tile[4][xx] += temp_av * temp_bv[4];
				reg_tile[5][xx] += temp_av * temp_bv[5];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	#pragma unroll 6
	for (int i = 0; i < 6; i++)
	{
		for (int j = 0; j < 6; j++)
		{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
		}
	}
}

// created by tc_gen_code_Kernel()
__global__ void kernel__3_1(double* dev_t3, 
double* dev_t2, 
double* dev_v2, 
int size_a1, int size_a2, int size_b, int size_c, int size_d, 
int numBlk_a1, int numBlk_a2, int numBlk_b, int numBlk_c, 
int stride_int_t2, int stride_int_v2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ double sm_a[16][96];
	__shared__ double sm_b[16][96];


	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 1
	// # of indices mapped on TB_Y: 1
	int idx_a1 = threadIdx.x;
	int idx_b = threadIdx.y;

	int tmp_blkIdx;
	int blk_idx_c = blockIdx.x / (numBlk_b * numBlk_a2 * numBlk_a1);
	tmp_blkIdx = blockIdx.x % (numBlk_b * numBlk_a2 * numBlk_a1);

	int blk_idx_b = tmp_blkIdx / (numBlk_a2 * numBlk_a1);
	tmp_blkIdx = tmp_blkIdx % (numBlk_a2 * numBlk_a1);

	int blk_idx_a2 = tmp_blkIdx / numBlk_a1;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a1);

	int  blk_idx_a1 = tmp_blkIdx;

	int t3_base_thread = blk_idx_a1 * SIZE_SLICE_1_A1 + idx_a1 + (blk_idx_a2 * SIZE_SLICE_1_A2 + (blk_idx_b * SIZE_SLICE_1_B + idx_b + (blk_idx_c * SIZE_SLICE_1_C) * size_b) * size_a2) * size_a1;

	// need to support partial tiles
	int rng_a1, rng_a2, rng_b, rng_c;
	if ((size_a1 - (blk_idx_a1 * SIZE_SLICE_1_A1)) >= SIZE_SLICE_1_A1)
	{
		rng_a1 = SIZE_SLICE_1_A1;
	}
	else
	{
		rng_a1 = size_a1 % SIZE_SLICE_1_A1;
	}
	if ((size_a2 - (blk_idx_a2 * SIZE_SLICE_1_A2)) >= SIZE_SLICE_1_A2)
	{
		rng_a2 = SIZE_SLICE_1_A2;
	}
	else
	{
		rng_a2 = size_a2 % SIZE_SLICE_1_A2;
	}
	if ((size_b - (blk_idx_b * SIZE_SLICE_1_B)) >= SIZE_SLICE_1_B)
	{
		rng_b = SIZE_SLICE_1_B;
	}
	else
	{
		rng_b = size_b % SIZE_SLICE_1_B;
	}
	if ((size_c - (blk_idx_c * SIZE_SLICE_1_C)) >= SIZE_SLICE_1_C)
	{
		rng_c = SIZE_SLICE_1_C;
	}
	else
	{
		rng_c = size_c % SIZE_SLICE_1_C;
	}

	double temp_av;
	double temp_bv[6];
	double reg_tile[6][6];

	for (int i = 0; i < 6; i++)
	for (int j = 0; j < 6; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'x', 't2', ['a1', 'a2', 'd']], [16, 'STR_SD2_V2_H7', 'y', 'v2', ['b', 'd', 'c']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (idx_a1 < rng_a1)
		for (int ll = 0; ll < rng_a2; ll++)
		{
			// ['a1', 'a2', 'd']
			// Exception: Temp. version!: threadIdx.y + l
			// Exception: Temp. version!: idx_a1 < rng_a1
			sm_a[threadIdx.y][threadIdx.x + ll * 16] = dev_t2[blk_idx_a1 * SIZE_SLICE_1_A1 + idx_a1 + (blk_idx_a2 * SIZE_SLICE_1_A2 + ll) * size_a1 + (threadIdx.y + l) * stride_int_t2];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (idx_a1 < rng_b)
		for (int ll = 0; ll < rng_c; ll++)
		{
			// ['b', 'd', 'c']
			// Exception: Temp. version!: threadIdx.y + l
			// Exception: Temp. version!: idx_a1 < rng_b
			sm_b[threadIdx.y][threadIdx.x + ll * 16] = dev_v2[blk_idx_b * SIZE_SLICE_1_B + idx_a1 + ((blk_idx_c * SIZE_SLICE_1_C + ll) * size_d) * size_b + (threadIdx.y + l) * stride_int_v2];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1; ll++)
		{
			temp_bv[0] = sm_b[ll][idx_b + 0];
			temp_bv[1] = sm_b[ll][idx_b + 16];
			temp_bv[2] = sm_b[ll][idx_b + 32];
			temp_bv[3] = sm_b[ll][idx_b + 48];
			temp_bv[4] = sm_b[ll][idx_b + 64];
			temp_bv[5] = sm_b[ll][idx_b + 80];

			for (int xx = 0; xx < 6; xx++) // (1)
			{
				temp_av = sm_a[ll][idx_a1 + (xx * 16)];

				reg_tile[0][xx] += temp_av * temp_bv[0];
				reg_tile[1][xx] += temp_av * temp_bv[1];
				reg_tile[2][xx] += temp_av * temp_bv[2];
				reg_tile[3][xx] += temp_av * temp_bv[3];
				reg_tile[4][xx] += temp_av * temp_bv[4];
				reg_tile[5][xx] += temp_av * temp_bv[5];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	if (idx_a1 < rng_a1 && idx_b < rng_b)
	for (int i = 0; i < 6; i++)
	{
		for (int j = 0; j < 6; j++)
		{
			if(i < rng_c && j < rng_a2)
			{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
			}
		}
	}
}

// created by tc_gen_code_Kernel()
__global__ void kernel__4_1(double* dev_t3, 
double* dev_t2, 
double* dev_v2, 
int size_a1, int size_a2, int size_b, int size_c, int size_d, 
int numBlk_a1, int numBlk_a2, int numBlk_b, int numBlk_c, 
int stride_int_t2, int stride_int_v2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ double sm_a[16][96];
	__shared__ double sm_b[16][96];


	int internal_upperbound   = 0;
	int internal_offset;

	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 1
	// # of indices mapped on TB_Y: 1
	int idx_a1 = threadIdx.x;
	int idx_b = threadIdx.y;

	int tmp_blkIdx;
	int blk_idx_c = blockIdx.x / (numBlk_b * numBlk_a2 * numBlk_a1);
	tmp_blkIdx = blockIdx.x % (numBlk_b * numBlk_a2 * numBlk_a1);

	int blk_idx_b = tmp_blkIdx / (numBlk_a2 * numBlk_a1);
	tmp_blkIdx = tmp_blkIdx % (numBlk_a2 * numBlk_a1);

	int blk_idx_a2 = tmp_blkIdx / numBlk_a1;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a1);

	int  blk_idx_a1 = tmp_blkIdx;

	int t3_base_thread = blk_idx_a1 * SIZE_SLICE_1_A1 + idx_a1 + (blk_idx_a2 * SIZE_SLICE_1_A2 + (blk_idx_b * SIZE_SLICE_1_B + idx_b + (blk_idx_c * SIZE_SLICE_1_C) * size_b) * size_a2) * size_a1;

	// need to support partial tiles
	int rng_a1, rng_a2, rng_b, rng_c;
	if ((size_a1 - (blk_idx_a1 * SIZE_SLICE_1_A1)) >= SIZE_SLICE_1_A1)
	{
		rng_a1 = SIZE_SLICE_1_A1;
	}
	else
	{
		rng_a1 = size_a1 % SIZE_SLICE_1_A1;
	}
	if ((size_a2 - (blk_idx_a2 * SIZE_SLICE_1_A2)) >= SIZE_SLICE_1_A2)
	{
		rng_a2 = SIZE_SLICE_1_A2;
	}
	else
	{
		rng_a2 = size_a2 % SIZE_SLICE_1_A2;
	}
	if ((size_b - (blk_idx_b * SIZE_SLICE_1_B)) >= SIZE_SLICE_1_B)
	{
		rng_b = SIZE_SLICE_1_B;
	}
	else
	{
		rng_b = size_b % SIZE_SLICE_1_B;
	}
	if ((size_c - (blk_idx_c * SIZE_SLICE_1_C)) >= SIZE_SLICE_1_C)
	{
		rng_c = SIZE_SLICE_1_C;
	}
	else
	{
		rng_c = size_c % SIZE_SLICE_1_C;
	}

	double temp_av;
	double temp_bv[6];
	double reg_tile[6][6];

	for (int i = 0; i < 6; i++)
	for (int j = 0; j < 6; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'x', 't2', ['a1', 'a2', 'd']], [16, 'STR_SD2_V2_H7', 'y', 'v2', ['b', 'd', 'c']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		// Part: Generalized Contraction Index (p7b)
		internal_offset = (l + SIZE_INT_UNIT_1) - size_internal;
		if (internal_offset > 0) internal_upperbound = internal_offset;

		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (idx_a1 < rng_a1 && threadIdx.y < SIZE_INT_UNIT_1 - internal_upperbound)
		for (int ll = 0; ll < rng_a2; ll++)
		{
			// ['a1', 'a2', 'd']
			// Exception: Temp. version!: threadIdx.y + l
			// Exception: Temp. version!: idx_a1 < rng_a1
			sm_a[threadIdx.y][threadIdx.x + ll * 16] = dev_t2[blk_idx_a1 * SIZE_SLICE_1_A1 + idx_a1 + (blk_idx_a2 * SIZE_SLICE_1_A2 + ll) * size_a1 + (threadIdx.y + l) * stride_int_t2];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (idx_a1 < rng_b && threadIdx.y < SIZE_INT_UNIT_1 - internal_upperbound)
		for (int ll = 0; ll < rng_c; ll++)
		{
			// ['b', 'd', 'c']
			// Exception: Temp. version!: threadIdx.y + l
			// Exception: Temp. version!: idx_a1 < rng_b
			sm_b[threadIdx.y][threadIdx.x + ll * 16] = dev_v2[blk_idx_b * SIZE_SLICE_1_B + idx_a1 + ((blk_idx_c * SIZE_SLICE_1_C + ll) * size_d) * size_b + (threadIdx.y + l) * stride_int_v2];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1 - internal_upperbound; ll++)
		{
			temp_bv[0] = sm_b[ll][idx_b + 0];
			temp_bv[1] = sm_b[ll][idx_b + 16];
			temp_bv[2] = sm_b[ll][idx_b + 32];
			temp_bv[3] = sm_b[ll][idx_b + 48];
			temp_bv[4] = sm_b[ll][idx_b + 64];
			temp_bv[5] = sm_b[ll][idx_b + 80];

			for (int xx = 0; xx < 6; xx++) // (1)
			{
				temp_av = sm_a[ll][idx_a1 + (xx * 16)];

				reg_tile[0][xx] += temp_av * temp_bv[0];
				reg_tile[1][xx] += temp_av * temp_bv[1];
				reg_tile[2][xx] += temp_av * temp_bv[2];
				reg_tile[3][xx] += temp_av * temp_bv[3];
				reg_tile[4][xx] += temp_av * temp_bv[4];
				reg_tile[5][xx] += temp_av * temp_bv[5];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	if (idx_a1 < rng_a1 && idx_b < rng_b)
	for (int i = 0; i < 6; i++)
	{
		for (int j = 0; j < 6; j++)
		{
			if(i < rng_c && j < rng_a2)
			{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
			}
		}
	}
}

// written by tc_interface.tc_gen_code_interface_Header()
extern "C"
void sd_t_d2_fusion(int size_a1, int size_a2, int size_b, int size_c, int size_d, double* t3, double* host_t2, double* host_v2, int cond_kernel_1, int opt_register_transpose)
{
	int num_thread_blocks_kernel_1;

	double* dev_t3;
	double* dev_t2;
	double* dev_v2;


	num_thread_blocks_kernel_1 = CEIL(size_a1, SIZE_SLICE_1_A1) * CEIL(size_a2, SIZE_SLICE_1_A2) * CEIL(size_b, SIZE_SLICE_1_B) * CEIL(size_c, SIZE_SLICE_1_C);
	// cudaMalloc()
	hipMalloc((void**) &dev_t3, sizeof(double) * size_a1 * size_a2 * size_b * size_c);
	hipMalloc((void**) &dev_t2, sizeof(double) * size_d * size_a2 * size_a1);
	hipMalloc((void**) &dev_v2, sizeof(double) * size_c * size_d * size_b);

	// cudaMemcpy()
	hipMemcpy(dev_t3, t3, sizeof(double) * size_a1 * size_a2 * size_b * size_c, hipMemcpyHostToDevice);
	hipMemcpy(dev_t2, host_t2, sizeof(double) * size_d * size_a2 * size_a1, hipMemcpyHostToDevice);
	hipMemcpy(dev_v2, host_v2, sizeof(double) * size_c * size_d * size_b, hipMemcpyHostToDevice);

	// Related to Kernels
	// There are 1 Basic Kernels
	long long int tmp_operations = 2 * (long long int)(size_a1 * size_a2 * size_b * size_c) * size_d;
	printf ("========================================= fusedKernels =============================================\n");
	printf ("		Grid Size  : %6d (1D)\n", num_thread_blocks_kernel_1);
	printf ("		Block-size : %2d, %2d (2D)\n", SIZE_TB_1_X, SIZE_TB_1_Y);
	printf ("		Reg.-size  : %2d, %2d (2D)\n", SIZE_REG_1_X, SIZE_REG_1_Y);
	printf ("		A thread deals with (%d x %d) elements (basically)\n", SIZE_TB_1_X * SIZE_REG_1_X, SIZE_TB_1_Y * SIZE_REG_1_Y);
	printf ("		# of Operations: %lld\n", tmp_operations);
	printf ("====================================================================================================\n");
	dim3 gridsize_1(num_thread_blocks_kernel_1);
	dim3 blocksize_1(SIZE_TB_1_X, SIZE_TB_1_Y);

	int stride_output_a1 = 1;
	int stride_output_a2 = stride_output_a1 * size_a1;
	int stride_output_b = stride_output_a2 * size_a2;
	int stride_output_c = stride_output_b * size_b;

	int stride_reg_x_1 = stride_output_a2;
	int stride_reg_y_1 = stride_output_c;

	int size_internal = size_d;

	int stride_int_t2 = size_a1 * size_a2;
	int stride_int_v2 = size_b;

	// Decision Tree for Kernel Types
	// No Chance to Utilize the Register Transpose
	if (size_a1 % SIZE_SLICE_1_A1 == 0 && size_a2 % SIZE_SLICE_1_A2 == 0 && size_b % SIZE_SLICE_1_B == 0 && size_c % SIZE_SLICE_1_C == 0)
	{
		// [2] Extenral Index: Full
		if (size_d % SIZE_SLICE_1_D == 0)
		{
			// [3] Internal Index: Full
			// >>> External: Full && Internal: Full
			printf ("External: Full, Internal: Full\n");
			kernel__1_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a1, size_a2, size_b, size_c, size_d, CEIL(size_a1, SIZE_SLICE_1_A1), CEIL(size_a2, SIZE_SLICE_1_A2), CEIL(size_b, SIZE_SLICE_1_B), CEIL(size_c, SIZE_SLICE_1_C), stride_int_t2, stride_int_v2, stride_reg_x_1, stride_reg_y_1, size_internal);
		}
		else
		{
			// [4] Internal Index: Partial
			// >>> External: Full && Internal: Partial
			printf ("External: Full, Internal: Partial\n");
			kernel__2_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a1, size_a2, size_b, size_c, size_d, CEIL(size_a1, SIZE_SLICE_1_A1), CEIL(size_a2, SIZE_SLICE_1_A2), CEIL(size_b, SIZE_SLICE_1_B), CEIL(size_c, SIZE_SLICE_1_C), stride_int_t2, stride_int_v2, stride_reg_x_1, stride_reg_y_1, size_internal);
		}
	}
	else
	{
		// [2] Extenral Index: Partial
		if (size_d % SIZE_SLICE_1_D == 0)
		{
			// [3] Internal Index: Full
			// >>> External: Partial && Internal: Full
			printf ("External: Partial, Internal: Full\n");
			kernel__3_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a1, size_a2, size_b, size_c, size_d, CEIL(size_a1, SIZE_SLICE_1_A1), CEIL(size_a2, SIZE_SLICE_1_A2), CEIL(size_b, SIZE_SLICE_1_B), CEIL(size_c, SIZE_SLICE_1_C), stride_int_t2, stride_int_v2, stride_reg_x_1, stride_reg_y_1, size_internal);
		}
		else
		{
			// [4] Internal Index: Partial
			// >>> External: Partial && Internal: Partial
			printf ("External: Partial, Internal: Partial\n");
			kernel__4_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a1, size_a2, size_b, size_c, size_d, CEIL(size_a1, SIZE_SLICE_1_A1), CEIL(size_a2, SIZE_SLICE_1_A2), CEIL(size_b, SIZE_SLICE_1_B), CEIL(size_c, SIZE_SLICE_1_C), stride_int_t2, stride_int_v2, stride_reg_x_1, stride_reg_y_1, size_internal);
		}
	}

	// Copy the Result from Device to Host
	hipMemcpy(t3, dev_t3, sizeof(double) * (size_a1 * size_a2 * size_b * size_c), hipMemcpyDeviceToHost);

	// cudaFree()
	hipFree(dev_t3);	hipFree(dev_t2);	hipFree(dev_v2);

	// Shoule be Fixed
	// HostFree

}

// This is written by tc_interface.tc_gen_code_interface()
// This Interface Should be Called to Run the Kernels
extern "C"
void sd_t_d2_fusion_(int size_a, int size_b, int size_c, int size_d, double* t3, double* t2, double* v2, int cond_kernel_1, int opt_register_transpose)
{
	// Pre-Processing for Split
	// Based on Tile-Sizes and Problem-Size
	// Currently, one index can be split into two indices
	int size_a1;
	int size_a2;

	if (size_a % 52 == 0)
	{
		//
		size_a1 = 52;
		size_a2 = size_a / 52;
	}
	else
	{
		//
		size_a1 = size_a;
		size_a2 = 1;
	}

	// Call An Application
	sd_t_d2_fusion(size_a1, size_a2, size_b, size_c, size_d, t3, t2, v2, cond_kernel_1, opt_register_transpose);
}
