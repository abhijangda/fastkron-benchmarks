// created by tc_code_include() in tc_code_include.py

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <sys/time.h>
#include <locale.h>
#include <algorithm>
using namespace std;

// created by tc_gen_definition_new()
#define SIZE_SLICE_1_G 8
#define SIZE_SLICE_1_A 16
#define SIZE_SLICE_1_E 4
#define SIZE_SLICE_1_C 1
#define SIZE_SLICE_1_D 8
#define SIZE_SLICE_1_F 8
#define SIZE_SLICE_1_B 1

#define SIZE_INT_UNIT_1 SIZE_SLICE_1_G

#define SIZE_TB_1_X 	SIZE_SLICE_1_A * SIZE_SLICE_1_C
#define SIZE_TB_1_Y 	SIZE_SLICE_1_D * SIZE_SLICE_1_B
#define SIZE_REG_1_X 	SIZE_SLICE_1_E
#define SIZE_REG_1_Y 	SIZE_SLICE_1_F

#define NUM_INDEX 		6
#define CEIL(a, b) 		(((a) + (b) - 1) / (b))

// created by tc_gen_code_Kernel()
__global__ void kernel__1_1(double* dev_t3, 
double* dev_t2, 
double* dev_v2, 
int size_a, int size_b, int size_c, int size_d, int size_e, int size_f, int size_g, 
int numBlk_a, int numBlk_b, int numBlk_c, int numBlk_d, int numBlk_e, int numBlk_f, 
int stride_int_t2, int stride_int_v2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ double sm_a[8][64];
	__shared__ double sm_b[8][64];


	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 2
	// # of indices mapped on TB_Y: 2
	int idx_a = threadIdx.x % SIZE_SLICE_1_A;
	int idx_c = threadIdx.x / SIZE_SLICE_1_A;
	int idx_d = threadIdx.y % SIZE_SLICE_1_D;
	int idx_b = threadIdx.y / SIZE_SLICE_1_D;

	int tmp_blkIdx;
	int blk_idx_f = blockIdx.x / (numBlk_e * numBlk_d * numBlk_c * numBlk_b * numBlk_a);
	tmp_blkIdx = blockIdx.x % (numBlk_e * numBlk_d * numBlk_c * numBlk_b * numBlk_a);

	int blk_idx_e = tmp_blkIdx / (numBlk_d * numBlk_c * numBlk_b * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_d * numBlk_c * numBlk_b * numBlk_a);

	int blk_idx_d = tmp_blkIdx / (numBlk_c * numBlk_b * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_c * numBlk_b * numBlk_a);

	int blk_idx_c = tmp_blkIdx / (numBlk_b * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_b * numBlk_a);

	int blk_idx_b = tmp_blkIdx / numBlk_a;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a);

	int  blk_idx_a = tmp_blkIdx;

	int t3_base_thread = blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_b * SIZE_SLICE_1_B + idx_b + (blk_idx_c * SIZE_SLICE_1_C + idx_c + (blk_idx_d * SIZE_SLICE_1_D + idx_d + (blk_idx_e * SIZE_SLICE_1_E + (blk_idx_f * SIZE_SLICE_1_F) * size_e) * size_d) * size_c) * size_b) * size_a;


	double temp_av;
	double temp_bv[8];
	double reg_tile[8][4];

	for (int i = 0; i < 8; i++)
	for (int j = 0; j < 4; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'y', 't2', ['d', 'f', 'g', 'b']], [16, 'STR_SD2_V2_H7', 'x', 'v2', ['g', 'e', 'a', 'c']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (threadIdx.x < 8)
		for (int ll = 0; ll < 8; ll++)
		{
			// ['d', 'f', 'g', 'b']
			// Exception: Temp. version!: threadIdx.y + l
			// Exception: Temp. version!: idx_a < rng_d
			sm_a[threadIdx.y][threadIdx.x + ll * 8] = dev_t2[blk_idx_d * SIZE_SLICE_1_D + idx_a + (blk_idx_f * SIZE_SLICE_1_F + ll + ((blk_idx_b * SIZE_SLICE_1_B + 0) * size_g) * size_f) * size_d + (threadIdx.y + l) * stride_int_t2];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (threadIdx.x < 8 && threadIdx.x < 8)
		for (int ll = 0; ll < 4; ll++)
		{
			// ['g', 'e', 'a', 'c']
			// Exception: Temp. version!: threadIdx.x + l
			// Exception: Temp. version!: idx_d + 0 < rng_a
			sm_b[threadIdx.x][threadIdx.y + 0 + ll * 16] = dev_v2[(blk_idx_e * SIZE_SLICE_1_E + ll + (blk_idx_a * SIZE_SLICE_1_A + idx_d + 0 + (blk_idx_c * SIZE_SLICE_1_C + 0) * size_a) * size_e) * size_g + (threadIdx.x + l)];
			// Exception: Temp. version!: threadIdx.x + l
			// Exception: Temp. version!: idx_d + 8 < rng_a
			// Exception: Full-Full
			sm_b[threadIdx.x][threadIdx.y + 8 + ll * 16] = dev_v2[(blk_idx_e * SIZE_SLICE_1_E + ll + (blk_idx_a * SIZE_SLICE_1_A + idx_d + 8 + (blk_idx_c * SIZE_SLICE_1_C + 0) * size_a) * size_e) * size_g + (threadIdx.x + l)];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1; ll++)
		{
			temp_bv[0] = sm_a[ll][idx_d + (idx_b) * SIZE_SLICE_1_D + 0];
			temp_bv[1] = sm_a[ll][idx_d + (idx_b) * SIZE_SLICE_1_D + 8];
			temp_bv[2] = sm_a[ll][idx_d + (idx_b) * SIZE_SLICE_1_D + 16];
			temp_bv[3] = sm_a[ll][idx_d + (idx_b) * SIZE_SLICE_1_D + 24];
			temp_bv[4] = sm_a[ll][idx_d + (idx_b) * SIZE_SLICE_1_D + 32];
			temp_bv[5] = sm_a[ll][idx_d + (idx_b) * SIZE_SLICE_1_D + 40];
			temp_bv[6] = sm_a[ll][idx_d + (idx_b) * SIZE_SLICE_1_D + 48];
			temp_bv[7] = sm_a[ll][idx_d + (idx_b) * SIZE_SLICE_1_D + 56];

			for (int xx = 0; xx < 4; xx++) // (1)
			{
				temp_av = sm_b[ll][idx_a + (idx_c) * SIZE_SLICE_1_A + (xx * 16)];

				reg_tile[0][xx] += temp_av * temp_bv[0];
				reg_tile[1][xx] += temp_av * temp_bv[1];
				reg_tile[2][xx] += temp_av * temp_bv[2];
				reg_tile[3][xx] += temp_av * temp_bv[3];
				reg_tile[4][xx] += temp_av * temp_bv[4];
				reg_tile[5][xx] += temp_av * temp_bv[5];
				reg_tile[6][xx] += temp_av * temp_bv[6];
				reg_tile[7][xx] += temp_av * temp_bv[7];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	#pragma unroll 8
	for (int i = 0; i < 8; i++)
	{
		for (int j = 0; j < 4; j++)
		{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
		}
	}
}

// created by tc_gen_code_Kernel()
__global__ void kernel__2_1(double* dev_t3, 
double* dev_t2, 
double* dev_v2, 
int size_a, int size_b, int size_c, int size_d, int size_e, int size_f, int size_g, 
int numBlk_a, int numBlk_b, int numBlk_c, int numBlk_d, int numBlk_e, int numBlk_f, 
int stride_int_t2, int stride_int_v2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ double sm_a[8][64];
	__shared__ double sm_b[8][64];


	int internal_upperbound   = 0;
	int internal_offset;

	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 2
	// # of indices mapped on TB_Y: 2
	int idx_a = threadIdx.x % SIZE_SLICE_1_A;
	int idx_c = threadIdx.x / SIZE_SLICE_1_A;
	int idx_d = threadIdx.y % SIZE_SLICE_1_D;
	int idx_b = threadIdx.y / SIZE_SLICE_1_D;

	int tmp_blkIdx;
	int blk_idx_f = blockIdx.x / (numBlk_e * numBlk_d * numBlk_c * numBlk_b * numBlk_a);
	tmp_blkIdx = blockIdx.x % (numBlk_e * numBlk_d * numBlk_c * numBlk_b * numBlk_a);

	int blk_idx_e = tmp_blkIdx / (numBlk_d * numBlk_c * numBlk_b * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_d * numBlk_c * numBlk_b * numBlk_a);

	int blk_idx_d = tmp_blkIdx / (numBlk_c * numBlk_b * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_c * numBlk_b * numBlk_a);

	int blk_idx_c = tmp_blkIdx / (numBlk_b * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_b * numBlk_a);

	int blk_idx_b = tmp_blkIdx / numBlk_a;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a);

	int  blk_idx_a = tmp_blkIdx;

	int t3_base_thread = blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_b * SIZE_SLICE_1_B + idx_b + (blk_idx_c * SIZE_SLICE_1_C + idx_c + (blk_idx_d * SIZE_SLICE_1_D + idx_d + (blk_idx_e * SIZE_SLICE_1_E + (blk_idx_f * SIZE_SLICE_1_F) * size_e) * size_d) * size_c) * size_b) * size_a;


	double temp_av;
	double temp_bv[8];
	double reg_tile[8][4];

	for (int i = 0; i < 8; i++)
	for (int j = 0; j < 4; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'y', 't2', ['d', 'f', 'g', 'b']], [16, 'STR_SD2_V2_H7', 'x', 'v2', ['g', 'e', 'a', 'c']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		// Part: Generalized Contraction Index (p7b)
		internal_offset = (l + SIZE_INT_UNIT_1) - size_internal;
		if (internal_offset > 0) internal_upperbound = internal_offset;

		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (threadIdx.y < SIZE_INT_UNIT_1 - internal_upperbound && threadIdx.x < 8)
		for (int ll = 0; ll < 8; ll++)
		{
			// ['d', 'f', 'g', 'b']
			// Exception: Temp. version!: threadIdx.y + l
			// Exception: Temp. version!: idx_a < rng_d
			sm_a[threadIdx.y][threadIdx.x + ll * 8] = dev_t2[blk_idx_d * SIZE_SLICE_1_D + idx_a + (blk_idx_f * SIZE_SLICE_1_F + ll + ((blk_idx_b * SIZE_SLICE_1_B + 0) * size_g) * size_f) * size_d + (threadIdx.y + l) * stride_int_t2];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (threadIdx.x < SIZE_INT_UNIT_1 - internal_upperbound && threadIdx.x < 8 && threadIdx.x < 8)
		for (int ll = 0; ll < 4; ll++)
		{
			// ['g', 'e', 'a', 'c']
			// Exception: Temp. version!: threadIdx.x + l
			// Exception: Temp. version!: idx_d + 0 < rng_a
			sm_b[threadIdx.x][threadIdx.y + 0 + ll * 16] = dev_v2[(blk_idx_e * SIZE_SLICE_1_E + ll + (blk_idx_a * SIZE_SLICE_1_A + idx_d + 0 + (blk_idx_c * SIZE_SLICE_1_C + 0) * size_a) * size_e) * size_g + (threadIdx.x + l)];
			// Exception: Temp. version!: threadIdx.x + l
			// Exception: Temp. version!: idx_d + 8 < rng_a
			if (threadIdx.x + l < size_internal) 
			sm_b[threadIdx.x][threadIdx.y + 8 + ll * 16] = dev_v2[(blk_idx_e * SIZE_SLICE_1_E + ll + (blk_idx_a * SIZE_SLICE_1_A + idx_d + 8 + (blk_idx_c * SIZE_SLICE_1_C + 0) * size_a) * size_e) * size_g + (threadIdx.x + l)];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1 - internal_upperbound; ll++)
		{
			temp_bv[0] = sm_a[ll][idx_d + (idx_b) * SIZE_SLICE_1_D + 0];
			temp_bv[1] = sm_a[ll][idx_d + (idx_b) * SIZE_SLICE_1_D + 8];
			temp_bv[2] = sm_a[ll][idx_d + (idx_b) * SIZE_SLICE_1_D + 16];
			temp_bv[3] = sm_a[ll][idx_d + (idx_b) * SIZE_SLICE_1_D + 24];
			temp_bv[4] = sm_a[ll][idx_d + (idx_b) * SIZE_SLICE_1_D + 32];
			temp_bv[5] = sm_a[ll][idx_d + (idx_b) * SIZE_SLICE_1_D + 40];
			temp_bv[6] = sm_a[ll][idx_d + (idx_b) * SIZE_SLICE_1_D + 48];
			temp_bv[7] = sm_a[ll][idx_d + (idx_b) * SIZE_SLICE_1_D + 56];

			for (int xx = 0; xx < 4; xx++) // (1)
			{
				temp_av = sm_b[ll][idx_a + (idx_c) * SIZE_SLICE_1_A + (xx * 16)];

				reg_tile[0][xx] += temp_av * temp_bv[0];
				reg_tile[1][xx] += temp_av * temp_bv[1];
				reg_tile[2][xx] += temp_av * temp_bv[2];
				reg_tile[3][xx] += temp_av * temp_bv[3];
				reg_tile[4][xx] += temp_av * temp_bv[4];
				reg_tile[5][xx] += temp_av * temp_bv[5];
				reg_tile[6][xx] += temp_av * temp_bv[6];
				reg_tile[7][xx] += temp_av * temp_bv[7];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	#pragma unroll 8
	for (int i = 0; i < 8; i++)
	{
		for (int j = 0; j < 4; j++)
		{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
		}
	}
}

// created by tc_gen_code_Kernel()
__global__ void kernel__3_1(double* dev_t3, 
double* dev_t2, 
double* dev_v2, 
int size_a, int size_b, int size_c, int size_d, int size_e, int size_f, int size_g, 
int numBlk_a, int numBlk_b, int numBlk_c, int numBlk_d, int numBlk_e, int numBlk_f, 
int stride_int_t2, int stride_int_v2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ double sm_a[8][64];
	__shared__ double sm_b[8][64];


	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 2
	// # of indices mapped on TB_Y: 2
	int idx_a = threadIdx.x % SIZE_SLICE_1_A;
	int idx_c = threadIdx.x / SIZE_SLICE_1_A;
	int idx_d = threadIdx.y % SIZE_SLICE_1_D;
	int idx_b = threadIdx.y / SIZE_SLICE_1_D;

	int tmp_blkIdx;
	int blk_idx_f = blockIdx.x / (numBlk_e * numBlk_d * numBlk_c * numBlk_b * numBlk_a);
	tmp_blkIdx = blockIdx.x % (numBlk_e * numBlk_d * numBlk_c * numBlk_b * numBlk_a);

	int blk_idx_e = tmp_blkIdx / (numBlk_d * numBlk_c * numBlk_b * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_d * numBlk_c * numBlk_b * numBlk_a);

	int blk_idx_d = tmp_blkIdx / (numBlk_c * numBlk_b * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_c * numBlk_b * numBlk_a);

	int blk_idx_c = tmp_blkIdx / (numBlk_b * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_b * numBlk_a);

	int blk_idx_b = tmp_blkIdx / numBlk_a;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a);

	int  blk_idx_a = tmp_blkIdx;

	int t3_base_thread = blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_b * SIZE_SLICE_1_B + idx_b + (blk_idx_c * SIZE_SLICE_1_C + idx_c + (blk_idx_d * SIZE_SLICE_1_D + idx_d + (blk_idx_e * SIZE_SLICE_1_E + (blk_idx_f * SIZE_SLICE_1_F) * size_e) * size_d) * size_c) * size_b) * size_a;

	// need to support partial tiles
	int rng_a, rng_b, rng_c, rng_d, rng_e, rng_f;
	if ((size_a - (blk_idx_a * SIZE_SLICE_1_A)) >= SIZE_SLICE_1_A)
	{
		rng_a = SIZE_SLICE_1_A;
	}
	else
	{
		rng_a = size_a % SIZE_SLICE_1_A;
	}
	if ((size_b - (blk_idx_b * SIZE_SLICE_1_B)) >= SIZE_SLICE_1_B)
	{
		rng_b = SIZE_SLICE_1_B;
	}
	else
	{
		rng_b = size_b % SIZE_SLICE_1_B;
	}
	if ((size_c - (blk_idx_c * SIZE_SLICE_1_C)) >= SIZE_SLICE_1_C)
	{
		rng_c = SIZE_SLICE_1_C;
	}
	else
	{
		rng_c = size_c % SIZE_SLICE_1_C;
	}
	if ((size_d - (blk_idx_d * SIZE_SLICE_1_D)) >= SIZE_SLICE_1_D)
	{
		rng_d = SIZE_SLICE_1_D;
	}
	else
	{
		rng_d = size_d % SIZE_SLICE_1_D;
	}
	if ((size_e - (blk_idx_e * SIZE_SLICE_1_E)) >= SIZE_SLICE_1_E)
	{
		rng_e = SIZE_SLICE_1_E;
	}
	else
	{
		rng_e = size_e % SIZE_SLICE_1_E;
	}
	if ((size_f - (blk_idx_f * SIZE_SLICE_1_F)) >= SIZE_SLICE_1_F)
	{
		rng_f = SIZE_SLICE_1_F;
	}
	else
	{
		rng_f = size_f % SIZE_SLICE_1_F;
	}

	double temp_av;
	double temp_bv[8];
	double reg_tile[8][4];

	for (int i = 0; i < 8; i++)
	for (int j = 0; j < 4; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'y', 't2', ['d', 'f', 'g', 'b']], [16, 'STR_SD2_V2_H7', 'x', 'v2', ['g', 'e', 'a', 'c']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (idx_a < rng_d && 0 < rng_b && threadIdx.x < 8)
		for (int ll = 0; ll < rng_f; ll++)
		{
			// ['d', 'f', 'g', 'b']
			// Exception: Temp. version!: threadIdx.y + l
			// Exception: Temp. version!: idx_a < rng_d
			sm_a[threadIdx.y][threadIdx.x + ll * 8] = dev_t2[blk_idx_d * SIZE_SLICE_1_D + idx_a + (blk_idx_f * SIZE_SLICE_1_F + ll + ((blk_idx_b * SIZE_SLICE_1_B + 0) * size_g) * size_f) * size_d + (threadIdx.y + l) * stride_int_t2];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (idx_d < rng_a && 0 < rng_c && threadIdx.x < 8 && threadIdx.x < 8)
		for (int ll = 0; ll < rng_e; ll++)
		{
			// ['g', 'e', 'a', 'c']
			// Exception: Temp. version!: threadIdx.x + l
			// Exception: Temp. version!: idx_d + 0 < rng_a
			sm_b[threadIdx.x][threadIdx.y + 0 + ll * 16] = dev_v2[(blk_idx_e * SIZE_SLICE_1_E + ll + (blk_idx_a * SIZE_SLICE_1_A + idx_d + 0 + (blk_idx_c * SIZE_SLICE_1_C + 0) * size_a) * size_e) * size_g + (threadIdx.x + l)];
			// Exception: Temp. version!: threadIdx.x + l
			// Exception: Temp. version!: idx_d + 8 < rng_a
			if (idx_d + 8 < rng_a) 
			sm_b[threadIdx.x][threadIdx.y + 8 + ll * 16] = dev_v2[(blk_idx_e * SIZE_SLICE_1_E + ll + (blk_idx_a * SIZE_SLICE_1_A + idx_d + 8 + (blk_idx_c * SIZE_SLICE_1_C + 0) * size_a) * size_e) * size_g + (threadIdx.x + l)];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1; ll++)
		{
			temp_bv[0] = sm_a[ll][idx_d + (idx_b) * SIZE_SLICE_1_D + 0];
			temp_bv[1] = sm_a[ll][idx_d + (idx_b) * SIZE_SLICE_1_D + 8];
			temp_bv[2] = sm_a[ll][idx_d + (idx_b) * SIZE_SLICE_1_D + 16];
			temp_bv[3] = sm_a[ll][idx_d + (idx_b) * SIZE_SLICE_1_D + 24];
			temp_bv[4] = sm_a[ll][idx_d + (idx_b) * SIZE_SLICE_1_D + 32];
			temp_bv[5] = sm_a[ll][idx_d + (idx_b) * SIZE_SLICE_1_D + 40];
			temp_bv[6] = sm_a[ll][idx_d + (idx_b) * SIZE_SLICE_1_D + 48];
			temp_bv[7] = sm_a[ll][idx_d + (idx_b) * SIZE_SLICE_1_D + 56];

			for (int xx = 0; xx < 4; xx++) // (1)
			{
				temp_av = sm_b[ll][idx_a + (idx_c) * SIZE_SLICE_1_A + (xx * 16)];

				reg_tile[0][xx] += temp_av * temp_bv[0];
				reg_tile[1][xx] += temp_av * temp_bv[1];
				reg_tile[2][xx] += temp_av * temp_bv[2];
				reg_tile[3][xx] += temp_av * temp_bv[3];
				reg_tile[4][xx] += temp_av * temp_bv[4];
				reg_tile[5][xx] += temp_av * temp_bv[5];
				reg_tile[6][xx] += temp_av * temp_bv[6];
				reg_tile[7][xx] += temp_av * temp_bv[7];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	if (idx_a < rng_a && idx_c < rng_c && idx_d < rng_d && idx_b < rng_b)
	for (int i = 0; i < 8; i++)
	{
		for (int j = 0; j < 4; j++)
		{
			if(i < rng_f && j < rng_e)
			{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
			}
		}
	}
}

// created by tc_gen_code_Kernel()
__global__ void kernel__4_1(double* dev_t3, 
double* dev_t2, 
double* dev_v2, 
int size_a, int size_b, int size_c, int size_d, int size_e, int size_f, int size_g, 
int numBlk_a, int numBlk_b, int numBlk_c, int numBlk_d, int numBlk_e, int numBlk_f, 
int stride_int_t2, int stride_int_v2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ double sm_a[8][64];
	__shared__ double sm_b[8][64];


	int internal_upperbound   = 0;
	int internal_offset;

	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 2
	// # of indices mapped on TB_Y: 2
	int idx_a = threadIdx.x % SIZE_SLICE_1_A;
	int idx_c = threadIdx.x / SIZE_SLICE_1_A;
	int idx_d = threadIdx.y % SIZE_SLICE_1_D;
	int idx_b = threadIdx.y / SIZE_SLICE_1_D;

	int tmp_blkIdx;
	int blk_idx_f = blockIdx.x / (numBlk_e * numBlk_d * numBlk_c * numBlk_b * numBlk_a);
	tmp_blkIdx = blockIdx.x % (numBlk_e * numBlk_d * numBlk_c * numBlk_b * numBlk_a);

	int blk_idx_e = tmp_blkIdx / (numBlk_d * numBlk_c * numBlk_b * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_d * numBlk_c * numBlk_b * numBlk_a);

	int blk_idx_d = tmp_blkIdx / (numBlk_c * numBlk_b * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_c * numBlk_b * numBlk_a);

	int blk_idx_c = tmp_blkIdx / (numBlk_b * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_b * numBlk_a);

	int blk_idx_b = tmp_blkIdx / numBlk_a;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a);

	int  blk_idx_a = tmp_blkIdx;

	int t3_base_thread = blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_b * SIZE_SLICE_1_B + idx_b + (blk_idx_c * SIZE_SLICE_1_C + idx_c + (blk_idx_d * SIZE_SLICE_1_D + idx_d + (blk_idx_e * SIZE_SLICE_1_E + (blk_idx_f * SIZE_SLICE_1_F) * size_e) * size_d) * size_c) * size_b) * size_a;

	// need to support partial tiles
	int rng_a, rng_b, rng_c, rng_d, rng_e, rng_f;
	if ((size_a - (blk_idx_a * SIZE_SLICE_1_A)) >= SIZE_SLICE_1_A)
	{
		rng_a = SIZE_SLICE_1_A;
	}
	else
	{
		rng_a = size_a % SIZE_SLICE_1_A;
	}
	if ((size_b - (blk_idx_b * SIZE_SLICE_1_B)) >= SIZE_SLICE_1_B)
	{
		rng_b = SIZE_SLICE_1_B;
	}
	else
	{
		rng_b = size_b % SIZE_SLICE_1_B;
	}
	if ((size_c - (blk_idx_c * SIZE_SLICE_1_C)) >= SIZE_SLICE_1_C)
	{
		rng_c = SIZE_SLICE_1_C;
	}
	else
	{
		rng_c = size_c % SIZE_SLICE_1_C;
	}
	if ((size_d - (blk_idx_d * SIZE_SLICE_1_D)) >= SIZE_SLICE_1_D)
	{
		rng_d = SIZE_SLICE_1_D;
	}
	else
	{
		rng_d = size_d % SIZE_SLICE_1_D;
	}
	if ((size_e - (blk_idx_e * SIZE_SLICE_1_E)) >= SIZE_SLICE_1_E)
	{
		rng_e = SIZE_SLICE_1_E;
	}
	else
	{
		rng_e = size_e % SIZE_SLICE_1_E;
	}
	if ((size_f - (blk_idx_f * SIZE_SLICE_1_F)) >= SIZE_SLICE_1_F)
	{
		rng_f = SIZE_SLICE_1_F;
	}
	else
	{
		rng_f = size_f % SIZE_SLICE_1_F;
	}

	double temp_av;
	double temp_bv[8];
	double reg_tile[8][4];

	for (int i = 0; i < 8; i++)
	for (int j = 0; j < 4; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'y', 't2', ['d', 'f', 'g', 'b']], [16, 'STR_SD2_V2_H7', 'x', 'v2', ['g', 'e', 'a', 'c']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		// Part: Generalized Contraction Index (p7b)
		internal_offset = (l + SIZE_INT_UNIT_1) - size_internal;
		if (internal_offset > 0) internal_upperbound = internal_offset;

		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (idx_a < rng_d && 0 < rng_b && threadIdx.y < SIZE_INT_UNIT_1 - internal_upperbound && threadIdx.x < 8)
		for (int ll = 0; ll < rng_f; ll++)
		{
			// ['d', 'f', 'g', 'b']
			// Exception: Temp. version!: threadIdx.y + l
			// Exception: Temp. version!: idx_a < rng_d
			sm_a[threadIdx.y][threadIdx.x + ll * 8] = dev_t2[blk_idx_d * SIZE_SLICE_1_D + idx_a + (blk_idx_f * SIZE_SLICE_1_F + ll + ((blk_idx_b * SIZE_SLICE_1_B + 0) * size_g) * size_f) * size_d + (threadIdx.y + l) * stride_int_t2];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (idx_d < rng_a && 0 < rng_c && threadIdx.x < SIZE_INT_UNIT_1 - internal_upperbound && threadIdx.x < 8 && threadIdx.x < 8)
		for (int ll = 0; ll < rng_e; ll++)
		{
			// ['g', 'e', 'a', 'c']
			// Exception: Temp. version!: threadIdx.x + l
			// Exception: Temp. version!: idx_d + 0 < rng_a
			sm_b[threadIdx.x][threadIdx.y + 0 + ll * 16] = dev_v2[(blk_idx_e * SIZE_SLICE_1_E + ll + (blk_idx_a * SIZE_SLICE_1_A + idx_d + 0 + (blk_idx_c * SIZE_SLICE_1_C + 0) * size_a) * size_e) * size_g + (threadIdx.x + l)];
			// Exception: Temp. version!: threadIdx.x + l
			// Exception: Temp. version!: idx_d + 8 < rng_a
			if (threadIdx.x + l < size_internal && idx_d + 8 < rng_a) 
			sm_b[threadIdx.x][threadIdx.y + 8 + ll * 16] = dev_v2[(blk_idx_e * SIZE_SLICE_1_E + ll + (blk_idx_a * SIZE_SLICE_1_A + idx_d + 8 + (blk_idx_c * SIZE_SLICE_1_C + 0) * size_a) * size_e) * size_g + (threadIdx.x + l)];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1 - internal_upperbound; ll++)
		{
			temp_bv[0] = sm_a[ll][idx_d + (idx_b) * SIZE_SLICE_1_D + 0];
			temp_bv[1] = sm_a[ll][idx_d + (idx_b) * SIZE_SLICE_1_D + 8];
			temp_bv[2] = sm_a[ll][idx_d + (idx_b) * SIZE_SLICE_1_D + 16];
			temp_bv[3] = sm_a[ll][idx_d + (idx_b) * SIZE_SLICE_1_D + 24];
			temp_bv[4] = sm_a[ll][idx_d + (idx_b) * SIZE_SLICE_1_D + 32];
			temp_bv[5] = sm_a[ll][idx_d + (idx_b) * SIZE_SLICE_1_D + 40];
			temp_bv[6] = sm_a[ll][idx_d + (idx_b) * SIZE_SLICE_1_D + 48];
			temp_bv[7] = sm_a[ll][idx_d + (idx_b) * SIZE_SLICE_1_D + 56];

			for (int xx = 0; xx < 4; xx++) // (1)
			{
				temp_av = sm_b[ll][idx_a + (idx_c) * SIZE_SLICE_1_A + (xx * 16)];

				reg_tile[0][xx] += temp_av * temp_bv[0];
				reg_tile[1][xx] += temp_av * temp_bv[1];
				reg_tile[2][xx] += temp_av * temp_bv[2];
				reg_tile[3][xx] += temp_av * temp_bv[3];
				reg_tile[4][xx] += temp_av * temp_bv[4];
				reg_tile[5][xx] += temp_av * temp_bv[5];
				reg_tile[6][xx] += temp_av * temp_bv[6];
				reg_tile[7][xx] += temp_av * temp_bv[7];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	if (idx_a < rng_a && idx_c < rng_c && idx_d < rng_d && idx_b < rng_b)
	for (int i = 0; i < 8; i++)
	{
		for (int j = 0; j < 4; j++)
		{
			if(i < rng_f && j < rng_e)
			{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
			}
		}
	}
}

// written by tc_interface.tc_gen_code_interface_Header()
extern "C"
void sd_t_d2_fusion(int size_a, int size_b, int size_c, int size_d, int size_e, int size_f, int size_g, double* t3, double* host_t2, double* host_v2, int cond_kernel_1, int opt_register_transpose)
{
	int num_thread_blocks_kernel_1;

	double* dev_t3;
	double* dev_t2;
	double* dev_v2;


	num_thread_blocks_kernel_1 = CEIL(size_a, SIZE_SLICE_1_A) * CEIL(size_b, SIZE_SLICE_1_B) * CEIL(size_c, SIZE_SLICE_1_C) * CEIL(size_d, SIZE_SLICE_1_D) * CEIL(size_e, SIZE_SLICE_1_E) * CEIL(size_f, SIZE_SLICE_1_F);
	// cudaMalloc()
	hipMalloc((void**) &dev_t3, sizeof(double) * size_a * size_b * size_c * size_d * size_e * size_f);
	hipMalloc((void**) &dev_t2, sizeof(double) * size_b * size_g * size_f * size_d);
	hipMalloc((void**) &dev_v2, sizeof(double) * size_c * size_a * size_e * size_g);

	// cudaMemcpy()
	hipMemcpy(dev_t3, t3, sizeof(double) * size_a * size_b * size_c * size_d * size_e * size_f, hipMemcpyHostToDevice);
	hipMemcpy(dev_t2, host_t2, sizeof(double) * size_b * size_g * size_f * size_d, hipMemcpyHostToDevice);
	hipMemcpy(dev_v2, host_v2, sizeof(double) * size_c * size_a * size_e * size_g, hipMemcpyHostToDevice);

	// Related to Kernels
	// There are 1 Basic Kernels
	long long int tmp_operations = 2 * (long long int)(size_a * size_b * size_c * size_d * size_e * size_f) * size_g;
	printf ("========================================= fusedKernels =============================================\n");
	printf ("		Grid Size  : %6d (1D)\n", num_thread_blocks_kernel_1);
	printf ("		Block-size : %2d, %2d (2D)\n", SIZE_TB_1_X, SIZE_TB_1_Y);
	printf ("		Reg.-size  : %2d, %2d (2D)\n", SIZE_REG_1_X, SIZE_REG_1_Y);
	printf ("		A thread deals with (%d x %d) elements (basically)\n", SIZE_TB_1_X * SIZE_REG_1_X, SIZE_TB_1_Y * SIZE_REG_1_Y);
	printf ("		# of Operations: %lld\n", tmp_operations);
	printf ("====================================================================================================\n");
	dim3 gridsize_1(num_thread_blocks_kernel_1);
	dim3 blocksize_1(SIZE_TB_1_X, SIZE_TB_1_Y);

	int stride_output_a = 1;
	int stride_output_b = stride_output_a * size_a;
	int stride_output_c = stride_output_b * size_b;
	int stride_output_d = stride_output_c * size_c;
	int stride_output_e = stride_output_d * size_d;
	int stride_output_f = stride_output_e * size_e;

	int stride_reg_x_1 = stride_output_e;
	int stride_reg_y_1 = stride_output_f;

	int size_internal = size_g;

	int stride_int_t2 = size_d * size_f;
	int stride_int_v2 = 1;

	// Decision Tree for Kernel Types
	// No Chance to Utilize the Register Transpose
	if (size_a % SIZE_SLICE_1_A == 0 && size_b % SIZE_SLICE_1_B == 0 && size_c % SIZE_SLICE_1_C == 0 && size_d % SIZE_SLICE_1_D == 0 && size_e % SIZE_SLICE_1_E == 0 && size_f % SIZE_SLICE_1_F == 0)
	{
		// [2] Extenral Index: Full
		if (size_g % SIZE_SLICE_1_G == 0)
		{
			// [3] Internal Index: Full
			// >>> External: Full && Internal: Full
			printf ("External: Full, Internal: Full\n");
			kernel__1_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a, size_b, size_c, size_d, size_e, size_f, size_g, CEIL(size_a, SIZE_SLICE_1_A), CEIL(size_b, SIZE_SLICE_1_B), CEIL(size_c, SIZE_SLICE_1_C), CEIL(size_d, SIZE_SLICE_1_D), CEIL(size_e, SIZE_SLICE_1_E), CEIL(size_f, SIZE_SLICE_1_F), stride_int_t2, stride_int_v2, stride_reg_x_1, stride_reg_y_1, size_internal);
		}
		else
		{
			// [4] Internal Index: Partial
			// >>> External: Full && Internal: Partial
			printf ("External: Full, Internal: Partial\n");
			kernel__2_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a, size_b, size_c, size_d, size_e, size_f, size_g, CEIL(size_a, SIZE_SLICE_1_A), CEIL(size_b, SIZE_SLICE_1_B), CEIL(size_c, SIZE_SLICE_1_C), CEIL(size_d, SIZE_SLICE_1_D), CEIL(size_e, SIZE_SLICE_1_E), CEIL(size_f, SIZE_SLICE_1_F), stride_int_t2, stride_int_v2, stride_reg_x_1, stride_reg_y_1, size_internal);
		}
	}
	else
	{
		// [2] Extenral Index: Partial
		if (size_g % SIZE_SLICE_1_G == 0)
		{
			// [3] Internal Index: Full
			// >>> External: Partial && Internal: Full
			printf ("External: Partial, Internal: Full\n");
			kernel__3_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a, size_b, size_c, size_d, size_e, size_f, size_g, CEIL(size_a, SIZE_SLICE_1_A), CEIL(size_b, SIZE_SLICE_1_B), CEIL(size_c, SIZE_SLICE_1_C), CEIL(size_d, SIZE_SLICE_1_D), CEIL(size_e, SIZE_SLICE_1_E), CEIL(size_f, SIZE_SLICE_1_F), stride_int_t2, stride_int_v2, stride_reg_x_1, stride_reg_y_1, size_internal);
		}
		else
		{
			// [4] Internal Index: Partial
			// >>> External: Partial && Internal: Partial
			printf ("External: Partial, Internal: Partial\n");
			kernel__4_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a, size_b, size_c, size_d, size_e, size_f, size_g, CEIL(size_a, SIZE_SLICE_1_A), CEIL(size_b, SIZE_SLICE_1_B), CEIL(size_c, SIZE_SLICE_1_C), CEIL(size_d, SIZE_SLICE_1_D), CEIL(size_e, SIZE_SLICE_1_E), CEIL(size_f, SIZE_SLICE_1_F), stride_int_t2, stride_int_v2, stride_reg_x_1, stride_reg_y_1, size_internal);
		}
	}

	// Copy the Result from Device to Host
	hipMemcpy(t3, dev_t3, sizeof(double) * (size_a * size_b * size_c * size_d * size_e * size_f), hipMemcpyDeviceToHost);

	// cudaFree()
	hipFree(dev_t3);	hipFree(dev_t2);	hipFree(dev_v2);

	// Shoule be Fixed
	// HostFree

}

// This is written by tc_interface.tc_gen_code_interface()
// This Interface Should be Called to Run the Kernels
extern "C"
void sd_t_d2_fusion_(int size_a, int size_b, int size_c, int size_d, int size_e, int size_f, int size_g, double* t3, double* t2, double* v2, int cond_kernel_1, int opt_register_transpose)
{
	// Pre-Processing for Split
	// Based on Tile-Sizes and Problem-Size
	// Currently, one index can be split into two indices

	// Call An Application
	sd_t_d2_fusion(size_a, size_b, size_c, size_d, size_e, size_f, size_g, t3, t2, v2, cond_kernel_1, opt_register_transpose);
}
