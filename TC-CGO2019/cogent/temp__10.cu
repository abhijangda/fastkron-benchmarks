// created by tc_code_include() in tc_code_include.py

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <sys/time.h>
#include <locale.h>
#include <algorithm>
using namespace std;

// created by tc_gen_definition_new()
#define SIZE_SLICE_1_E 16
#define SIZE_SLICE_1_A 16
#define SIZE_SLICE_1_C 6
#define SIZE_SLICE_1_D 1
#define SIZE_SLICE_1_B1 16
#define SIZE_SLICE_1_B2 6

#define SIZE_INT_UNIT_1 SIZE_SLICE_1_E

#define SIZE_TB_1_X 	SIZE_SLICE_1_A * SIZE_SLICE_1_D
#define SIZE_TB_1_Y 	SIZE_SLICE_1_B1
#define SIZE_REG_1_X 	SIZE_SLICE_1_C
#define SIZE_REG_1_Y 	SIZE_SLICE_1_B2

#define NUM_INDEX 		5
#define CEIL(a, b) 		(((a) + (b) - 1) / (b))

// created by tc_gen_code_Kernel()
__global__ void kernel__1_1(double* dev_t3, 
double* dev_t2, 
double* dev_v2, 
int size_a, int size_b1, int size_b2, int size_c, int size_d, int size_e, 
int numBlk_a, int numBlk_b1, int numBlk_b2, int numBlk_c, int numBlk_d, 
int stride_int_t2, int stride_int_v2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ double sm_a[16][96];
	__shared__ double sm_b[16][96];


	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 2
	// # of indices mapped on TB_Y: 1
	int idx_a = threadIdx.x % SIZE_SLICE_1_A;
	int idx_d = threadIdx.x / SIZE_SLICE_1_A;
	int idx_b1 = threadIdx.y;

	int tmp_blkIdx;
	int blk_idx_d = blockIdx.x / (numBlk_c * numBlk_b2 * numBlk_b1 * numBlk_a);
	tmp_blkIdx = blockIdx.x % (numBlk_c * numBlk_b2 * numBlk_b1 * numBlk_a);

	int blk_idx_c = tmp_blkIdx / (numBlk_b2 * numBlk_b1 * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_b2 * numBlk_b1 * numBlk_a);

	int blk_idx_b2 = tmp_blkIdx / (numBlk_b1 * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_b1 * numBlk_a);

	int blk_idx_b1 = tmp_blkIdx / numBlk_a;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a);

	int  blk_idx_a = tmp_blkIdx;

	int t3_base_thread = blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_b1 * SIZE_SLICE_1_B1 + idx_b1 + (blk_idx_b2 * SIZE_SLICE_1_B2 + (blk_idx_c * SIZE_SLICE_1_C + (blk_idx_d * SIZE_SLICE_1_D + idx_d) * size_c) * size_b2) * size_b1) * size_a;


	double temp_av;
	double temp_bv[6];
	double reg_tile[6][6];

	for (int i = 0; i < 6; i++)
	for (int j = 0; j < 6; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'y', 't2', ['e', 'b1', 'b2']], [16, 'STR_SD2_V2_H7', 'x', 'v2', ['a', 'e', 'c', 'd']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		// No Need to Put Boundary-Checks before For-Statement: : 
		for (int ll = 0; ll < 6; ll++)
		{
			// ['e', 'b1', 'b2']
			// Exception: Temp. version!: threadIdx.x + l
			// Exception: Temp. version!: idx_b1 < rng_b1
			sm_a[threadIdx.x][threadIdx.y + ll * 16] = dev_t2[(blk_idx_b1 * SIZE_SLICE_1_B1 + idx_b1 + (blk_idx_b2 * SIZE_SLICE_1_B2 + ll) * size_b1) * size_e + (threadIdx.x + l)];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		// No Need to Put Boundary-Checks before For-Statement: : 
		for (int ll = 0; ll < 6; ll++)
		{
			// ['a', 'e', 'c', 'd']
			// Exception: Temp. version!: threadIdx.y + l
			// Exception: Temp. version!: idx_a < rng_a
			sm_b[threadIdx.y][threadIdx.x + ll * 16] = dev_v2[blk_idx_a * SIZE_SLICE_1_A + idx_a + ((blk_idx_c * SIZE_SLICE_1_C + ll + (blk_idx_d * SIZE_SLICE_1_D + 0) * size_c) * size_e) * size_a + (threadIdx.y + l) * stride_int_v2];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1; ll++)
		{
			temp_bv[0] = sm_a[ll][idx_b1 + 0];
			temp_bv[1] = sm_a[ll][idx_b1 + 16];
			temp_bv[2] = sm_a[ll][idx_b1 + 32];
			temp_bv[3] = sm_a[ll][idx_b1 + 48];
			temp_bv[4] = sm_a[ll][idx_b1 + 64];
			temp_bv[5] = sm_a[ll][idx_b1 + 80];

			for (int xx = 0; xx < 6; xx++) // (1)
			{
				temp_av = sm_b[ll][idx_a + (idx_d) * SIZE_SLICE_1_A + (xx * 16)];

				reg_tile[0][xx] += temp_av * temp_bv[0];
				reg_tile[1][xx] += temp_av * temp_bv[1];
				reg_tile[2][xx] += temp_av * temp_bv[2];
				reg_tile[3][xx] += temp_av * temp_bv[3];
				reg_tile[4][xx] += temp_av * temp_bv[4];
				reg_tile[5][xx] += temp_av * temp_bv[5];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	#pragma unroll 6
	for (int i = 0; i < 6; i++)
	{
		for (int j = 0; j < 6; j++)
		{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
		}
	}
}

// created by tc_gen_code_Kernel()
__global__ void kernel__2_1(double* dev_t3, 
double* dev_t2, 
double* dev_v2, 
int size_a, int size_b1, int size_b2, int size_c, int size_d, int size_e, 
int numBlk_a, int numBlk_b1, int numBlk_b2, int numBlk_c, int numBlk_d, 
int stride_int_t2, int stride_int_v2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ double sm_a[16][96];
	__shared__ double sm_b[16][96];


	int internal_upperbound   = 0;
	int internal_offset;

	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 2
	// # of indices mapped on TB_Y: 1
	int idx_a = threadIdx.x % SIZE_SLICE_1_A;
	int idx_d = threadIdx.x / SIZE_SLICE_1_A;
	int idx_b1 = threadIdx.y;

	int tmp_blkIdx;
	int blk_idx_d = blockIdx.x / (numBlk_c * numBlk_b2 * numBlk_b1 * numBlk_a);
	tmp_blkIdx = blockIdx.x % (numBlk_c * numBlk_b2 * numBlk_b1 * numBlk_a);

	int blk_idx_c = tmp_blkIdx / (numBlk_b2 * numBlk_b1 * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_b2 * numBlk_b1 * numBlk_a);

	int blk_idx_b2 = tmp_blkIdx / (numBlk_b1 * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_b1 * numBlk_a);

	int blk_idx_b1 = tmp_blkIdx / numBlk_a;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a);

	int  blk_idx_a = tmp_blkIdx;

	int t3_base_thread = blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_b1 * SIZE_SLICE_1_B1 + idx_b1 + (blk_idx_b2 * SIZE_SLICE_1_B2 + (blk_idx_c * SIZE_SLICE_1_C + (blk_idx_d * SIZE_SLICE_1_D + idx_d) * size_c) * size_b2) * size_b1) * size_a;


	double temp_av;
	double temp_bv[6];
	double reg_tile[6][6];

	for (int i = 0; i < 6; i++)
	for (int j = 0; j < 6; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'y', 't2', ['e', 'b1', 'b2']], [16, 'STR_SD2_V2_H7', 'x', 'v2', ['a', 'e', 'c', 'd']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		// Part: Generalized Contraction Index (p7b)
		internal_offset = (l + SIZE_INT_UNIT_1) - size_internal;
		if (internal_offset > 0) internal_upperbound = internal_offset;

		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (threadIdx.x < SIZE_INT_UNIT_1 - internal_upperbound)
		for (int ll = 0; ll < 6; ll++)
		{
			// ['e', 'b1', 'b2']
			// Exception: Temp. version!: threadIdx.x + l
			// Exception: Temp. version!: idx_b1 < rng_b1
			sm_a[threadIdx.x][threadIdx.y + ll * 16] = dev_t2[(blk_idx_b1 * SIZE_SLICE_1_B1 + idx_b1 + (blk_idx_b2 * SIZE_SLICE_1_B2 + ll) * size_b1) * size_e + (threadIdx.x + l)];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (threadIdx.y < SIZE_INT_UNIT_1 - internal_upperbound)
		for (int ll = 0; ll < 6; ll++)
		{
			// ['a', 'e', 'c', 'd']
			// Exception: Temp. version!: threadIdx.y + l
			// Exception: Temp. version!: idx_a < rng_a
			sm_b[threadIdx.y][threadIdx.x + ll * 16] = dev_v2[blk_idx_a * SIZE_SLICE_1_A + idx_a + ((blk_idx_c * SIZE_SLICE_1_C + ll + (blk_idx_d * SIZE_SLICE_1_D + 0) * size_c) * size_e) * size_a + (threadIdx.y + l) * stride_int_v2];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1 - internal_upperbound; ll++)
		{
			temp_bv[0] = sm_a[ll][idx_b1 + 0];
			temp_bv[1] = sm_a[ll][idx_b1 + 16];
			temp_bv[2] = sm_a[ll][idx_b1 + 32];
			temp_bv[3] = sm_a[ll][idx_b1 + 48];
			temp_bv[4] = sm_a[ll][idx_b1 + 64];
			temp_bv[5] = sm_a[ll][idx_b1 + 80];

			for (int xx = 0; xx < 6; xx++) // (1)
			{
				temp_av = sm_b[ll][idx_a + (idx_d) * SIZE_SLICE_1_A + (xx * 16)];

				reg_tile[0][xx] += temp_av * temp_bv[0];
				reg_tile[1][xx] += temp_av * temp_bv[1];
				reg_tile[2][xx] += temp_av * temp_bv[2];
				reg_tile[3][xx] += temp_av * temp_bv[3];
				reg_tile[4][xx] += temp_av * temp_bv[4];
				reg_tile[5][xx] += temp_av * temp_bv[5];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	#pragma unroll 6
	for (int i = 0; i < 6; i++)
	{
		for (int j = 0; j < 6; j++)
		{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
		}
	}
}

// created by tc_gen_code_Kernel()
__global__ void kernel__3_1(double* dev_t3, 
double* dev_t2, 
double* dev_v2, 
int size_a, int size_b1, int size_b2, int size_c, int size_d, int size_e, 
int numBlk_a, int numBlk_b1, int numBlk_b2, int numBlk_c, int numBlk_d, 
int stride_int_t2, int stride_int_v2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ double sm_a[16][96];
	__shared__ double sm_b[16][96];


	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 2
	// # of indices mapped on TB_Y: 1
	int idx_a = threadIdx.x % SIZE_SLICE_1_A;
	int idx_d = threadIdx.x / SIZE_SLICE_1_A;
	int idx_b1 = threadIdx.y;

	int tmp_blkIdx;
	int blk_idx_d = blockIdx.x / (numBlk_c * numBlk_b2 * numBlk_b1 * numBlk_a);
	tmp_blkIdx = blockIdx.x % (numBlk_c * numBlk_b2 * numBlk_b1 * numBlk_a);

	int blk_idx_c = tmp_blkIdx / (numBlk_b2 * numBlk_b1 * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_b2 * numBlk_b1 * numBlk_a);

	int blk_idx_b2 = tmp_blkIdx / (numBlk_b1 * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_b1 * numBlk_a);

	int blk_idx_b1 = tmp_blkIdx / numBlk_a;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a);

	int  blk_idx_a = tmp_blkIdx;

	int t3_base_thread = blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_b1 * SIZE_SLICE_1_B1 + idx_b1 + (blk_idx_b2 * SIZE_SLICE_1_B2 + (blk_idx_c * SIZE_SLICE_1_C + (blk_idx_d * SIZE_SLICE_1_D + idx_d) * size_c) * size_b2) * size_b1) * size_a;

	// need to support partial tiles
	int rng_a, rng_b1, rng_b2, rng_c, rng_d;
	if ((size_a - (blk_idx_a * SIZE_SLICE_1_A)) >= SIZE_SLICE_1_A)
	{
		rng_a = SIZE_SLICE_1_A;
	}
	else
	{
		rng_a = size_a % SIZE_SLICE_1_A;
	}
	if ((size_b1 - (blk_idx_b1 * SIZE_SLICE_1_B1)) >= SIZE_SLICE_1_B1)
	{
		rng_b1 = SIZE_SLICE_1_B1;
	}
	else
	{
		rng_b1 = size_b1 % SIZE_SLICE_1_B1;
	}
	if ((size_b2 - (blk_idx_b2 * SIZE_SLICE_1_B2)) >= SIZE_SLICE_1_B2)
	{
		rng_b2 = SIZE_SLICE_1_B2;
	}
	else
	{
		rng_b2 = size_b2 % SIZE_SLICE_1_B2;
	}
	if ((size_c - (blk_idx_c * SIZE_SLICE_1_C)) >= SIZE_SLICE_1_C)
	{
		rng_c = SIZE_SLICE_1_C;
	}
	else
	{
		rng_c = size_c % SIZE_SLICE_1_C;
	}
	if ((size_d - (blk_idx_d * SIZE_SLICE_1_D)) >= SIZE_SLICE_1_D)
	{
		rng_d = SIZE_SLICE_1_D;
	}
	else
	{
		rng_d = size_d % SIZE_SLICE_1_D;
	}

	double temp_av;
	double temp_bv[6];
	double reg_tile[6][6];

	for (int i = 0; i < 6; i++)
	for (int j = 0; j < 6; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'y', 't2', ['e', 'b1', 'b2']], [16, 'STR_SD2_V2_H7', 'x', 'v2', ['a', 'e', 'c', 'd']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (idx_b1 < rng_b1)
		for (int ll = 0; ll < rng_b2; ll++)
		{
			// ['e', 'b1', 'b2']
			// Exception: Temp. version!: threadIdx.x + l
			// Exception: Temp. version!: idx_b1 < rng_b1
			sm_a[threadIdx.x][threadIdx.y + ll * 16] = dev_t2[(blk_idx_b1 * SIZE_SLICE_1_B1 + idx_b1 + (blk_idx_b2 * SIZE_SLICE_1_B2 + ll) * size_b1) * size_e + (threadIdx.x + l)];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (idx_a < rng_a && 0 < rng_d)
		for (int ll = 0; ll < rng_c; ll++)
		{
			// ['a', 'e', 'c', 'd']
			// Exception: Temp. version!: threadIdx.y + l
			// Exception: Temp. version!: idx_a < rng_a
			sm_b[threadIdx.y][threadIdx.x + ll * 16] = dev_v2[blk_idx_a * SIZE_SLICE_1_A + idx_a + ((blk_idx_c * SIZE_SLICE_1_C + ll + (blk_idx_d * SIZE_SLICE_1_D + 0) * size_c) * size_e) * size_a + (threadIdx.y + l) * stride_int_v2];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1; ll++)
		{
			temp_bv[0] = sm_a[ll][idx_b1 + 0];
			temp_bv[1] = sm_a[ll][idx_b1 + 16];
			temp_bv[2] = sm_a[ll][idx_b1 + 32];
			temp_bv[3] = sm_a[ll][idx_b1 + 48];
			temp_bv[4] = sm_a[ll][idx_b1 + 64];
			temp_bv[5] = sm_a[ll][idx_b1 + 80];

			for (int xx = 0; xx < 6; xx++) // (1)
			{
				temp_av = sm_b[ll][idx_a + (idx_d) * SIZE_SLICE_1_A + (xx * 16)];

				reg_tile[0][xx] += temp_av * temp_bv[0];
				reg_tile[1][xx] += temp_av * temp_bv[1];
				reg_tile[2][xx] += temp_av * temp_bv[2];
				reg_tile[3][xx] += temp_av * temp_bv[3];
				reg_tile[4][xx] += temp_av * temp_bv[4];
				reg_tile[5][xx] += temp_av * temp_bv[5];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	if (idx_a < rng_a && idx_d < rng_d && idx_b1 < rng_b1)
	for (int i = 0; i < 6; i++)
	{
		for (int j = 0; j < 6; j++)
		{
			if(i < rng_b2 && j < rng_c)
			{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
			}
		}
	}
}

// created by tc_gen_code_Kernel()
__global__ void kernel__4_1(double* dev_t3, 
double* dev_t2, 
double* dev_v2, 
int size_a, int size_b1, int size_b2, int size_c, int size_d, int size_e, 
int numBlk_a, int numBlk_b1, int numBlk_b2, int numBlk_c, int numBlk_d, 
int stride_int_t2, int stride_int_v2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ double sm_a[16][96];
	__shared__ double sm_b[16][96];


	int internal_upperbound   = 0;
	int internal_offset;

	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 2
	// # of indices mapped on TB_Y: 1
	int idx_a = threadIdx.x % SIZE_SLICE_1_A;
	int idx_d = threadIdx.x / SIZE_SLICE_1_A;
	int idx_b1 = threadIdx.y;

	int tmp_blkIdx;
	int blk_idx_d = blockIdx.x / (numBlk_c * numBlk_b2 * numBlk_b1 * numBlk_a);
	tmp_blkIdx = blockIdx.x % (numBlk_c * numBlk_b2 * numBlk_b1 * numBlk_a);

	int blk_idx_c = tmp_blkIdx / (numBlk_b2 * numBlk_b1 * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_b2 * numBlk_b1 * numBlk_a);

	int blk_idx_b2 = tmp_blkIdx / (numBlk_b1 * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_b1 * numBlk_a);

	int blk_idx_b1 = tmp_blkIdx / numBlk_a;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a);

	int  blk_idx_a = tmp_blkIdx;

	int t3_base_thread = blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_b1 * SIZE_SLICE_1_B1 + idx_b1 + (blk_idx_b2 * SIZE_SLICE_1_B2 + (blk_idx_c * SIZE_SLICE_1_C + (blk_idx_d * SIZE_SLICE_1_D + idx_d) * size_c) * size_b2) * size_b1) * size_a;

	// need to support partial tiles
	int rng_a, rng_b1, rng_b2, rng_c, rng_d;
	if ((size_a - (blk_idx_a * SIZE_SLICE_1_A)) >= SIZE_SLICE_1_A)
	{
		rng_a = SIZE_SLICE_1_A;
	}
	else
	{
		rng_a = size_a % SIZE_SLICE_1_A;
	}
	if ((size_b1 - (blk_idx_b1 * SIZE_SLICE_1_B1)) >= SIZE_SLICE_1_B1)
	{
		rng_b1 = SIZE_SLICE_1_B1;
	}
	else
	{
		rng_b1 = size_b1 % SIZE_SLICE_1_B1;
	}
	if ((size_b2 - (blk_idx_b2 * SIZE_SLICE_1_B2)) >= SIZE_SLICE_1_B2)
	{
		rng_b2 = SIZE_SLICE_1_B2;
	}
	else
	{
		rng_b2 = size_b2 % SIZE_SLICE_1_B2;
	}
	if ((size_c - (blk_idx_c * SIZE_SLICE_1_C)) >= SIZE_SLICE_1_C)
	{
		rng_c = SIZE_SLICE_1_C;
	}
	else
	{
		rng_c = size_c % SIZE_SLICE_1_C;
	}
	if ((size_d - (blk_idx_d * SIZE_SLICE_1_D)) >= SIZE_SLICE_1_D)
	{
		rng_d = SIZE_SLICE_1_D;
	}
	else
	{
		rng_d = size_d % SIZE_SLICE_1_D;
	}

	double temp_av;
	double temp_bv[6];
	double reg_tile[6][6];

	for (int i = 0; i < 6; i++)
	for (int j = 0; j < 6; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'y', 't2', ['e', 'b1', 'b2']], [16, 'STR_SD2_V2_H7', 'x', 'v2', ['a', 'e', 'c', 'd']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		// Part: Generalized Contraction Index (p7b)
		internal_offset = (l + SIZE_INT_UNIT_1) - size_internal;
		if (internal_offset > 0) internal_upperbound = internal_offset;

		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (idx_b1 < rng_b1 && threadIdx.x < SIZE_INT_UNIT_1 - internal_upperbound)
		for (int ll = 0; ll < rng_b2; ll++)
		{
			// ['e', 'b1', 'b2']
			// Exception: Temp. version!: threadIdx.x + l
			// Exception: Temp. version!: idx_b1 < rng_b1
			sm_a[threadIdx.x][threadIdx.y + ll * 16] = dev_t2[(blk_idx_b1 * SIZE_SLICE_1_B1 + idx_b1 + (blk_idx_b2 * SIZE_SLICE_1_B2 + ll) * size_b1) * size_e + (threadIdx.x + l)];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (idx_a < rng_a && 0 < rng_d && threadIdx.y < SIZE_INT_UNIT_1 - internal_upperbound)
		for (int ll = 0; ll < rng_c; ll++)
		{
			// ['a', 'e', 'c', 'd']
			// Exception: Temp. version!: threadIdx.y + l
			// Exception: Temp. version!: idx_a < rng_a
			sm_b[threadIdx.y][threadIdx.x + ll * 16] = dev_v2[blk_idx_a * SIZE_SLICE_1_A + idx_a + ((blk_idx_c * SIZE_SLICE_1_C + ll + (blk_idx_d * SIZE_SLICE_1_D + 0) * size_c) * size_e) * size_a + (threadIdx.y + l) * stride_int_v2];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1 - internal_upperbound; ll++)
		{
			temp_bv[0] = sm_a[ll][idx_b1 + 0];
			temp_bv[1] = sm_a[ll][idx_b1 + 16];
			temp_bv[2] = sm_a[ll][idx_b1 + 32];
			temp_bv[3] = sm_a[ll][idx_b1 + 48];
			temp_bv[4] = sm_a[ll][idx_b1 + 64];
			temp_bv[5] = sm_a[ll][idx_b1 + 80];

			for (int xx = 0; xx < 6; xx++) // (1)
			{
				temp_av = sm_b[ll][idx_a + (idx_d) * SIZE_SLICE_1_A + (xx * 16)];

				reg_tile[0][xx] += temp_av * temp_bv[0];
				reg_tile[1][xx] += temp_av * temp_bv[1];
				reg_tile[2][xx] += temp_av * temp_bv[2];
				reg_tile[3][xx] += temp_av * temp_bv[3];
				reg_tile[4][xx] += temp_av * temp_bv[4];
				reg_tile[5][xx] += temp_av * temp_bv[5];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	if (idx_a < rng_a && idx_d < rng_d && idx_b1 < rng_b1)
	for (int i = 0; i < 6; i++)
	{
		for (int j = 0; j < 6; j++)
		{
			if(i < rng_b2 && j < rng_c)
			{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
			}
		}
	}
}

// written by tc_interface.tc_gen_code_interface_Header()
extern "C"
void sd_t_d2_fusion(int size_a, int size_b1, int size_b2, int size_c, int size_d, int size_e, double* t3, double* host_t2, double* host_v2, int cond_kernel_1, int opt_register_transpose)
{
	int num_thread_blocks_kernel_1;

	double* dev_t3;
	double* dev_t2;
	double* dev_v2;


	num_thread_blocks_kernel_1 = CEIL(size_a, SIZE_SLICE_1_A) * CEIL(size_b1, SIZE_SLICE_1_B1) * CEIL(size_b2, SIZE_SLICE_1_B2) * CEIL(size_c, SIZE_SLICE_1_C) * CEIL(size_d, SIZE_SLICE_1_D);
	// cudaMalloc()
	hipMalloc((void**) &dev_t3, sizeof(double) * size_a * size_b1 * size_b2 * size_c * size_d);
	hipMalloc((void**) &dev_t2, sizeof(double) * size_b2 * size_b1 * size_e);
	hipMalloc((void**) &dev_v2, sizeof(double) * size_d * size_c * size_e * size_a);

	// cudaMemcpy()
	hipMemcpy(dev_t3, t3, sizeof(double) * size_a * size_b1 * size_b2 * size_c * size_d, hipMemcpyHostToDevice);
	hipMemcpy(dev_t2, host_t2, sizeof(double) * size_b2 * size_b1 * size_e, hipMemcpyHostToDevice);
	hipMemcpy(dev_v2, host_v2, sizeof(double) * size_d * size_c * size_e * size_a, hipMemcpyHostToDevice);

	// Related to Kernels
	// There are 1 Basic Kernels
	long long int tmp_operations = 2 * (long long int)(size_a * size_b1 * size_b2 * size_c * size_d) * size_e;
	printf ("========================================= fusedKernels =============================================\n");
	printf ("		Grid Size  : %6d (1D)\n", num_thread_blocks_kernel_1);
	printf ("		Block-size : %2d, %2d (2D)\n", SIZE_TB_1_X, SIZE_TB_1_Y);
	printf ("		Reg.-size  : %2d, %2d (2D)\n", SIZE_REG_1_X, SIZE_REG_1_Y);
	printf ("		A thread deals with (%d x %d) elements (basically)\n", SIZE_TB_1_X * SIZE_REG_1_X, SIZE_TB_1_Y * SIZE_REG_1_Y);
	printf ("		# of Operations: %lld\n", tmp_operations);
	printf ("====================================================================================================\n");
	dim3 gridsize_1(num_thread_blocks_kernel_1);
	dim3 blocksize_1(SIZE_TB_1_X, SIZE_TB_1_Y);

	int stride_output_a = 1;
	int stride_output_b1 = stride_output_a * size_a;
	int stride_output_b2 = stride_output_b1 * size_b1;
	int stride_output_c = stride_output_b2 * size_b2;
	int stride_output_d = stride_output_c * size_c;

	int stride_reg_x_1 = stride_output_c;
	int stride_reg_y_1 = stride_output_b2;

	int size_internal = size_e;

	int stride_int_t2 = 1;
	int stride_int_v2 = size_a;

	// Decision Tree for Kernel Types
	// No Chance to Utilize the Register Transpose
	if (size_a % SIZE_SLICE_1_A == 0 && size_b1 % SIZE_SLICE_1_B1 == 0 && size_b2 % SIZE_SLICE_1_B2 == 0 && size_c % SIZE_SLICE_1_C == 0 && size_d % SIZE_SLICE_1_D == 0)
	{
		// [2] Extenral Index: Full
		if (size_e % SIZE_SLICE_1_E == 0)
		{
			// [3] Internal Index: Full
			// >>> External: Full && Internal: Full
			printf ("External: Full, Internal: Full\n");
			kernel__1_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a, size_b1, size_b2, size_c, size_d, size_e, CEIL(size_a, SIZE_SLICE_1_A), CEIL(size_b1, SIZE_SLICE_1_B1), CEIL(size_b2, SIZE_SLICE_1_B2), CEIL(size_c, SIZE_SLICE_1_C), CEIL(size_d, SIZE_SLICE_1_D), stride_int_t2, stride_int_v2, stride_reg_x_1, stride_reg_y_1, size_internal);
		}
		else
		{
			// [4] Internal Index: Partial
			// >>> External: Full && Internal: Partial
			printf ("External: Full, Internal: Partial\n");
			kernel__2_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a, size_b1, size_b2, size_c, size_d, size_e, CEIL(size_a, SIZE_SLICE_1_A), CEIL(size_b1, SIZE_SLICE_1_B1), CEIL(size_b2, SIZE_SLICE_1_B2), CEIL(size_c, SIZE_SLICE_1_C), CEIL(size_d, SIZE_SLICE_1_D), stride_int_t2, stride_int_v2, stride_reg_x_1, stride_reg_y_1, size_internal);
		}
	}
	else
	{
		// [2] Extenral Index: Partial
		if (size_e % SIZE_SLICE_1_E == 0)
		{
			// [3] Internal Index: Full
			// >>> External: Partial && Internal: Full
			printf ("External: Partial, Internal: Full\n");
			kernel__3_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a, size_b1, size_b2, size_c, size_d, size_e, CEIL(size_a, SIZE_SLICE_1_A), CEIL(size_b1, SIZE_SLICE_1_B1), CEIL(size_b2, SIZE_SLICE_1_B2), CEIL(size_c, SIZE_SLICE_1_C), CEIL(size_d, SIZE_SLICE_1_D), stride_int_t2, stride_int_v2, stride_reg_x_1, stride_reg_y_1, size_internal);
		}
		else
		{
			// [4] Internal Index: Partial
			// >>> External: Partial && Internal: Partial
			printf ("External: Partial, Internal: Partial\n");
			kernel__4_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a, size_b1, size_b2, size_c, size_d, size_e, CEIL(size_a, SIZE_SLICE_1_A), CEIL(size_b1, SIZE_SLICE_1_B1), CEIL(size_b2, SIZE_SLICE_1_B2), CEIL(size_c, SIZE_SLICE_1_C), CEIL(size_d, SIZE_SLICE_1_D), stride_int_t2, stride_int_v2, stride_reg_x_1, stride_reg_y_1, size_internal);
		}
	}

	// Copy the Result from Device to Host
	hipMemcpy(t3, dev_t3, sizeof(double) * (size_a * size_b1 * size_b2 * size_c * size_d), hipMemcpyDeviceToHost);

	// cudaFree()
	hipFree(dev_t3);	hipFree(dev_t2);	hipFree(dev_v2);

	// Shoule be Fixed
	// HostFree

}

// This is written by tc_interface.tc_gen_code_interface()
// This Interface Should be Called to Run the Kernels
extern "C"
void sd_t_d2_fusion_(int size_a, int size_b, int size_c, int size_d, int size_e, double* t3, double* t2, double* v2, int cond_kernel_1, int opt_register_transpose)
{
	// Pre-Processing for Split
	// Based on Tile-Sizes and Problem-Size
	// Currently, one index can be split into two indices
	int size_b1;
	int size_b2;

	if (size_b % 12 == 0)
	{
		//
		size_b1 = 12;
		size_b2 = size_b / 12;
	}
	else
	{
		//
		size_b1 = size_b;
		size_b2 = 1;
	}

	// Call An Application
	sd_t_d2_fusion(size_a, size_b1, size_b2, size_c, size_d, size_e, t3, t2, v2, cond_kernel_1, opt_register_transpose);
}
