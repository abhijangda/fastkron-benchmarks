// created by tc_code_include() in tc_code_include.py

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <sys/time.h>
#include <locale.h>
#include <algorithm>
using namespace std;

// created by tc_gen_definition_new()
#define SIZE_SLICE_1_C 16
#define SIZE_SLICE_1_D 1
#define SIZE_SLICE_1_A1 16
#define SIZE_SLICE_1_A2 4
#define SIZE_SLICE_1_B1 8
#define SIZE_SLICE_1_B2 8

#define SIZE_INT_UNIT_1 SIZE_SLICE_1_C * SIZE_SLICE_1_D

#define SIZE_TB_1_X 	SIZE_SLICE_1_A1
#define SIZE_TB_1_Y 	SIZE_SLICE_1_B1
#define SIZE_REG_1_X 	SIZE_SLICE_1_A2
#define SIZE_REG_1_Y 	SIZE_SLICE_1_B2

#define NUM_INDEX 		4
#define CEIL(a, b) 		(((a) + (b) - 1) / (b))

// Not Yet: Multiple Tensor Contractions.
// |Constant Memory| = 64KB, 16K Words(Integer), which means |K| <= 8192
#define MAX_CONST_LEN 		8192
__constant__ int const_internal_t2_offset[MAX_CONST_LEN];
__constant__ int const_internal_v2_offset[MAX_CONST_LEN];

// created by tc_gen_code_Kernel()
__global__ void kernel__1_1(double* dev_t3, 
double* dev_t2, 
double* dev_v2, 
int size_a1, int size_a2, int size_b1, int size_b2, int size_c, int size_d, 
int numBlk_a1, int numBlk_a2, int numBlk_b1, int numBlk_b2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ double sm_a[16][64];
	__shared__ double sm_b[16][64];


	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 1
	// # of indices mapped on TB_Y: 1
	int idx_a1 = threadIdx.x;
	int idx_b1 = threadIdx.y;

	int tmp_blkIdx;
	int blk_idx_b2 = blockIdx.x / (numBlk_b1 * numBlk_a2 * numBlk_a1);
	tmp_blkIdx = blockIdx.x % (numBlk_b1 * numBlk_a2 * numBlk_a1);

	int blk_idx_b1 = tmp_blkIdx / (numBlk_a2 * numBlk_a1);
	tmp_blkIdx = tmp_blkIdx % (numBlk_a2 * numBlk_a1);

	int blk_idx_a2 = tmp_blkIdx / numBlk_a1;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a1);

	int  blk_idx_a1 = tmp_blkIdx;

	int t3_base_thread = blk_idx_a1 * SIZE_SLICE_1_A1 + idx_a1 + (blk_idx_a2 * SIZE_SLICE_1_A2 + (blk_idx_b1 * SIZE_SLICE_1_B1 + idx_b1 + (blk_idx_b2 * SIZE_SLICE_1_B2) * size_b1) * size_a2) * size_a1;


	double temp_av;
	double temp_bv[8];
	double reg_tile[8][4];

	for (int i = 0; i < 8; i++)
	for (int j = 0; j < 4; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'x', 't2', ['c', 'a1', 'a2', 'd']], [16, 'STR_SD2_V2_H7', 'y', 'v2', ['d', 'c', 'b1', 'b2']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		// No Need to Put Boundary-Checks before For-Statement: : 
		for (int ll = 0; ll < 4; ll++)
		{
			// ['c', 'a1', 'a2', 'd']
			// Exception: Temp. version!: threadIdx.y + l + 0
			// Exception: Temp. version!: idx_a1 < rng_a1
			sm_a[threadIdx.y + 0][threadIdx.x + ll * 16] = dev_t2[(blk_idx_a1 * SIZE_SLICE_1_A1 + idx_a1 + (blk_idx_a2 * SIZE_SLICE_1_A2 + ll) * size_a1) * size_c + const_internal_t2_offset[threadIdx.y + l + 0]];
			// Exception: Temp. version!: threadIdx.y + l + 8
			// Exception: Temp. version!: idx_a1 < rng_a1
			// Exception: Full-Full
			sm_a[threadIdx.y + 8][threadIdx.x + ll * 16] = dev_t2[(blk_idx_a1 * SIZE_SLICE_1_A1 + idx_a1 + (blk_idx_a2 * SIZE_SLICE_1_A2 + ll) * size_a1) * size_c + const_internal_t2_offset[threadIdx.y + l + 8]];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (threadIdx.x < 8)
		for (int ll = 0; ll < 8; ll++)
		{
			// ['d', 'c', 'b1', 'b2']
			// Exception: Temp. version!: threadIdx.y + l + 0
			// Exception: Temp. version!: idx_a1 < rng_b1
			sm_b[threadIdx.y + 0][threadIdx.x + ll * 8] = dev_v2[((blk_idx_b1 * SIZE_SLICE_1_B1 + idx_a1 + (blk_idx_b2 * SIZE_SLICE_1_B2 + ll) * size_b1) * size_c) * size_d + const_internal_v2_offset[threadIdx.y + l + 0]];
			// Exception: Temp. version!: threadIdx.y + l + 8
			// Exception: Temp. version!: idx_a1 < rng_b1
			// Exception: Full-Full
			sm_b[threadIdx.y + 8][threadIdx.x + ll * 8] = dev_v2[((blk_idx_b1 * SIZE_SLICE_1_B1 + idx_a1 + (blk_idx_b2 * SIZE_SLICE_1_B2 + ll) * size_b1) * size_c) * size_d + const_internal_v2_offset[threadIdx.y + l + 8]];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1; ll++)
		{
			temp_bv[0] = sm_b[ll][idx_b1 + 0];
			temp_bv[1] = sm_b[ll][idx_b1 + 8];
			temp_bv[2] = sm_b[ll][idx_b1 + 16];
			temp_bv[3] = sm_b[ll][idx_b1 + 24];
			temp_bv[4] = sm_b[ll][idx_b1 + 32];
			temp_bv[5] = sm_b[ll][idx_b1 + 40];
			temp_bv[6] = sm_b[ll][idx_b1 + 48];
			temp_bv[7] = sm_b[ll][idx_b1 + 56];

			for (int xx = 0; xx < 4; xx++) // (1)
			{
				temp_av = sm_a[ll][idx_a1 + (xx * 16)];

				reg_tile[0][xx] += temp_av * temp_bv[0];
				reg_tile[1][xx] += temp_av * temp_bv[1];
				reg_tile[2][xx] += temp_av * temp_bv[2];
				reg_tile[3][xx] += temp_av * temp_bv[3];
				reg_tile[4][xx] += temp_av * temp_bv[4];
				reg_tile[5][xx] += temp_av * temp_bv[5];
				reg_tile[6][xx] += temp_av * temp_bv[6];
				reg_tile[7][xx] += temp_av * temp_bv[7];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	#pragma unroll 8
	for (int i = 0; i < 8; i++)
	{
		for (int j = 0; j < 4; j++)
		{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
		}
	}
}

// created by tc_gen_code_Kernel()
__global__ void kernel__2_1(double* dev_t3, 
double* dev_t2, 
double* dev_v2, 
int size_a1, int size_a2, int size_b1, int size_b2, int size_c, int size_d, 
int numBlk_a1, int numBlk_a2, int numBlk_b1, int numBlk_b2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ double sm_a[16][64];
	__shared__ double sm_b[16][64];


	int internal_upperbound   = 0;
	int internal_offset;

	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 1
	// # of indices mapped on TB_Y: 1
	int idx_a1 = threadIdx.x;
	int idx_b1 = threadIdx.y;

	int tmp_blkIdx;
	int blk_idx_b2 = blockIdx.x / (numBlk_b1 * numBlk_a2 * numBlk_a1);
	tmp_blkIdx = blockIdx.x % (numBlk_b1 * numBlk_a2 * numBlk_a1);

	int blk_idx_b1 = tmp_blkIdx / (numBlk_a2 * numBlk_a1);
	tmp_blkIdx = tmp_blkIdx % (numBlk_a2 * numBlk_a1);

	int blk_idx_a2 = tmp_blkIdx / numBlk_a1;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a1);

	int  blk_idx_a1 = tmp_blkIdx;

	int t3_base_thread = blk_idx_a1 * SIZE_SLICE_1_A1 + idx_a1 + (blk_idx_a2 * SIZE_SLICE_1_A2 + (blk_idx_b1 * SIZE_SLICE_1_B1 + idx_b1 + (blk_idx_b2 * SIZE_SLICE_1_B2) * size_b1) * size_a2) * size_a1;


	double temp_av;
	double temp_bv[8];
	double reg_tile[8][4];

	for (int i = 0; i < 8; i++)
	for (int j = 0; j < 4; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'x', 't2', ['c', 'a1', 'a2', 'd']], [16, 'STR_SD2_V2_H7', 'y', 'v2', ['d', 'c', 'b1', 'b2']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		// Part: Generalized Contraction Index (p7b)
		internal_offset = (l + SIZE_INT_UNIT_1) - size_internal;
		if (internal_offset > 0) internal_upperbound = internal_offset;

		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (threadIdx.y < SIZE_INT_UNIT_1 - internal_upperbound)
		for (int ll = 0; ll < 4; ll++)
		{
			// ['c', 'a1', 'a2', 'd']
			// Exception: Temp. version!: threadIdx.y + l + 0
			// Exception: Temp. version!: idx_a1 < rng_a1
			sm_a[threadIdx.y + 0][threadIdx.x + ll * 16] = dev_t2[(blk_idx_a1 * SIZE_SLICE_1_A1 + idx_a1 + (blk_idx_a2 * SIZE_SLICE_1_A2 + ll) * size_a1) * size_c + const_internal_t2_offset[threadIdx.y + l + 0]];
			// Exception: Temp. version!: threadIdx.y + l + 8
			// Exception: Temp. version!: idx_a1 < rng_a1
			if (threadIdx.y + l + 8 < size_internal) 
			sm_a[threadIdx.y + 8][threadIdx.x + ll * 16] = dev_t2[(blk_idx_a1 * SIZE_SLICE_1_A1 + idx_a1 + (blk_idx_a2 * SIZE_SLICE_1_A2 + ll) * size_a1) * size_c + const_internal_t2_offset[threadIdx.y + l + 8]];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (threadIdx.y < SIZE_INT_UNIT_1 - internal_upperbound && threadIdx.x < 8)
		for (int ll = 0; ll < 8; ll++)
		{
			// ['d', 'c', 'b1', 'b2']
			// Exception: Temp. version!: threadIdx.y + l + 0
			// Exception: Temp. version!: idx_a1 < rng_b1
			sm_b[threadIdx.y + 0][threadIdx.x + ll * 8] = dev_v2[((blk_idx_b1 * SIZE_SLICE_1_B1 + idx_a1 + (blk_idx_b2 * SIZE_SLICE_1_B2 + ll) * size_b1) * size_c) * size_d + const_internal_v2_offset[threadIdx.y + l + 0]];
			// Exception: Temp. version!: threadIdx.y + l + 8
			// Exception: Temp. version!: idx_a1 < rng_b1
			if (threadIdx.y + l + 8 < size_internal) 
			sm_b[threadIdx.y + 8][threadIdx.x + ll * 8] = dev_v2[((blk_idx_b1 * SIZE_SLICE_1_B1 + idx_a1 + (blk_idx_b2 * SIZE_SLICE_1_B2 + ll) * size_b1) * size_c) * size_d + const_internal_v2_offset[threadIdx.y + l + 8]];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1 - internal_upperbound; ll++)
		{
			temp_bv[0] = sm_b[ll][idx_b1 + 0];
			temp_bv[1] = sm_b[ll][idx_b1 + 8];
			temp_bv[2] = sm_b[ll][idx_b1 + 16];
			temp_bv[3] = sm_b[ll][idx_b1 + 24];
			temp_bv[4] = sm_b[ll][idx_b1 + 32];
			temp_bv[5] = sm_b[ll][idx_b1 + 40];
			temp_bv[6] = sm_b[ll][idx_b1 + 48];
			temp_bv[7] = sm_b[ll][idx_b1 + 56];

			for (int xx = 0; xx < 4; xx++) // (1)
			{
				temp_av = sm_a[ll][idx_a1 + (xx * 16)];

				reg_tile[0][xx] += temp_av * temp_bv[0];
				reg_tile[1][xx] += temp_av * temp_bv[1];
				reg_tile[2][xx] += temp_av * temp_bv[2];
				reg_tile[3][xx] += temp_av * temp_bv[3];
				reg_tile[4][xx] += temp_av * temp_bv[4];
				reg_tile[5][xx] += temp_av * temp_bv[5];
				reg_tile[6][xx] += temp_av * temp_bv[6];
				reg_tile[7][xx] += temp_av * temp_bv[7];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	#pragma unroll 8
	for (int i = 0; i < 8; i++)
	{
		for (int j = 0; j < 4; j++)
		{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
		}
	}
}

// created by tc_gen_code_Kernel()
__global__ void kernel__3_1(double* dev_t3, 
double* dev_t2, 
double* dev_v2, 
int size_a1, int size_a2, int size_b1, int size_b2, int size_c, int size_d, 
int numBlk_a1, int numBlk_a2, int numBlk_b1, int numBlk_b2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ double sm_a[16][64];
	__shared__ double sm_b[16][64];


	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 1
	// # of indices mapped on TB_Y: 1
	int idx_a1 = threadIdx.x;
	int idx_b1 = threadIdx.y;

	int tmp_blkIdx;
	int blk_idx_b2 = blockIdx.x / (numBlk_b1 * numBlk_a2 * numBlk_a1);
	tmp_blkIdx = blockIdx.x % (numBlk_b1 * numBlk_a2 * numBlk_a1);

	int blk_idx_b1 = tmp_blkIdx / (numBlk_a2 * numBlk_a1);
	tmp_blkIdx = tmp_blkIdx % (numBlk_a2 * numBlk_a1);

	int blk_idx_a2 = tmp_blkIdx / numBlk_a1;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a1);

	int  blk_idx_a1 = tmp_blkIdx;

	int t3_base_thread = blk_idx_a1 * SIZE_SLICE_1_A1 + idx_a1 + (blk_idx_a2 * SIZE_SLICE_1_A2 + (blk_idx_b1 * SIZE_SLICE_1_B1 + idx_b1 + (blk_idx_b2 * SIZE_SLICE_1_B2) * size_b1) * size_a2) * size_a1;

	// need to support partial tiles
	int rng_a1, rng_a2, rng_b1, rng_b2;
	if ((size_a1 - (blk_idx_a1 * SIZE_SLICE_1_A1)) >= SIZE_SLICE_1_A1)
	{
		rng_a1 = SIZE_SLICE_1_A1;
	}
	else
	{
		rng_a1 = size_a1 % SIZE_SLICE_1_A1;
	}
	if ((size_a2 - (blk_idx_a2 * SIZE_SLICE_1_A2)) >= SIZE_SLICE_1_A2)
	{
		rng_a2 = SIZE_SLICE_1_A2;
	}
	else
	{
		rng_a2 = size_a2 % SIZE_SLICE_1_A2;
	}
	if ((size_b1 - (blk_idx_b1 * SIZE_SLICE_1_B1)) >= SIZE_SLICE_1_B1)
	{
		rng_b1 = SIZE_SLICE_1_B1;
	}
	else
	{
		rng_b1 = size_b1 % SIZE_SLICE_1_B1;
	}
	if ((size_b2 - (blk_idx_b2 * SIZE_SLICE_1_B2)) >= SIZE_SLICE_1_B2)
	{
		rng_b2 = SIZE_SLICE_1_B2;
	}
	else
	{
		rng_b2 = size_b2 % SIZE_SLICE_1_B2;
	}

	double temp_av;
	double temp_bv[8];
	double reg_tile[8][4];

	for (int i = 0; i < 8; i++)
	for (int j = 0; j < 4; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'x', 't2', ['c', 'a1', 'a2', 'd']], [16, 'STR_SD2_V2_H7', 'y', 'v2', ['d', 'c', 'b1', 'b2']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (idx_a1 < rng_a1)
		for (int ll = 0; ll < rng_a2; ll++)
		{
			// ['c', 'a1', 'a2', 'd']
			// Exception: Temp. version!: threadIdx.y + l + 0
			// Exception: Temp. version!: idx_a1 < rng_a1
			sm_a[threadIdx.y + 0][threadIdx.x + ll * 16] = dev_t2[(blk_idx_a1 * SIZE_SLICE_1_A1 + idx_a1 + (blk_idx_a2 * SIZE_SLICE_1_A2 + ll) * size_a1) * size_c + const_internal_t2_offset[threadIdx.y + l + 0]];
			// Exception: Temp. version!: threadIdx.y + l + 8
			// Exception: Temp. version!: idx_a1 < rng_a1
			if (idx_a1 < rng_a1) 
			sm_a[threadIdx.y + 8][threadIdx.x + ll * 16] = dev_t2[(blk_idx_a1 * SIZE_SLICE_1_A1 + idx_a1 + (blk_idx_a2 * SIZE_SLICE_1_A2 + ll) * size_a1) * size_c + const_internal_t2_offset[threadIdx.y + l + 8]];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (idx_a1 < rng_b1 && threadIdx.x < 8)
		for (int ll = 0; ll < rng_b2; ll++)
		{
			// ['d', 'c', 'b1', 'b2']
			// Exception: Temp. version!: threadIdx.y + l + 0
			// Exception: Temp. version!: idx_a1 < rng_b1
			sm_b[threadIdx.y + 0][threadIdx.x + ll * 8] = dev_v2[((blk_idx_b1 * SIZE_SLICE_1_B1 + idx_a1 + (blk_idx_b2 * SIZE_SLICE_1_B2 + ll) * size_b1) * size_c) * size_d + const_internal_v2_offset[threadIdx.y + l + 0]];
			// Exception: Temp. version!: threadIdx.y + l + 8
			// Exception: Temp. version!: idx_a1 < rng_b1
			if (idx_a1 < rng_b1) 
			sm_b[threadIdx.y + 8][threadIdx.x + ll * 8] = dev_v2[((blk_idx_b1 * SIZE_SLICE_1_B1 + idx_a1 + (blk_idx_b2 * SIZE_SLICE_1_B2 + ll) * size_b1) * size_c) * size_d + const_internal_v2_offset[threadIdx.y + l + 8]];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1; ll++)
		{
			temp_bv[0] = sm_b[ll][idx_b1 + 0];
			temp_bv[1] = sm_b[ll][idx_b1 + 8];
			temp_bv[2] = sm_b[ll][idx_b1 + 16];
			temp_bv[3] = sm_b[ll][idx_b1 + 24];
			temp_bv[4] = sm_b[ll][idx_b1 + 32];
			temp_bv[5] = sm_b[ll][idx_b1 + 40];
			temp_bv[6] = sm_b[ll][idx_b1 + 48];
			temp_bv[7] = sm_b[ll][idx_b1 + 56];

			for (int xx = 0; xx < 4; xx++) // (1)
			{
				temp_av = sm_a[ll][idx_a1 + (xx * 16)];

				reg_tile[0][xx] += temp_av * temp_bv[0];
				reg_tile[1][xx] += temp_av * temp_bv[1];
				reg_tile[2][xx] += temp_av * temp_bv[2];
				reg_tile[3][xx] += temp_av * temp_bv[3];
				reg_tile[4][xx] += temp_av * temp_bv[4];
				reg_tile[5][xx] += temp_av * temp_bv[5];
				reg_tile[6][xx] += temp_av * temp_bv[6];
				reg_tile[7][xx] += temp_av * temp_bv[7];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	if (idx_a1 < rng_a1 && idx_b1 < rng_b1)
	for (int i = 0; i < 8; i++)
	{
		for (int j = 0; j < 4; j++)
		{
			if(i < rng_b2 && j < rng_a2)
			{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
			}
		}
	}
}

// created by tc_gen_code_Kernel()
__global__ void kernel__4_1(double* dev_t3, 
double* dev_t2, 
double* dev_v2, 
int size_a1, int size_a2, int size_b1, int size_b2, int size_c, int size_d, 
int numBlk_a1, int numBlk_a2, int numBlk_b1, int numBlk_b2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ double sm_a[16][64];
	__shared__ double sm_b[16][64];


	int internal_upperbound   = 0;
	int internal_offset;

	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 1
	// # of indices mapped on TB_Y: 1
	int idx_a1 = threadIdx.x;
	int idx_b1 = threadIdx.y;

	int tmp_blkIdx;
	int blk_idx_b2 = blockIdx.x / (numBlk_b1 * numBlk_a2 * numBlk_a1);
	tmp_blkIdx = blockIdx.x % (numBlk_b1 * numBlk_a2 * numBlk_a1);

	int blk_idx_b1 = tmp_blkIdx / (numBlk_a2 * numBlk_a1);
	tmp_blkIdx = tmp_blkIdx % (numBlk_a2 * numBlk_a1);

	int blk_idx_a2 = tmp_blkIdx / numBlk_a1;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a1);

	int  blk_idx_a1 = tmp_blkIdx;

	int t3_base_thread = blk_idx_a1 * SIZE_SLICE_1_A1 + idx_a1 + (blk_idx_a2 * SIZE_SLICE_1_A2 + (blk_idx_b1 * SIZE_SLICE_1_B1 + idx_b1 + (blk_idx_b2 * SIZE_SLICE_1_B2) * size_b1) * size_a2) * size_a1;

	// need to support partial tiles
	int rng_a1, rng_a2, rng_b1, rng_b2;
	if ((size_a1 - (blk_idx_a1 * SIZE_SLICE_1_A1)) >= SIZE_SLICE_1_A1)
	{
		rng_a1 = SIZE_SLICE_1_A1;
	}
	else
	{
		rng_a1 = size_a1 % SIZE_SLICE_1_A1;
	}
	if ((size_a2 - (blk_idx_a2 * SIZE_SLICE_1_A2)) >= SIZE_SLICE_1_A2)
	{
		rng_a2 = SIZE_SLICE_1_A2;
	}
	else
	{
		rng_a2 = size_a2 % SIZE_SLICE_1_A2;
	}
	if ((size_b1 - (blk_idx_b1 * SIZE_SLICE_1_B1)) >= SIZE_SLICE_1_B1)
	{
		rng_b1 = SIZE_SLICE_1_B1;
	}
	else
	{
		rng_b1 = size_b1 % SIZE_SLICE_1_B1;
	}
	if ((size_b2 - (blk_idx_b2 * SIZE_SLICE_1_B2)) >= SIZE_SLICE_1_B2)
	{
		rng_b2 = SIZE_SLICE_1_B2;
	}
	else
	{
		rng_b2 = size_b2 % SIZE_SLICE_1_B2;
	}

	double temp_av;
	double temp_bv[8];
	double reg_tile[8][4];

	for (int i = 0; i < 8; i++)
	for (int j = 0; j < 4; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'x', 't2', ['c', 'a1', 'a2', 'd']], [16, 'STR_SD2_V2_H7', 'y', 'v2', ['d', 'c', 'b1', 'b2']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		// Part: Generalized Contraction Index (p7b)
		internal_offset = (l + SIZE_INT_UNIT_1) - size_internal;
		if (internal_offset > 0) internal_upperbound = internal_offset;

		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (idx_a1 < rng_a1 && threadIdx.y < SIZE_INT_UNIT_1 - internal_upperbound)
		for (int ll = 0; ll < rng_a2; ll++)
		{
			// ['c', 'a1', 'a2', 'd']
			// Exception: Temp. version!: threadIdx.y + l + 0
			// Exception: Temp. version!: idx_a1 < rng_a1
			sm_a[threadIdx.y + 0][threadIdx.x + ll * 16] = dev_t2[(blk_idx_a1 * SIZE_SLICE_1_A1 + idx_a1 + (blk_idx_a2 * SIZE_SLICE_1_A2 + ll) * size_a1) * size_c + const_internal_t2_offset[threadIdx.y + l + 0]];
			// Exception: Temp. version!: threadIdx.y + l + 8
			// Exception: Temp. version!: idx_a1 < rng_a1
			if (threadIdx.y + l + 8 < size_internal && idx_a1 < rng_a1) 
			sm_a[threadIdx.y + 8][threadIdx.x + ll * 16] = dev_t2[(blk_idx_a1 * SIZE_SLICE_1_A1 + idx_a1 + (blk_idx_a2 * SIZE_SLICE_1_A2 + ll) * size_a1) * size_c + const_internal_t2_offset[threadIdx.y + l + 8]];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (idx_a1 < rng_b1 && threadIdx.y < SIZE_INT_UNIT_1 - internal_upperbound && threadIdx.x < 8)
		for (int ll = 0; ll < rng_b2; ll++)
		{
			// ['d', 'c', 'b1', 'b2']
			// Exception: Temp. version!: threadIdx.y + l + 0
			// Exception: Temp. version!: idx_a1 < rng_b1
			sm_b[threadIdx.y + 0][threadIdx.x + ll * 8] = dev_v2[((blk_idx_b1 * SIZE_SLICE_1_B1 + idx_a1 + (blk_idx_b2 * SIZE_SLICE_1_B2 + ll) * size_b1) * size_c) * size_d + const_internal_v2_offset[threadIdx.y + l + 0]];
			// Exception: Temp. version!: threadIdx.y + l + 8
			// Exception: Temp. version!: idx_a1 < rng_b1
			if (threadIdx.y + l + 8 < size_internal && idx_a1 < rng_b1) 
			sm_b[threadIdx.y + 8][threadIdx.x + ll * 8] = dev_v2[((blk_idx_b1 * SIZE_SLICE_1_B1 + idx_a1 + (blk_idx_b2 * SIZE_SLICE_1_B2 + ll) * size_b1) * size_c) * size_d + const_internal_v2_offset[threadIdx.y + l + 8]];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1 - internal_upperbound; ll++)
		{
			temp_bv[0] = sm_b[ll][idx_b1 + 0];
			temp_bv[1] = sm_b[ll][idx_b1 + 8];
			temp_bv[2] = sm_b[ll][idx_b1 + 16];
			temp_bv[3] = sm_b[ll][idx_b1 + 24];
			temp_bv[4] = sm_b[ll][idx_b1 + 32];
			temp_bv[5] = sm_b[ll][idx_b1 + 40];
			temp_bv[6] = sm_b[ll][idx_b1 + 48];
			temp_bv[7] = sm_b[ll][idx_b1 + 56];

			for (int xx = 0; xx < 4; xx++) // (1)
			{
				temp_av = sm_a[ll][idx_a1 + (xx * 16)];

				reg_tile[0][xx] += temp_av * temp_bv[0];
				reg_tile[1][xx] += temp_av * temp_bv[1];
				reg_tile[2][xx] += temp_av * temp_bv[2];
				reg_tile[3][xx] += temp_av * temp_bv[3];
				reg_tile[4][xx] += temp_av * temp_bv[4];
				reg_tile[5][xx] += temp_av * temp_bv[5];
				reg_tile[6][xx] += temp_av * temp_bv[6];
				reg_tile[7][xx] += temp_av * temp_bv[7];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	if (idx_a1 < rng_a1 && idx_b1 < rng_b1)
	for (int i = 0; i < 8; i++)
	{
		for (int j = 0; j < 4; j++)
		{
			if(i < rng_b2 && j < rng_a2)
			{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
			}
		}
	}
}

// created by tc_gen_code_Kernel()
__global__ void kernel__1_tex_1(double* dev_t3, 
double* dev_t2, 
double* dev_v2, 
int size_a1, int size_a2, int size_b1, int size_b2, int size_c, int size_d, 
int numBlk_a1, int numBlk_a2, int numBlk_b1, int numBlk_b2, 
int* dev_internal_offset_t2, int* dev_internal_offset_v2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ double sm_a[16][64];
	__shared__ double sm_b[16][64];


	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 1
	// # of indices mapped on TB_Y: 1
	int idx_a1 = threadIdx.x;
	int idx_b1 = threadIdx.y;

	int tmp_blkIdx;
	int blk_idx_b2 = blockIdx.x / (numBlk_b1 * numBlk_a2 * numBlk_a1);
	tmp_blkIdx = blockIdx.x % (numBlk_b1 * numBlk_a2 * numBlk_a1);

	int blk_idx_b1 = tmp_blkIdx / (numBlk_a2 * numBlk_a1);
	tmp_blkIdx = tmp_blkIdx % (numBlk_a2 * numBlk_a1);

	int blk_idx_a2 = tmp_blkIdx / numBlk_a1;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a1);

	int  blk_idx_a1 = tmp_blkIdx;

	int t3_base_thread = blk_idx_a1 * SIZE_SLICE_1_A1 + idx_a1 + (blk_idx_a2 * SIZE_SLICE_1_A2 + (blk_idx_b1 * SIZE_SLICE_1_B1 + idx_b1 + (blk_idx_b2 * SIZE_SLICE_1_B2) * size_b1) * size_a2) * size_a1;


	double temp_av;
	double temp_bv[8];
	double reg_tile[8][4];

	for (int i = 0; i < 8; i++)
	for (int j = 0; j < 4; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'x', 't2', ['c', 'a1', 'a2', 'd']], [16, 'STR_SD2_V2_H7', 'y', 'v2', ['d', 'c', 'b1', 'b2']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		// No Need to Put Boundary-Checks before For-Statement: : 
		for (int ll = 0; ll < 4; ll++)
		{
			// ['c', 'a1', 'a2', 'd']
			// Exception: Temp. version!: threadIdx.y + l + 0
			// Exception: Temp. version!: idx_a1 < rng_a1
			sm_a[threadIdx.y + 0][threadIdx.x + ll * 16] = dev_t2[(blk_idx_a1 * SIZE_SLICE_1_A1 + idx_a1 + (blk_idx_a2 * SIZE_SLICE_1_A2 + ll) * size_a1) * size_c + dev_internal_offset_t2[threadIdx.y + l + 0]];
			// Exception: Temp. version!: threadIdx.y + l + 8
			// Exception: Temp. version!: idx_a1 < rng_a1
			// Exception: Full-Full
			sm_a[threadIdx.y + 8][threadIdx.x + ll * 16] = dev_t2[(blk_idx_a1 * SIZE_SLICE_1_A1 + idx_a1 + (blk_idx_a2 * SIZE_SLICE_1_A2 + ll) * size_a1) * size_c + dev_internal_offset_t2[threadIdx.y + l + 8]];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (threadIdx.x < 8)
		for (int ll = 0; ll < 8; ll++)
		{
			// ['d', 'c', 'b1', 'b2']
			// Exception: Temp. version!: threadIdx.y + l + 0
			// Exception: Temp. version!: idx_a1 < rng_b1
			sm_b[threadIdx.y + 0][threadIdx.x + ll * 8] = dev_v2[((blk_idx_b1 * SIZE_SLICE_1_B1 + idx_a1 + (blk_idx_b2 * SIZE_SLICE_1_B2 + ll) * size_b1) * size_c) * size_d + dev_internal_offset_v2[threadIdx.y + l + 0]];
			// Exception: Temp. version!: threadIdx.y + l + 8
			// Exception: Temp. version!: idx_a1 < rng_b1
			// Exception: Full-Full
			sm_b[threadIdx.y + 8][threadIdx.x + ll * 8] = dev_v2[((blk_idx_b1 * SIZE_SLICE_1_B1 + idx_a1 + (blk_idx_b2 * SIZE_SLICE_1_B2 + ll) * size_b1) * size_c) * size_d + dev_internal_offset_v2[threadIdx.y + l + 8]];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1; ll++)
		{
			temp_bv[0] = sm_b[ll][idx_b1 + 0];
			temp_bv[1] = sm_b[ll][idx_b1 + 8];
			temp_bv[2] = sm_b[ll][idx_b1 + 16];
			temp_bv[3] = sm_b[ll][idx_b1 + 24];
			temp_bv[4] = sm_b[ll][idx_b1 + 32];
			temp_bv[5] = sm_b[ll][idx_b1 + 40];
			temp_bv[6] = sm_b[ll][idx_b1 + 48];
			temp_bv[7] = sm_b[ll][idx_b1 + 56];

			for (int xx = 0; xx < 4; xx++) // (1)
			{
				temp_av = sm_a[ll][idx_a1 + (xx * 16)];

				reg_tile[0][xx] += temp_av * temp_bv[0];
				reg_tile[1][xx] += temp_av * temp_bv[1];
				reg_tile[2][xx] += temp_av * temp_bv[2];
				reg_tile[3][xx] += temp_av * temp_bv[3];
				reg_tile[4][xx] += temp_av * temp_bv[4];
				reg_tile[5][xx] += temp_av * temp_bv[5];
				reg_tile[6][xx] += temp_av * temp_bv[6];
				reg_tile[7][xx] += temp_av * temp_bv[7];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	#pragma unroll 8
	for (int i = 0; i < 8; i++)
	{
		for (int j = 0; j < 4; j++)
		{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
		}
	}
}

// created by tc_gen_code_Kernel()
__global__ void kernel__2_tex_1(double* dev_t3, 
double* dev_t2, 
double* dev_v2, 
int size_a1, int size_a2, int size_b1, int size_b2, int size_c, int size_d, 
int numBlk_a1, int numBlk_a2, int numBlk_b1, int numBlk_b2, 
int* dev_internal_offset_t2, int* dev_internal_offset_v2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ double sm_a[16][64];
	__shared__ double sm_b[16][64];


	int internal_upperbound   = 0;
	int internal_offset;

	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 1
	// # of indices mapped on TB_Y: 1
	int idx_a1 = threadIdx.x;
	int idx_b1 = threadIdx.y;

	int tmp_blkIdx;
	int blk_idx_b2 = blockIdx.x / (numBlk_b1 * numBlk_a2 * numBlk_a1);
	tmp_blkIdx = blockIdx.x % (numBlk_b1 * numBlk_a2 * numBlk_a1);

	int blk_idx_b1 = tmp_blkIdx / (numBlk_a2 * numBlk_a1);
	tmp_blkIdx = tmp_blkIdx % (numBlk_a2 * numBlk_a1);

	int blk_idx_a2 = tmp_blkIdx / numBlk_a1;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a1);

	int  blk_idx_a1 = tmp_blkIdx;

	int t3_base_thread = blk_idx_a1 * SIZE_SLICE_1_A1 + idx_a1 + (blk_idx_a2 * SIZE_SLICE_1_A2 + (blk_idx_b1 * SIZE_SLICE_1_B1 + idx_b1 + (blk_idx_b2 * SIZE_SLICE_1_B2) * size_b1) * size_a2) * size_a1;


	double temp_av;
	double temp_bv[8];
	double reg_tile[8][4];

	for (int i = 0; i < 8; i++)
	for (int j = 0; j < 4; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'x', 't2', ['c', 'a1', 'a2', 'd']], [16, 'STR_SD2_V2_H7', 'y', 'v2', ['d', 'c', 'b1', 'b2']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		// Part: Generalized Contraction Index (p7b)
		internal_offset = (l + SIZE_INT_UNIT_1) - size_internal;
		if (internal_offset > 0) internal_upperbound = internal_offset;

		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (threadIdx.y < SIZE_INT_UNIT_1 - internal_upperbound)
		for (int ll = 0; ll < 4; ll++)
		{
			// ['c', 'a1', 'a2', 'd']
			// Exception: Temp. version!: threadIdx.y + l + 0
			// Exception: Temp. version!: idx_a1 < rng_a1
			sm_a[threadIdx.y + 0][threadIdx.x + ll * 16] = dev_t2[(blk_idx_a1 * SIZE_SLICE_1_A1 + idx_a1 + (blk_idx_a2 * SIZE_SLICE_1_A2 + ll) * size_a1) * size_c + dev_internal_offset_t2[threadIdx.y + l + 0]];
			// Exception: Temp. version!: threadIdx.y + l + 8
			// Exception: Temp. version!: idx_a1 < rng_a1
			if (threadIdx.y + l + 8 < size_internal) 
			sm_a[threadIdx.y + 8][threadIdx.x + ll * 16] = dev_t2[(blk_idx_a1 * SIZE_SLICE_1_A1 + idx_a1 + (blk_idx_a2 * SIZE_SLICE_1_A2 + ll) * size_a1) * size_c + dev_internal_offset_t2[threadIdx.y + l + 8]];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (threadIdx.y < SIZE_INT_UNIT_1 - internal_upperbound && threadIdx.x < 8)
		for (int ll = 0; ll < 8; ll++)
		{
			// ['d', 'c', 'b1', 'b2']
			// Exception: Temp. version!: threadIdx.y + l + 0
			// Exception: Temp. version!: idx_a1 < rng_b1
			sm_b[threadIdx.y + 0][threadIdx.x + ll * 8] = dev_v2[((blk_idx_b1 * SIZE_SLICE_1_B1 + idx_a1 + (blk_idx_b2 * SIZE_SLICE_1_B2 + ll) * size_b1) * size_c) * size_d + dev_internal_offset_v2[threadIdx.y + l + 0]];
			// Exception: Temp. version!: threadIdx.y + l + 8
			// Exception: Temp. version!: idx_a1 < rng_b1
			if (threadIdx.y + l + 8 < size_internal) 
			sm_b[threadIdx.y + 8][threadIdx.x + ll * 8] = dev_v2[((blk_idx_b1 * SIZE_SLICE_1_B1 + idx_a1 + (blk_idx_b2 * SIZE_SLICE_1_B2 + ll) * size_b1) * size_c) * size_d + dev_internal_offset_v2[threadIdx.y + l + 8]];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1 - internal_upperbound; ll++)
		{
			temp_bv[0] = sm_b[ll][idx_b1 + 0];
			temp_bv[1] = sm_b[ll][idx_b1 + 8];
			temp_bv[2] = sm_b[ll][idx_b1 + 16];
			temp_bv[3] = sm_b[ll][idx_b1 + 24];
			temp_bv[4] = sm_b[ll][idx_b1 + 32];
			temp_bv[5] = sm_b[ll][idx_b1 + 40];
			temp_bv[6] = sm_b[ll][idx_b1 + 48];
			temp_bv[7] = sm_b[ll][idx_b1 + 56];

			for (int xx = 0; xx < 4; xx++) // (1)
			{
				temp_av = sm_a[ll][idx_a1 + (xx * 16)];

				reg_tile[0][xx] += temp_av * temp_bv[0];
				reg_tile[1][xx] += temp_av * temp_bv[1];
				reg_tile[2][xx] += temp_av * temp_bv[2];
				reg_tile[3][xx] += temp_av * temp_bv[3];
				reg_tile[4][xx] += temp_av * temp_bv[4];
				reg_tile[5][xx] += temp_av * temp_bv[5];
				reg_tile[6][xx] += temp_av * temp_bv[6];
				reg_tile[7][xx] += temp_av * temp_bv[7];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	#pragma unroll 8
	for (int i = 0; i < 8; i++)
	{
		for (int j = 0; j < 4; j++)
		{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
		}
	}
}

// created by tc_gen_code_Kernel()
__global__ void kernel__3_tex_1(double* dev_t3, 
double* dev_t2, 
double* dev_v2, 
int size_a1, int size_a2, int size_b1, int size_b2, int size_c, int size_d, 
int numBlk_a1, int numBlk_a2, int numBlk_b1, int numBlk_b2, 
int* dev_internal_offset_t2, int* dev_internal_offset_v2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ double sm_a[16][64];
	__shared__ double sm_b[16][64];


	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 1
	// # of indices mapped on TB_Y: 1
	int idx_a1 = threadIdx.x;
	int idx_b1 = threadIdx.y;

	int tmp_blkIdx;
	int blk_idx_b2 = blockIdx.x / (numBlk_b1 * numBlk_a2 * numBlk_a1);
	tmp_blkIdx = blockIdx.x % (numBlk_b1 * numBlk_a2 * numBlk_a1);

	int blk_idx_b1 = tmp_blkIdx / (numBlk_a2 * numBlk_a1);
	tmp_blkIdx = tmp_blkIdx % (numBlk_a2 * numBlk_a1);

	int blk_idx_a2 = tmp_blkIdx / numBlk_a1;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a1);

	int  blk_idx_a1 = tmp_blkIdx;

	int t3_base_thread = blk_idx_a1 * SIZE_SLICE_1_A1 + idx_a1 + (blk_idx_a2 * SIZE_SLICE_1_A2 + (blk_idx_b1 * SIZE_SLICE_1_B1 + idx_b1 + (blk_idx_b2 * SIZE_SLICE_1_B2) * size_b1) * size_a2) * size_a1;

	// need to support partial tiles
	int rng_a1, rng_a2, rng_b1, rng_b2;
	if ((size_a1 - (blk_idx_a1 * SIZE_SLICE_1_A1)) >= SIZE_SLICE_1_A1)
	{
		rng_a1 = SIZE_SLICE_1_A1;
	}
	else
	{
		rng_a1 = size_a1 % SIZE_SLICE_1_A1;
	}
	if ((size_a2 - (blk_idx_a2 * SIZE_SLICE_1_A2)) >= SIZE_SLICE_1_A2)
	{
		rng_a2 = SIZE_SLICE_1_A2;
	}
	else
	{
		rng_a2 = size_a2 % SIZE_SLICE_1_A2;
	}
	if ((size_b1 - (blk_idx_b1 * SIZE_SLICE_1_B1)) >= SIZE_SLICE_1_B1)
	{
		rng_b1 = SIZE_SLICE_1_B1;
	}
	else
	{
		rng_b1 = size_b1 % SIZE_SLICE_1_B1;
	}
	if ((size_b2 - (blk_idx_b2 * SIZE_SLICE_1_B2)) >= SIZE_SLICE_1_B2)
	{
		rng_b2 = SIZE_SLICE_1_B2;
	}
	else
	{
		rng_b2 = size_b2 % SIZE_SLICE_1_B2;
	}

	double temp_av;
	double temp_bv[8];
	double reg_tile[8][4];

	for (int i = 0; i < 8; i++)
	for (int j = 0; j < 4; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'x', 't2', ['c', 'a1', 'a2', 'd']], [16, 'STR_SD2_V2_H7', 'y', 'v2', ['d', 'c', 'b1', 'b2']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (idx_a1 < rng_a1)
		for (int ll = 0; ll < rng_a2; ll++)
		{
			// ['c', 'a1', 'a2', 'd']
			// Exception: Temp. version!: threadIdx.y + l + 0
			// Exception: Temp. version!: idx_a1 < rng_a1
			sm_a[threadIdx.y + 0][threadIdx.x + ll * 16] = dev_t2[(blk_idx_a1 * SIZE_SLICE_1_A1 + idx_a1 + (blk_idx_a2 * SIZE_SLICE_1_A2 + ll) * size_a1) * size_c + dev_internal_offset_t2[threadIdx.y + l + 0]];
			// Exception: Temp. version!: threadIdx.y + l + 8
			// Exception: Temp. version!: idx_a1 < rng_a1
			if (idx_a1 < rng_a1) 
			sm_a[threadIdx.y + 8][threadIdx.x + ll * 16] = dev_t2[(blk_idx_a1 * SIZE_SLICE_1_A1 + idx_a1 + (blk_idx_a2 * SIZE_SLICE_1_A2 + ll) * size_a1) * size_c + dev_internal_offset_t2[threadIdx.y + l + 8]];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (idx_a1 < rng_b1 && threadIdx.x < 8)
		for (int ll = 0; ll < rng_b2; ll++)
		{
			// ['d', 'c', 'b1', 'b2']
			// Exception: Temp. version!: threadIdx.y + l + 0
			// Exception: Temp. version!: idx_a1 < rng_b1
			sm_b[threadIdx.y + 0][threadIdx.x + ll * 8] = dev_v2[((blk_idx_b1 * SIZE_SLICE_1_B1 + idx_a1 + (blk_idx_b2 * SIZE_SLICE_1_B2 + ll) * size_b1) * size_c) * size_d + dev_internal_offset_v2[threadIdx.y + l + 0]];
			// Exception: Temp. version!: threadIdx.y + l + 8
			// Exception: Temp. version!: idx_a1 < rng_b1
			if (idx_a1 < rng_b1) 
			sm_b[threadIdx.y + 8][threadIdx.x + ll * 8] = dev_v2[((blk_idx_b1 * SIZE_SLICE_1_B1 + idx_a1 + (blk_idx_b2 * SIZE_SLICE_1_B2 + ll) * size_b1) * size_c) * size_d + dev_internal_offset_v2[threadIdx.y + l + 8]];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1; ll++)
		{
			temp_bv[0] = sm_b[ll][idx_b1 + 0];
			temp_bv[1] = sm_b[ll][idx_b1 + 8];
			temp_bv[2] = sm_b[ll][idx_b1 + 16];
			temp_bv[3] = sm_b[ll][idx_b1 + 24];
			temp_bv[4] = sm_b[ll][idx_b1 + 32];
			temp_bv[5] = sm_b[ll][idx_b1 + 40];
			temp_bv[6] = sm_b[ll][idx_b1 + 48];
			temp_bv[7] = sm_b[ll][idx_b1 + 56];

			for (int xx = 0; xx < 4; xx++) // (1)
			{
				temp_av = sm_a[ll][idx_a1 + (xx * 16)];

				reg_tile[0][xx] += temp_av * temp_bv[0];
				reg_tile[1][xx] += temp_av * temp_bv[1];
				reg_tile[2][xx] += temp_av * temp_bv[2];
				reg_tile[3][xx] += temp_av * temp_bv[3];
				reg_tile[4][xx] += temp_av * temp_bv[4];
				reg_tile[5][xx] += temp_av * temp_bv[5];
				reg_tile[6][xx] += temp_av * temp_bv[6];
				reg_tile[7][xx] += temp_av * temp_bv[7];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	if (idx_a1 < rng_a1 && idx_b1 < rng_b1)
	for (int i = 0; i < 8; i++)
	{
		for (int j = 0; j < 4; j++)
		{
			if(i < rng_b2 && j < rng_a2)
			{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
			}
		}
	}
}

// created by tc_gen_code_Kernel()
__global__ void kernel__4_tex_1(double* dev_t3, 
double* dev_t2, 
double* dev_v2, 
int size_a1, int size_a2, int size_b1, int size_b2, int size_c, int size_d, 
int numBlk_a1, int numBlk_a2, int numBlk_b1, int numBlk_b2, 
int* dev_internal_offset_t2, int* dev_internal_offset_v2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ double sm_a[16][64];
	__shared__ double sm_b[16][64];


	int internal_upperbound   = 0;
	int internal_offset;

	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 1
	// # of indices mapped on TB_Y: 1
	int idx_a1 = threadIdx.x;
	int idx_b1 = threadIdx.y;

	int tmp_blkIdx;
	int blk_idx_b2 = blockIdx.x / (numBlk_b1 * numBlk_a2 * numBlk_a1);
	tmp_blkIdx = blockIdx.x % (numBlk_b1 * numBlk_a2 * numBlk_a1);

	int blk_idx_b1 = tmp_blkIdx / (numBlk_a2 * numBlk_a1);
	tmp_blkIdx = tmp_blkIdx % (numBlk_a2 * numBlk_a1);

	int blk_idx_a2 = tmp_blkIdx / numBlk_a1;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a1);

	int  blk_idx_a1 = tmp_blkIdx;

	int t3_base_thread = blk_idx_a1 * SIZE_SLICE_1_A1 + idx_a1 + (blk_idx_a2 * SIZE_SLICE_1_A2 + (blk_idx_b1 * SIZE_SLICE_1_B1 + idx_b1 + (blk_idx_b2 * SIZE_SLICE_1_B2) * size_b1) * size_a2) * size_a1;

	// need to support partial tiles
	int rng_a1, rng_a2, rng_b1, rng_b2;
	if ((size_a1 - (blk_idx_a1 * SIZE_SLICE_1_A1)) >= SIZE_SLICE_1_A1)
	{
		rng_a1 = SIZE_SLICE_1_A1;
	}
	else
	{
		rng_a1 = size_a1 % SIZE_SLICE_1_A1;
	}
	if ((size_a2 - (blk_idx_a2 * SIZE_SLICE_1_A2)) >= SIZE_SLICE_1_A2)
	{
		rng_a2 = SIZE_SLICE_1_A2;
	}
	else
	{
		rng_a2 = size_a2 % SIZE_SLICE_1_A2;
	}
	if ((size_b1 - (blk_idx_b1 * SIZE_SLICE_1_B1)) >= SIZE_SLICE_1_B1)
	{
		rng_b1 = SIZE_SLICE_1_B1;
	}
	else
	{
		rng_b1 = size_b1 % SIZE_SLICE_1_B1;
	}
	if ((size_b2 - (blk_idx_b2 * SIZE_SLICE_1_B2)) >= SIZE_SLICE_1_B2)
	{
		rng_b2 = SIZE_SLICE_1_B2;
	}
	else
	{
		rng_b2 = size_b2 % SIZE_SLICE_1_B2;
	}

	double temp_av;
	double temp_bv[8];
	double reg_tile[8][4];

	for (int i = 0; i < 8; i++)
	for (int j = 0; j < 4; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'x', 't2', ['c', 'a1', 'a2', 'd']], [16, 'STR_SD2_V2_H7', 'y', 'v2', ['d', 'c', 'b1', 'b2']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		// Part: Generalized Contraction Index (p7b)
		internal_offset = (l + SIZE_INT_UNIT_1) - size_internal;
		if (internal_offset > 0) internal_upperbound = internal_offset;

		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (idx_a1 < rng_a1 && threadIdx.y < SIZE_INT_UNIT_1 - internal_upperbound)
		for (int ll = 0; ll < rng_a2; ll++)
		{
			// ['c', 'a1', 'a2', 'd']
			// Exception: Temp. version!: threadIdx.y + l + 0
			// Exception: Temp. version!: idx_a1 < rng_a1
			sm_a[threadIdx.y + 0][threadIdx.x + ll * 16] = dev_t2[(blk_idx_a1 * SIZE_SLICE_1_A1 + idx_a1 + (blk_idx_a2 * SIZE_SLICE_1_A2 + ll) * size_a1) * size_c + dev_internal_offset_t2[threadIdx.y + l + 0]];
			// Exception: Temp. version!: threadIdx.y + l + 8
			// Exception: Temp. version!: idx_a1 < rng_a1
			if (threadIdx.y + l + 8 < size_internal && idx_a1 < rng_a1) 
			sm_a[threadIdx.y + 8][threadIdx.x + ll * 16] = dev_t2[(blk_idx_a1 * SIZE_SLICE_1_A1 + idx_a1 + (blk_idx_a2 * SIZE_SLICE_1_A2 + ll) * size_a1) * size_c + dev_internal_offset_t2[threadIdx.y + l + 8]];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (idx_a1 < rng_b1 && threadIdx.y < SIZE_INT_UNIT_1 - internal_upperbound && threadIdx.x < 8)
		for (int ll = 0; ll < rng_b2; ll++)
		{
			// ['d', 'c', 'b1', 'b2']
			// Exception: Temp. version!: threadIdx.y + l + 0
			// Exception: Temp. version!: idx_a1 < rng_b1
			sm_b[threadIdx.y + 0][threadIdx.x + ll * 8] = dev_v2[((blk_idx_b1 * SIZE_SLICE_1_B1 + idx_a1 + (blk_idx_b2 * SIZE_SLICE_1_B2 + ll) * size_b1) * size_c) * size_d + dev_internal_offset_v2[threadIdx.y + l + 0]];
			// Exception: Temp. version!: threadIdx.y + l + 8
			// Exception: Temp. version!: idx_a1 < rng_b1
			if (threadIdx.y + l + 8 < size_internal && idx_a1 < rng_b1) 
			sm_b[threadIdx.y + 8][threadIdx.x + ll * 8] = dev_v2[((blk_idx_b1 * SIZE_SLICE_1_B1 + idx_a1 + (blk_idx_b2 * SIZE_SLICE_1_B2 + ll) * size_b1) * size_c) * size_d + dev_internal_offset_v2[threadIdx.y + l + 8]];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1 - internal_upperbound; ll++)
		{
			temp_bv[0] = sm_b[ll][idx_b1 + 0];
			temp_bv[1] = sm_b[ll][idx_b1 + 8];
			temp_bv[2] = sm_b[ll][idx_b1 + 16];
			temp_bv[3] = sm_b[ll][idx_b1 + 24];
			temp_bv[4] = sm_b[ll][idx_b1 + 32];
			temp_bv[5] = sm_b[ll][idx_b1 + 40];
			temp_bv[6] = sm_b[ll][idx_b1 + 48];
			temp_bv[7] = sm_b[ll][idx_b1 + 56];

			for (int xx = 0; xx < 4; xx++) // (1)
			{
				temp_av = sm_a[ll][idx_a1 + (xx * 16)];

				reg_tile[0][xx] += temp_av * temp_bv[0];
				reg_tile[1][xx] += temp_av * temp_bv[1];
				reg_tile[2][xx] += temp_av * temp_bv[2];
				reg_tile[3][xx] += temp_av * temp_bv[3];
				reg_tile[4][xx] += temp_av * temp_bv[4];
				reg_tile[5][xx] += temp_av * temp_bv[5];
				reg_tile[6][xx] += temp_av * temp_bv[6];
				reg_tile[7][xx] += temp_av * temp_bv[7];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	if (idx_a1 < rng_a1 && idx_b1 < rng_b1)
	for (int i = 0; i < 8; i++)
	{
		for (int j = 0; j < 4; j++)
		{
			if(i < rng_b2 && j < rng_a2)
			{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
			}
		}
	}
}

// written by tc_interface.tc_gen_code_interface_Header()
extern "C"
void sd_t_d2_fusion(int size_a1, int size_a2, int size_b1, int size_b2, int size_c, int size_d, double* t3, double* host_t2, double* host_v2, int cond_kernel_1, int opt_register_transpose)
{
	int num_thread_blocks_kernel_1;

	double* dev_t3;
	double* dev_t2;
	double* dev_v2;

	int* host_internal_left_offset;
	int* host_internal_right_offset;

	num_thread_blocks_kernel_1 = CEIL(size_a1, SIZE_SLICE_1_A1) * CEIL(size_a2, SIZE_SLICE_1_A2) * CEIL(size_b1, SIZE_SLICE_1_B1) * CEIL(size_b2, SIZE_SLICE_1_B2);
	// cudaMalloc()
	hipMalloc((void**) &dev_t3, sizeof(double) * size_a1 * size_a2 * size_b1 * size_b2);
	hipMalloc((void**) &dev_t2, sizeof(double) * size_d * size_a2 * size_a1 * size_c);
	hipMalloc((void**) &dev_v2, sizeof(double) * size_b2 * size_b1 * size_c * size_d);

	// cudaMemcpy()
	hipMemcpy(dev_t3, t3, sizeof(double) * size_a1 * size_a2 * size_b1 * size_b2, hipMemcpyHostToDevice);
	hipMemcpy(dev_t2, host_t2, sizeof(double) * size_d * size_a2 * size_a1 * size_c, hipMemcpyHostToDevice);
	hipMemcpy(dev_v2, host_v2, sizeof(double) * size_b2 * size_b1 * size_c * size_d, hipMemcpyHostToDevice);

	// Related to Kernels
	// There are 1 Basic Kernels
	long long int tmp_operations = 2 * (long long int)((long long int)(size_a1 * size_a2 * size_b1 * size_b2) * size_c) * size_d;
	printf ("========================================= fusedKernels =============================================\n");
	printf ("		Grid Size  : %6d (1D)\n", num_thread_blocks_kernel_1);
	printf ("		Block-size : %2d, %2d (2D)\n", SIZE_TB_1_X, SIZE_TB_1_Y);
	printf ("		Reg.-size  : %2d, %2d (2D)\n", SIZE_REG_1_X, SIZE_REG_1_Y);
	printf ("		A thread deals with (%d x %d) elements (basically)\n", SIZE_TB_1_X * SIZE_REG_1_X, SIZE_TB_1_Y * SIZE_REG_1_Y);
	printf ("		# of Operations: %lld\n", tmp_operations);
	printf ("====================================================================================================\n");
	dim3 gridsize_1(num_thread_blocks_kernel_1);
	dim3 blocksize_1(SIZE_TB_1_X, SIZE_TB_1_Y);

	int stride_output_a1 = 1;
	int stride_output_a2 = stride_output_a1 * size_a1;
	int stride_output_b1 = stride_output_a2 * size_a2;
	int stride_output_b2 = stride_output_b1 * size_b1;

	int stride_reg_x_1 = stride_output_a2;
	int stride_reg_y_1 = stride_output_b2;

	int size_internal = size_c * size_d;

	// (manually) ['c', 'd']
	host_internal_left_offset 	= (int*)malloc(sizeof(int) * size_internal);
	host_internal_right_offset 	= (int*)malloc(sizeof(int) * size_internal);
	for (int idx_d = 0; idx_d < size_d; idx_d++)
	for (int idx_c = 0; idx_c < size_c; idx_c++)
	{
		host_internal_left_offset[idx_c + (idx_d) * size_c] 	= idx_c + (((idx_d) * size_a2) * size_a1) * size_c;
		host_internal_right_offset[idx_c + (idx_d) * size_c] 	= idx_d + (idx_c) * size_d;
	}

	hipMemcpyToSymbol(HIP_SYMBOL(const_internal_t2_offset), host_internal_left_offset, sizeof(int) * size_internal);
	hipMemcpyToSymbol(HIP_SYMBOL(const_internal_v2_offset), host_internal_right_offset, sizeof(int) * size_internal);

	int* dev_internal_offset_t2;
	int* dev_internal_offset_v2;
	// cudaMalloc()
	hipMalloc((void**) &dev_internal_offset_t2, sizeof(int) * size_internal);
	hipMalloc((void**) &dev_internal_offset_v2, sizeof(int) * size_internal);

	// cudaMemcpy()
	hipMemcpy(dev_internal_offset_t2, host_internal_left_offset, sizeof(int) * size_internal, hipMemcpyHostToDevice);
	hipMemcpy(dev_internal_offset_v2, host_internal_right_offset, sizeof(int) * size_internal, hipMemcpyHostToDevice);

	// Decision Tree for Kernel Types
	// No Chance to Utilize the Register Transpose
	if (size_a1 % SIZE_SLICE_1_A1 == 0 && size_a2 % SIZE_SLICE_1_A2 == 0 && size_b1 % SIZE_SLICE_1_B1 == 0 && size_b2 % SIZE_SLICE_1_B2 == 0)
	{
		// [2] Extenral Index: Full
		if (size_c % SIZE_SLICE_1_C == 0 && size_d % SIZE_SLICE_1_D == 0)
		{
			// [3] Internal Index: Full
			// >>> External: Full && Internal: Full
			printf ("External: Full, Internal: Full\n");
			if (size_internal > MAX_CONST_LEN)
			{
				kernel__1_tex_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a1, size_a2, size_b1, size_b2, size_c, size_d, CEIL(size_a1, SIZE_SLICE_1_A1), CEIL(size_a2, SIZE_SLICE_1_A2), CEIL(size_b1, SIZE_SLICE_1_B1), CEIL(size_b2, SIZE_SLICE_1_B2), dev_internal_offset_t2, dev_internal_offset_v2, stride_reg_x_1, stride_reg_y_1, size_internal);
			}
			else
			{
				kernel__1_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a1, size_a2, size_b1, size_b2, size_c, size_d, CEIL(size_a1, SIZE_SLICE_1_A1), CEIL(size_a2, SIZE_SLICE_1_A2), CEIL(size_b1, SIZE_SLICE_1_B1), CEIL(size_b2, SIZE_SLICE_1_B2), stride_reg_x_1, stride_reg_y_1, size_internal);
			}
		}
		else
		{
			// [4] Internal Index: Partial
			// >>> External: Full && Internal: Partial
			printf ("External: Full, Internal: Partial\n");
			if (size_internal > MAX_CONST_LEN)
			{
				kernel__2_tex_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a1, size_a2, size_b1, size_b2, size_c, size_d, CEIL(size_a1, SIZE_SLICE_1_A1), CEIL(size_a2, SIZE_SLICE_1_A2), CEIL(size_b1, SIZE_SLICE_1_B1), CEIL(size_b2, SIZE_SLICE_1_B2), dev_internal_offset_t2, dev_internal_offset_v2, stride_reg_x_1, stride_reg_y_1, size_internal);
			}
			else
			{
				kernel__2_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a1, size_a2, size_b1, size_b2, size_c, size_d, CEIL(size_a1, SIZE_SLICE_1_A1), CEIL(size_a2, SIZE_SLICE_1_A2), CEIL(size_b1, SIZE_SLICE_1_B1), CEIL(size_b2, SIZE_SLICE_1_B2), stride_reg_x_1, stride_reg_y_1, size_internal);
			}
		}
	}
	else
	{
		// [2] Extenral Index: Partial
		if (size_c % SIZE_SLICE_1_C == 0 && size_d % SIZE_SLICE_1_D == 0)
		{
			// [3] Internal Index: Full
			// >>> External: Partial && Internal: Full
			printf ("External: Partial, Internal: Full\n");
			if (size_internal > MAX_CONST_LEN)
			{
				kernel__3_tex_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a1, size_a2, size_b1, size_b2, size_c, size_d, CEIL(size_a1, SIZE_SLICE_1_A1), CEIL(size_a2, SIZE_SLICE_1_A2), CEIL(size_b1, SIZE_SLICE_1_B1), CEIL(size_b2, SIZE_SLICE_1_B2), dev_internal_offset_t2, dev_internal_offset_v2, stride_reg_x_1, stride_reg_y_1, size_internal);
			}
			else
			{
				kernel__3_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a1, size_a2, size_b1, size_b2, size_c, size_d, CEIL(size_a1, SIZE_SLICE_1_A1), CEIL(size_a2, SIZE_SLICE_1_A2), CEIL(size_b1, SIZE_SLICE_1_B1), CEIL(size_b2, SIZE_SLICE_1_B2), stride_reg_x_1, stride_reg_y_1, size_internal);
			}
		}
		else
		{
			// [4] Internal Index: Partial
			// >>> External: Partial && Internal: Partial
			printf ("External: Partial, Internal: Partial\n");
			if (size_internal > MAX_CONST_LEN)
			{
				kernel__4_tex_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a1, size_a2, size_b1, size_b2, size_c, size_d, CEIL(size_a1, SIZE_SLICE_1_A1), CEIL(size_a2, SIZE_SLICE_1_A2), CEIL(size_b1, SIZE_SLICE_1_B1), CEIL(size_b2, SIZE_SLICE_1_B2), dev_internal_offset_t2, dev_internal_offset_v2, stride_reg_x_1, stride_reg_y_1, size_internal);
			}
			else
			{
				kernel__4_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a1, size_a2, size_b1, size_b2, size_c, size_d, CEIL(size_a1, SIZE_SLICE_1_A1), CEIL(size_a2, SIZE_SLICE_1_A2), CEIL(size_b1, SIZE_SLICE_1_B1), CEIL(size_b2, SIZE_SLICE_1_B2), stride_reg_x_1, stride_reg_y_1, size_internal);
			}
		}
	}

	// Copy the Result from Device to Host
	hipMemcpy(t3, dev_t3, sizeof(double) * (size_a1 * size_a2 * size_b1 * size_b2), hipMemcpyDeviceToHost);

	// cudaFree()
	hipFree(dev_t3);	hipFree(dev_t2);	hipFree(dev_v2);

	// Shoule be Fixed
	// HostFree

}

// This is written by tc_interface.tc_gen_code_interface()
// This Interface Should be Called to Run the Kernels
extern "C"
void sd_t_d2_fusion_(int size_a, int size_b, int size_c, int size_d, double* t3, double* t2, double* v2, int cond_kernel_1, int opt_register_transpose)
{
	// Pre-Processing for Split
	// Based on Tile-Sizes and Problem-Size
	// Currently, one index can be split into two indices
	int size_a1;
	int size_a2;

	if (size_a % 78 == 0)
	{
		//
		size_a1 = 78;
		size_a2 = size_a / 78;
	}
	else
	{
		//
		size_a1 = size_a;
		size_a2 = 1;
	}
	int size_b1;
	int size_b2;

	if (size_b % 37 == 0)
	{
		//
		size_b1 = 37;
		size_b2 = size_b / 37;
	}
	else
	{
		//
		size_b1 = size_b;
		size_b2 = 1;
	}

	// Call An Application
	sd_t_d2_fusion(size_a1, size_a2, size_b1, size_b2, size_c, size_d, t3, t2, v2, cond_kernel_1, opt_register_transpose);
}
