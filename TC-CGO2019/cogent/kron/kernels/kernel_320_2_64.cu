// created by tc_code_include() in tc_code_include.py

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <sys/time.h>
#include <locale.h>
#include <algorithm>
using namespace std;

// created by tc_gen_definition_new()
#define SIZE_SLICE_1_D 16
#define SIZE_SLICE_1_A 16
#define SIZE_SLICE_1_B 4
#define SIZE_SLICE_1_C1 8
#define SIZE_SLICE_1_C2 8

#define SIZE_INT_UNIT_1 SIZE_SLICE_1_D

#define SIZE_TB_1_X 	SIZE_SLICE_1_A
#define SIZE_TB_1_Y 	SIZE_SLICE_1_C1
#define SIZE_REG_1_X 	SIZE_SLICE_1_B
#define SIZE_REG_1_Y 	SIZE_SLICE_1_C2

#define NUM_INDEX 		4
#define CEIL(a, b) 		(((a) + (b) - 1) / (b))

// created by tc_gen_code_Kernel()
__global__ void kernel__1_1(float* dev_t3, 
float* dev_t2, 
float* dev_v2, 
int size_a, int size_c1, int size_c2, int size_b, int size_d, 
int numBlk_a, int numBlk_c1, int numBlk_c2, int numBlk_b, 
int stride_int_t2, int stride_int_v2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ float sm_a[16][64];
	__shared__ float sm_b[16][64];


	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 1
	// # of indices mapped on TB_Y: 1
	int idx_a = threadIdx.x;
	int idx_c1 = threadIdx.y;

	int tmp_blkIdx;
	int blk_idx_b = blockIdx.x / (numBlk_c2 * numBlk_c1 * numBlk_a);
	tmp_blkIdx = blockIdx.x % (numBlk_c2 * numBlk_c1 * numBlk_a);

	int blk_idx_c2 = tmp_blkIdx / (numBlk_c1 * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_c1 * numBlk_a);

	int blk_idx_c1 = tmp_blkIdx / numBlk_a;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a);

	int  blk_idx_a = tmp_blkIdx;

	int t3_base_thread = blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_c1 * SIZE_SLICE_1_C1 + idx_c1 + (blk_idx_c2 * SIZE_SLICE_1_C2 + (blk_idx_b * SIZE_SLICE_1_B) * size_c2) * size_c1) * size_a;


	float temp_av;
	float temp_bv[8];
	float reg_tile[8][4];

	for (int i = 0; i < 8; i++)
	for (int j = 0; j < 4; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'x', 't2', ['a', 'b', 'd']], [16, 'STR_SD2_V2_H7', 'y', 'v2', ['d', 'c1', 'c2']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		// No Need to Put Boundary-Checks before For-Statement: : 
		for (int ll = 0; ll < 4; ll++)
		{
			// ['a', 'b', 'd']
			// Exception: Temp. version!: threadIdx.y + l + 0
			// Exception: Temp. version!: idx_a < rng_a
			sm_a[threadIdx.y + 0][threadIdx.x + ll * 16] = dev_t2[blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_b * SIZE_SLICE_1_B + ll) * size_a + (threadIdx.y + l + 0) * stride_int_t2];
			// Exception: Temp. version!: threadIdx.y + l + 8
			// Exception: Temp. version!: idx_a < rng_a
			// Exception: Full-Full
			sm_a[threadIdx.y + 8][threadIdx.x + ll * 16] = dev_t2[blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_b * SIZE_SLICE_1_B + ll) * size_a + (threadIdx.y + l + 8) * stride_int_t2];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		// No Need to Put Boundary-Checks before For-Statement: : 
		for (int ll = 0; ll < 8; ll++)
		{
			// ['d', 'c1', 'c2']
			// Exception: Temp. version!: threadIdx.x + l
			// Exception: Temp. version!: idx_c1 < rng_c1
			sm_b[threadIdx.x][threadIdx.y + ll * 8] = dev_v2[(blk_idx_c1 * SIZE_SLICE_1_C1 + idx_c1 + (blk_idx_c2 * SIZE_SLICE_1_C2 + ll) * size_c1) * size_d + (threadIdx.x + l)];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1; ll++)
		{
			temp_bv[0] = sm_b[ll][idx_c1 + 0];
			temp_bv[1] = sm_b[ll][idx_c1 + 8];
			temp_bv[2] = sm_b[ll][idx_c1 + 16];
			temp_bv[3] = sm_b[ll][idx_c1 + 24];
			temp_bv[4] = sm_b[ll][idx_c1 + 32];
			temp_bv[5] = sm_b[ll][idx_c1 + 40];
			temp_bv[6] = sm_b[ll][idx_c1 + 48];
			temp_bv[7] = sm_b[ll][idx_c1 + 56];

			for (int xx = 0; xx < 4; xx++) // (1)
			{
				temp_av = sm_a[ll][idx_a + (xx * 16)];

				reg_tile[0][xx] += temp_av * temp_bv[0];
				reg_tile[1][xx] += temp_av * temp_bv[1];
				reg_tile[2][xx] += temp_av * temp_bv[2];
				reg_tile[3][xx] += temp_av * temp_bv[3];
				reg_tile[4][xx] += temp_av * temp_bv[4];
				reg_tile[5][xx] += temp_av * temp_bv[5];
				reg_tile[6][xx] += temp_av * temp_bv[6];
				reg_tile[7][xx] += temp_av * temp_bv[7];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	#pragma unroll 8
	for (int i = 0; i < 8; i++)
	{
		for (int j = 0; j < 4; j++)
		{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
		}
	}
}

// created by tc_gen_code_Kernel()
__global__ void kernel__2_1(float* dev_t3, 
float* dev_t2, 
float* dev_v2, 
int size_a, int size_c1, int size_c2, int size_b, int size_d, 
int numBlk_a, int numBlk_c1, int numBlk_c2, int numBlk_b, 
int stride_int_t2, int stride_int_v2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ float sm_a[16][64];
	__shared__ float sm_b[16][64];


	int internal_upperbound   = 0;
	int internal_offset;

	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 1
	// # of indices mapped on TB_Y: 1
	int idx_a = threadIdx.x;
	int idx_c1 = threadIdx.y;

	int tmp_blkIdx;
	int blk_idx_b = blockIdx.x / (numBlk_c2 * numBlk_c1 * numBlk_a);
	tmp_blkIdx = blockIdx.x % (numBlk_c2 * numBlk_c1 * numBlk_a);

	int blk_idx_c2 = tmp_blkIdx / (numBlk_c1 * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_c1 * numBlk_a);

	int blk_idx_c1 = tmp_blkIdx / numBlk_a;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a);

	int  blk_idx_a = tmp_blkIdx;

	int t3_base_thread = blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_c1 * SIZE_SLICE_1_C1 + idx_c1 + (blk_idx_c2 * SIZE_SLICE_1_C2 + (blk_idx_b * SIZE_SLICE_1_B) * size_c2) * size_c1) * size_a;


	float temp_av;
	float temp_bv[8];
	float reg_tile[8][4];

	for (int i = 0; i < 8; i++)
	for (int j = 0; j < 4; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'x', 't2', ['a', 'b', 'd']], [16, 'STR_SD2_V2_H7', 'y', 'v2', ['d', 'c1', 'c2']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		// Part: Generalized Contraction Index (p7b)
		internal_offset = (l + SIZE_INT_UNIT_1) - size_internal;
		if (internal_offset > 0) internal_upperbound = internal_offset;

		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (threadIdx.y < SIZE_INT_UNIT_1 - internal_upperbound)
		for (int ll = 0; ll < 4; ll++)
		{
			// ['a', 'b', 'd']
			// Exception: Temp. version!: threadIdx.y + l + 0
			// Exception: Temp. version!: idx_a < rng_a
			sm_a[threadIdx.y + 0][threadIdx.x + ll * 16] = dev_t2[blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_b * SIZE_SLICE_1_B + ll) * size_a + (threadIdx.y + l + 0) * stride_int_t2];
			// Exception: Temp. version!: threadIdx.y + l + 8
			// Exception: Temp. version!: idx_a < rng_a
			if (threadIdx.y + l + 8 < size_internal) 
			sm_a[threadIdx.y + 8][threadIdx.x + ll * 16] = dev_t2[blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_b * SIZE_SLICE_1_B + ll) * size_a + (threadIdx.y + l + 8) * stride_int_t2];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (threadIdx.x < SIZE_INT_UNIT_1 - internal_upperbound)
		for (int ll = 0; ll < 8; ll++)
		{
			// ['d', 'c1', 'c2']
			// Exception: Temp. version!: threadIdx.x + l
			// Exception: Temp. version!: idx_c1 < rng_c1
			sm_b[threadIdx.x][threadIdx.y + ll * 8] = dev_v2[(blk_idx_c1 * SIZE_SLICE_1_C1 + idx_c1 + (blk_idx_c2 * SIZE_SLICE_1_C2 + ll) * size_c1) * size_d + (threadIdx.x + l)];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1 - internal_upperbound; ll++)
		{
			temp_bv[0] = sm_b[ll][idx_c1 + 0];
			temp_bv[1] = sm_b[ll][idx_c1 + 8];
			temp_bv[2] = sm_b[ll][idx_c1 + 16];
			temp_bv[3] = sm_b[ll][idx_c1 + 24];
			temp_bv[4] = sm_b[ll][idx_c1 + 32];
			temp_bv[5] = sm_b[ll][idx_c1 + 40];
			temp_bv[6] = sm_b[ll][idx_c1 + 48];
			temp_bv[7] = sm_b[ll][idx_c1 + 56];

			for (int xx = 0; xx < 4; xx++) // (1)
			{
				temp_av = sm_a[ll][idx_a + (xx * 16)];

				reg_tile[0][xx] += temp_av * temp_bv[0];
				reg_tile[1][xx] += temp_av * temp_bv[1];
				reg_tile[2][xx] += temp_av * temp_bv[2];
				reg_tile[3][xx] += temp_av * temp_bv[3];
				reg_tile[4][xx] += temp_av * temp_bv[4];
				reg_tile[5][xx] += temp_av * temp_bv[5];
				reg_tile[6][xx] += temp_av * temp_bv[6];
				reg_tile[7][xx] += temp_av * temp_bv[7];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	#pragma unroll 8
	for (int i = 0; i < 8; i++)
	{
		for (int j = 0; j < 4; j++)
		{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
		}
	}
}

// created by tc_gen_code_Kernel()
__global__ void kernel__3_1(float* dev_t3, 
float* dev_t2, 
float* dev_v2, 
int size_a, int size_c1, int size_c2, int size_b, int size_d, 
int numBlk_a, int numBlk_c1, int numBlk_c2, int numBlk_b, 
int stride_int_t2, int stride_int_v2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ float sm_a[16][64];
	__shared__ float sm_b[16][64];


	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 1
	// # of indices mapped on TB_Y: 1
	int idx_a = threadIdx.x;
	int idx_c1 = threadIdx.y;

	int tmp_blkIdx;
	int blk_idx_b = blockIdx.x / (numBlk_c2 * numBlk_c1 * numBlk_a);
	tmp_blkIdx = blockIdx.x % (numBlk_c2 * numBlk_c1 * numBlk_a);

	int blk_idx_c2 = tmp_blkIdx / (numBlk_c1 * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_c1 * numBlk_a);

	int blk_idx_c1 = tmp_blkIdx / numBlk_a;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a);

	int  blk_idx_a = tmp_blkIdx;

	int t3_base_thread = blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_c1 * SIZE_SLICE_1_C1 + idx_c1 + (blk_idx_c2 * SIZE_SLICE_1_C2 + (blk_idx_b * SIZE_SLICE_1_B) * size_c2) * size_c1) * size_a;

	// need to support partial tiles
	int rng_a, rng_c1, rng_c2, rng_b;
	if ((size_a - (blk_idx_a * SIZE_SLICE_1_A)) >= SIZE_SLICE_1_A)
	{
		rng_a = SIZE_SLICE_1_A;
	}
	else
	{
		rng_a = size_a % SIZE_SLICE_1_A;
	}
	if ((size_c1 - (blk_idx_c1 * SIZE_SLICE_1_C1)) >= SIZE_SLICE_1_C1)
	{
		rng_c1 = SIZE_SLICE_1_C1;
	}
	else
	{
		rng_c1 = size_c1 % SIZE_SLICE_1_C1;
	}
	if ((size_c2 - (blk_idx_c2 * SIZE_SLICE_1_C2)) >= SIZE_SLICE_1_C2)
	{
		rng_c2 = SIZE_SLICE_1_C2;
	}
	else
	{
		rng_c2 = size_c2 % SIZE_SLICE_1_C2;
	}
	if ((size_b - (blk_idx_b * SIZE_SLICE_1_B)) >= SIZE_SLICE_1_B)
	{
		rng_b = SIZE_SLICE_1_B;
	}
	else
	{
		rng_b = size_b % SIZE_SLICE_1_B;
	}

	float temp_av;
	float temp_bv[8];
	float reg_tile[8][4];

	for (int i = 0; i < 8; i++)
	for (int j = 0; j < 4; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'x', 't2', ['a', 'b', 'd']], [16, 'STR_SD2_V2_H7', 'y', 'v2', ['d', 'c1', 'c2']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (idx_a < rng_a)
		for (int ll = 0; ll < rng_b; ll++)
		{
			// ['a', 'b', 'd']
			// Exception: Temp. version!: threadIdx.y + l + 0
			// Exception: Temp. version!: idx_a < rng_a
			sm_a[threadIdx.y + 0][threadIdx.x + ll * 16] = dev_t2[blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_b * SIZE_SLICE_1_B + ll) * size_a + (threadIdx.y + l + 0) * stride_int_t2];
			// Exception: Temp. version!: threadIdx.y + l + 8
			// Exception: Temp. version!: idx_a < rng_a
			if (idx_a < rng_a) 
			sm_a[threadIdx.y + 8][threadIdx.x + ll * 16] = dev_t2[blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_b * SIZE_SLICE_1_B + ll) * size_a + (threadIdx.y + l + 8) * stride_int_t2];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (idx_c1 < rng_c1)
		for (int ll = 0; ll < rng_c2; ll++)
		{
			// ['d', 'c1', 'c2']
			// Exception: Temp. version!: threadIdx.x + l
			// Exception: Temp. version!: idx_c1 < rng_c1
			sm_b[threadIdx.x][threadIdx.y + ll * 8] = dev_v2[(blk_idx_c1 * SIZE_SLICE_1_C1 + idx_c1 + (blk_idx_c2 * SIZE_SLICE_1_C2 + ll) * size_c1) * size_d + (threadIdx.x + l)];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1; ll++)
		{
			temp_bv[0] = sm_b[ll][idx_c1 + 0];
			temp_bv[1] = sm_b[ll][idx_c1 + 8];
			temp_bv[2] = sm_b[ll][idx_c1 + 16];
			temp_bv[3] = sm_b[ll][idx_c1 + 24];
			temp_bv[4] = sm_b[ll][idx_c1 + 32];
			temp_bv[5] = sm_b[ll][idx_c1 + 40];
			temp_bv[6] = sm_b[ll][idx_c1 + 48];
			temp_bv[7] = sm_b[ll][idx_c1 + 56];

			for (int xx = 0; xx < 4; xx++) // (1)
			{
				temp_av = sm_a[ll][idx_a + (xx * 16)];

				reg_tile[0][xx] += temp_av * temp_bv[0];
				reg_tile[1][xx] += temp_av * temp_bv[1];
				reg_tile[2][xx] += temp_av * temp_bv[2];
				reg_tile[3][xx] += temp_av * temp_bv[3];
				reg_tile[4][xx] += temp_av * temp_bv[4];
				reg_tile[5][xx] += temp_av * temp_bv[5];
				reg_tile[6][xx] += temp_av * temp_bv[6];
				reg_tile[7][xx] += temp_av * temp_bv[7];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	if (idx_a < rng_a && idx_c1 < rng_c1)
	for (int i = 0; i < 8; i++)
	{
		for (int j = 0; j < 4; j++)
		{
			if(i < rng_c2 && j < rng_b)
			{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
			}
		}
	}
}

// created by tc_gen_code_Kernel()
__global__ void kernel__4_1(float* dev_t3, 
float* dev_t2, 
float* dev_v2, 
int size_a, int size_c1, int size_c2, int size_b, int size_d, 
int numBlk_a, int numBlk_c1, int numBlk_c2, int numBlk_b, 
int stride_int_t2, int stride_int_v2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ float sm_a[16][64];
	__shared__ float sm_b[16][64];


	int internal_upperbound   = 0;
	int internal_offset;

	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 1
	// # of indices mapped on TB_Y: 1
	int idx_a = threadIdx.x;
	int idx_c1 = threadIdx.y;

	int tmp_blkIdx;
	int blk_idx_b = blockIdx.x / (numBlk_c2 * numBlk_c1 * numBlk_a);
	tmp_blkIdx = blockIdx.x % (numBlk_c2 * numBlk_c1 * numBlk_a);

	int blk_idx_c2 = tmp_blkIdx / (numBlk_c1 * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_c1 * numBlk_a);

	int blk_idx_c1 = tmp_blkIdx / numBlk_a;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a);

	int  blk_idx_a = tmp_blkIdx;

	int t3_base_thread = blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_c1 * SIZE_SLICE_1_C1 + idx_c1 + (blk_idx_c2 * SIZE_SLICE_1_C2 + (blk_idx_b * SIZE_SLICE_1_B) * size_c2) * size_c1) * size_a;

	// need to support partial tiles
	int rng_a, rng_c1, rng_c2, rng_b;
	if ((size_a - (blk_idx_a * SIZE_SLICE_1_A)) >= SIZE_SLICE_1_A)
	{
		rng_a = SIZE_SLICE_1_A;
	}
	else
	{
		rng_a = size_a % SIZE_SLICE_1_A;
	}
	if ((size_c1 - (blk_idx_c1 * SIZE_SLICE_1_C1)) >= SIZE_SLICE_1_C1)
	{
		rng_c1 = SIZE_SLICE_1_C1;
	}
	else
	{
		rng_c1 = size_c1 % SIZE_SLICE_1_C1;
	}
	if ((size_c2 - (blk_idx_c2 * SIZE_SLICE_1_C2)) >= SIZE_SLICE_1_C2)
	{
		rng_c2 = SIZE_SLICE_1_C2;
	}
	else
	{
		rng_c2 = size_c2 % SIZE_SLICE_1_C2;
	}
	if ((size_b - (blk_idx_b * SIZE_SLICE_1_B)) >= SIZE_SLICE_1_B)
	{
		rng_b = SIZE_SLICE_1_B;
	}
	else
	{
		rng_b = size_b % SIZE_SLICE_1_B;
	}

	float temp_av;
	float temp_bv[8];
	float reg_tile[8][4];

	for (int i = 0; i < 8; i++)
	for (int j = 0; j < 4; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'x', 't2', ['a', 'b', 'd']], [16, 'STR_SD2_V2_H7', 'y', 'v2', ['d', 'c1', 'c2']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		// Part: Generalized Contraction Index (p7b)
		internal_offset = (l + SIZE_INT_UNIT_1) - size_internal;
		if (internal_offset > 0) internal_upperbound = internal_offset;

		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (idx_a < rng_a && threadIdx.y < SIZE_INT_UNIT_1 - internal_upperbound)
		for (int ll = 0; ll < rng_b; ll++)
		{
			// ['a', 'b', 'd']
			// Exception: Temp. version!: threadIdx.y + l + 0
			// Exception: Temp. version!: idx_a < rng_a
			sm_a[threadIdx.y + 0][threadIdx.x + ll * 16] = dev_t2[blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_b * SIZE_SLICE_1_B + ll) * size_a + (threadIdx.y + l + 0) * stride_int_t2];
			// Exception: Temp. version!: threadIdx.y + l + 8
			// Exception: Temp. version!: idx_a < rng_a
			if (threadIdx.y + l + 8 < size_internal && idx_a < rng_a) 
			sm_a[threadIdx.y + 8][threadIdx.x + ll * 16] = dev_t2[blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_b * SIZE_SLICE_1_B + ll) * size_a + (threadIdx.y + l + 8) * stride_int_t2];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (idx_c1 < rng_c1 && threadIdx.x < SIZE_INT_UNIT_1 - internal_upperbound)
		for (int ll = 0; ll < rng_c2; ll++)
		{
			// ['d', 'c1', 'c2']
			// Exception: Temp. version!: threadIdx.x + l
			// Exception: Temp. version!: idx_c1 < rng_c1
			sm_b[threadIdx.x][threadIdx.y + ll * 8] = dev_v2[(blk_idx_c1 * SIZE_SLICE_1_C1 + idx_c1 + (blk_idx_c2 * SIZE_SLICE_1_C2 + ll) * size_c1) * size_d + (threadIdx.x + l)];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1 - internal_upperbound; ll++)
		{
			temp_bv[0] = sm_b[ll][idx_c1 + 0];
			temp_bv[1] = sm_b[ll][idx_c1 + 8];
			temp_bv[2] = sm_b[ll][idx_c1 + 16];
			temp_bv[3] = sm_b[ll][idx_c1 + 24];
			temp_bv[4] = sm_b[ll][idx_c1 + 32];
			temp_bv[5] = sm_b[ll][idx_c1 + 40];
			temp_bv[6] = sm_b[ll][idx_c1 + 48];
			temp_bv[7] = sm_b[ll][idx_c1 + 56];

			for (int xx = 0; xx < 4; xx++) // (1)
			{
				temp_av = sm_a[ll][idx_a + (xx * 16)];

				reg_tile[0][xx] += temp_av * temp_bv[0];
				reg_tile[1][xx] += temp_av * temp_bv[1];
				reg_tile[2][xx] += temp_av * temp_bv[2];
				reg_tile[3][xx] += temp_av * temp_bv[3];
				reg_tile[4][xx] += temp_av * temp_bv[4];
				reg_tile[5][xx] += temp_av * temp_bv[5];
				reg_tile[6][xx] += temp_av * temp_bv[6];
				reg_tile[7][xx] += temp_av * temp_bv[7];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	if (idx_a < rng_a && idx_c1 < rng_c1)
	for (int i = 0; i < 8; i++)
	{
		for (int j = 0; j < 4; j++)
		{
			if(i < rng_c2 && j < rng_b)
			{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
			}
		}
	}
}

// written by tc_interface.tc_gen_code_interface_Header()
extern "C"
void sd_t_d2_fusion(int size_a, int size_c1, int size_c2, int size_b, int size_d, float* t3, float* host_t2, float* host_v2, int cond_kernel_1, int opt_register_transpose)
{
	int num_thread_blocks_kernel_1;

	float* dev_t3;
	float* dev_t2;
	float* dev_v2;


	num_thread_blocks_kernel_1 = CEIL(size_a, SIZE_SLICE_1_A) * CEIL(size_c1, SIZE_SLICE_1_C1) * CEIL(size_c2, SIZE_SLICE_1_C2) * CEIL(size_b, SIZE_SLICE_1_B);
	// cudaMalloc()
	hipMalloc((void**) &dev_t3, sizeof(float) * size_a * size_c1 * size_c2 * size_b);
	hipMalloc((void**) &dev_t2, sizeof(float) * size_d * size_b * size_a);
	hipMalloc((void**) &dev_v2, sizeof(float) * size_c2 * size_c1 * size_d);

	// cudaMemcpy()
	hipMemcpy(dev_t3, t3, sizeof(float) * size_a * size_c1 * size_c2 * size_b, hipMemcpyHostToDevice);
	hipMemcpy(dev_t2, host_t2, sizeof(float) * size_d * size_b * size_a, hipMemcpyHostToDevice);
	hipMemcpy(dev_v2, host_v2, sizeof(float) * size_c2 * size_c1 * size_d, hipMemcpyHostToDevice);

	// Related to Kernels
	// There are 1 Basic Kernels
	long long int tmp_operations = (long long int)(size_a * size_c1 * size_c2 * size_b) * size_d;
	printf ("========================================= fusedKernels =============================================\n");
	printf ("		Grid Size  : %6d (1D)\n", num_thread_blocks_kernel_1);
	printf ("		Block-size : %2d, %2d (2D)\n", SIZE_TB_1_X, SIZE_TB_1_Y);
	printf ("		Reg.-size  : %2d, %2d (2D)\n", SIZE_REG_1_X, SIZE_REG_1_Y);
	printf ("		A thread deals with (%d x %d) elements (basically)\n", SIZE_TB_1_X * SIZE_REG_1_X, SIZE_TB_1_Y * SIZE_REG_1_Y);
	printf ("		# of Operations: %lld\n", tmp_operations);
	printf ("====================================================================================================\n");
	dim3 gridsize_1(num_thread_blocks_kernel_1);
	dim3 blocksize_1(SIZE_TB_1_X, SIZE_TB_1_Y);

	int stride_output_a = 1;
	int stride_output_c1 = stride_output_a * size_a;
	int stride_output_c2 = stride_output_c1 * size_c1;
	int stride_output_b = stride_output_c2 * size_c2;

	int stride_reg_x_1 = stride_output_b;
	int stride_reg_y_1 = stride_output_c2;

	int size_internal = size_d;

	int stride_int_t2 = size_a * size_b;
	int stride_int_v2 = 1;

	 hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);	for (int i = 0; i < 110; i++) {
	if (i==9)hipEventRecord(start);
	// Decision Tree for Kernel Types
	// No Chance to Utilize the Register Transpose
	if (size_a % SIZE_SLICE_1_A == 0 && size_c1 % SIZE_SLICE_1_C1 == 0 && size_c2 % SIZE_SLICE_1_C2 == 0 && size_b % SIZE_SLICE_1_B == 0)
	{
		// [2] Extenral Index: Full
		if (size_d % SIZE_SLICE_1_D == 0)
		{
			// [3] Internal Index: Full
			// >>> External: Full && Internal: Full
			printf ("External: Full, Internal: Full\n");
			kernel__1_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a, size_c1, size_c2, size_b, size_d, CEIL(size_a, SIZE_SLICE_1_A), CEIL(size_c1, SIZE_SLICE_1_C1), CEIL(size_c2, SIZE_SLICE_1_C2), CEIL(size_b, SIZE_SLICE_1_B), stride_int_t2, stride_int_v2, stride_reg_x_1, stride_reg_y_1, size_internal);
		}
		else
		{
			// [4] Internal Index: Partial
			// >>> External: Full && Internal: Partial
			printf ("External: Full, Internal: Partial\n");
			kernel__2_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a, size_c1, size_c2, size_b, size_d, CEIL(size_a, SIZE_SLICE_1_A), CEIL(size_c1, SIZE_SLICE_1_C1), CEIL(size_c2, SIZE_SLICE_1_C2), CEIL(size_b, SIZE_SLICE_1_B), stride_int_t2, stride_int_v2, stride_reg_x_1, stride_reg_y_1, size_internal);
		}
	}
	else
	{
		// [2] Extenral Index: Partial
		if (size_d % SIZE_SLICE_1_D == 0)
		{
			// [3] Internal Index: Full
			// >>> External: Partial && Internal: Full
			printf ("External: Partial, Internal: Full\n");
			kernel__3_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a, size_c1, size_c2, size_b, size_d, CEIL(size_a, SIZE_SLICE_1_A), CEIL(size_c1, SIZE_SLICE_1_C1), CEIL(size_c2, SIZE_SLICE_1_C2), CEIL(size_b, SIZE_SLICE_1_B), stride_int_t2, stride_int_v2, stride_reg_x_1, stride_reg_y_1, size_internal);
		}
		else
		{
			// [4] Internal Index: Partial
			// >>> External: Partial && Internal: Partial
			printf ("External: Partial, Internal: Partial\n");
			kernel__4_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a, size_c1, size_c2, size_b, size_d, CEIL(size_a, SIZE_SLICE_1_A), CEIL(size_c1, SIZE_SLICE_1_C1), CEIL(size_c2, SIZE_SLICE_1_C2), CEIL(size_b, SIZE_SLICE_1_B), stride_int_t2, stride_int_v2, stride_reg_x_1, stride_reg_y_1, size_internal);
		}
	}

	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);

	float milliseconds = 0;

	hipEventElapsedTime(&milliseconds, start, stop);

	printf("Time of one iteration: %f milliseconds\n", milliseconds/100.);
	// Copy the Result from Device to Host
	hipMemcpy(t3, dev_t3, sizeof(float) * (size_a * size_c1 * size_c2 * size_b), hipMemcpyDeviceToHost);

	// cudaFree()
	hipFree(dev_t3);	hipFree(dev_t2);	hipFree(dev_v2);

	// Shoule be Fixed
	// HostFree

}

// This is written by tc_interface.tc_gen_code_interface()
// This Interface Should be Called to Run the Kernels
extern "C"
void sd_t_d2_fusion_(int size_a, int size_c, int size_b, int size_d, float* t3, float* t2, float* v2, int cond_kernel_1, int opt_register_transpose)
{
	// Pre-Processing for Split
	// Based on Tile-Sizes and Problem-Size
	// Currently, one index can be split into two indices
	int size_c1;
	int size_c2;

	if (size_c % 8 == 0)
	{
		//
		size_c1 = 8;
		size_c2 = size_c / 8;
	}
	else
	{
		//
		size_c1 = size_c;
		size_c2 = 1;
	}

	// Call An Application
	sd_t_d2_fusion(size_a, size_c1, size_c2, size_b, size_d, t3, t2, v2, cond_kernel_1, opt_register_transpose);
}
