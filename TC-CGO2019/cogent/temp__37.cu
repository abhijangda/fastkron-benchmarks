// created by tc_code_include() in tc_code_include.py

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <sys/time.h>
#include <locale.h>
#include <algorithm>
using namespace std;

// created by tc_gen_definition_new()
#define SIZE_SLICE_1_G 16
#define SIZE_SLICE_1_A 16
#define SIZE_SLICE_1_E 6
#define SIZE_SLICE_1_F 1
#define SIZE_SLICE_1_D 16
#define SIZE_SLICE_1_B 6
#define SIZE_SLICE_1_C 1

#define SIZE_INT_UNIT_1 SIZE_SLICE_1_G

#define SIZE_TB_1_X 	SIZE_SLICE_1_A * SIZE_SLICE_1_F
#define SIZE_TB_1_Y 	SIZE_SLICE_1_D * SIZE_SLICE_1_C
#define SIZE_REG_1_X 	SIZE_SLICE_1_E
#define SIZE_REG_1_Y 	SIZE_SLICE_1_B

#define NUM_INDEX 		6
#define CEIL(a, b) 		(((a) + (b) - 1) / (b))

// created by tc_gen_code_Kernel()
__global__ void kernel__1_1(double* dev_t3, 
double* dev_t2, 
double* dev_v2, 
int size_a, int size_b, int size_c, int size_d, int size_e, int size_f, int size_g, 
int numBlk_a, int numBlk_b, int numBlk_c, int numBlk_d, int numBlk_e, int numBlk_f, 
int stride_int_t2, int stride_int_v2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ double sm_a[16][96];
	__shared__ double sm_b[16][96];


	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 2
	// # of indices mapped on TB_Y: 2
	int idx_a = threadIdx.x % SIZE_SLICE_1_A;
	int idx_f = threadIdx.x / SIZE_SLICE_1_A;
	int idx_d = threadIdx.y % SIZE_SLICE_1_D;
	int idx_c = threadIdx.y / SIZE_SLICE_1_D;

	int tmp_blkIdx;
	int blk_idx_f = blockIdx.x / (numBlk_e * numBlk_d * numBlk_c * numBlk_b * numBlk_a);
	tmp_blkIdx = blockIdx.x % (numBlk_e * numBlk_d * numBlk_c * numBlk_b * numBlk_a);

	int blk_idx_e = tmp_blkIdx / (numBlk_d * numBlk_c * numBlk_b * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_d * numBlk_c * numBlk_b * numBlk_a);

	int blk_idx_d = tmp_blkIdx / (numBlk_c * numBlk_b * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_c * numBlk_b * numBlk_a);

	int blk_idx_c = tmp_blkIdx / (numBlk_b * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_b * numBlk_a);

	int blk_idx_b = tmp_blkIdx / numBlk_a;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a);

	int  blk_idx_a = tmp_blkIdx;

	int t3_base_thread = blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_b * SIZE_SLICE_1_B + (blk_idx_c * SIZE_SLICE_1_C + idx_c + (blk_idx_d * SIZE_SLICE_1_D + idx_d + (blk_idx_e * SIZE_SLICE_1_E + (blk_idx_f * SIZE_SLICE_1_F + idx_f) * size_e) * size_d) * size_c) * size_b) * size_a;


	double temp_av;
	double temp_bv[6];
	double reg_tile[6][6];

	for (int i = 0; i < 6; i++)
	for (int j = 0; j < 6; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'x', 't2', ['e', 'f', 'g', 'a']], [16, 'STR_SD2_V2_H7', 'y', 'v2', ['g', 'd', 'b', 'c']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		// No Need to Put Boundary-Checks before For-Statement: : 
		for (int ll = 0; ll < 6; ll++)
		{
			// ['e', 'f', 'g', 'a']
			// Exception: Temp. version!: threadIdx.y + l
			// Exception: Temp. version!: 0 < rng_f
			sm_a[threadIdx.y][threadIdx.x + ll * 16] = dev_t2[blk_idx_e * SIZE_SLICE_1_E + ll + (blk_idx_f * SIZE_SLICE_1_F + 0 + ((blk_idx_a * SIZE_SLICE_1_A + idx_a) * size_g) * size_f) * size_e + (threadIdx.y + l) * stride_int_t2];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		// No Need to Put Boundary-Checks before For-Statement: : 
		for (int ll = 0; ll < 6; ll++)
		{
			// ['g', 'd', 'b', 'c']
			// Exception: Temp. version!: threadIdx.x + l
			// Exception: Temp. version!: idx_d < rng_d
			sm_b[threadIdx.x][threadIdx.y + ll * 16] = dev_v2[(blk_idx_d * SIZE_SLICE_1_D + idx_d + (blk_idx_b * SIZE_SLICE_1_B + ll + (blk_idx_c * SIZE_SLICE_1_C + 0) * size_b) * size_d) * size_g + (threadIdx.x + l)];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1; ll++)
		{
			temp_bv[0] = sm_b[ll][idx_d + (idx_c) * SIZE_SLICE_1_D + 0];
			temp_bv[1] = sm_b[ll][idx_d + (idx_c) * SIZE_SLICE_1_D + 16];
			temp_bv[2] = sm_b[ll][idx_d + (idx_c) * SIZE_SLICE_1_D + 32];
			temp_bv[3] = sm_b[ll][idx_d + (idx_c) * SIZE_SLICE_1_D + 48];
			temp_bv[4] = sm_b[ll][idx_d + (idx_c) * SIZE_SLICE_1_D + 64];
			temp_bv[5] = sm_b[ll][idx_d + (idx_c) * SIZE_SLICE_1_D + 80];

			for (int xx = 0; xx < 6; xx++) // (1)
			{
				temp_av = sm_a[ll][idx_f + (idx_a) * SIZE_SLICE_1_F + (xx * 16)];

				reg_tile[0][xx] += temp_av * temp_bv[0];
				reg_tile[1][xx] += temp_av * temp_bv[1];
				reg_tile[2][xx] += temp_av * temp_bv[2];
				reg_tile[3][xx] += temp_av * temp_bv[3];
				reg_tile[4][xx] += temp_av * temp_bv[4];
				reg_tile[5][xx] += temp_av * temp_bv[5];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	#pragma unroll 6
	for (int i = 0; i < 6; i++)
	{
		for (int j = 0; j < 6; j++)
		{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
		}
	}
}

// created by tc_gen_code_Kernel()
__global__ void kernel__2_1(double* dev_t3, 
double* dev_t2, 
double* dev_v2, 
int size_a, int size_b, int size_c, int size_d, int size_e, int size_f, int size_g, 
int numBlk_a, int numBlk_b, int numBlk_c, int numBlk_d, int numBlk_e, int numBlk_f, 
int stride_int_t2, int stride_int_v2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ double sm_a[16][96];
	__shared__ double sm_b[16][96];


	int internal_upperbound   = 0;
	int internal_offset;

	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 2
	// # of indices mapped on TB_Y: 2
	int idx_a = threadIdx.x % SIZE_SLICE_1_A;
	int idx_f = threadIdx.x / SIZE_SLICE_1_A;
	int idx_d = threadIdx.y % SIZE_SLICE_1_D;
	int idx_c = threadIdx.y / SIZE_SLICE_1_D;

	int tmp_blkIdx;
	int blk_idx_f = blockIdx.x / (numBlk_e * numBlk_d * numBlk_c * numBlk_b * numBlk_a);
	tmp_blkIdx = blockIdx.x % (numBlk_e * numBlk_d * numBlk_c * numBlk_b * numBlk_a);

	int blk_idx_e = tmp_blkIdx / (numBlk_d * numBlk_c * numBlk_b * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_d * numBlk_c * numBlk_b * numBlk_a);

	int blk_idx_d = tmp_blkIdx / (numBlk_c * numBlk_b * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_c * numBlk_b * numBlk_a);

	int blk_idx_c = tmp_blkIdx / (numBlk_b * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_b * numBlk_a);

	int blk_idx_b = tmp_blkIdx / numBlk_a;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a);

	int  blk_idx_a = tmp_blkIdx;

	int t3_base_thread = blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_b * SIZE_SLICE_1_B + (blk_idx_c * SIZE_SLICE_1_C + idx_c + (blk_idx_d * SIZE_SLICE_1_D + idx_d + (blk_idx_e * SIZE_SLICE_1_E + (blk_idx_f * SIZE_SLICE_1_F + idx_f) * size_e) * size_d) * size_c) * size_b) * size_a;


	double temp_av;
	double temp_bv[6];
	double reg_tile[6][6];

	for (int i = 0; i < 6; i++)
	for (int j = 0; j < 6; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'x', 't2', ['e', 'f', 'g', 'a']], [16, 'STR_SD2_V2_H7', 'y', 'v2', ['g', 'd', 'b', 'c']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		// Part: Generalized Contraction Index (p7b)
		internal_offset = (l + SIZE_INT_UNIT_1) - size_internal;
		if (internal_offset > 0) internal_upperbound = internal_offset;

		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (threadIdx.y < SIZE_INT_UNIT_1 - internal_upperbound)
		for (int ll = 0; ll < 6; ll++)
		{
			// ['e', 'f', 'g', 'a']
			// Exception: Temp. version!: threadIdx.y + l
			// Exception: Temp. version!: 0 < rng_f
			sm_a[threadIdx.y][threadIdx.x + ll * 16] = dev_t2[blk_idx_e * SIZE_SLICE_1_E + ll + (blk_idx_f * SIZE_SLICE_1_F + 0 + ((blk_idx_a * SIZE_SLICE_1_A + idx_a) * size_g) * size_f) * size_e + (threadIdx.y + l) * stride_int_t2];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (threadIdx.x < SIZE_INT_UNIT_1 - internal_upperbound)
		for (int ll = 0; ll < 6; ll++)
		{
			// ['g', 'd', 'b', 'c']
			// Exception: Temp. version!: threadIdx.x + l
			// Exception: Temp. version!: idx_d < rng_d
			sm_b[threadIdx.x][threadIdx.y + ll * 16] = dev_v2[(blk_idx_d * SIZE_SLICE_1_D + idx_d + (blk_idx_b * SIZE_SLICE_1_B + ll + (blk_idx_c * SIZE_SLICE_1_C + 0) * size_b) * size_d) * size_g + (threadIdx.x + l)];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1 - internal_upperbound; ll++)
		{
			temp_bv[0] = sm_b[ll][idx_d + (idx_c) * SIZE_SLICE_1_D + 0];
			temp_bv[1] = sm_b[ll][idx_d + (idx_c) * SIZE_SLICE_1_D + 16];
			temp_bv[2] = sm_b[ll][idx_d + (idx_c) * SIZE_SLICE_1_D + 32];
			temp_bv[3] = sm_b[ll][idx_d + (idx_c) * SIZE_SLICE_1_D + 48];
			temp_bv[4] = sm_b[ll][idx_d + (idx_c) * SIZE_SLICE_1_D + 64];
			temp_bv[5] = sm_b[ll][idx_d + (idx_c) * SIZE_SLICE_1_D + 80];

			for (int xx = 0; xx < 6; xx++) // (1)
			{
				temp_av = sm_a[ll][idx_f + (idx_a) * SIZE_SLICE_1_F + (xx * 16)];

				reg_tile[0][xx] += temp_av * temp_bv[0];
				reg_tile[1][xx] += temp_av * temp_bv[1];
				reg_tile[2][xx] += temp_av * temp_bv[2];
				reg_tile[3][xx] += temp_av * temp_bv[3];
				reg_tile[4][xx] += temp_av * temp_bv[4];
				reg_tile[5][xx] += temp_av * temp_bv[5];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	#pragma unroll 6
	for (int i = 0; i < 6; i++)
	{
		for (int j = 0; j < 6; j++)
		{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
		}
	}
}

// created by tc_gen_code_Kernel()
__global__ void kernel__3_1(double* dev_t3, 
double* dev_t2, 
double* dev_v2, 
int size_a, int size_b, int size_c, int size_d, int size_e, int size_f, int size_g, 
int numBlk_a, int numBlk_b, int numBlk_c, int numBlk_d, int numBlk_e, int numBlk_f, 
int stride_int_t2, int stride_int_v2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ double sm_a[16][96];
	__shared__ double sm_b[16][96];


	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 2
	// # of indices mapped on TB_Y: 2
	int idx_a = threadIdx.x % SIZE_SLICE_1_A;
	int idx_f = threadIdx.x / SIZE_SLICE_1_A;
	int idx_d = threadIdx.y % SIZE_SLICE_1_D;
	int idx_c = threadIdx.y / SIZE_SLICE_1_D;

	int tmp_blkIdx;
	int blk_idx_f = blockIdx.x / (numBlk_e * numBlk_d * numBlk_c * numBlk_b * numBlk_a);
	tmp_blkIdx = blockIdx.x % (numBlk_e * numBlk_d * numBlk_c * numBlk_b * numBlk_a);

	int blk_idx_e = tmp_blkIdx / (numBlk_d * numBlk_c * numBlk_b * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_d * numBlk_c * numBlk_b * numBlk_a);

	int blk_idx_d = tmp_blkIdx / (numBlk_c * numBlk_b * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_c * numBlk_b * numBlk_a);

	int blk_idx_c = tmp_blkIdx / (numBlk_b * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_b * numBlk_a);

	int blk_idx_b = tmp_blkIdx / numBlk_a;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a);

	int  blk_idx_a = tmp_blkIdx;

	int t3_base_thread = blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_b * SIZE_SLICE_1_B + (blk_idx_c * SIZE_SLICE_1_C + idx_c + (blk_idx_d * SIZE_SLICE_1_D + idx_d + (blk_idx_e * SIZE_SLICE_1_E + (blk_idx_f * SIZE_SLICE_1_F + idx_f) * size_e) * size_d) * size_c) * size_b) * size_a;

	// need to support partial tiles
	int rng_a, rng_b, rng_c, rng_d, rng_e, rng_f;
	if ((size_a - (blk_idx_a * SIZE_SLICE_1_A)) >= SIZE_SLICE_1_A)
	{
		rng_a = SIZE_SLICE_1_A;
	}
	else
	{
		rng_a = size_a % SIZE_SLICE_1_A;
	}
	if ((size_b - (blk_idx_b * SIZE_SLICE_1_B)) >= SIZE_SLICE_1_B)
	{
		rng_b = SIZE_SLICE_1_B;
	}
	else
	{
		rng_b = size_b % SIZE_SLICE_1_B;
	}
	if ((size_c - (blk_idx_c * SIZE_SLICE_1_C)) >= SIZE_SLICE_1_C)
	{
		rng_c = SIZE_SLICE_1_C;
	}
	else
	{
		rng_c = size_c % SIZE_SLICE_1_C;
	}
	if ((size_d - (blk_idx_d * SIZE_SLICE_1_D)) >= SIZE_SLICE_1_D)
	{
		rng_d = SIZE_SLICE_1_D;
	}
	else
	{
		rng_d = size_d % SIZE_SLICE_1_D;
	}
	if ((size_e - (blk_idx_e * SIZE_SLICE_1_E)) >= SIZE_SLICE_1_E)
	{
		rng_e = SIZE_SLICE_1_E;
	}
	else
	{
		rng_e = size_e % SIZE_SLICE_1_E;
	}
	if ((size_f - (blk_idx_f * SIZE_SLICE_1_F)) >= SIZE_SLICE_1_F)
	{
		rng_f = SIZE_SLICE_1_F;
	}
	else
	{
		rng_f = size_f % SIZE_SLICE_1_F;
	}

	double temp_av;
	double temp_bv[6];
	double reg_tile[6][6];

	for (int i = 0; i < 6; i++)
	for (int j = 0; j < 6; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'x', 't2', ['e', 'f', 'g', 'a']], [16, 'STR_SD2_V2_H7', 'y', 'v2', ['g', 'd', 'b', 'c']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (0 < rng_f && idx_a < rng_a)
		for (int ll = 0; ll < rng_e; ll++)
		{
			// ['e', 'f', 'g', 'a']
			// Exception: Temp. version!: threadIdx.y + l
			// Exception: Temp. version!: 0 < rng_f
			sm_a[threadIdx.y][threadIdx.x + ll * 16] = dev_t2[blk_idx_e * SIZE_SLICE_1_E + ll + (blk_idx_f * SIZE_SLICE_1_F + 0 + ((blk_idx_a * SIZE_SLICE_1_A + idx_a) * size_g) * size_f) * size_e + (threadIdx.y + l) * stride_int_t2];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (idx_d < rng_d && 0 < rng_c)
		for (int ll = 0; ll < rng_b; ll++)
		{
			// ['g', 'd', 'b', 'c']
			// Exception: Temp. version!: threadIdx.x + l
			// Exception: Temp. version!: idx_d < rng_d
			sm_b[threadIdx.x][threadIdx.y + ll * 16] = dev_v2[(blk_idx_d * SIZE_SLICE_1_D + idx_d + (blk_idx_b * SIZE_SLICE_1_B + ll + (blk_idx_c * SIZE_SLICE_1_C + 0) * size_b) * size_d) * size_g + (threadIdx.x + l)];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1; ll++)
		{
			temp_bv[0] = sm_b[ll][idx_d + (idx_c) * SIZE_SLICE_1_D + 0];
			temp_bv[1] = sm_b[ll][idx_d + (idx_c) * SIZE_SLICE_1_D + 16];
			temp_bv[2] = sm_b[ll][idx_d + (idx_c) * SIZE_SLICE_1_D + 32];
			temp_bv[3] = sm_b[ll][idx_d + (idx_c) * SIZE_SLICE_1_D + 48];
			temp_bv[4] = sm_b[ll][idx_d + (idx_c) * SIZE_SLICE_1_D + 64];
			temp_bv[5] = sm_b[ll][idx_d + (idx_c) * SIZE_SLICE_1_D + 80];

			for (int xx = 0; xx < 6; xx++) // (1)
			{
				temp_av = sm_a[ll][idx_f + (idx_a) * SIZE_SLICE_1_F + (xx * 16)];

				reg_tile[0][xx] += temp_av * temp_bv[0];
				reg_tile[1][xx] += temp_av * temp_bv[1];
				reg_tile[2][xx] += temp_av * temp_bv[2];
				reg_tile[3][xx] += temp_av * temp_bv[3];
				reg_tile[4][xx] += temp_av * temp_bv[4];
				reg_tile[5][xx] += temp_av * temp_bv[5];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	if (idx_a < rng_a && idx_f < rng_f && idx_d < rng_d && idx_c < rng_c)
	for (int i = 0; i < 6; i++)
	{
		for (int j = 0; j < 6; j++)
		{
			if(i < rng_b && j < rng_e)
			{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
			}
		}
	}
}

// created by tc_gen_code_Kernel()
__global__ void kernel__4_1(double* dev_t3, 
double* dev_t2, 
double* dev_v2, 
int size_a, int size_b, int size_c, int size_d, int size_e, int size_f, int size_g, 
int numBlk_a, int numBlk_b, int numBlk_c, int numBlk_d, int numBlk_e, int numBlk_f, 
int stride_int_t2, int stride_int_v2, 
int stride_reg_x, int stride_reg_y, 
int size_internal)
{
	// For Shared Memory,
	__shared__ double sm_a[16][96];
	__shared__ double sm_b[16][96];


	int internal_upperbound   = 0;
	int internal_offset;

	// when opt_pre_computed == -1, all indices will be calculated manually
	// # of indices mapped on TB_X: 2
	// # of indices mapped on TB_Y: 2
	int idx_a = threadIdx.x % SIZE_SLICE_1_A;
	int idx_f = threadIdx.x / SIZE_SLICE_1_A;
	int idx_d = threadIdx.y % SIZE_SLICE_1_D;
	int idx_c = threadIdx.y / SIZE_SLICE_1_D;

	int tmp_blkIdx;
	int blk_idx_f = blockIdx.x / (numBlk_e * numBlk_d * numBlk_c * numBlk_b * numBlk_a);
	tmp_blkIdx = blockIdx.x % (numBlk_e * numBlk_d * numBlk_c * numBlk_b * numBlk_a);

	int blk_idx_e = tmp_blkIdx / (numBlk_d * numBlk_c * numBlk_b * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_d * numBlk_c * numBlk_b * numBlk_a);

	int blk_idx_d = tmp_blkIdx / (numBlk_c * numBlk_b * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_c * numBlk_b * numBlk_a);

	int blk_idx_c = tmp_blkIdx / (numBlk_b * numBlk_a);
	tmp_blkIdx = tmp_blkIdx % (numBlk_b * numBlk_a);

	int blk_idx_b = tmp_blkIdx / numBlk_a;
	tmp_blkIdx = tmp_blkIdx % (numBlk_a);

	int  blk_idx_a = tmp_blkIdx;

	int t3_base_thread = blk_idx_a * SIZE_SLICE_1_A + idx_a + (blk_idx_b * SIZE_SLICE_1_B + (blk_idx_c * SIZE_SLICE_1_C + idx_c + (blk_idx_d * SIZE_SLICE_1_D + idx_d + (blk_idx_e * SIZE_SLICE_1_E + (blk_idx_f * SIZE_SLICE_1_F + idx_f) * size_e) * size_d) * size_c) * size_b) * size_a;

	// need to support partial tiles
	int rng_a, rng_b, rng_c, rng_d, rng_e, rng_f;
	if ((size_a - (blk_idx_a * SIZE_SLICE_1_A)) >= SIZE_SLICE_1_A)
	{
		rng_a = SIZE_SLICE_1_A;
	}
	else
	{
		rng_a = size_a % SIZE_SLICE_1_A;
	}
	if ((size_b - (blk_idx_b * SIZE_SLICE_1_B)) >= SIZE_SLICE_1_B)
	{
		rng_b = SIZE_SLICE_1_B;
	}
	else
	{
		rng_b = size_b % SIZE_SLICE_1_B;
	}
	if ((size_c - (blk_idx_c * SIZE_SLICE_1_C)) >= SIZE_SLICE_1_C)
	{
		rng_c = SIZE_SLICE_1_C;
	}
	else
	{
		rng_c = size_c % SIZE_SLICE_1_C;
	}
	if ((size_d - (blk_idx_d * SIZE_SLICE_1_D)) >= SIZE_SLICE_1_D)
	{
		rng_d = SIZE_SLICE_1_D;
	}
	else
	{
		rng_d = size_d % SIZE_SLICE_1_D;
	}
	if ((size_e - (blk_idx_e * SIZE_SLICE_1_E)) >= SIZE_SLICE_1_E)
	{
		rng_e = SIZE_SLICE_1_E;
	}
	else
	{
		rng_e = size_e % SIZE_SLICE_1_E;
	}
	if ((size_f - (blk_idx_f * SIZE_SLICE_1_F)) >= SIZE_SLICE_1_F)
	{
		rng_f = SIZE_SLICE_1_F;
	}
	else
	{
		rng_f = size_f % SIZE_SLICE_1_F;
	}

	double temp_av;
	double temp_bv[6];
	double reg_tile[6][6];

	for (int i = 0; i < 6; i++)
	for (int j = 0; j < 6; j++)
	reg_tile[i][j] = 0.0;

	// tensor contraction: [[16, 'STR_SD2_T2_H7', 'x', 't2', ['e', 'f', 'g', 'a']], [16, 'STR_SD2_V2_H7', 'y', 'v2', ['g', 'd', 'b', 'c']], '+=']
	#pragma unroll 1
	for (int l = 0; l < size_internal; l += SIZE_INT_UNIT_1)
	{
		// Part: Generalized Contraction Index (p7b)
		internal_offset = (l + SIZE_INT_UNIT_1) - size_internal;
		if (internal_offset > 0) internal_upperbound = internal_offset;

		//---------------------------------------------------------------------------------------------------
		// This is for the new version
		// This Part is for Loading Input-Left
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (0 < rng_f && idx_a < rng_a && threadIdx.y < SIZE_INT_UNIT_1 - internal_upperbound)
		for (int ll = 0; ll < rng_e; ll++)
		{
			// ['e', 'f', 'g', 'a']
			// Exception: Temp. version!: threadIdx.y + l
			// Exception: Temp. version!: 0 < rng_f
			sm_a[threadIdx.y][threadIdx.x + ll * 16] = dev_t2[blk_idx_e * SIZE_SLICE_1_E + ll + (blk_idx_f * SIZE_SLICE_1_F + 0 + ((blk_idx_a * SIZE_SLICE_1_A + idx_a) * size_g) * size_f) * size_e + (threadIdx.y + l) * stride_int_t2];
		}
		
		// This Part is for Loading Input-Right
		// tc_gen_code_Kernel_Load_Inputs_Abstracts()
		if (idx_d < rng_d && 0 < rng_c && threadIdx.x < SIZE_INT_UNIT_1 - internal_upperbound)
		for (int ll = 0; ll < rng_b; ll++)
		{
			// ['g', 'd', 'b', 'c']
			// Exception: Temp. version!: threadIdx.x + l
			// Exception: Temp. version!: idx_d < rng_d
			sm_b[threadIdx.x][threadIdx.y + ll * 16] = dev_v2[(blk_idx_d * SIZE_SLICE_1_D + idx_d + (blk_idx_b * SIZE_SLICE_1_B + ll + (blk_idx_c * SIZE_SLICE_1_C + 0) * size_b) * size_d) * size_g + (threadIdx.x + l)];
		}
		__syncthreads();
		//---------------------------------------------------------------------------------------------------
		

		// Part: Generalized Threads
		for (int ll = 0; ll < SIZE_INT_UNIT_1 - internal_upperbound; ll++)
		{
			temp_bv[0] = sm_b[ll][idx_d + (idx_c) * SIZE_SLICE_1_D + 0];
			temp_bv[1] = sm_b[ll][idx_d + (idx_c) * SIZE_SLICE_1_D + 16];
			temp_bv[2] = sm_b[ll][idx_d + (idx_c) * SIZE_SLICE_1_D + 32];
			temp_bv[3] = sm_b[ll][idx_d + (idx_c) * SIZE_SLICE_1_D + 48];
			temp_bv[4] = sm_b[ll][idx_d + (idx_c) * SIZE_SLICE_1_D + 64];
			temp_bv[5] = sm_b[ll][idx_d + (idx_c) * SIZE_SLICE_1_D + 80];

			for (int xx = 0; xx < 6; xx++) // (1)
			{
				temp_av = sm_a[ll][idx_f + (idx_a) * SIZE_SLICE_1_F + (xx * 16)];

				reg_tile[0][xx] += temp_av * temp_bv[0];
				reg_tile[1][xx] += temp_av * temp_bv[1];
				reg_tile[2][xx] += temp_av * temp_bv[2];
				reg_tile[3][xx] += temp_av * temp_bv[3];
				reg_tile[4][xx] += temp_av * temp_bv[4];
				reg_tile[5][xx] += temp_av * temp_bv[5];
			}
		}
		__syncthreads();
	}


	// Store Results (Registers) to Global Memory
	// Part: Generalized Threads
	// Part: Generalized Register-Tiling
	if (idx_a < rng_a && idx_f < rng_f && idx_d < rng_d && idx_c < rng_c)
	for (int i = 0; i < 6; i++)
	{
		for (int j = 0; j < 6; j++)
		{
			if(i < rng_b && j < rng_e)
			{
			dev_t3[t3_base_thread + (i * stride_reg_y) + (j * stride_reg_x)] = reg_tile[i][j];
			}
		}
	}
}

// written by tc_interface.tc_gen_code_interface_Header()
extern "C"
void sd_t_d2_fusion(int size_a, int size_b, int size_c, int size_d, int size_e, int size_f, int size_g, double* t3, double* host_t2, double* host_v2, int cond_kernel_1, int opt_register_transpose)
{
	int num_thread_blocks_kernel_1;

	double* dev_t3;
	double* dev_t2;
	double* dev_v2;


	num_thread_blocks_kernel_1 = CEIL(size_a, SIZE_SLICE_1_A) * CEIL(size_b, SIZE_SLICE_1_B) * CEIL(size_c, SIZE_SLICE_1_C) * CEIL(size_d, SIZE_SLICE_1_D) * CEIL(size_e, SIZE_SLICE_1_E) * CEIL(size_f, SIZE_SLICE_1_F);
	// cudaMalloc()
	hipMalloc((void**) &dev_t3, sizeof(double) * size_a * size_b * size_c * size_d * size_e * size_f);
	hipMalloc((void**) &dev_t2, sizeof(double) * size_a * size_g * size_f * size_e);
	hipMalloc((void**) &dev_v2, sizeof(double) * size_c * size_b * size_d * size_g);

	// cudaMemcpy()
	hipMemcpy(dev_t3, t3, sizeof(double) * size_a * size_b * size_c * size_d * size_e * size_f, hipMemcpyHostToDevice);
	hipMemcpy(dev_t2, host_t2, sizeof(double) * size_a * size_g * size_f * size_e, hipMemcpyHostToDevice);
	hipMemcpy(dev_v2, host_v2, sizeof(double) * size_c * size_b * size_d * size_g, hipMemcpyHostToDevice);

	// Related to Kernels
	// There are 1 Basic Kernels
	long long int tmp_operations = 2 * (long long int)(size_a * size_b * size_c * size_d * size_e * size_f) * size_g;
	printf ("========================================= fusedKernels =============================================\n");
	printf ("		Grid Size  : %6d (1D)\n", num_thread_blocks_kernel_1);
	printf ("		Block-size : %2d, %2d (2D)\n", SIZE_TB_1_X, SIZE_TB_1_Y);
	printf ("		Reg.-size  : %2d, %2d (2D)\n", SIZE_REG_1_X, SIZE_REG_1_Y);
	printf ("		A thread deals with (%d x %d) elements (basically)\n", SIZE_TB_1_X * SIZE_REG_1_X, SIZE_TB_1_Y * SIZE_REG_1_Y);
	printf ("		# of Operations: %lld\n", tmp_operations);
	printf ("====================================================================================================\n");
	dim3 gridsize_1(num_thread_blocks_kernel_1);
	dim3 blocksize_1(SIZE_TB_1_X, SIZE_TB_1_Y);

	int stride_output_a = 1;
	int stride_output_b = stride_output_a * size_a;
	int stride_output_c = stride_output_b * size_b;
	int stride_output_d = stride_output_c * size_c;
	int stride_output_e = stride_output_d * size_d;
	int stride_output_f = stride_output_e * size_e;

	int stride_reg_x_1 = stride_output_e;
	int stride_reg_y_1 = stride_output_b;

	int size_internal = size_g;

	int stride_int_t2 = size_e * size_f;
	int stride_int_v2 = 1;

	// Decision Tree for Kernel Types
	// No Chance to Utilize the Register Transpose
	if (size_a % SIZE_SLICE_1_A == 0 && size_b % SIZE_SLICE_1_B == 0 && size_c % SIZE_SLICE_1_C == 0 && size_d % SIZE_SLICE_1_D == 0 && size_e % SIZE_SLICE_1_E == 0 && size_f % SIZE_SLICE_1_F == 0)
	{
		// [2] Extenral Index: Full
		if (size_g % SIZE_SLICE_1_G == 0)
		{
			// [3] Internal Index: Full
			// >>> External: Full && Internal: Full
			printf ("External: Full, Internal: Full\n");
			kernel__1_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a, size_b, size_c, size_d, size_e, size_f, size_g, CEIL(size_a, SIZE_SLICE_1_A), CEIL(size_b, SIZE_SLICE_1_B), CEIL(size_c, SIZE_SLICE_1_C), CEIL(size_d, SIZE_SLICE_1_D), CEIL(size_e, SIZE_SLICE_1_E), CEIL(size_f, SIZE_SLICE_1_F), stride_int_t2, stride_int_v2, stride_reg_x_1, stride_reg_y_1, size_internal);
		}
		else
		{
			// [4] Internal Index: Partial
			// >>> External: Full && Internal: Partial
			printf ("External: Full, Internal: Partial\n");
			kernel__2_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a, size_b, size_c, size_d, size_e, size_f, size_g, CEIL(size_a, SIZE_SLICE_1_A), CEIL(size_b, SIZE_SLICE_1_B), CEIL(size_c, SIZE_SLICE_1_C), CEIL(size_d, SIZE_SLICE_1_D), CEIL(size_e, SIZE_SLICE_1_E), CEIL(size_f, SIZE_SLICE_1_F), stride_int_t2, stride_int_v2, stride_reg_x_1, stride_reg_y_1, size_internal);
		}
	}
	else
	{
		// [2] Extenral Index: Partial
		if (size_g % SIZE_SLICE_1_G == 0)
		{
			// [3] Internal Index: Full
			// >>> External: Partial && Internal: Full
			printf ("External: Partial, Internal: Full\n");
			kernel__3_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a, size_b, size_c, size_d, size_e, size_f, size_g, CEIL(size_a, SIZE_SLICE_1_A), CEIL(size_b, SIZE_SLICE_1_B), CEIL(size_c, SIZE_SLICE_1_C), CEIL(size_d, SIZE_SLICE_1_D), CEIL(size_e, SIZE_SLICE_1_E), CEIL(size_f, SIZE_SLICE_1_F), stride_int_t2, stride_int_v2, stride_reg_x_1, stride_reg_y_1, size_internal);
		}
		else
		{
			// [4] Internal Index: Partial
			// >>> External: Partial && Internal: Partial
			printf ("External: Partial, Internal: Partial\n");
			kernel__4_1<<<gridsize_1, blocksize_1>>>(dev_t3, dev_t2, dev_v2, size_a, size_b, size_c, size_d, size_e, size_f, size_g, CEIL(size_a, SIZE_SLICE_1_A), CEIL(size_b, SIZE_SLICE_1_B), CEIL(size_c, SIZE_SLICE_1_C), CEIL(size_d, SIZE_SLICE_1_D), CEIL(size_e, SIZE_SLICE_1_E), CEIL(size_f, SIZE_SLICE_1_F), stride_int_t2, stride_int_v2, stride_reg_x_1, stride_reg_y_1, size_internal);
		}
	}

	// Copy the Result from Device to Host
	hipMemcpy(t3, dev_t3, sizeof(double) * (size_a * size_b * size_c * size_d * size_e * size_f), hipMemcpyDeviceToHost);

	// cudaFree()
	hipFree(dev_t3);	hipFree(dev_t2);	hipFree(dev_v2);

	// Shoule be Fixed
	// HostFree

}

// This is written by tc_interface.tc_gen_code_interface()
// This Interface Should be Called to Run the Kernels
extern "C"
void sd_t_d2_fusion_(int size_a, int size_b, int size_c, int size_d, int size_e, int size_f, int size_g, double* t3, double* t2, double* v2, int cond_kernel_1, int opt_register_transpose)
{
	// Pre-Processing for Split
	// Based on Tile-Sizes and Problem-Size
	// Currently, one index can be split into two indices

	// Call An Application
	sd_t_d2_fusion(size_a, size_b, size_c, size_d, size_e, size_f, size_g, t3, t2, v2, cond_kernel_1, opt_register_transpose);
}
